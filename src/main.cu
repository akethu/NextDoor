
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <string.h>
#include <assert.h>
#include <tuple>

#define LINE_SIZE 1024*1024
//#define USE_FIXED_THREADS
#define MAX_CUDA_THREADS (96*96)
#define THREAD_BLOCK_SIZE 256
#define WARP_SIZE 32
#define ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING true
//#define USE_CSR_IN_SHARED
//#define EMBEDDING_IN_SHARED_MEM_PER_VERTEX
//#define USE_EMBEDDING_IN_GLOBAL_MEM
//#define USE_EMBEDDING_IN_SHARED_MEM
//#define ALL_THREAD_BLOCK_EMBEDDINGS_IN_SHARED_MEM_PER_VERTEX
#define USE_EMBEDDING_IN_LOCAL_MEM
#define PROCESS_EMBEDDINGS_PER_VERTEX
//#define EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK
#define GPU_QUERY_WAIT_TIME 1000UL

//#define ADD_TO_OUTPUT
//#define SHARED_MEM_NON_COALESCING
/**
  * The commit performing better is 698368fa19d023e3cb09705d820d333f79d0bf46.
  */
#ifdef SHARED_MEM_NON_COALESCING
  #ifndef USE_EMBEDDING_IN_SHARED_MEM
    #error "USE_EMBEDDING_IN_SHARED_MEM must be enabled with SHARED_MEM_NON_COALESCING"
  #endif
#endif
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #ifdef USE_FIXED_THREADS
    #error "USE_FIXED_THREADS cannot be enabled with USE_EMBEDDING_IN_SHARED_MEM"
  #endif
#endif

#define NEW_EMBEDDING_BUFFER_SIZE 10*1024 //Size in terms of Bytes //Setting it to 128 MB makes citeseer performs a lot better

//#define USE_CONSTANT_MEM

typedef uint8_t SharedMemElem;

//citeseer.graph
const int N = 3312;
const int N_EDGES = 9074;

//micro.graph
//const int N = 100000;
//const int N_EDGES = 2160312;

enum BUFFER_STATUS {
  GPU_USING,
  CPU_COPYING,
  FREE,
};

class Vertex
{
private:
  int id;
  int label;
  std::vector <int> edges;

public:
  Vertex (int _id, int _label) : label(_label), id (_id)
  {
  }

  int set_id (int _id) {id = _id;}
  int get_id () {return id;}
  int get_label () {return label;}
  void add_edge (int vertexID) {edges.push_back (vertexID);}
  void sort_edges () {std::sort (edges.begin(), edges.end ());}
  std::vector <int>& get_edges () {return edges;}
  void print (std::ostream& os)
  {
    os << id << " " << label << " ";
    for (auto edge : edges) {
      os << edge << " ";
    }

    os << std::endl;
  }

  static bool compare_vertex (Vertex& v1, Vertex& v2) 
  {
    return v1.edges.size () < v2.edges.size ();
  }
};

int chars_in_int (int num)
{
  if (num == 0) return sizeof(char);
  return (int)((ceil(log10(num))+1)*sizeof(char));
}

class Graph
{
private:
  std::vector<Vertex> vertices;
  int n_edges;

public:
  Graph (std::vector<Vertex> _vertices, int _n_edges) :
    vertices (_vertices), n_edges(_n_edges)
  {}

  const std::vector<Vertex>& get_vertices () {return vertices;}
  int get_n_edges () {return n_edges;}
};

class CSR
{
public:
  struct Vertex
  {
    int id;
    int label;
    int start_edge_id;
    int end_edge_id;
    __host__ __device__
    Vertex ()
    {
      id = -1;
      label = -1;
      start_edge_id = -1;
      end_edge_id = -1;
    }

    void set_from_graph_vertex (::Vertex& vertex)
    {
      id = vertex.get_id ();
      label = vertex.get_label ();
    }

    void set_start_edge_id (int start) {start_edge_id = start;}
    void set_end_edge_id (int end) {end_edge_id = end;}
  };

  typedef int Edge;

public:
  CSR::Vertex vertices[N];
  CSR::Edge edges[N_EDGES];
  int n_vertices;
  int n_edges;

public:
  CSR (int _n_vertices, int _n_edges)
  {
    n_vertices = _n_vertices;
    n_edges = _n_edges;
  }

  __host__ __device__
  CSR ()
  {
    n_vertices = N;
    n_edges = N_EDGES;
  }

  void print (std::ostream& os)
  {
    for (int i = 0; i < n_vertices; i++) {
      os << vertices[i].id << " " << vertices[i].label << " ";
      for (int edge_iter = vertices[i].start_edge_id;
           edge_iter <= vertices[i].end_edge_id; edge_iter++) {
        os << edges[edge_iter] << " ";
      }
      os << std::endl;
    }
  }

  __host__ __device__
  int get_start_edge_idx (int vertex_id)
  {
    if (!(vertex_id < n_vertices && 0 <= vertex_id)) {
      printf ("vertex_id %d, n_vertices %d\n", vertex_id, n_vertices);
      assert (false);
    }
    return vertices[vertex_id].start_edge_id;
  }

  __host__ __device__
  int get_end_edge_idx (int vertex_id)
  {
    assert (vertex_id < n_vertices && 0 <= vertex_id);
    return vertices[vertex_id].end_edge_id;
  }

  __host__ __device__
  bool has_edge (int u, int v)
  {
    //TODO: Since graph is sorted, do this using binary search
    for (int e = get_start_edge_idx (u); e <= get_end_edge_idx (u); e++) {
      if (edges[e] == v) {
        return true;
      }
    }

    return false;
  }

  __host__ __device__
  const CSR::Edge* get_edges () {return &edges[0];}

  __host__ __device__
  const CSR::Vertex* get_vertices () {return &vertices[0];}

  __host__ __device__
  int get_n_vertices () {return n_vertices;}

  __host__ __device__
  void copy_vertices (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      vertices[i] = src->get_vertices()[i];
    }
  }

  __host__ __device__
  void copy_edges (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      edges[i] = src->get_edges ()[i];
    }
  }

  __host__ __device__
  int get_n_edges () {return n_edges;}
};

#ifdef USE_CONSTANT_MEM
  __constant__ unsigned char csr_constant_buff[sizeof(CSR)];
#endif

void csr_from_graph (CSR* csr, Graph& graph)
{
  int edge_iterator = 0;
  auto graph_vertices = graph.get_vertices ();
  for (int i = 0; i < graph_vertices.size (); i++) {
    ::Vertex& vertex = graph_vertices[i];
    csr->vertices[i].set_from_graph_vertex (graph_vertices[i]);
    csr->vertices[i].set_start_edge_id (edge_iterator);
    for (auto edge : vertex.get_edges ()) {
      csr->edges[edge_iterator] = edge;
      edge_iterator++;
    }

    csr->vertices[i].set_end_edge_id (edge_iterator-1);
  }
}

//template <size_t N> using VertexEmbedding = std::bitset<N>;

#define CVT_TO_NEXT_MULTIPLE(n,k) ((n) %(k) ==0 ? (n) : ((n)/(k)+1)*(k))
class BitVectorVertexEmbedding
{
private:
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  unsigned char array[CVT_TO_NEXT_MULTIPLE(N/8, 32*sizeof(SharedMemElem))];
#else
  unsigned char array[(N/8)];
#endif

public:
  __device__ __host__
  BitVectorVertexEmbedding ()
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    assert (array != nullptr);
    reset ();
    assert (all_false () == true);
  }

  __host__ __device__
  size_t convert_to_bytes_multiple (size_t n)
  {
    return (n/8)*8;
  }

  __host__ __device__
  BitVectorVertexEmbedding (const BitVectorVertexEmbedding& embedding)
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    for (int i = 0; i <  convert_to_bytes_multiple(N)/8; i++) {
      array[i] = embedding.array[i];
    }
  }

  __host__ __device__
  void set (int index)
  {
    assert (index >= 0 and index < N);
    array[index/8] = array[index/8] | (1 << (index %8));
  }

  __host__ __device__
  void set ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = (unsigned char) (~(0UL));
    }
  }

  __host__ __device__
  void reset ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = 0;
    }
  }

  __host__ __device__
  void reset (int index)
  {
    assert (index >= 0 and index < N);
    array[index/8] = array[index/8] & (~(1UL << (index %8)));
  }

  __host__ __device__
  bool test (int index)
  {
    return (bool) ((array[index/8] >> (index % 8))&1);
  }

  __host__ __device__
  bool all_false ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      if (array[i] != 0UL) {
        return false;
      }
    }

    return true;
  }
  
  __host__ __device__
  int get_n_vertices () 
  {
    int n_vertices = 0;
    for (int i = 0; i < N; i++) {
      if (test(i) == true) {
        n_vertices++;
      }
    }
    
    return n_vertices;
  }
  
  __host__ __device__
  ~BitVectorVertexEmbedding ()
  {
    //delete[] array;
  }
};

//typedef BitVectorVertexEmbedding VertexEmbedding;

template <uint32_t size> 
class VectorVertexEmbedding
{
private:
  uint32_t array[size];
  uint32_t filled_size;
  
public:
  __device__ __host__
  VectorVertexEmbedding ()
  {
    filled_size = 0;
  }

  __host__ __device__
  VectorVertexEmbedding (const VectorVertexEmbedding<size>& embedding)
  {
  #if DEBUG
    assert (embedding.get_max_size () <= get_max_size ());
  #endif
    filled_size = 0;
    for (int i = 0; i < embedding.get_n_vertices (); i++) {
      add (embedding.get_vertex (i));
    }
  }
  
  __host__ __device__
  void add (int v)
  {
  #if DEBUG
    if (!(size != 0 and filled_size < size)) {
      printf ("filled_size %d, size %d\n", filled_size, size);
      //assert (size != 0 and filled_size < size);
      assert (false);
    }
  #endif
  
    add_unsorted (v);
    return;
    int pos = 0;
    
    for (int i = 0; i < filled_size; i++) {
      if (array[i] > v) {
        pos = i;
        break;
      }
    }
    
    for (int i = filled_size-1; i >= pos ; i--) {
      array[i+1] = array[i];
    }
    
    array[pos] = v;
    filled_size++;
  }

  __host__ __device__
  void add_last_in_sort_order () 
  {
    int v = array[filled_size-1];
    remove_last ();
    add (v);
  }

  __host__ __device__
  void add_unsorted (int v) 
  {
    array[filled_size++] = v;
  }
  
  __host__ __device__
  void remove (int v)
  {
    printf ("Do not support remove\n");
    assert (false);
  }
  
  __host__ __device__
  const bool has_logn (int v)
  {
    int l = 0;
    int r = filled_size-1;
    
    while (l <= r) {
      int m = l+(r-l)/2;
      
      if (array[m] == v)
        return true;
      
      if (array[m] < v)
        l = m + 1;
      else
        r = m - 1;
    }
    
    return false;
  }
  
  __host__ __device__
  const bool has (int v)
  {
    for (int i = 0; i < filled_size; i++) {
      if (array[i] == v) {
        return true;
      }
    }
    
    return false;
  }
  
  __host__ __device__
  const size_t get_n_vertices () const
  {
    return filled_size;
  }
  
  __host__ __device__
  const int get_vertex (int index) const
  {
    return array[index];
  }
  
  __host__ __device__
  const int get_last_vertex () const
  {
    return array[filled_size-1];
  }
  
  __host__ __device__
  size_t get_max_size () const
  {
    return size;
  }
  
  __host__ __device__
  void clear ()
  {
    filled_size = 0;
  }
  
  __host__ __device__
  void remove_last () 
  {
    assert (filled_size > 0);
    filled_size--;
  }
  __host__ __device__
  ~VectorVertexEmbedding ()
  {
    //delete[] array;
  }
};

template <uint32_t size>
__host__ __device__
void vector_embedding_from_one_less_size (VectorVertexEmbedding<size> const & vec_emb1,
                                          VectorVertexEmbedding<size+1>& vec_emb2)
{
  //TODO: Optimize here, filled_size++ in add is being called several times
  //but can be called only once too
  //if  (false and vec_emb1.get_n_vertices () != size) {
  //  printf ("vec_emb1.get_n_vertices () %ld != size %d\n", vec_emb1.get_n_vertices (), size);
  //  assert (false);
  //}
  for (int i = 0; i < vec_emb1.get_n_vertices (); i++) {
    vec_emb2.add (vec_emb1.get_vertex (i));
  }
}

template <uint32_t size> 
void bitvector_to_vector_embedding (BitVectorVertexEmbedding& bit_emb, 
                                    VectorVertexEmbedding<size>& vec_emb)
{
  for (int u = 0; u < N; u++) {
    if (bit_emb.test(u)) {
      vec_emb.add (u);
    }
  }
}

void print_embedding (BitVectorVertexEmbedding embedding, std::ostream& os);

std::vector<BitVectorVertexEmbedding> get_extensions_bitvector (BitVectorVertexEmbedding& embedding, CSR* csr)
{
  std::vector<BitVectorVertexEmbedding> extensions;

  if (embedding.all_false ()) {
    for (int u = 0; u < N; u++) {
      BitVectorVertexEmbedding extension;
      extension.set(u);
      //print_embedding (extension, std::cout);
      //std::cout << " " << u << std::endl;
      extensions.push_back (extension);
    }
  } else {
    for (int u = 0; u < N; u++) {
      if (embedding.test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding.test (v) == false) {
            BitVectorVertexEmbedding extension = BitVectorVertexEmbedding(embedding);
            extension.set(v);
            extensions.push_back(extension);
          }
        }
      }
    }
  }

  return extensions;
}

template <uint32_t size>
std::vector<VectorVertexEmbedding<size+1>> get_extensions_vector (VectorVertexEmbedding<size>& embedding, CSR* csr)
{
  std::vector<VectorVertexEmbedding<size+1>> extensions;

  if (embedding.get_n_vertices () == 0) {
    for (int u = 0; u < N; u++) {
      VectorVertexEmbedding<size+1> extension;
      extension.add(u);
      //print_embedding (extension, std::cout);
      //std::cout << " " << u << std::endl;
      extensions.push_back (extension);
    }
  } else {
    for (int i = 0; i < embedding.get_n_vertices (); i++) {
      int u = embedding.get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        int v = csr->get_edges () [e];
        if (embedding.has (v) == false) {
          VectorVertexEmbedding<size+1> extension;
          vector_embedding_from_one_less_size (embedding, extension);
          extension.add(v);
          extensions.push_back(extension);
        }
      }
    }
  }

  return extensions;
}

std::vector<BitVectorVertexEmbedding> get_initial_embedding_bitvector (CSR* csr)
{
  BitVectorVertexEmbedding embedding;
  std::vector <BitVectorVertexEmbedding> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

std::vector<VectorVertexEmbedding<0>> get_initial_embedding_vector (CSR* csr)
{
  VectorVertexEmbedding<0> embedding;
  std::vector <VectorVertexEmbedding<0>> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

bool (*filter) (CSR* csr, BitVectorVertexEmbedding& embedding);
void (*process) (std::vector<BitVectorVertexEmbedding>& output, BitVectorVertexEmbedding& embedding);

__host__ __device__
bool clique_filter (CSR* csr, BitVectorVertexEmbedding* embedding)
{
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      for (int v = 0; v < N; v++) {
        if (u != v and embedding->test(v)) {
          if (!csr->has_edge (u, v)) {
            return false;
          }
        }
      }
    }
  }

  return true;
}

template <int size>
void print_embedding (VectorVertexEmbedding<size>* embedding)
{
  std::cout << "{";
  for (int i = 0; i < size; i++) {
    std::cout << embedding->get_vertex (i) << ", ";
  }
  std::cout << "}";
}

template <int size>
std::string embedding_to_string (VectorVertexEmbedding<size>* embedding)
{
  std::string s = "";
  s = "{";
  for (int i = 0; i < size; i++) {
    s += embedding->get_vertex (i) + ", ";
  }
  s += "}";
  return s;
}

template <uint32_t size>
__host__ __device__
bool clique_filter_vector (CSR* csr, VectorVertexEmbedding<size>* embedding)
{
  for (int i = 0; i < embedding->get_n_vertices (); i++) {
    int u = embedding->get_vertex (i);
    for (int j = 0; j < embedding->get_n_vertices (); j++) {
      int v = embedding->get_vertex (j);
      if (u != v) {
        if (!csr->has_edge (u, v)) {
          return false;
        }
      }
    }
  }

  return true;
}

template <uint32_t size>
__host__ __device__
bool clique_filter_vector_optimized (CSR* csr, VectorVertexEmbedding<size> const * embedding, int last_vertex)
{
  for (int i = 0; i < embedding->get_n_vertices (); i++) {
    int u = embedding->get_vertex (i);
    if (!csr->has_edge (u, last_vertex)) {
        return false;
    }
  }

  return true;
}

void clique_process_bit_vector (std::vector<BitVectorVertexEmbedding>& output, BitVectorVertexEmbedding& embedding)
{
  output.push_back (embedding);
}

template <uint32_t size>
void clique_process_vector (std::vector<VectorVertexEmbedding<size>>& output, VectorVertexEmbedding<size>& embedding)
{
  output.push_back (embedding);
}

void run_single_step_initial_bitvector (void* input, int n_embeddings, CSR* csr,
                      std::vector<BitVectorVertexEmbedding>& output,
                      std::vector<BitVectorVertexEmbedding>& next_step)
{
  BitVectorVertexEmbedding* embeddings = (BitVectorVertexEmbedding*)input;

  for (int i = 0; i < n_embeddings; i++) {
    BitVectorVertexEmbedding embedding = embeddings[i];
    std::vector<BitVectorVertexEmbedding> extensions = get_extensions_bitvector (embedding, csr);

    for (auto extension : extensions) {
      if (clique_filter (csr, &extension)) {
        clique_process_bit_vector (output, extension);
        next_step.push_back (extension);
      }
    }
  }
}

void run_single_step_initial_vector (void* input, int n_embeddings, CSR* csr,
                      std::vector<VectorVertexEmbedding<1>>& output,
                      std::vector<VectorVertexEmbedding<1>>& next_step)
{
  VectorVertexEmbedding<0>* embeddings = (VectorVertexEmbedding<0>*)input;

  for (int i = 0; i < n_embeddings; i++) {
    VectorVertexEmbedding<0> embedding = embeddings[i];
    std::vector<VectorVertexEmbedding<1>> extensions = get_extensions_vector (embedding, csr);
    std::cout << "extensions " << extensions.size () << std::endl;
    for (auto extension : extensions) {
      if (clique_filter_vector (csr, &extension)) {
        clique_process_vector (output, extension);
        next_step.push_back (extension);
      }
    }
  }
}

__device__
void printf_embedding (BitVectorVertexEmbedding* embedding)
{
  printf ("[");
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      printf ("%d, ", u);
    }
  }

  printf ("]\n");
}

/*__global__
void run_single_step_bitvector_embedding (void* input, int n_embeddings, CSR* csr,
                      void* output_ptr,
                      int* n_output,
                      void* next_step, int* n_next_step,
                      int* n_output_1, int* n_next_step_1)
{
  int id;

#ifdef USE_CSR_IN_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread,
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread,
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#else
#ifdef USE_CONSTANT_MEM
  csr = (CSR*) csr_constant_buff;
#endif

#endif

  BitVectorVertexEmbedding* embeddings = (BitVectorVertexEmbedding*)input;
  BitVectorVertexEmbedding* new_embeddings = (BitVectorVertexEmbedding*)next_step;
  BitVectorVertexEmbedding* output = ((BitVectorVertexEmbedding*)output_ptr);
#ifdef USE_EMBEDDING_IN_LOCAL_MEM
  unsigned char temp_buffer [sizeof(BitVectorVertexEmbedding)];
#endif
  id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
#ifdef USE_FIXED_THREADS
  if (n_embeddings >= MAX_CUDA_THREADS) {
    int embeddings_per_thread = n_embeddings/MAX_CUDA_THREADS+1;

    start = id*embeddings_per_thread;
    end = (id+1)*embeddings_per_thread < n_embeddings ? (id+1)*embeddings_per_thread : n_embeddings;
  } else {
    if (id >= n_embeddings)
      return;

    start = id;
    end = id+1;
  }
#else
  if (id >= n_embeddings)
      return;

  start = id;
  end = id+1;
#endif

  int q[1000] = {0};

#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #if 0
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[per_thread_shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[0];
  #else
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
  #endif
#endif

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int lane_id = threadIdx.x % WARP_SIZE;
  for (int i = start; i < end; i++) {
    #ifdef USE_EMBEDDING_IN_SHARED_MEM
      #ifdef SHARED_MEM_NON_COALESCING
        memcpy (local_shared_buff, &embeddings[i], sizeof(BitVectorVertexEmbedding));

        BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*) local_shared_buff;
      #else
        const int thread_block_size = WARP_SIZE;
        const int last_emb = WARP_SIZE*(warp_id+1);
        if (blockIdx.x*blockDim.x + (warp_id+1)*WARP_SIZE > n_embeddings) {
          assert (false);
          //thread_block_size = n_embeddings - blockIdx.x*blockDim.x -
          //                    warp_id*WARP_SIZE;
          //last_emb = warp_id*WARP_SIZE + thread_block_size;
        }

        for (int emb = WARP_SIZE*warp_id; emb < last_emb; emb++) {
          SharedMemElem* embedding_buff = (SharedMemElem*) &embeddings[emb+blockIdx.x*blockDim.x];

          for (int j = 0; j < sizeof (BitVectorVertexEmbedding)/sizeof (SharedMemElem);
               j += thread_block_size) {
            int idx = per_thread_shared_mem_size/sizeof(SharedMemElem)*emb;
            if (true or j + lane_id  < sizeof (BitVectorVertexEmbedding)/sizeof (SharedMemElem)) { //TODO: Remove this if by doing padding with VertexEmbedding
              shared_buff[idx + j + lane_id] = embedding_buff[j + lane_id];
            }
          }
        }

        BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*) local_shared_buff;
      #endif
      //embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_LOCAL_MEM)
      //memcpy (&temp[0], &embeddings[i], sizeof (VertexEmbedding));
      memcpy (&temp_buffer[0], &embeddings[i], sizeof(BitVectorVertexEmbedding));
      BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*)&temp_buffer[0];
      //VertexEmbedding* embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_GLOBAL_MEM)
      BitVectorVertexEmbedding* embedding = &embeddings[i];
    #else
      #error "None of USE_EMBEDDING_IN_*_MEM option defined"
    #endif

  #if 1
    typedef uint32_t VertexEmbeddingChange;
    const uint32_t max_changes = 32; //max changes per warp
    //__shared__ int32_t shared_mem [max_changes*THREAD_BLOCK_SIZE/WARP_SIZE+THREAD_BLOCK_SIZE/WARP_SIZE+1];
    __shared__ VertexEmbeddingChange changes[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t changed_thread_ids[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    //For each warp record new embeddings (or changes) using atomic operations
    __shared__ int32_t n_extensions[THREAD_BLOCK_SIZE/WARP_SIZE];
    n_extensions[warp_id] = 0;
    __shared__ uint32_t prev_n_outputs [THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t prev_n_next_steps [THREAD_BLOCK_SIZE/WARP_SIZE];
    
    const uint32_t mask = ~0U;
    for (int u = 0; u < N; u++) {
      int e = csr->get_start_edge_idx(u);
      
      while (true) {
        
        int predicate = 1;
        //n_extensions[warp_id] = 0;

        if (e <= csr->get_end_edge_idx(u)) {
          int v = csr->get_edges () [e];
          if (embedding->test(u) and embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              int prev_n_extensions = atomicAdd (&n_extensions[warp_id], 1);
              changes[warp_id*max_changes+prev_n_extensions] = v;
              changed_thread_ids[warp_id*max_changes+prev_n_extensions] = id;
              //memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (VertexEmbedding));
              //memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (VertexEmbedding));
            }

            extension->reset(v);
          }

          e++;

          if (e > csr->get_end_edge_idx(u)) {
            predicate = 0;
          }
        } else {
          predicate = 0;
        }

        int32_t n_changes = n_extensions[warp_id];

        if (n_changes > max_changes)
          assert (false);

        //assert (__activemask () == ~0U);
        uint32_t orig_prev_n_output = 0;
        uint32_t orig_prev_n_next_step = 0;
        
        if (lane_id == 0) {
          if (n_changes > 0) {
            n_extensions[warp_id] = 0;
            orig_prev_n_output = atomicAdd (n_output, n_changes);
            orig_prev_n_next_step = atomicAdd (n_next_step, n_changes);
            //if (warp_id == 2 && (threadIdx.x == 64 || threadIdx.x == 65) && blockIdx.x == 0) {
            //  printf ("prev_n_next_steps[warp_id]: %d\n", prev_n_next_steps[warp_id]);
            //}
          }
        }
        
        //__syncwarp ();
        
        if (n_changes > 0) {
          
          uint32_t prev_n_output = __shfl_sync (__activemask (), orig_prev_n_output, 0);
          uint32_t prev_n_next_step = __shfl_sync (__activemask (), orig_prev_n_next_step, 0);
          
          for (int i = 0; i < n_changes; i++) {
            
            int changes_idx = warp_id*max_changes + i;
            uint32_t expected_thread_id = changed_thread_ids[changes_idx];
            if (expected_thread_id == id) {
              int v = changes[changes_idx];
              BitVectorVertexEmbedding* extension = embedding;
              extension->set (v);
              
              memcpy (&output[prev_n_output + i], extension, 
                      sizeof(BitVectorVertexEmbedding));
              memcpy (&new_embeddings[prev_n_next_step + i], extension,
                      sizeof (BitVectorVertexEmbedding));
              extension->reset (v);
            }
          }
        }

        if (n_extensions[warp_id] != 0) {
          printf ("n_extensions[warp_id] not zero but is %d\n", n_extensions[warp_id]);
        }
        if (!__any_sync (__activemask (), predicate)) {
          break;
        }
        //n_extensions[warp_id] = 0;
      }
    }
  #else
    for (int u = 0; u < N; u++) {
      if (embedding->test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (BitVectorVertexEmbedding));
              memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (BitVectorVertexEmbedding));
            }
            extension->reset(v);
          }
        }
      }
    }
  #endif

  }

  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}*/


template <size_t size>
__host__ __device__
inline bool is_embedding_canonical (CSR* csr, VectorVertexEmbedding<size> const * embedding, int v)
{
  if (embedding->get_vertex (0) > v)
    return false;
  
  //if (size <= 2)
  //  return true;
  
  bool found_neighbor = false;
  for (int j = 0; j < embedding->get_n_vertices (); j++) {
    int v_j = embedding->get_vertex (j);
    if (found_neighbor == false && csr->has_edge (v_j, v)) {
      found_neighbor = true;
    } else if (found_neighbor == true && v_j > v) {
      return false;
    }
  }

  return true;
}

template <size_t embedding_size> 
__global__
void run_single_step_vectorvertex_embedding (void* input, int n_embeddings, CSR* csr,
                      void* output_ptr,
                      int* n_output,
                      void* next_step_1, int* n_next_step_1, volatile BUFFER_STATUS* buff_1_status, //Should be volatile because of the busy wait loop
                      void* next_step_2, int* n_next_step_2, volatile BUFFER_STATUS* buff_2_status, //Should be volatile because of the busy wait loop
                      volatile int* curr_step_storage_id, //Should be volatile because threads coordinate based on this value
                      int only_copy_change)
{
  int id;

#ifdef USE_CSR_IN_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread,
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread,
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#else
#ifdef USE_CONSTANT_MEM
  csr = (CSR*) csr_constant_buff;
#endif

#endif

  VectorVertexEmbedding<embedding_size>* embeddings = (VectorVertexEmbedding<embedding_size>*)input;
  
#ifdef USE_EMBEDDING_IN_LOCAL_MEM
  unsigned char temp_buffer [sizeof(VectorVertexEmbedding<embedding_size+1>)];
#endif
  id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
#ifdef USE_FIXED_THREADS
  if (n_embeddings >= MAX_CUDA_THREADS) {
    int embeddings_per_thread = n_embeddings/MAX_CUDA_THREADS+1;

    start = id*embeddings_per_thread;
    end = (id+1)*embeddings_per_thread < n_embeddings ? (id+1)*embeddings_per_thread : n_embeddings;
  } else {
    if (id >= n_embeddings)
      return;

    start = id;
    end = id+1;
  }
#else
  if (id >= n_embeddings)
      return;

  start = id;
  end = id+1;
#endif

  int q[1000] = {0};

#ifdef USE_EMBEDDING_IN_SHARED_MEM
//TODO: Support VectorVertexEmbedding
  #if 0
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    assert (per_thread_shared_mem_size >= sizeof (VectorVertexEmbedding<embedding_size>));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[per_thread_shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[0];
  #else
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
  #endif
#endif

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int lane_id = threadIdx.x % WARP_SIZE;
  for (int i = start; i < end; i++) {
    #ifdef USE_EMBEDDING_IN_SHARED_MEM
    //TODO: Support VectorVertexEmbedding, size+1
      #ifdef SHARED_MEM_NON_COALESCING
        memcpy (local_shared_buff, &embeddings[i], sizeof(VectorVertexEmbedding<embedding_size>));

        VectorVertexEmbedding<embedding_size>* embedding = (VectorVertexEmbedding<embedding_size>*) local_shared_buff;
      #else
        const int thread_block_size = WARP_SIZE;
        const int last_emb = WARP_SIZE*(warp_id+1);
        if (blockIdx.x*blockDim.x + (warp_id+1)*WARP_SIZE > n_embeddings) {
          assert (false);
          /*thread_block_size = n_embeddings - blockIdx.x*blockDim.x -
                              warp_id*WARP_SIZE;
          last_emb = warp_id*WARP_SIZE + thread_block_size;*/
        }

        for (int emb = WARP_SIZE*warp_id; emb < last_emb; emb++) {
          SharedMemElem* embedding_buff = (SharedMemElem*) &embeddings[emb+blockIdx.x*blockDim.x];

          for (int j = 0; j < sizeof (VectorVertexEmbedding<embedding_size>)/sizeof (SharedMemElem);
               j += thread_block_size) {
            int idx = per_thread_shared_mem_size/sizeof(SharedMemElem)*emb;
            if (true or j + lane_id  < sizeof (VectorVertexEmbedding<embedding_size>)/sizeof (SharedMemElem)) { //TODO: Remove this if by doing padding with VertexEmbedding
              shared_buff[idx + j + lane_id] = embedding_buff[j + lane_id];
            }
          }
        }

        VectorVertexEmbedding<embedding_size>* embedding = (VectorVertexEmbedding<embedding_size>*) local_shared_buff;
      #endif
      //embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_LOCAL_MEM)
      //memcpy (&temp[0], &embeddings[i], sizeof (VertexEmbedding));
      //memcpy (&temp_buffer[0], &embeddings[i], sizeof(VectorVertexEmbedding<embedding_size>));
      VectorVertexEmbedding<embedding_size+1>* embedding = (VectorVertexEmbedding<embedding_size+1>*)&temp_buffer[0];
      embedding->clear ();
      vector_embedding_from_one_less_size (embeddings[i], *embedding);
      //VertexEmbedding* embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_GLOBAL_MEM)
    //TODO: Support VectorVertexEmbedding with size + 1, below is wrong
      VectorVertexEmbedding<embedding_size+1>* embedding = &embeddings[i];
    #else
      #error "None of USE_EMBEDDING_IN_*_MEM option defined"
    #endif

  #if 0
  //TODO: Support VectorVertexEmbedding with size + 1.
    typedef uint32_t VertexEmbeddingChange;
    const uint32_t max_changes = 32; //max changes per warp
    //__shared__ int32_t shared_mem [max_changes*THREAD_BLOCK_SIZE/WARP_SIZE+THREAD_BLOCK_SIZE/WARP_SIZE+1];
    __shared__ VertexEmbeddingChange changes[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t changed_thread_ids[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    //For each warp record new embeddings (or changes) using atomic operations
    __shared__ int32_t n_extensions[THREAD_BLOCK_SIZE/WARP_SIZE];
    n_extensions[warp_id] = 0;
    __shared__ uint32_t prev_n_outputs [THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t prev_n_next_steps [THREAD_BLOCK_SIZE/WARP_SIZE];
    
    const uint32_t mask = ~0U;
    for (int u = 0; u < N; u++) {
      int e = csr->get_start_edge_idx(u);
      
      while (true) {
        
        int predicate = 1;
        //n_extensions[warp_id] = 0;

        if (e <= csr->get_end_edge_idx(u)) {
          int v = csr->get_edges () [e];
          if (embedding->test(u) and embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              int prev_n_extensions = atomicAdd (&n_extensions[warp_id], 1);
              changes[warp_id*max_changes+prev_n_extensions] = v;
              changed_thread_ids[warp_id*max_changes+prev_n_extensions] = id;
              //memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (VertexEmbedding));
              //memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (VertexEmbedding));
            }

            extension->reset(v);
          }

          e++;

          if (e > csr->get_end_edge_idx(u)) {
            predicate = 0;
          }
        } else {
          predicate = 0;
        }

        int32_t n_changes = n_extensions[warp_id];

        if (n_changes > max_changes)
          assert (false);

        //assert (__activemask () == ~0U);
        uint32_t orig_prev_n_output = 0;
        uint32_t orig_prev_n_next_step = 0;
        
        if (lane_id == 0) {
          if (n_changes > 0) {
            n_extensions[warp_id] = 0;
            orig_prev_n_output = atomicAdd (n_output, n_changes);
            orig_prev_n_next_step = atomicAdd (n_next_step, n_changes);
            //if (warp_id == 2 && (threadIdx.x == 64 || threadIdx.x == 65) && blockIdx.x == 0) {
            //  printf ("prev_n_next_steps[warp_id]: %d\n", prev_n_next_steps[warp_id]);
            //}
          }
        }
        
        //__syncwarp ();
        
        if (n_changes > 0) {
          
          uint32_t prev_n_output = __shfl_sync (__activemask (), orig_prev_n_output, 0);
          uint32_t prev_n_next_step = __shfl_sync (__activemask (), orig_prev_n_next_step, 0);
          
          for (int i = 0; i < n_changes; i++) {
            
            int changes_idx = warp_id*max_changes + i;
            uint32_t expected_thread_id = changed_thread_ids[changes_idx];
            if (expected_thread_id == id) {
              int v = changes[changes_idx];
              BitVectorVertexEmbedding* extension = embedding;
              extension->set (v);
              
              memcpy (&output[prev_n_output + i], extension, 
                      sizeof(BitVectorVertexEmbedding));
              memcpy (&new_embeddings[prev_n_next_step + i], extension,
                      sizeof (BitVectorVertexEmbedding));
              extension->reset (v);
            }
          }
        }

        if (n_extensions[warp_id] != 0) {
          printf ("n_extensions[warp_id] not zero but is %d\n", n_extensions[warp_id]);
        }
        if (!__any_sync (__activemask (), predicate)) {
          break;
        }
        //n_extensions[warp_id] = 0;
      }
    }
  #else
    for (int i = 0; i < embedding->get_n_vertices (); i++) {
      int u = embedding->get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        
        int v = csr->get_edges () [e];

        if (is_embedding_canonical<embedding_size+1> (csr, embedding, v) && embedding->has (v) == false) {
          VectorVertexEmbedding<embedding_size+1>* extension = embedding;
          extension->add_unsorted (v);
          
          if (clique_filter_vector (csr, extension)) {
            //VectorVertexEmbedding<embedding_size+1> extension = *embedding;
            //extension.add_last_in_sort_order ();
            //int o = atomicAdd(n_output,1);
            //int n = atomicAdd(n_next_step_1,1);
            
            if (only_copy_change) {
              int o = atomicAdd(n_output, 1);
              int n = atomicAdd(n_next_step_1, 1);
              int* new_embeddings = (int*) next_step_1;
              int* output = (int*) output_ptr;

              new_embeddings[2*n] = id;
              new_embeddings[2*n+1] = v;
              output[2*o] = id;
              output[2*o+1] = v;
            }
            else {
              int storage_id = *curr_step_storage_id;
              const size_t max_n_embeddings = NEW_EMBEDDING_BUFFER_SIZE/sizeof (VectorVertexEmbedding<embedding_size+1>);
              //const int storage_id = 0;
              int n = 0;
              switch (storage_id) {
                case 0: {
                  int o = atomicAdd(n_output, 1);
                  n = atomicAdd(n_next_step_1, 1);
                  //Switch from buff1 to buff2
                  while (n >= max_n_embeddings) {//TODO: change it to do-while 
                    if (*curr_step_storage_id == 0) {
                      n = atomicSub (n_next_step_1, 1); //TODO: can remove that
                      while (*buff_2_status == BUFFER_STATUS::CPU_COPYING) {
                        /*unsigned long i = 0;
                        while (i <= (1UL<<30)) {
                          i++;
                        }*/
                      }
                      *curr_step_storage_id = 1;
                      *buff_1_status = BUFFER_STATUS::CPU_COPYING;
                      *buff_2_status = BUFFER_STATUS::GPU_USING;
                      n = atomicAdd(n_next_step_2, 1);
                    } else {
                      n = atomicSub (n_next_step_2, 1); //TODO: can remove that
                      while (*buff_1_status == BUFFER_STATUS::CPU_COPYING) {
                        /*unsigned long i = 0;
                        while (i <= (1UL<<30)) {
                          i++;
                        }*/
                      }
                      *curr_step_storage_id = 0;
                      *buff_2_status = BUFFER_STATUS::CPU_COPYING;
                      *buff_1_status = BUFFER_STATUS::GPU_USING;
                      n = atomicAdd(n_next_step_1, 1);
                    }
                  }
                  
                  if (*curr_step_storage_id == 1) {
                    //n = atomicAdd (n_next_step_2, 1);
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_2;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    //memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  } else {
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_1;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    //memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  }
                  break;
                }

                case 1: {
                  int o = atomicAdd(n_output, 1);
                  n = atomicAdd(n_next_step_2, 1);
                  if (n >= max_n_embeddings) {
                    //Switch from buff2 to buff1
                    atomicSub (n_next_step_2, 1); //TODO: can remove that
                    *curr_step_storage_id = 0;
                    *buff_2_status = BUFFER_STATUS::CPU_COPYING;
                    while (*buff_1_status == BUFFER_STATUS::CPU_COPYING) {
                      /*unsigned long i = 0;
                      while (i <= (1UL<<30)) {
                        i++;
                      }*/
                    }
                    *buff_1_status = BUFFER_STATUS::GPU_USING;
                    n = atomicAdd (n_next_step_1, 1);
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_1;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    //memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  } else {
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_2;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    //memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  }
                }
              }
            }
            //output[o].add_last_in_sort_order ();
            //new_embeddings[n].add_last_in_sort_order ();
          }
          extension->remove_last ();
        }
      }
    }
  #endif
  }

  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}

__host__ __device__ struct PairEmbIdxVertexIdx {
  int emb_idx;
  int vertex_idx;

  PairEmbIdxVertexIdx (long _emb_idx, int _vertex_idx) {
    emb_idx = _emb_idx;
    vertex_idx = _vertex_idx;
  }
};

template <size_t embedding_size> 
__global__
void run_single_step_vectorvertex_embedding_per_vertex (void* input, int n_embeddings, CSR* csr,
                      void* embs_per_parts, 
                      void* output_ptr,
                      int* n_output,
                      void* next_step_1, int* n_next_step_1, volatile BUFFER_STATUS* buff_1_status, //Should be volatile because of the busy wait loop
                      void* next_step_2, int* n_next_step_2, volatile BUFFER_STATUS* buff_2_status, //Should be volatile because of the busy wait loop
                      volatile int* curr_step_storage_id, //Should be volatile because threads coordinate based on this value
                      int* n_next_step_1_done, int* n_next_step_2_done, 
                      int only_copy_change)
{
  int id;

#ifdef USE_CSR_IN_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread,
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread,
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#else
#ifdef USE_CONSTANT_MEM
  csr = (CSR*) csr_constant_buff;
#endif

#endif

  VectorVertexEmbedding<embedding_size>* embeddings = (VectorVertexEmbedding<embedding_size>*)input;
  PairEmbIdxVertexIdx* embeddings_per_partitions = (PairEmbIdxVertexIdx*) embs_per_parts;

id = blockIdx.x*blockDim.x + threadIdx.x;

#if defined(EMBEDDING_IN_SHARED_MEM_PER_VERTEX)
//TODO: Support VectorVertexEmbedding
  #if 0
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    assert (per_thread_shared_mem_size >= sizeof (VectorVertexEmbedding<embedding_size>));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[per_thread_shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[0];
  #else
    const int shared_mem_size = THREAD_BLOCK_SIZE*sizeof(VectorVertexEmbedding<embedding_size+1>);
    //std::static_assert (shared_mem_size <= 49152);
    assert (shared_mem_size <= 49152);
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[shared_mem_size];///sizeof (SharedMemElem)];

    /*SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
    memcpy (local_shared_buff, &embeddings[id], embedding_size);
    __syncthreads ();
    */
  #endif
#else 
#  if defined(ALL_THREAD_BLOCK_EMBEDDINGS_IN_SHARED_MEM_PER_VERTEX)
  const int shared_mem_size = THREAD_BLOCK_SIZE*sizeof(VectorVertexEmbedding<embedding_size>);
  //std::static_assert (shared_mem_size <= 49152);
  assert (shared_mem_size <= 49152);
  const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

  //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
  //per_thread_shared_mem_size = sizeof (VertexEmbedding);
  __shared__ SharedMemElem shared_buff[shared_mem_size];

  SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
  memcpy (local_shared_buff, &embeddings[id], sizeof (VectorVertexEmbedding<embedding_size>));
  __syncthreads ();

#  else //#ifdef USE_EMBEDDING_IN_LOCAL_MEM
  unsigned char temp_buffer [sizeof(VectorVertexEmbedding<embedding_size>)];
#  endif
#endif
  const int warp_id = threadIdx.x / WARP_SIZE;
  const int lane_id = threadIdx.x % WARP_SIZE;
  
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
#ifndef EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK 
  if (id >= n_embeddings)
      return;
#endif

  int thread_block_start_idx = blockIdx.x*blockDim.x * embedding_size;
  const bool enable_edge_pulling = false;
  for (int load = 0; load < embedding_size; load++) {
    //printf ("load %d embedding_size %d\n", load, embedding_size);
    int load_ids[2];
    
#if defined (EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK)
    load_ids[0] = thread_block_start_idx + threadIdx.x + load*blockDim.x;
    load_ids[1] = thread_block_start_idx + embedding_size*THREAD_BLOCK_SIZE - 1 - (threadIdx.x + load*blockDim.x);
    
    //blockIdx.x*blockDim.x + load*n_embeddings + (load+1)*blockDim.x - 1 - threadIdx.x;
    /*if (load == 0)
      printf ("P1 id: %d blockIdx: %d threadIdx: %d load_ids[0]: %d load: %d\n",
            id, blockIdx.x, threadIdx.x, load_ids[0], load);
            */
    if (load_ids[0] >= n_embeddings*embedding_size)
            return;
    /*if (load == 0)
      printf ("P2 id: %d blockIdx: %d threadIdx: %d load_ids[0]: %d load: %d\n",
            id, blockIdx.x, threadIdx.x, load_ids[0], load);
    */
#else
    load_ids[0] = id + load*n_embeddings;
    load_ids[1] = (load+1)*n_embeddings - 1 - id;
#endif
    //printf ("t-id %d ; load_ids[0] %d ; load_ids[1] %d\n", id, load_ids[0], load_ids[1]);
    int n_loads;
#if defined (EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK)
    if (enable_edge_pulling && threadIdx.x + load*blockDim.x < embedding_size*THREAD_BLOCK_SIZE/2 && 
        load_ids[1] < n_embeddings*embedding_size) {
#else
    if (enable_edge_pulling && id < n_embeddings/2) {
#endif

      //Vertices are sorted in increasing order. a vertex in first half will pull edges from 
      //vertex in second half.
      n_loads = 2;
    } else {
      n_loads = 1;
    }

    for (int i = 0; i < n_loads; i++) {
#if defined (EMBEDDING_IN_SHARED_MEM_PER_VERTEX)
      int emb_idx = embeddings_per_partitions[load_ids[i]].emb_idx;
      VectorVertexEmbedding<embedding_size>* embedding = &((VectorVertexEmbedding<embedding_size>*)&shared_buff[0])[threadIdx.x];
      memcpy (embedding, &embeddings[embeddings_per_partitions[load_ids[i]].emb_idx], sizeof (VectorVertexEmbedding<embedding_size>));
      //embedding->clear ();
      //vector_embedding_from_one_less_size (embeddings[embeddings_per_partitions[load_ids[i]].emb_idx], *embedding);
#else
#  if defined (ALL_THREAD_BLOCK_EMBEDDINGS_IN_SHARED_MEM_PER_VERTEX)
      int emb_idx = embeddings_per_partitions[load_ids[i]].emb_idx;
      VectorVertexEmbedding<embedding_size> const * embedding = &((VectorVertexEmbedding<embedding_size>*)&shared_buff[0])[emb_idx%256];
      //embedding->clear ();
      //vector_embedding_from_one_less_size (embeddings[emb_idx], *embedding);
#  else
      VectorVertexEmbedding<embedding_size>* embedding = (VectorVertexEmbedding<embedding_size>*)&temp_buffer[0];
      embedding->clear ();
      memcpy (embedding, &embeddings[embeddings_per_partitions[load_ids[i]].emb_idx], sizeof (VectorVertexEmbedding<embedding_size>));
#  endif
#endif

      int u = embedding->get_vertex (embeddings_per_partitions[load_ids[i]].vertex_idx);
      int start_edge_idx;
      int end_edge_idx;

      if (enable_edge_pulling && i == 1) {
        //Can only happen when vertex is in first half
        assert (n_loads == 2);
        int v = embeddings[embeddings_per_partitions[load_ids[0]].emb_idx].get_vertex (embeddings_per_partitions[load_ids[0]].vertex_idx);
        int first_n_edges = csr->get_end_edge_idx (v) - csr->get_start_edge_idx (v) + 1;
        int second_n_edges = csr->get_end_edge_idx (u) - csr->get_start_edge_idx (u) + 1;
        /*if (!(first_n_edges <= second_n_edges)) {
          printf ("id: %d blockIdx: %d threadIdx: %d load_ids[0]: %d load_idx[1]: %d first %d second %d\n",
          id, blockIdx.x, threadIdx.x, load_ids[0], load_ids[1], first_n_edges, second_n_edges);
        }*/
        assert (first_n_edges <= second_n_edges);
        int total_edges_to_process = (second_n_edges + first_n_edges)/2;
        int remaining_edges = total_edges_to_process - first_n_edges; //first n edges have already been done
        start_edge_idx = csr->get_end_edge_idx (u) - remaining_edges + 1;
        end_edge_idx = csr->get_end_edge_idx (u);
      } else {
        if (enable_edge_pulling && n_loads == 1 && load_ids[1] < n_embeddings*embedding_size) {
          //Can only happen when vertex is in second half
          int v = embeddings[embeddings_per_partitions[load_ids[1]].emb_idx].get_vertex (embeddings_per_partitions[load_ids[1]].vertex_idx);
          int first_n_edges = csr->get_end_edge_idx(u) - csr->get_start_edge_idx(u) + 1;
          int second_n_edges = csr->get_end_edge_idx(v) - csr->get_start_edge_idx(v) + 1;
          if (!(first_n_edges >= second_n_edges)) {
            printf ("id: %d blockIdx: %d threadIdx: %d load_ids[0]: %d load_idx[1]: %d first %d second %d\n",
            id, blockIdx.x, threadIdx.x, load_ids[0], load_ids[1], first_n_edges, second_n_edges);
          }
          assert (first_n_edges >= second_n_edges);
          int edges_to_process = ((second_n_edges + first_n_edges) % 2 == 0) ? (second_n_edges + first_n_edges)/2 : (second_n_edges + first_n_edges)/2+1;
          //int edges_to_process = (second_n_edges + first_n_edges)/2;
          start_edge_idx = csr->get_start_edge_idx (u);
          end_edge_idx = start_edge_idx + edges_to_process - 1;
        }
        else {
          //Can only happen when vertex is in first half
          start_edge_idx = csr->get_start_edge_idx(u);
          end_edge_idx = csr->get_end_edge_idx(u);
        }
      }

      for (int e = start_edge_idx; e <= end_edge_idx; e++) {
        
        int v = csr->get_edges () [e];

        if (is_embedding_canonical<embedding_size> (csr, embedding, v) && 
            ((VectorVertexEmbedding<embedding_size>*)embedding)->has (v) == false) { //TODO: Make both these checks in same loop
          VectorVertexEmbedding<embedding_size> const * extension = embedding;
          //extension->add_unsorted (v);
          
          if (clique_filter_vector_optimized (csr, extension, v)) {
            //VectorVertexEmbedding<embedding_size+1> extension = *embedding;
            //extension.add_last_in_sort_order ();
            //int o = atomicAdd(n_output,1);
            //int n = atomicAdd(n_next_step_1,1);
            
            if (only_copy_change) {
              int o = atomicAdd(n_output, 1);
              int n = atomicAdd(n_next_step_1, 1);
              int* new_embeddings = (int*) next_step_1;
              int* output = (int*) output_ptr;

              new_embeddings[2*n] = id;
              new_embeddings[2*n+1] = v;
              output[2*o] = id;
              output[2*o+1] = v;
            }
            else {
              int storage_id = *curr_step_storage_id;
              /*assert (! (*buff_1_status == BUFFER_STATUS::CPU_COPYING) && (*buff_2_status == BUFFER_STATUS::CPU_COPYING));
              assert (! (*buff_1_status == BUFFER_STATUS::GPU_USING) && (*buff_2_status == BUFFER_STATUS::GPU_USING));
              if ((*buff_1_status == BUFFER_STATUS::FREE) && (*buff_2_status == BUFFER_STATUS::FREE)) {
                storage_id = 0;
              } else {
                
              }*/
              
              const size_t max_n_embeddings = NEW_EMBEDDING_BUFFER_SIZE/sizeof (VectorVertexEmbedding<embedding_size+1>);
              //const int storage_id = 0;
              int n = 0;
              switch (storage_id) {
                case 0: {
                  n = atomicAdd(n_next_step_1, 1);
                  //Switch from buff1 to buff2
                  while (n >= max_n_embeddings) {//TODO: change it to do-while 
                    if (storage_id == 0) {
                      n = atomicSub (n_next_step_1, 1); //TODO: can remove that
                      while (*buff_2_status == BUFFER_STATUS::CPU_COPYING) {
                        /*unsigned long i = 0;
                        while (i <= (1UL<<30)) {
                          i++;
                        }*/
                      }
                      *curr_step_storage_id = 1;
                      storage_id = 1;
                      *buff_2_status = BUFFER_STATUS::GPU_USING;
                      n = atomicAdd(n_next_step_2, 1);
                    } else {
                      n = atomicSub (n_next_step_2, 1); //TODO: can remove that
                      while (*buff_1_status == BUFFER_STATUS::CPU_COPYING) {
                        /*unsigned long i = 0;
                        while (i <= (1UL<<30)) {
                          i++;
                        }*/
                      }
                      *curr_step_storage_id = 0;
                      storage_id = 0;
                      
                      *buff_1_status = BUFFER_STATUS::GPU_USING;
                      n = atomicAdd(n_next_step_1, 1);
                    }
                  }
                  
                  if (storage_id == 1) {
                    //n = atomicAdd (n_next_step_2, 1);
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_2;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    #ifdef ADD_TO_OUTPUT
                    int o = atomicAdd(n_output, 1);
                    memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    #endif
                    //memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    vector_embedding_from_one_less_size (*extension, new_embeddings[n]);
                    new_embeddings[n].add_unsorted (v);
                    if (atomicAdd ((int*)n_next_step_2_done, 1) >= max_n_embeddings) {
                      *buff_2_status = BUFFER_STATUS::CPU_COPYING;
                    }
                  } else {
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_1;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    #ifdef ADD_TO_OUTPUT
                    int o = atomicAdd(n_output, 1);
                    memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    #endif
                    //memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    vector_embedding_from_one_less_size (*extension, new_embeddings[n]);
                    new_embeddings[n].add_unsorted (v);
                    if (atomicAdd ((int*)n_next_step_1_done, 1) >= max_n_embeddings) {
                      *buff_1_status = BUFFER_STATUS::CPU_COPYING;
                    }
                  }
                  break;
                }

                case 1: {
                  n = atomicAdd(n_next_step_2, 1);

                  while (n >= max_n_embeddings) {//TODO: change it to do-while 
                    if (storage_id == 0) {
                      n = atomicSub (n_next_step_1, 1); //TODO: can remove that
                      
                      while (*buff_2_status == BUFFER_STATUS::CPU_COPYING) {
                        /*unsigned long i = 0;
                        while (i <= (1UL<<30)) {
                          i++;
                        }*/
                      }
                      *buff_2_status = BUFFER_STATUS::GPU_USING;
                      *curr_step_storage_id = 1;
                      storage_id = 1;
                      //*buff_1_status = BUFFER_STATUS::CPU_COPYING;
                      n = atomicAdd(n_next_step_2, 1);
                    } else {
                      n = atomicSub (n_next_step_2, 1); //TODO: can remove that
                      while (*buff_1_status == BUFFER_STATUS::CPU_COPYING) {
                        /*unsigned long i = 0;
                        while (i <= (1UL<<30)) {
                          i++;
                        }*/
                      }
                      *curr_step_storage_id = 0;
                      storage_id = 0;
                      //*buff_2_status = BUFFER_STATUS::CPU_COPYING;
                      *buff_1_status = BUFFER_STATUS::GPU_USING;
                      n = atomicAdd(n_next_step_1, 1);
                    }
                  }
                  
                  if (storage_id == 1) {
                    //n = atomicAdd (n_next_step_2, 1);
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_2;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    #ifdef ADD_TO_OUTPUT
                    int o = atomicAdd(n_output, 1);
                    memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    #endif
                    //memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    vector_embedding_from_one_less_size (*extension, new_embeddings[n]);
                    new_embeddings[n].add_unsorted (v);
                    if (atomicAdd ((int*)n_next_step_2_done, 1) >= max_n_embeddings) {
                      *buff_2_status = BUFFER_STATUS::CPU_COPYING;
                    }
                  } else {
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_1;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    #ifdef ADD_TO_OUTPUT
                    int o = atomicAdd(n_output, 1);
                    memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    #endif
                    //memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    vector_embedding_from_one_less_size (*extension, new_embeddings[n]);
                    new_embeddings[n].add_unsorted (v);
                    if (atomicAdd ((int*)n_next_step_1_done, 1) >= max_n_embeddings) {
                      *buff_1_status = BUFFER_STATUS::CPU_COPYING;
                    }
                  }

                  // if (n >= max_n_embeddings) {
                  //   //Switch from buff2 to buff1
                  //   atomicSub (n_next_step_2, 1); //TODO: can remove that
                  //   *curr_step_storage_id = 0;
                  //   *buff_2_status = BUFFER_STATUS::CPU_COPYING;
                  //   while (*buff_1_status == BUFFER_STATUS::CPU_COPYING) {
                  //     /*unsigned long i = 0;
                  //     while (i <= (1UL<<30)) {
                  //       i++;
                  //     }*/
                  //   }
                  //   *buff_1_status = BUFFER_STATUS::GPU_USING;
                  //   n = atomicAdd (n_next_step_1, 1);
                  //   VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_1;
                  //   VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                  //   #ifdef ADD_TO_OUTPUT
                  //   int o = atomicAdd(n_output, 1);
                  //   memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  //   #endif
                  //   //memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  //   vector_embedding_from_one_less_size (*extension, new_embeddings[n]);
                  //   new_embeddings[n].add_unsorted (v);
                  // } else {
                  //   VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_2;
                  //   VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                  //   #ifdef ADD_TO_OUTPUT
                  //   int o = atomicAdd(n_output, 1);
                  //   memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  //   #endif
                  //   //memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  //   vector_embedding_from_one_less_size (*extension, new_embeddings[n]);
                  //   new_embeddings[n].add_unsorted (v);
                  // }
                }
              }
            }
            //output[o].add_last_in_sort_order ();
            //new_embeddings[n].add_last_in_sort_order ();
          }
          //extension->remove_last ();
        }
      }
    }
  }

  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}


void print_embedding (BitVectorVertexEmbedding embedding, std::ostream& os)
{
  os << "[";
  for (int u = 0; u < N; u++) {
    if (embedding.test(u)) {
      os << u << ", ";
    }
  }
  os << "]";
}

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

double_t convertTimeValToDouble (struct timeval _time)
{
  return ((double_t)_time.tv_sec) + ((double_t)_time.tv_usec)/1000000.0f;
}

struct timeval getTimeOfDay ()
{
  struct timeval _time;

  if (gettimeofday (&_time, NULL) == -1) {
    fprintf (stderr, "gettimeofday returned -1\n");
    perror ("");
    abort ();
  }

  return _time;
}

enum EmbeddingType {
  VectorVertex,
  BitVector,
};

class Partition
{
private:
  int start_vertex;
  int end_vertex;
  int id;
  int start_idx;
  int end_idx;

public:
  Partition (int s, int e) : start_vertex (s), end_vertex (e)
  {}
  
};

size_t get_emb_per_parts_size (size_t n_embeddings, int vertices_per_embedding) 
{
  return vertices_per_embedding * n_embeddings * sizeof(PairEmbIdxVertexIdx);
}

template <uint size>
void collect_vertex_embeddings_in_partitions (CSR* csr, VectorVertexEmbedding<size>* embeddings, size_t start_idx, 
                                              size_t n_embeddings,
                                              std::vector<std::vector<int>>& partitions, std::vector<int>& vertex_to_partition,
                                              PairEmbIdxVertexIdx* embs_for_parts, size_t embs_for_parts_size)
{
  std::vector<int> embs_per_parts_size(partitions.size (), 0);
  
  for (int i = start_idx; i < start_idx + n_embeddings; i++) {
    VectorVertexEmbedding<size>& emb = embeddings[i];
    //print_embedding<size> (&emb);
    //std::cout << std::endl;
    for (int v_i = 0; v_i < size; v_i++) {
      int v = emb.get_vertex (v_i);
      int part_idx = vertex_to_partition [v];
      embs_per_parts_size[part_idx]++;
    }
  }

  /*for (int i = 0; i < embs_per_parts_size.size (); i++) {
    std::cout << i << ":" << embs_per_parts_size[i] << std::endl;
  }*/
  std::vector<int> parts_pushed_idx (partitions.size (), 0);
  std::vector<int> part_idx_in_embs_for_parts (partitions.size (), 0);
  for (int i = 1; i < embs_per_parts_size.size (); i++) {
    part_idx_in_embs_for_parts[i] = part_idx_in_embs_for_parts[i-1] + embs_per_parts_size[i-1];
  }

  /*for (int i = 0; i < embs_per_parts_size.size (); i++) {
    std::cout << "part_idx_in_embs_for_parts " << i << ":" << part_idx_in_embs_for_parts[i] << std::endl;
  }*/

  //std::cout << "embs_for_parts_size " << embs_for_parts_size <<  ":" << get_emb_per_parts_size (n_embeddings, size) << std::endl;
  assert (get_emb_per_parts_size (n_embeddings, size) == embs_for_parts_size);

  for (int i = start_idx; i < start_idx + n_embeddings; i++) {
    VectorVertexEmbedding<size>& emb = embeddings[i];
    for (int v_i = 0; v_i < size; v_i++) {
      int v = emb.get_vertex (v_i);
      int part_idx = vertex_to_partition [v];
      embs_for_parts[part_idx_in_embs_for_parts[part_idx] + parts_pushed_idx[part_idx]] = PairEmbIdxVertexIdx (i, v_i);
      parts_pushed_idx[part_idx]++;
    }
  }
}

template <uint size>
void collect_vertex_embeddings_in_partitions_per_threadblock (CSR* csr, VectorVertexEmbedding<size>* embeddings, size_t n_embeddings,
                                                              std::vector<std::vector<int>>& partitions, std::vector<int>& vertex_to_partition,
                                                              PairEmbIdxVertexIdx* embs_for_parts, size_t embs_for_parts_size)
{
  int n_threadblocks = n_embeddings%THREAD_BLOCK_SIZE == 0 ? n_embeddings/THREAD_BLOCK_SIZE : n_embeddings/THREAD_BLOCK_SIZE + 1;
  size_t embs_for_parts_size_per_threadblock = get_emb_per_parts_size (THREAD_BLOCK_SIZE, size);
  size_t embs_for_parts_processed = 0;
  size_t n_embeddings_processed = 0;
  for (int i = 0; i < ((n_embeddings%THREAD_BLOCK_SIZE == 0) ? n_threadblocks : n_threadblocks - 1); i++) {
    collect_vertex_embeddings_in_partitions (csr, embeddings, n_embeddings_processed, 
                                             THREAD_BLOCK_SIZE, partitions, vertex_to_partition,
                                             embs_for_parts + embs_for_parts_processed, 
                                             embs_for_parts_size_per_threadblock);
    //std::cout << "embs_for_parts_processed " << embs_for_parts_processed << std::endl;
    embs_for_parts_processed += embs_for_parts_size_per_threadblock/sizeof (PairEmbIdxVertexIdx);
    n_embeddings_processed += THREAD_BLOCK_SIZE;    
  }

  if (n_embeddings%THREAD_BLOCK_SIZE != 0) {
    size_t remaining = n_embeddings - n_embeddings_processed;
    collect_vertex_embeddings_in_partitions (csr, embeddings, n_embeddings_processed, 
                                             remaining, partitions, vertex_to_partition,
                                             embs_for_parts + embs_for_parts_processed, 
                                             get_emb_per_parts_size (remaining, size));
    embs_for_parts_processed += embs_for_parts_size_per_threadblock;
  }


  //for (int i = 0; i < n_embeddings*size; i++) {
  //  PairEmbIdxVertexIdx p = embs_for_parts[i];
  //  printf ("%d: [%d, %d]:\n", i, p.emb_idx, p.vertex_idx);
    //VectorVertexEmbedding<2>* emb = &((VectorVertexEmbedding<2>*)input_embeddings_ptr)[p.emb_idx];
    //print_embedding<2> (emb);
    //std::cout << std::endl;
  //}
}

int main (int argc, char* argv[])
{
  std::vector<Vertex> vertices;
  int n_edges = 0;

  if (argc < 2) {
    std::cout << "Arguments: graph-file" << std::endl;
    return -1;
  }

  char* graph_file = argv[1];
  FILE* fp = fopen (graph_file, "r+");
  if (fp == nullptr) {
    std::cout << "File '" << graph_file << "' not found" << std::endl;
    return 1;
  }

  while (true) {
    char line[LINE_SIZE];
    char num_str[LINE_SIZE];
    size_t line_size;

    if (fgets (line, LINE_SIZE, fp) == nullptr) {
      break;
    }

    int id, label;
    int bytes_read;

    bytes_read = sscanf (line, "%d %d", &id, &label);
    Vertex vertex (id, label);
    char* _line = line + chars_in_int (id) + chars_in_int (label);
    do {
      int num;

      bytes_read = sscanf (_line, "%d", &num);
      if (bytes_read > 0) {
        vertex.add_edge (num);
        _line += chars_in_int (num);
        n_edges++;
      }

    } while (bytes_read > 0);

    vertex.sort_edges ();

    vertices.push_back (vertex);
  }

  fclose (fp);

  std::cout << "n_edges "<<n_edges <<std::endl;
  std::cout << "vertices " << vertices.size () << std::endl; 

#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
  std::cout << "Sorting " << std::endl;
  std::vector <Vertex> old_vertices = vertices;
  std::vector <int> new_to_old_vertex_ids;
  std::vector <int> old_to_new_vertex_ids;
  for (size_t i = 0; i < vertices.size (); i++) {
    new_to_old_vertex_ids.push_back (vertices[i].get_id ());
    old_to_new_vertex_ids.push_back (vertices[i].get_id ());
  }

  std::sort (vertices.begin (), vertices.end (), Vertex::compare_vertex);
  assert (Vertex::compare_vertex(vertices[0], vertices[vertices.size () - 1]));
  for (size_t i = 0; i < vertices.size (); i++) {
    int old_id = vertices[i].get_id ();
    int new_id = i;
    vertices[i].set_id (new_id);
    new_to_old_vertex_ids[new_id] = old_id;
    old_to_new_vertex_ids[old_id] = new_id;
  }

  for (size_t i = 0; i < vertices.size (); i++) {
    std::vector <int>& edges = vertices[i].get_edges ();
    for (size_t j = 0; j < edges.size (); j++) {
      edges[j] = old_to_new_vertex_ids[edges[j]];
    }
  }

  int partition_size = 1;
  assert (vertices.size () % partition_size == 0);
  int num_partition = vertices.size ()/partition_size;
  std::vector <std::vector <int> > partitions;
  std::vector <int> vertex_to_partition (vertices.size (), 0);

  for (int i = 0; i < vertices.size (); i++) {
    partitions.push_back (std::vector<int> {i});
    vertex_to_partition[i] = i;
  }
#endif

  Graph graph (vertices, n_edges);

  CSR* csr = new CSR(N, N_EDGES);
  std::cout << "sizeof(CSR)"<< sizeof(CSR)<<std::endl;
  std::cout <<"sizeof(VertexEmbedding)" << sizeof(BitVectorVertexEmbedding) << std::endl;
  csr_from_graph (csr, graph);
  
#ifdef USE_CONSTANT_MEM
  cudaMemcpyToSymbol (csr_constant_buff, csr, sizeof(CSR));
  //~ CSR* csr_constant = (CSR*) &csr_constant_buff[0];
  //~ csr_constant->n_vertices = csr->get_n_vertices ();
  //~ printf ("csr->get_n_vertices () = %d\n", csr->get_n_vertices ());
  //~ csr_constant->n_edges = csr->get_n_edges ();
  //~ csr_constant->copy_vertices (csr, 0, csr->get_n_vertices ());
  //~ csr_constant->copy_edges (csr, 0, csr->get_n_edges ());
#endif

  std::vector<VectorVertexEmbedding<0>> initial_embeddings = get_initial_embedding_vector (csr);
  std::vector<VectorVertexEmbedding<1>> output_1;
  std::vector<VectorVertexEmbedding<2>> output_2;
  std::vector<VectorVertexEmbedding<3>> output_3;
  std::vector<VectorVertexEmbedding<4>> output_4;
  std::vector<VectorVertexEmbedding<5>> output_5;
  std::vector<VectorVertexEmbedding<6>> output_6;
  std::vector<VectorVertexEmbedding<7>> output_7;
  std::vector<VectorVertexEmbedding<8>> output_8;
  std::vector<std::pair<void*, size_t>> embeddings;

  //filter = clique_filter;
  //process = clique_process;
  size_t new_embeddings_size = 0;
  int iter = 0;
  {
    std::vector<VectorVertexEmbedding<1>> new_embeddings;
    run_single_step_initial_vector (&initial_embeddings[0], 1, csr, 
                                    output_1, new_embeddings);
    new_embeddings_size = new_embeddings.size ();
    embeddings.push_back (std::make_pair (malloc (sizeof (VectorVertexEmbedding<1>)*new_embeddings_size), new_embeddings_size));
    for (int i = 0; i < new_embeddings_size; i++) {
      ((VectorVertexEmbedding<1>*)embeddings[0].first)[i] = new_embeddings[i];
      int v = ((VectorVertexEmbedding<1>*)embeddings[0].first)[i].get_vertex (0);
      assert (v >= 0);
    }
  }

  iter = 1;
  double total_stream_time = 0;
  size_t global_mem_size = 15*1024*1024*1024UL;
#define PINNED_MEMORY
#ifdef PINNED_MEMORY
  char* global_mem_ptr;
  hipError_t malloc_error = hipHostMalloc ((void**)&global_mem_ptr, global_mem_size, hipHostMallocDefault);
  assert (malloc_error == hipSuccess);
#else
  char* global_mem_ptr = new char[global_mem_size];
#endif

  const size_t max_embedding_size_per_iter = (12000000/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
  double_t kernelTotalTime = 0.0;
  for (iter; iter < 8 && new_embeddings_size > 0; iter++) {
    std::cout << "iter " << iter << " embeddings " << new_embeddings_size << std::endl;
    
    size_t remaining_embeddings = new_embeddings_size;
    size_t n_embeddings = new_embeddings_size;
    #ifdef DEBUG
      memset (global_mem_ptr, 0, global_mem_size);
    #endif

    //Copy all embeddings to global memory
    size_t embedding_size = 0;
    size_t new_embedding_size = 0;
    size_t global_mem_iter = 0;
    switch (iter) {
      case 1: {
        embedding_size = sizeof (VectorVertexEmbedding<1>);
        new_embedding_size = sizeof (VectorVertexEmbedding<2>);
        
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<1>*)global_mem_ptr)[global_mem_iter] = ((VectorVertexEmbedding<1>*) iter.first)[i];
            int v = ((VectorVertexEmbedding<1>*)global_mem_ptr)[global_mem_iter].get_vertex (0);
            global_mem_iter++;
            assert (v >= 0);
          }
        }
        break;
      }      
      case 2: {
        embedding_size = sizeof (VectorVertexEmbedding<2>);
        new_embedding_size = sizeof (VectorVertexEmbedding<3>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<2>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<2>*)iter.first)[i];
          }
        }
        break;
      }
      
      case 3: {
        embedding_size = sizeof (VectorVertexEmbedding<3>);
        new_embedding_size = sizeof (VectorVertexEmbedding<4>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<3>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<3>*)iter.first)[i];
          }
        }
        break;
      }
      
      case 4: {
          embedding_size = sizeof (VectorVertexEmbedding<4>);
          new_embedding_size = sizeof (VectorVertexEmbedding<5>);
          for (auto iter: embeddings) {
            for (int i = 0; i < iter.second; i++) {
              ((VectorVertexEmbedding<4>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<4>*)iter.first)[i];
            }
          }
        break;
      }
      case 5: {
        embedding_size = sizeof (VectorVertexEmbedding<5>);
        new_embedding_size = sizeof (VectorVertexEmbedding<6>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<5>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<5>*)iter.first)[i];
          }
        }
        break;
      }
      case 6: {
        embedding_size = sizeof (VectorVertexEmbedding<6>);
        new_embedding_size = sizeof (VectorVertexEmbedding<7>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<6>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<6>*)iter.first)[i];
          }
        }
        break;
      }
      case 7: {
        embedding_size = sizeof (VectorVertexEmbedding<7>);
        new_embedding_size = sizeof (VectorVertexEmbedding<8>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<7>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<7>*)iter.first)[i];
          }
        }
        break;
      }
      case 8: {
        embedding_size = sizeof (VectorVertexEmbedding<8>);
        new_embedding_size = sizeof (VectorVertexEmbedding<9>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<8>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<8>*)iter.first)[i];
          }
        }
        break;
      }
    }

    //delete embeddings too because there is a memory leak?
    if (iter > 1) {
      for (auto iter: embeddings) {
        free(iter.first);
      }
    }

    embeddings.clear ();
    std::cout << "Copying to global_mem_ptr done. global mem used " << global_mem_iter*embedding_size << std::endl;
    
    void* embeddings_ptr = global_mem_ptr;

    size_t n_next_step_embeddings = 0;
    n_embeddings = 0;

    void* orig_new_embeddings_ptr = ((char*)global_mem_ptr) + (global_mem_iter)*(new_embedding_size); //Size of next embedding will be one more
    //size_t max_embeddings = 40000000; //There is something with this value which makes it perform better, may be alignment?
    size_t max_embeddings = NEW_EMBEDDING_BUFFER_SIZE/(new_embedding_size);
    printf ("new_embedding_size %ld\n", new_embedding_size);
    void* orig_output_ptr = (char*)orig_new_embeddings_ptr + (max_embeddings)*(new_embedding_size);
    size_t orig_outputs_size = max_embeddings*(new_embedding_size);
    hipError_t error;
    double stream_time_1 = convertTimeValToDouble (getTimeOfDay ());
    
    while (remaining_embeddings != 0) {
      n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
      std::cout << "iter " << iter << " n_embeddings " << new_embeddings_size << " remaining_embeddings " << remaining_embeddings << std::endl;
      embeddings_ptr = ((char*)global_mem_ptr) + embedding_size*(new_embeddings_size - remaining_embeddings);
      //printf ("embeddings_ptr %x\n", embeddings_ptr);
      n_embeddings = std::min (remaining_embeddings, max_embedding_size_per_iter);

      remaining_embeddings -= n_embeddings;
      //n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
      
      const int N_STREAMS = 1;
      //assert (max_embeddings/N_STREAMS >= 30000000);
      int only_copy_change = 0;
      assert (only_copy_change == 0); //TODO: Streams with only copy change
      void* new_embeddings_ptr[N_STREAMS];
      assert (max_embeddings%N_STREAMS == 0);
      for (int i = 0; i < N_STREAMS; i++) {
        new_embeddings_ptr[i] = (char*)orig_new_embeddings_ptr + i*new_embedding_size*max_embeddings/N_STREAMS;
      }

      void* output_ptr[N_STREAMS];
      for (int i = 0; i < N_STREAMS; i++) {
        output_ptr[i] = (char*)orig_output_ptr + i*orig_outputs_size/N_STREAMS;
      }

      int n_new_embeddings_1[N_STREAMS] = {0};
      int n_new_embeddings_2[N_STREAMS] = {0};
      int n_output[N_STREAMS] = {0};
      int n_output_1[N_STREAMS] = {0};
      int curr_new_embeddings_idx[N_STREAMS] = {0};
      char* device_embeddings[N_STREAMS];
      char *device_new_embeddings_1[N_STREAMS];
      char *device_new_embeddings_2[N_STREAMS];
      BUFFER_STATUS *device_new_embeddings_1_status[N_STREAMS];
      BUFFER_STATUS *device_new_embeddings_2_status[N_STREAMS];
      int* device_n_embeddings_1[N_STREAMS];
      int* device_n_embeddings_2[N_STREAMS];
      int* device_curr_new_embeddings_idx[N_STREAMS];
      char *device_outputs[N_STREAMS];
      int* device_n_outputs[N_STREAMS];
      int* device_n_outputs_1[N_STREAMS];
      BUFFER_STATUS new_embeddings_1_status[N_STREAMS];
      BUFFER_STATUS new_embeddings_2_status[N_STREAMS];
      char *device_embeddings_per_partitions[N_STREAMS];
      for (int i = 0; i < N_STREAMS; i++) {
        new_embeddings_1_status[i] = BUFFER_STATUS::FREE;
        new_embeddings_2_status[i] = BUFFER_STATUS::FREE;
      }
      
      CSR* device_csr[N_STREAMS];
      
      assert (N_STREAMS >= 1);

      hipStream_t streams[N_STREAMS];
      for (int i = 0; i < N_STREAMS; i++) {
        hipError_t err = hipStreamCreate (&streams[i]);
        assert (err == hipSuccess);
      }

      const bool stream_synchronize = true;
      size_t per_stream_embeddings_done = 0;
      size_t per_stream_n_embeddings = n_embeddings/N_STREAMS;
      assert (N_STREAMS == 1);
      for (int i = 0; i < N_STREAMS; i++) {
        const bool unified_mem = false;
        if (false && iter == 2) {
          if (i == 0) {
            per_stream_n_embeddings = (n_embeddings*9)/10;
          } else {
            per_stream_n_embeddings = (n_embeddings*1)/10;
          }
        }

#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
        assert (N_STREAMS == 1);
        char* embeddings_per_partitions_ptr = (char*)orig_output_ptr + orig_outputs_size;
        size_t embeddings_per_partitions_ptr_size = get_emb_per_parts_size (per_stream_n_embeddings, iter);
        assert ((size_t)embeddings_per_partitions_ptr + embeddings_per_partitions_ptr_size < (size_t)global_mem_ptr + global_mem_size);
        std::cout << "embeddings_per_partitions_ptr_size: " << embeddings_per_partitions_ptr_size << std::endl;
        char* input_embeddings_ptr = (char*)embeddings_ptr + per_stream_embeddings_done*embedding_size;
        switch (iter) {
          case 1: {
#   ifdef EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK
            collect_vertex_embeddings_in_partitions_per_threadblock (csr, (VectorVertexEmbedding<1>*)input_embeddings_ptr, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                                    (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   else
            collect_vertex_embeddings_in_partitions (csr, (VectorVertexEmbedding<1>*)input_embeddings_ptr, 0, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                     (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   endif
            break;
          }
          case 2: {
#   ifdef EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK
            collect_vertex_embeddings_in_partitions_per_threadblock (csr, (VectorVertexEmbedding<2>*)input_embeddings_ptr, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                                    (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   else
            collect_vertex_embeddings_in_partitions (csr, (VectorVertexEmbedding<2>*)input_embeddings_ptr, 0, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                     (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   endif
            std::cout << "elements in embs_per_parts: " << per_stream_n_embeddings*iter << std::endl;
            
            for (int i = 0; i < per_stream_n_embeddings*iter; i++) {
              //PairEmbIdxVertexIdx p = ((PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr)[i];
              //printf ("%d: [%d, %d]:", i, p.emb_idx, p.vertex_idx);
              //VectorVertexEmbedding<2>* emb = &((VectorVertexEmbedding<2>*)input_embeddings_ptr)[p.emb_idx];
              //print_embedding<2> (emb);
              //std::cout << std::endl;
            }
            break;
          } 
          case 3: {
#   ifdef EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK
            collect_vertex_embeddings_in_partitions_per_threadblock (csr, (VectorVertexEmbedding<3>*)input_embeddings_ptr, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                                    (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   else
            collect_vertex_embeddings_in_partitions (csr, (VectorVertexEmbedding<3>*)input_embeddings_ptr, 0, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                     (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   endif
            std::cout << "iter = 3 elements in embs_per_parts: " << per_stream_n_embeddings*iter << std::endl;
            
            for (int i = 0; i < per_stream_n_embeddings*iter; i++) {
              PairEmbIdxVertexIdx p = ((PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr)[i];
              if (i == 1416192) {
                std::cout << "vertex_idx: " << p.vertex_idx << std::endl;
              }
              assert (p.vertex_idx < iter);
              //printf ("%d: [%d, %d]:", i, p.emb_idx, p.vertex_idx);
              //VectorVertexEmbedding<3>* emb = &((VectorVertexEmbedding<2>*)input_embeddings_ptr)[p.emb_idx];
              //print_embedding<2> (emb);
              //std::cout << std::endl;
            }
            
            break;
          }
          case 4: {
#   ifdef EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK
            collect_vertex_embeddings_in_partitions_per_threadblock (csr, (VectorVertexEmbedding<4>*)input_embeddings_ptr, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                                    (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   else
            collect_vertex_embeddings_in_partitions (csr, (VectorVertexEmbedding<4>*)input_embeddings_ptr, 0, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                     (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   endif
            std::cout << "elements in embs_per_parts: " << per_stream_n_embeddings*iter << std::endl;
            
            for (int i = 0; i < per_stream_n_embeddings*iter; i++) {
              //PairEmbIdxVertexIdx p = ((PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr)[i];
              //printf ("%d: [%d, %d]:", i, p.emb_idx, p.vertex_idx);
              //VectorVertexEmbedding<2>* emb = &((VectorVertexEmbedding<2>*)input_embeddings_ptr)[p.emb_idx];
              //print_embedding<2> (emb);
              //std::cout << std::endl;
            }
            break;
          }
          case 5: {
#   ifdef EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK
            collect_vertex_embeddings_in_partitions_per_threadblock (csr, (VectorVertexEmbedding<5>*)input_embeddings_ptr, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                                    (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   else
            collect_vertex_embeddings_in_partitions (csr, (VectorVertexEmbedding<5>*)input_embeddings_ptr, 0, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                     (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   endif
            std::cout << "elements in embs_per_parts: " << per_stream_n_embeddings*iter << std::endl;
            
            for (int i = 0; i < per_stream_n_embeddings*iter; i++) {
              //PairEmbIdxVertexIdx p = ((PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr)[i];
              //printf ("%d: [%d, %d]:", i, p.emb_idx, p.vertex_idx);
              //VectorVertexEmbedding<2>* emb = &((VectorVertexEmbedding<2>*)input_embeddings_ptr)[p.emb_idx];
              //print_embedding<2> (emb);
              //std::cout << std::endl;
            }
            break;
          }
          case 6: {
#   ifdef EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK
            collect_vertex_embeddings_in_partitions_per_threadblock (csr, (VectorVertexEmbedding<6>*)input_embeddings_ptr, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                                    (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   else
            collect_vertex_embeddings_in_partitions (csr, (VectorVertexEmbedding<6>*)input_embeddings_ptr, 0, per_stream_n_embeddings, partitions, vertex_to_partition, 
                                                     (PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr, embeddings_per_partitions_ptr_size);
#   endif
            std::cout << "elements in embs_per_parts: " << per_stream_n_embeddings*iter << std::endl;
            
            for (int i = 0; i < per_stream_n_embeddings*iter; i++) {
              //PairEmbIdxVertexIdx p = ((PairEmbIdxVertexIdx*)embeddings_per_partitions_ptr)[i];
              //printf ("%d: [%d, %d]:", i, p.emb_idx, p.vertex_idx);
              //VectorVertexEmbedding<2>* emb = &((VectorVertexEmbedding<2>*)input_embeddings_ptr)[p.emb_idx];
              //print_embedding<2> (emb);
              //std::cout << std::endl;
            }
            break;
          }
          default: {
            assert (false);
          }
        }
#endif

        if (unified_mem == true) {
          //cudaMallocManaged (embeddings_ptr, n_embeddings*embedding_size);
          //device_embeddings = (char*)embeddings_ptr;
          assert(false);
        } else {
          hipMalloc (&device_embeddings[i], per_stream_n_embeddings*embedding_size);
          hipMemcpyAsync (device_embeddings[i], (char*)embeddings_ptr + per_stream_embeddings_done*embedding_size,
                           per_stream_n_embeddings*embedding_size, hipMemcpyHostToDevice, streams[i]);
#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
          hipMalloc (&device_embeddings_per_partitions[i], embeddings_per_partitions_ptr_size);
          hipMemcpyAsync (device_embeddings_per_partitions[i], 
                           (char*)embeddings_per_partitions_ptr, 
                           embeddings_per_partitions_ptr_size, hipMemcpyHostToDevice, streams[i]);                           
          {
            hipError_t error = hipGetLastError ();
            if (error != hipSuccess) {
              const char* error_string = hipGetErrorString (error);
              std::cout << __LINE__<<" embeddings_per_partitions Cuda host to device copy error " << error_string << std::endl;
            } else {
              std::cout << "Cuda host to device copy success " << std::endl;
            }
          }
#endif
        }

        hipMalloc (&device_new_embeddings_1[i], max_embeddings/N_STREAMS*(new_embedding_size));
        hipMalloc (&device_new_embeddings_2[i], max_embeddings/N_STREAMS*(new_embedding_size));
        hipMalloc (&device_n_embeddings_1[i], sizeof(int));
        hipMalloc (&device_outputs[i], orig_outputs_size/N_STREAMS);
        hipMalloc (&device_n_embeddings_1[i], sizeof (int));
        hipMalloc (&device_n_embeddings_2[i], sizeof (int));
        hipMalloc (&device_n_outputs[i], sizeof (int));
        hipMalloc (&device_n_outputs_1[i], sizeof (int));
        hipMalloc (&device_csr[i], sizeof(CSR)); //TODO: Remove copying CSR graph again and again
        hipMalloc (&device_curr_new_embeddings_idx[i], sizeof (int));
        hipMalloc (&device_new_embeddings_1_status[i], sizeof (BUFFER_STATUS));
        hipMalloc (&device_new_embeddings_2_status[i], sizeof (BUFFER_STATUS));
        assert (N_STREAMS == 1); //TODO: for more than 1 N_STREAMS
        hipMemcpyAsync (device_new_embeddings_1_status[i], &new_embeddings_1_status[i], 
                         sizeof (new_embeddings_1_status[i]), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync (device_new_embeddings_2_status[i], &new_embeddings_2_status[i], 
                         sizeof (new_embeddings_2_status[i]), hipMemcpyHostToDevice, streams[i]);
        {
          hipError_t error = hipGetLastError ();
          if (error != hipSuccess) {
            const char* error_string = hipGetErrorString (error);
            std::cout << __LINE__<<" Cuda host to device copy error " << error_string << std::endl;
          } else {
            std::cout << __LINE__<< " Cuda host to device copy success " << std::endl;
          }
        }
        hipMemcpyAsync (device_n_embeddings_1[i], &n_new_embeddings_1[i],
                         sizeof (n_new_embeddings_1[i]), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync (device_n_outputs[i], &n_output[i], sizeof (n_output[i]),
                         hipMemcpyHostToDevice, streams[i]);
        
        {
          hipError_t error = hipGetLastError ();
          if (error != hipSuccess) {
            const char* error_string = hipGetErrorString (error);
            std::cout << __LINE__<<" Cuda host to device copy error " << error_string << std::endl;
          } else {
            std::cout << __LINE__<< " Cuda host to device copy success " << std::endl;
          }
        }                         
        
        assert (hipMemcpyAsync (device_n_embeddings_2[i], &n_new_embeddings_2[i],
                         sizeof (n_new_embeddings_2[i]), hipMemcpyHostToDevice, streams[i]) == hipSuccess);
        hipMemcpyAsync (device_n_outputs_1[i], &n_output_1[i], sizeof (n_output_1[i]),
                         hipMemcpyHostToDevice, streams[i]);
        
        {
          hipError_t error = hipGetLastError ();
          if (error != hipSuccess) {
            const char* error_string = hipGetErrorString (error);
            std::cout << __LINE__<<" Cuda host to device copy error " << error_string << std::endl;
          } else {
            std::cout << __LINE__<<" Cuda host to device copy success " << std::endl;
          }
        }
        hipMemcpyAsync (device_csr[i], csr, sizeof (CSR), hipMemcpyHostToDevice, streams[i]);
        
        hipMemcpyAsync (device_curr_new_embeddings_idx[i], &curr_new_embeddings_idx[i], 
                         sizeof (curr_new_embeddings_idx[i]), hipMemcpyHostToDevice,  
                         streams[i]);
        
        hipError_t error = hipGetLastError ();
        if (error != hipSuccess) {
          const char* error_string = hipGetErrorString (error);
          std::cout << __LINE__<<" Cuda host to device copy error " << error_string << std::endl;
        } else {
          std::cout << __LINE__<<" Cuda host to device copy success " << std::endl;
        }

        per_stream_embeddings_done += per_stream_n_embeddings;
      }

      for (int i = 0; i < N_STREAMS; i++) {
        double t1 = convertTimeValToDouble (getTimeOfDay ());
        if (stream_synchronize)
          hipStreamSynchronize (streams[i]);
        
        std::cout << "starting kernel with n_embeddings: " << per_stream_n_embeddings ;
    #ifdef USE_FIXED_THREADS
        //std::cout << " threads: " << MAX_CUDA_THREADS/THREAD_BLOCK_SIZE << std::endl;
        int thread_blocks = MAX_CUDA_THREADS/THREAD_BLOCK_SIZE;
    #else
        #ifdef PROCESS_EMBEDDINGS_PER_VERTEX
          int thread_blocks = (per_stream_n_embeddings%THREAD_BLOCK_SIZE != 0) ? ((per_stream_n_embeddings)/THREAD_BLOCK_SIZE+1) : (per_stream_n_embeddings)/THREAD_BLOCK_SIZE;
        #else
          int thread_blocks = (per_stream_n_embeddings%THREAD_BLOCK_SIZE != 0) ? (per_stream_n_embeddings/THREAD_BLOCK_SIZE+1) : per_stream_n_embeddings/THREAD_BLOCK_SIZE;
        #endif
    #endif
        std::cout << " threads: " << thread_blocks << std::endl;
        int* device_n_next_step_1_done;
        int* device_n_next_step_2_done;
        hipMalloc (&device_n_next_step_1_done, sizeof(int));
        hipMalloc (&device_n_next_step_2_done, sizeof(int));
        int qq = 0;
        hipMemcpyAsync (device_n_next_step_1_done, &qq, 
          sizeof (qq), hipMemcpyHostToDevice, streams[i]);
          hipMemcpyAsync (device_n_next_step_2_done, &qq, 
            sizeof (qq), hipMemcpyHostToDevice, streams[i]);
        switch (iter) {
          case 1: {
#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
            run_single_step_vectorvertex_embedding_per_vertex <1><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
              device_embeddings_per_partitions[i], device_outputs[i], device_n_outputs[i],
              device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
              device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i], 
              device_curr_new_embeddings_idx[i], 
              device_n_next_step_1_done,
              device_n_next_step_2_done,
              only_copy_change);
#else
            run_single_step_vectorvertex_embedding<1><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i], 
                                  device_curr_new_embeddings_idx[i], only_copy_change);
#endif                                  
            break;
          }
          case 2: {
#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
            run_single_step_vectorvertex_embedding_per_vertex <2><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
              device_embeddings_per_partitions[i], device_outputs[i], device_n_outputs[i],
              device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
              device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i], 
              device_curr_new_embeddings_idx[i], device_n_next_step_1_done,
              device_n_next_step_2_done, only_copy_change);
#else
            run_single_step_vectorvertex_embedding<2><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
#endif
            break;
          }
          case 3: {
#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
            run_single_step_vectorvertex_embedding_per_vertex <3><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
              device_embeddings_per_partitions[i], device_outputs[i], device_n_outputs[i],
              device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
              device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i], 
              device_curr_new_embeddings_idx[i], device_n_next_step_1_done,
              device_n_next_step_2_done, only_copy_change);
#else
            run_single_step_vectorvertex_embedding<3><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
#endif                                  
            break;
          }
          case 4: {
#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
            run_single_step_vectorvertex_embedding_per_vertex <4><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
              device_embeddings_per_partitions[i], device_outputs[i], device_n_outputs[i],
              device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
              device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i], 
              device_curr_new_embeddings_idx[i], device_n_next_step_1_done,
              device_n_next_step_2_done, only_copy_change);
#else
            run_single_step_vectorvertex_embedding<4><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
#endif
            break;
          }
          case 5: {
#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
            run_single_step_vectorvertex_embedding_per_vertex <5><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
              device_embeddings_per_partitions[i], device_outputs[i], device_n_outputs[i],
              device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
              device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i], 
              device_curr_new_embeddings_idx[i], device_n_next_step_1_done,
              device_n_next_step_2_done, only_copy_change);
#else
            run_single_step_vectorvertex_embedding<5><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
#endif
            break;
          }
          case 6: {
#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
            run_single_step_vectorvertex_embedding_per_vertex <6><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
              device_embeddings_per_partitions[i], device_outputs[i], device_n_outputs[i],
              device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
              device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i], 
              device_curr_new_embeddings_idx[i], device_n_next_step_1_done,
              device_n_next_step_2_done, only_copy_change);
#else
            run_single_step_vectorvertex_embedding<6><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
#endif
            break;
          }
          case 7: {
            run_single_step_vectorvertex_embedding<7><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
          case 8: {
            run_single_step_vectorvertex_embedding<8><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
        }
        
        //cudaDeviceSynchronize ();
        if (iter >= 1 && ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING) {
          int curr_step_storage_id = 0;
          std::cout << "copying to n_outputs_1" << std::endl; 
          hipStream_t t;
          int prev_curr_step_storage_id = 0;
          hipStreamCreate (&t);

          while (hipStreamQuery (streams[i]) == hipErrorNotReady) {
            BUFFER_STATUS __buff_1_status, __buff_2_status;
            do {
              assert (hipMemcpyAsync (&__buff_1_status, device_new_embeddings_1_status[i], sizeof (__buff_1_status), hipMemcpyDeviceToHost, t) == hipSuccess);
              assert (hipMemcpyAsync (&__buff_2_status, device_new_embeddings_2_status[i], sizeof (__buff_2_status), hipMemcpyDeviceToHost, t) == hipSuccess);
              hipStreamSynchronize (t);
              usleep (GPU_QUERY_WAIT_TIME);
            } while (__buff_1_status != BUFFER_STATUS::CPU_COPYING &&
                     __buff_2_status != BUFFER_STATUS::CPU_COPYING &&
                     hipStreamQuery (streams[i]) == hipErrorNotReady);
            //Above loop ends only when the storage id has changed or 
            //the stream's kernel has ended.
            usleep (1000000UL);
            if (__buff_1_status == BUFFER_STATUS::CPU_COPYING)
              prev_curr_step_storage_id = 0;
            else if (__buff_2_status == BUFFER_STATUS::CPU_COPYING)
              prev_curr_step_storage_id = 1;

            std::cout << "prev_curr_step_storage_id = " << prev_curr_step_storage_id <<
                         " curr_step_storage_id = " << curr_step_storage_id << 
                         " hipStreamQuery (streams[i]) == hipErrorNotReady " << 
                         (hipStreamQuery (streams[i]) == hipErrorNotReady) << std::endl;
            if (hipStreamQuery (streams[i]) == hipSuccess) {
              //Stream's kernel has completed, get out of loop.
              break;
            }
            std::cout << "n_new_embeddings_1[i] + n_new_embeddings_2[i] " << n_new_embeddings_1[i] + n_new_embeddings_2[i] << " * new_embedding_size " << 
            (n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size << std::endl;
            //copy
            switch (prev_curr_step_storage_id) {
              case 0: {
                //new_embeddings_1_status[i] = BUFFER_STATUS::CPU_COPYING;
                //assert (cudaMemcpyAsync (&device_new_embeddings_1_status[i], &new_embeddings_1_status[i], 
                //        sizeof (new_embeddings_1_status[i]), cudaMemcpyHostToDevice, t) == 0);
                int p = 0;
                assert (hipMemcpyAsync (&p, device_n_embeddings_1[i], sizeof (p), hipMemcpyDeviceToHost, t) == hipSuccess);
                std::cout << "Copying " << p << " new embeddings each of size " << new_embedding_size << std::endl;
                
                if (p != max_embeddings/N_STREAMS) {
                  std::cout << "SYNCHRONIZATION ISSUE: device_n_embeddings_1[i] " << p << " > per_stream_n_embeddings " << max_embeddings/N_STREAMS << std::endl;
                } 
                if (true) {
                  hipError_t err = hipMemcpyAsync ((char*)new_embeddings_ptr[i] + (n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size, device_new_embeddings_1[i], max_embeddings/N_STREAMS*new_embedding_size, hipMemcpyDeviceToHost, t);
                  hipStreamSynchronize (t);
                  char* __new_ptr = (char*)malloc (p*new_embedding_size);
                  hipMemcpyAsync (__new_ptr, device_new_embeddings_1[i], p*new_embedding_size, hipMemcpyDeviceToHost, t);
                  for (int ii = 0; ii < p; ii++) {
                    VectorVertexEmbedding<2>& e = ((VectorVertexEmbedding<2>*)__new_ptr)[ii];
                    print_embedding<2> (&e);
                    std::cout << std::endl;
                  }
                  free (__new_ptr);
                  hipStreamSynchronize (t);
                  //if (n_new_embeddings_1[i] + n_new_embeddings_2[i] >= 778462776)
                  //    exit(EXIT_SUCCESS);
                  //std::cout << "total embeddings found: " << n_new_embeddings_1[i] + n_new_embeddings_2[i] << std::endl;
                  if (err != hipSuccess) {
                    //if (n_new_embeddings_1[i] + n_new_embeddings_2[i] >= 778462776)
                    //  exit(EXIT_SUCCESS);
                    std::cout << hipGetErrorString (err) << std::endl;
                    assert (false);
                  }

                  // char* __new_ptr = (char*)new_embeddings_ptr[i] + (n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size;
                  //exit (EXIT_SUCCESS);
                }
                n_new_embeddings_1[i] += p;
                p = 0;
                assert (hipMemcpyAsync (device_n_embeddings_1[i], &p, sizeof (p), hipMemcpyHostToDevice, t) == hipSuccess);
                new_embeddings_1_status[i] = BUFFER_STATUS::FREE;
                hipError_t e = hipMemcpyAsync (device_new_embeddings_1_status[i], &new_embeddings_1_status[i], 
                                                 sizeof (new_embeddings_1_status[i]), hipMemcpyHostToDevice, t);
                //std::cout << cudaGetErrorString (e) << std::endl;
                assert (e == hipSuccess);
                break;
              }

              case 1: {
                int p = 0;
                //new_embeddings_2_status[i] = BUFFER_STATUS::CPU_COPYING;
                //assert (cudaMemcpyAsync (&device_new_embeddings_2_status[i], &new_embeddings_2_status[i], 
                //        sizeof (new_embeddings_2_status[i]), cudaMemcpyHostToDevice, t) == 0);
                assert (hipMemcpyAsync (&p, device_n_embeddings_2[i], sizeof (p), hipMemcpyDeviceToHost, t) == hipSuccess);
                std::cout << "Copying " << p << " new embeddings each of size " << new_embedding_size << std::endl;
                if (p > max_embeddings/N_STREAMS) {
                  std::cout << "SYNCHRONIZATION ISSUE: device_n_embeddings_1[i] " << p << " > per_stream_n_embeddings " << max_embeddings/N_STREAMS << std::endl;
                } 
                if (true) {
                  hipError_t err = hipMemcpyAsync ((char*)new_embeddings_ptr[i]+(n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size, device_new_embeddings_2[i], max_embeddings/N_STREAMS*new_embedding_size, hipMemcpyDeviceToHost, t);
                  hipStreamSynchronize (t);
                  char* __new_ptr = (char*)malloc (p*new_embedding_size);
                  hipMemcpyAsync (__new_ptr, device_new_embeddings_2[i], p*new_embedding_size, hipMemcpyDeviceToHost, t);
                  for (int ii = 0; ii < p; ii++) {
                    VectorVertexEmbedding<2>& e = ((VectorVertexEmbedding<2>*)__new_ptr)[ii];
                    print_embedding<2> (&e);
                    std::cout << std::endl;
                  }
                  free (__new_ptr);
                  hipStreamSynchronize (t);
                  //if (n_new_embeddings_1[i] + n_new_embeddings_2[i] >= 778462776)
                   //   exit(EXIT_SUCCESS);
                  //std::cout << "total embeddings found: " << n_new_embeddings_1[i] + n_new_embeddings_2[i] << std::endl;
                  if (err != hipSuccess) {
                   // if (n_new_embeddings_1[i] + n_new_embeddings_2[i] >= 778462776)
                   //   exit(EXIT_SUCCESS);
                    std::cout << "total embeddings found: " << n_new_embeddings_1[i] + n_new_embeddings_2[i] << std::endl;
                    std::cout << hipGetErrorString (err) << std::endl;
                    assert (false);
                  }

                  // char* __new_ptr = (char*)new_embeddings_ptr[i] + (n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size;
                }
                n_new_embeddings_2[i] += p;
                p = 0;
                assert (hipMemcpyAsync (device_n_embeddings_2[i], &p, sizeof (p), hipMemcpyHostToDevice, t) == hipSuccess);
                new_embeddings_2_status[i] = BUFFER_STATUS::FREE;
                assert (hipMemcpyAsync (device_new_embeddings_2_status[i], &new_embeddings_2_status[i], 
                        sizeof (new_embeddings_2_status[i]), hipMemcpyHostToDevice, t) == 0);
                break;
              }
            }
            prev_curr_step_storage_id = curr_step_storage_id;
            hipStreamSynchronize (t);
            std::cout << "Copying done " << std::endl;
            //cudaStreamQuery (streams[i]);
          }

          hipStreamDestroy (t);
        }

        //cudaDeviceSynchronize ();
        if (stream_synchronize)
          hipStreamSynchronize (streams[i]);

      double t2 = convertTimeValToDouble (getTimeOfDay ());

      std::cout << "Execution time " << (t2-t1) << " secs" << std::endl;
      kernelTotalTime += (t2-t1);

        error = hipGetLastError ();
        if (error != hipSuccess) {
          const char* error_string = hipGetErrorString (error);
          std::cout << "Cuda kernel error " << error_string << std::endl;
        } else {
          std::cout << "Cuda success " << std::endl;
        }
      }

      // char* __new_ptr = (char*)new_embeddings_ptr[0];

      // for (int ii = 0; ii < n_new_embeddings_1[0] + n_new_embeddings_2[0]; ii++) {
      //   VectorVertexEmbedding<2>& e = ((VectorVertexEmbedding<2>*)__new_ptr)[ii];
      //   print_embedding<2> (&e);
      //   std::cout << ii << std::endl;
      // }

      for (int i = 0; i < N_STREAMS; i++) 
      {
        //cudaStreamSynchronize (streams[i]);

        if (only_copy_change) {
          assert (false);
          //TODO: Change this to make an array of such ptrs
          hipMemcpyAsync (new_embeddings_ptr, device_new_embeddings_1[i], n_new_embeddings_1[i]*2*sizeof(int), hipMemcpyDeviceToHost, streams[i]);
          hipMemcpyAsync (output_ptr, device_outputs[i], n_output[i]*2*sizeof(int), hipMemcpyDeviceToHost, streams[i]);
        }
        else {
          std::cout << "n_output[" << i << "] = " << n_output[i] << std::endl;
          int _n_new = 0;
          assert (hipMemcpyAsync (&_n_new, device_n_embeddings_1[i], sizeof(0), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
          std::cout << "_n_new " << _n_new << std::endl;
          if (_n_new > 0 ) {
            assert (hipMemcpyAsync ((char*)new_embeddings_ptr[i] + (n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size,
                                     device_new_embeddings_1[i], _n_new*(new_embedding_size), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
            int p = _n_new;
            char* __new_ptr = (char*)malloc (p*new_embedding_size);
            hipMemcpyAsync (__new_ptr, device_new_embeddings_1[i], p*new_embedding_size, hipMemcpyDeviceToHost, streams[i]);
            for (int ii = 0; ii < p; ii++) {
              VectorVertexEmbedding<2>& e = ((VectorVertexEmbedding<2>*)__new_ptr)[ii];
              print_embedding<2> (&e);
              std::cout << std::endl;
            }
            free (__new_ptr);
            hipStreamSynchronize(streams[i]);
            n_new_embeddings_1[i] += _n_new;
          }
         
          assert (hipMemcpyAsync (&_n_new, device_n_embeddings_2[i], sizeof(0), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
          std::cout << "_n_new " << _n_new << std::endl;
          if (_n_new > 0 ) {
            assert (hipMemcpyAsync ((char*)new_embeddings_ptr[i] + (n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size,
                                     device_new_embeddings_2[i], _n_new*(new_embedding_size), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
            int p = _n_new;
            char* __new_ptr = (char*)malloc (p*new_embedding_size);
            hipMemcpyAsync (__new_ptr, device_new_embeddings_2[i], p*new_embedding_size, hipMemcpyDeviceToHost, streams[i]);
            for (int ii = 0; ii < p; ii++) {
              VectorVertexEmbedding<2>& e = ((VectorVertexEmbedding<2>*)__new_ptr)[ii];
              print_embedding<2> (&e);
              std::cout << std::endl;
            }
            free (__new_ptr);
            hipStreamSynchronize(streams[i]);
            n_new_embeddings_2[i] += _n_new;
          }
          
          assert (hipMemcpyAsync (&n_output[i], device_n_outputs[i], sizeof(0), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);

          //assert (cudaMemcpyAsync (output_ptr[i], device_outputs[i], n_output[i]*(new_embedding_size), cudaMemcpyDeviceToHost, streams[i])== cudaSuccess);
          n_output[i] = 0;
        }
        //cudaMemcpyAsync (&n_new_embeddings_1[i], device_n_embeddings_1[i], sizeof(0), cudaMemcpyDeviceToHost, streams[i]);
        //cudaMemcpyAsync (&n_output_1[i], device_n_outputs_1[i], sizeof(0), cudaMemcpyDeviceToHost, streams[i]);

        error = hipGetLastError ();
        if (error != hipSuccess) {
          const char* error_string = hipGetErrorString (error);
          std::cout << "Cuda device to host copy error " << error_string << std::endl;
        } else {
          std::cout << "Cuda device to host copy success " << std::endl;
        }

        std::cout << "Stream " << i << std::endl;
        std::cout << "n_new_embeddings "<<n_new_embeddings_1[i] + n_new_embeddings_2[i]<<std::endl;
        std::cout << "n_new_embeddings_1 "<<n_new_embeddings_1[i];
        std::cout << " n_output "<<n_output[i];
        std::cout << " n_output_1 "<<n_output_1[i]<<std::endl;
      }
      
      for (int i = 0; i < N_STREAMS; i++) {
        hipStreamSynchronize (streams[i]);
        hipError_t err = hipStreamDestroy (streams[i]);
        assert (err == hipSuccess);
      }

      double stream_time_2 = convertTimeValToDouble (getTimeOfDay ());

      total_stream_time += (stream_time_2-stream_time_1);

      //TODO: wait for all kernels and data transfers to finish
      for (int i = 0; i < N_STREAMS; i++) {
        n_next_step_embeddings += n_new_embeddings_1[i]+ n_new_embeddings_2[i];
      }
      switch (iter) {
        case 1: {
          VectorVertexEmbedding<2>* new_embeddings = (VectorVertexEmbedding<2>*)malloc (sizeof (VectorVertexEmbedding<2>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream] + n_new_embeddings_2[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<2> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<1>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [j] = embedding;
                j++;
              }
              else {
                VectorVertexEmbedding<2> embedding = ((VectorVertexEmbedding<2>*)(new_embeddings_ptr[stream]))[i];
                new_embeddings [j] = embedding;
                //print_embedding<2> (&embedding);
                //std::cout << std::endl;
                
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
                j++;
              }
            }
          }
          //return;
          assert (j == n_next_step_embeddings);
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          exit (EXIT_SUCCESS);
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<2> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<1>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_2.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_2.push_back (((VectorVertexEmbedding<2>*)output_ptr[stream])[i]);
              }
            }
          }
          
          break;
        }
        
        case 2: {
          VectorVertexEmbedding<3>* new_embeddings = (VectorVertexEmbedding<3>*)malloc (sizeof (VectorVertexEmbedding<3>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream] + n_new_embeddings_2[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<3> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<2>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                assert (false);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<3> embedding = ((VectorVertexEmbedding<3>*)(new_embeddings_ptr[stream]))[i];
                new_embeddings [j] = embedding;
                //print_embedding<3> (&embedding);
                //std::cout << std::endl;
                //s = embedding_to_string<3> (&embedding);
                //#ifdef DEBUG
                //if (embedding.get_n_vertices () != (iter + 1)) {
                //  printf ("embedding has %ld vertices\n", embedding.get_n_vertices ());
                //}
                //#endif
                j++;
              }
            }
          }

          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<3> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<2>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_3.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_3.push_back (((VectorVertexEmbedding<3>*)output_ptr[stream])[i]);
              }
            }
          }

          break;
        }
        
        case 3: {
          VectorVertexEmbedding<4>* new_embeddings = (VectorVertexEmbedding<4>*)malloc (sizeof (VectorVertexEmbedding<4>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<4> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<3>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<4> embedding = ((VectorVertexEmbedding<4>*)(new_embeddings_ptr[stream]))[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));

          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<4> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<3>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_4.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_4.push_back (((VectorVertexEmbedding<4>*)output_ptr[stream])[i]);
              }
            }
          }
          break;
        }
        
        case 4: {
          VectorVertexEmbedding<5>* new_embeddings = (VectorVertexEmbedding<5>*)malloc (sizeof (VectorVertexEmbedding<5>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<5> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<4>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<5> embedding = ((VectorVertexEmbedding<5>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<5> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<4>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_5.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_5.push_back (((VectorVertexEmbedding<5>*)output_ptr[stream])[i]);
              }
            }
          }

          break;
        }
        
        case 5: {
          VectorVertexEmbedding<6>* new_embeddings = (VectorVertexEmbedding<6>*)malloc (sizeof (VectorVertexEmbedding<6>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<6> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<5>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<6> embedding = ((VectorVertexEmbedding<6>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<6> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<5>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_6.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_6.push_back (((VectorVertexEmbedding<6>*)output_ptr[stream])[i]);
              }
            }
          }
          break;
        }
        
        case 6: {
          VectorVertexEmbedding<7>* new_embeddings = (VectorVertexEmbedding<7>*)malloc (sizeof (VectorVertexEmbedding<7>)*n_next_step_embeddings);
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<7> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<6>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<7> embedding = ((VectorVertexEmbedding<7>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<7> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<6>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_7.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_7.push_back (((VectorVertexEmbedding<7>*)output_ptr[stream])[i]);
              }
            }
          }
          break;
        }
        
        case 7: {
          VectorVertexEmbedding<8>* new_embeddings = (VectorVertexEmbedding<8>*)malloc (sizeof(VectorVertexEmbedding<8>)*n_next_step_embeddings);
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<8> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<7>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<8> embedding = ((VectorVertexEmbedding<8>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<8> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<7>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_8.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_8.push_back (((VectorVertexEmbedding<8>*)output_ptr[stream])[i]);
              }
            }
          }

          break;
        }
      }
      
      //embeddings = new_embeddings;

      for (int i = 0; i < N_STREAMS; i++) {
        hipFree (device_embeddings_per_partitions[i]);
        hipFree (device_embeddings[i]);
        hipFree (device_new_embeddings_1[i]);
        hipFree (device_new_embeddings_2[i]);
        hipFree (device_n_embeddings_1[i]);
        hipFree (device_outputs[i]);
        hipFree (device_n_outputs[i]);
        hipFree (device_csr[i]);
      }
    }

    new_embeddings_size = n_next_step_embeddings;
  }

#ifdef PINNED_MEMORY
  hipFree (global_mem_ptr);
#else
  delete[] global_mem_ptr;
#endif
  std::cout << "Number of embeddings found "<< (output_1.size () + output_2.size () + output_3.size () + output_4.size () + output_5.size () + output_6.size () + output_7.size () + output_8.size ()) << std::endl;
  std::cout << "Time spent in GPU kernel execution " << kernelTotalTime << std::endl;
  std::cout << "Time spent in Streams " << total_stream_time << std::endl;
  
  /* For BitVectorVertexEmbedding
   for (iter; iter < 10 && embeddings.size () > 0; iter++) {
    std::cout << "iter " << iter << " embeddings " << embeddings.size () << std::endl;
    size_t global_mem_size = 3*1024*1024*1024UL;
    char* global_mem_ptr = new char[global_mem_size];
  #ifdef DEBUG
    memset (global_mem_ptr, 0, global_mem_size);
  #endif
    int n_embeddings = embeddings.size ();
    //n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
    std::cout << "iter " << iter << " n_embeddings " << n_embeddings << std::endl;
  
    for (int i = 0; i < n_embeddings; i++) {
      ((BitVectorVertexEmbedding*)global_mem_ptr)[i] = embeddings[i];
    }
    void* embeddings_ptr = global_mem_ptr;

    int n_new_embeddings = 0;
    int n_new_embeddings_1 = 0;
    void* new_embeddings_ptr = (char*)embeddings_ptr + (n_embeddings)*sizeof(BitVectorVertexEmbedding);
    int max_embeddings = 1000000;
    void* output_ptr = (char*)new_embeddings_ptr + (max_embeddings)*sizeof(BitVectorVertexEmbedding);
    int n_output = 0;
    int n_output_1 = 0;
    char* device_embeddings;
    char *device_new_embeddings;
    int* device_n_embeddings;
    int* device_n_embeddings_1;
    char *device_outputs;
    int* device_n_outputs;
    int* device_n_outputs_1;
    CSR* device_csr;

    cudaMalloc (&device_embeddings, n_embeddings*sizeof(BitVectorVertexEmbedding));
    cudaMemcpy (device_embeddings, embeddings_ptr,
                n_embeddings*sizeof(BitVectorVertexEmbedding),
                cudaMemcpyHostToDevice);
    cudaMalloc (&device_new_embeddings, max_embeddings*sizeof (BitVectorVertexEmbedding));
    cudaMalloc (&device_outputs, max_embeddings*sizeof (BitVectorVertexEmbedding));
    cudaMalloc (&device_n_embeddings, sizeof (0));
    cudaMalloc (&device_n_embeddings_1, sizeof (0));
    cudaMalloc (&device_n_outputs, sizeof (0));
    cudaMalloc (&device_n_outputs_1, sizeof (0));
    cudaMalloc (&device_csr, sizeof(CSR));

    cudaMemcpy (device_n_embeddings, &n_new_embeddings,
                sizeof (n_new_embeddings), cudaMemcpyHostToDevice);
    cudaMemcpy (device_n_outputs, &n_output, sizeof (n_output),
                cudaMemcpyHostToDevice);

    cudaMemcpy (device_n_embeddings_1, &n_new_embeddings_1,
                sizeof (n_new_embeddings_1), cudaMemcpyHostToDevice);
    cudaMemcpy (device_n_outputs_1, &n_output_1, sizeof (n_output_1),
                cudaMemcpyHostToDevice);

    cudaMemcpy (device_csr, csr, sizeof (CSR), cudaMemcpyHostToDevice);

    std::cout << "starting kernel with n_embeddings: " << n_embeddings;

    double t1 = convertTimeValToDouble (getTimeOfDay ());
#ifdef USE_FIXED_THREADS
    std::cout << " threads: " << MAX_CUDA_THREADS/THREAD_BLOCK_SIZE << std::endl;
      run_single_step_bitvector_embedding<<<MAX_CUDA_THREADS/THREAD_BLOCK_SIZE,THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
#else
    int thread_blocks = (n_embeddings%THREAD_BLOCK_SIZE != 0) ? (n_embeddings/THREAD_BLOCK_SIZE+1) : n_embeddings/THREAD_BLOCK_SIZE;
    std::cout << " threads: " << n_embeddings/THREAD_BLOCK_SIZE << std::endl;
    run_single_step_bitvector_embedding<<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
#endif

    cudaDeviceSynchronize ();

    double t2 = convertTimeValToDouble (getTimeOfDay ());

    std::cout << "Execution time " << (t2-t1) << " secs" << std::endl;
    kernelTotalTime += (t2-t1);

    cudaError_t error = cudaGetLastError ();
    if (error != cudaSuccess) {
      const char* error_string = cudaGetErrorString (error);
      std::cout << error_string << std::endl;
    } else {
      std::cout << "Cuda success " << std::endl;
    }

    cudaMemcpy (new_embeddings_ptr, device_new_embeddings, max_embeddings*sizeof(BitVectorVertexEmbedding), cudaMemcpyDeviceToHost);
    cudaMemcpy (output_ptr, device_outputs, max_embeddings*sizeof(BitVectorVertexEmbedding), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_new_embeddings, device_n_embeddings, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_output, device_n_outputs, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_new_embeddings_1, device_n_embeddings_1, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_output_1, device_n_outputs_1, sizeof(0), cudaMemcpyDeviceToHost);

    std::cout << "n_new_embeddings "<<n_new_embeddings<<std::endl;
    std::cout << "n_new_embeddings_1 "<<n_new_embeddings_1;
    std::cout << " n_output "<<n_output;
    std::cout << " n_output_1 "<<n_output_1<<std::endl;
    std::vector<BitVectorVertexEmbedding> new_embeddings;
  
    for (int i = 0; i < n_new_embeddings; i++) {
      BitVectorVertexEmbedding embedding = ((BitVectorVertexEmbedding*)new_embeddings_ptr)[i];
      new_embeddings.push_back (embedding);
    #ifdef DEBUG
      if (embedding.get_n_vertices () != (iter + 1)) {
        printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
      }
    #endif
    }
    for (int i = 0; i < n_output; i++) {
      output.push_back (((BitVectorVertexEmbedding*)output_ptr)[i]);
    }
    embeddings = new_embeddings;

    cudaFree (device_embeddings);
    cudaFree (device_new_embeddings);
    cudaFree (device_n_embeddings);
    cudaFree (device_outputs);
    cudaFree (device_n_outputs);
    cudaFree (device_csr);
    delete[] global_mem_ptr;
    #endif
  } 
   */
}

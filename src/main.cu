
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>

#include <string.h>
#include <assert.h>
#include <tuple>

#define LINE_SIZE 1024*1024
//#define USE_FIXED_THREADS
#define MAX_CUDA_THREADS (96*96)
#define THREAD_BLOCK_SIZE 256
#define WARP_SIZE 32
//#define USE_CSR_IN_SHARED
//#define USE_EMBEDDING_IN_SHARED_MEM
//#define USE_EMBEDDING_IN_GLOBAL_MEM
#define USE_EMBEDDING_IN_LOCAL_MEM
//#define SHARED_MEM_NON_COALESCING
/**
  * The commit performing better is 698368fa19d023e3cb09705d820d333f79d0bf46.
  */
#ifdef SHARED_MEM_NON_COALESCING
  #ifndef USE_EMBEDDING_IN_SHARED_MEM
    #error "USE_EMBEDDING_IN_SHARED_MEM must be enabled with SHARED_MEM_NON_COALESCING"
  #endif
#endif
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #ifdef USE_FIXED_THREADS
    #error "USE_FIXED_THREADS cannot be enabled with USE_EMBEDDING_IN_SHARED_MEM"
  #endif
#endif

//#define USE_CONSTANT_MEM

typedef uint8_t SharedMemElem;

//citeseer.graph
const int N = 3312;
const int N_EDGES = 9074;

//micro.graph
//const int N = 100000;
//const int N_EDGES = 2160312;

class Vertex
{
private:
  int id;
  int label;
  std::vector <int> edges;

public:
  Vertex (int _id, int _label) : label(_label), id (_id)
  {
  }

  int get_id () {return id;}
  int get_label () {return label;}
  void add_edge (int vertexID) {edges.push_back (vertexID);}
  void sort_edges () {std::sort (edges.begin(), edges.end ());}
  std::vector <int>& get_edges () {return edges;}
  void print (std::ostream& os)
  {
    os << id << " " << label << " ";
    for (auto edge : edges) {
      os << edge << " ";
    }

    os << std::endl;
  }
};

int chars_in_int (int num)
{
  if (num == 0) return sizeof(char);
  return (int)((ceil(log10(num))+1)*sizeof(char));
}

class Graph
{
private:
  std::vector<Vertex> vertices;
  int n_edges;

public:
  Graph (std::vector<Vertex> _vertices, int _n_edges) :
    vertices (_vertices), n_edges(_n_edges)
  {}

  const std::vector<Vertex>& get_vertices () {return vertices;}
  int get_n_edges () {return n_edges;}
};

class CSR
{
public:
  struct Vertex
  {
    int id;
    int label;
    int start_edge_id;
    int end_edge_id;
    __host__ __device__
    Vertex ()
    {
      id = -1;
      label = -1;
      start_edge_id = -1;
      end_edge_id = -1;
    }

    void set_from_graph_vertex (::Vertex& vertex)
    {
      id = vertex.get_id ();
      label = vertex.get_label ();
    }

    void set_start_edge_id (int start) {start_edge_id = start;}
    void set_end_edge_id (int end) {end_edge_id = end;}
  };

  typedef int Edge;

public:
  CSR::Vertex vertices[N];
  CSR::Edge edges[N_EDGES];
  int n_vertices;
  int n_edges;

public:
  CSR (int _n_vertices, int _n_edges)
  {
    n_vertices = _n_vertices;
    n_edges = _n_edges;
  }

  __host__ __device__
  CSR ()
  {
    n_vertices = N;
    n_edges = N_EDGES;
  }

  void print (std::ostream& os)
  {
    for (int i = 0; i < n_vertices; i++) {
      os << vertices[i].id << " " << vertices[i].label << " ";
      for (int edge_iter = vertices[i].start_edge_id;
           edge_iter <= vertices[i].end_edge_id; edge_iter++) {
        os << edges[edge_iter] << " ";
      }
      os << std::endl;
    }
  }

  __host__ __device__
  int get_start_edge_idx (int vertex_id)
  {
    if (!(vertex_id < n_vertices && 0 <= vertex_id)) {
      printf ("vertex_id %d, n_vertices %d\n", vertex_id, n_vertices);
      assert (false);
    }
    return vertices[vertex_id].start_edge_id;
  }

  __host__ __device__
  int get_end_edge_idx (int vertex_id)
  {
    assert (vertex_id < n_vertices && 0 <= vertex_id);
    return vertices[vertex_id].end_edge_id;
  }

  __host__ __device__
  bool has_edge (int u, int v)
  {
    //TODO: Since graph is sorted, do this using binary search
    for (int e = get_start_edge_idx (u); e <= get_end_edge_idx (u); e++) {
      if (edges[e] == v) {
        return true;
      }
    }

    return false;
  }

  __host__ __device__
  const CSR::Edge* get_edges () {return &edges[0];}

  __host__ __device__
  const CSR::Vertex* get_vertices () {return &vertices[0];}

  __host__ __device__
  int get_n_vertices () {return n_vertices;}

  __host__ __device__
  void copy_vertices (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      vertices[i] = src->get_vertices()[i];
    }
  }

  __host__ __device__
  void copy_edges (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      edges[i] = src->get_edges ()[i];
    }
  }

  __host__ __device__
  int get_n_edges () {return n_edges;}
};

#ifdef USE_CONSTANT_MEM
  __constant__ unsigned char csr_constant_buff[sizeof(CSR)];
#endif

void csr_from_graph (CSR* csr, Graph& graph)
{
  int edge_iterator = 0;
  auto graph_vertices = graph.get_vertices ();
  for (int i = 0; i < graph_vertices.size (); i++) {
    ::Vertex& vertex = graph_vertices[i];
    csr->vertices[i].set_from_graph_vertex (graph_vertices[i]);
    csr->vertices[i].set_start_edge_id (edge_iterator);
    for (auto edge : vertex.get_edges ()) {
      csr->edges[edge_iterator] = edge;
      edge_iterator++;
    }

    csr->vertices[i].set_end_edge_id (edge_iterator-1);
  }
}

//template <size_t N> using VertexEmbedding = std::bitset<N>;

#define CVT_TO_NEXT_MULTIPLE(n,k) ((n) %(k) ==0 ? (n) : ((n)/(k)+1)*(k))
class BitVectorVertexEmbedding
{
private:
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  unsigned char array[CVT_TO_NEXT_MULTIPLE(N/8, 32*sizeof(SharedMemElem))];
#else
  unsigned char array[(N/8)];
#endif

public:
  __device__ __host__
  BitVectorVertexEmbedding ()
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    assert (array != nullptr);
    reset ();
    assert (all_false () == true);
  }

  __host__ __device__
  size_t convert_to_bytes_multiple (size_t n)
  {
    return (n/8)*8;
  }

  __host__ __device__
  BitVectorVertexEmbedding (const BitVectorVertexEmbedding& embedding)
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    for (int i = 0; i <  convert_to_bytes_multiple(N)/8; i++) {
      array[i] = embedding.array[i];
    }
  }

  __host__ __device__
  void set (int index)
  {
    assert (index >= 0 and index < N);
    array[index/8] = array[index/8] | (1 << (index %8));
  }

  __host__ __device__
  void set ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = (unsigned char) (~(0UL));
    }
  }

  __host__ __device__
  void reset ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = 0;
    }
  }

  __host__ __device__
  void reset (int index)
  {
    assert (index >= 0 and index < N);
    array[index/8] = array[index/8] & (~(1UL << (index %8)));
  }

  __host__ __device__
  bool test (int index)
  {
    return (bool) ((array[index/8] >> (index % 8))&1);
  }

  __host__ __device__
  bool all_false ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      if (array[i] != 0UL) {
        return false;
      }
    }

    return true;
  }
  
  __host__ __device__
  int get_n_vertices () 
  {
    int n_vertices = 0;
    for (int i = 0; i < N; i++) {
      if (test(i) == true) {
        n_vertices++;
      }
    }
    
    return n_vertices;
  }
  
  __host__ __device__
  ~BitVectorVertexEmbedding ()
  {
    //delete[] array;
  }
};

//typedef BitVectorVertexEmbedding VertexEmbedding;

template <uint32_t size> 
class VectorVertexEmbedding
{
private:
  uint32_t array[size];
  uint32_t filled_size;
  
public:
  __device__ __host__
  VectorVertexEmbedding ()
  {
    filled_size = 0;
  }

  __host__ __device__
  VectorVertexEmbedding (const VectorVertexEmbedding<size>& embedding)
  {
  #if DEBUG
    assert (embedding.get_max_size () <= get_max_size ());
  #endif
    filled_size = 0;
    for (int i = 0; i < embedding.get_n_vertices (); i++) {
      add (embedding.get_vertex (i));
    }
  }
  
  __host__ __device__
  void add (int v)
  {
  #if DEBUG
    if (!(size != 0 and filled_size < size)) {
      printf ("filled_size %d, size %d\n", filled_size, size);
      //assert (size != 0 and filled_size < size);
      assert (false);
    }
  #endif
  
    add_unsorted (v);
    return;
    int pos = 0;
    
    for (int i = 0; i < filled_size; i++) {
      if (array[i] > v) {
        pos = i;
        break;
      }
    }
    
    for (int i = filled_size-1; i >= pos ; i--) {
      array[i+1] = array[i];
    }
    
    array[pos] = v;
    filled_size++;
  }

  __host__ __device__
  void add_last_in_sort_order () 
  {
    int v = array[filled_size-1];
    remove_last ();
    add (v);
  }

  __host__ __device__
  void add_unsorted (int v) 
  {
    array[filled_size++] = v;
  }
  
  __host__ __device__
  void remove (int v)
  {
    printf ("Do not support remove\n");
    assert (false);
  }
  
  __host__ __device__
  bool has_logn (int v)
  {
    int l = 0;
    int r = filled_size-1;
    
    while (l <= r) {
      int m = l+(r-l)/2;
      
      if (array[m] == v)
        return true;
      
      if (array[m] < v)
        l = m + 1;
      else
        r = m - 1;
    }
    
    return false;
  }
  
  __host__ __device__
  bool has (int v)
  {
    for (int i = 0; i < filled_size; i++) {
      if (array[i] == v) {
        return true;
      }
    }
    
    return false;
  }
  
  __host__ __device__
  size_t get_n_vertices () const
  {
    return filled_size;
  }
  
  __host__ __device__
  int get_vertex (int index) const
  {
    return array[index];
  }
  
  __host__ __device__
  int get_last_vertex () const
  {
    return array[filled_size-1];
  }
  
  __host__ __device__
  size_t get_max_size () const
  {
    return size;
  }
  
  __host__ __device__
  void clear ()
  {
    filled_size = 0;
  }
  
  __host__ __device__
  void remove_last () 
  {
    assert (filled_size > 0);
    filled_size--;
  }
  __host__ __device__
  ~VectorVertexEmbedding ()
  {
    //delete[] array;
  }
};

template <uint32_t size>
__host__ __device__
void vector_embedding_from_one_less_size (VectorVertexEmbedding<size>& vec_emb1,
                                          VectorVertexEmbedding<size+1>& vec_emb2)
{
  //TODO: Optimize here, filled_size++ in add is being called several times
  //but can be called only once too
  //if  (false and vec_emb1.get_n_vertices () != size) {
  //  printf ("vec_emb1.get_n_vertices () %ld != size %d\n", vec_emb1.get_n_vertices (), size);
  //  assert (false);
  //}
  for (int i = 0; i < vec_emb1.get_n_vertices (); i++) {
    vec_emb2.add (vec_emb1.get_vertex (i));
  }
}

template <uint32_t size> 
void bitvector_to_vector_embedding (BitVectorVertexEmbedding& bit_emb, 
                                    VectorVertexEmbedding<size>& vec_emb)
{
  for (int u = 0; u < N; u++) {
    if (bit_emb.test(u)) {
      vec_emb.add (u);
    }
  }
}

void print_embedding (BitVectorVertexEmbedding embedding, std::ostream& os);

std::vector<BitVectorVertexEmbedding> get_extensions_bitvector (BitVectorVertexEmbedding& embedding, CSR* csr)
{
  std::vector<BitVectorVertexEmbedding> extensions;

  if (embedding.all_false ()) {
    for (int u = 0; u < N; u++) {
      BitVectorVertexEmbedding extension;
      extension.set(u);
      //print_embedding (extension, std::cout);
      //std::cout << " " << u << std::endl;
      extensions.push_back (extension);
    }
  } else {
    for (int u = 0; u < N; u++) {
      if (embedding.test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding.test (v) == false) {
            BitVectorVertexEmbedding extension = BitVectorVertexEmbedding(embedding);
            extension.set(v);
            extensions.push_back(extension);
          }
        }
      }
    }
  }

  return extensions;
}

template <uint32_t size>
std::vector<VectorVertexEmbedding<size+1>> get_extensions_vector (VectorVertexEmbedding<size>& embedding, CSR* csr)
{
  std::vector<VectorVertexEmbedding<size+1>> extensions;

  if (embedding.get_n_vertices () == 0) {
    for (int u = 0; u < N; u++) {
      VectorVertexEmbedding<size+1> extension;
      extension.add(u);
      //print_embedding (extension, std::cout);
      //std::cout << " " << u << std::endl;
      extensions.push_back (extension);
    }
  } else {
    for (int i = 0; i < embedding.get_n_vertices (); i++) {
      int u = embedding.get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        int v = csr->get_edges () [e];
        if (embedding.has (v) == false) {
          VectorVertexEmbedding<size+1> extension;
          vector_embedding_from_one_less_size (embedding, extension);
          extension.add(v);
          extensions.push_back(extension);
        }
      }
    }
  }

  return extensions;
}

std::vector<BitVectorVertexEmbedding> get_initial_embedding_bitvector (CSR* csr)
{
  BitVectorVertexEmbedding embedding;
  std::vector <BitVectorVertexEmbedding> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

std::vector<VectorVertexEmbedding<0>> get_initial_embedding_vector (CSR* csr)
{
  VectorVertexEmbedding<0> embedding;
  std::vector <VectorVertexEmbedding<0>> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

bool (*filter) (CSR* csr, BitVectorVertexEmbedding& embedding);
void (*process) (std::vector<BitVectorVertexEmbedding>& output, BitVectorVertexEmbedding& embedding);

__host__ __device__
bool clique_filter (CSR* csr, BitVectorVertexEmbedding* embedding)
{
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      for (int v = 0; v < N; v++) {
        if (u != v and embedding->test(v)) {
          if (!csr->has_edge (u, v)) {
            return false;
          }
        }
      }
    }
  }

  return true;
}


template <uint32_t size>
__host__ __device__
bool clique_filter_vector (CSR* csr, VectorVertexEmbedding<size>* embedding)
{
  for (int i = 0; i < embedding->get_n_vertices (); i++) {
    int u = embedding->get_vertex (i);
    for (int j = 0; j < embedding->get_n_vertices (); j++) {
      int v = embedding->get_vertex (j);
      if (u != v and embedding->has (v)) {
        if (!csr->has_edge (u, v)) {
          return false;
        }
      }
    }
  }

  return true;
}

void clique_process_bit_vector (std::vector<BitVectorVertexEmbedding>& output, BitVectorVertexEmbedding& embedding)
{
  output.push_back (embedding);
}

template <uint32_t size>
void clique_process_vector (std::vector<VectorVertexEmbedding<size>>& output, VectorVertexEmbedding<size>& embedding)
{
  output.push_back (embedding);
}

void run_single_step_initial_bitvector (void* input, int n_embeddings, CSR* csr,
                      std::vector<BitVectorVertexEmbedding>& output,
                      std::vector<BitVectorVertexEmbedding>& next_step)
{
  BitVectorVertexEmbedding* embeddings = (BitVectorVertexEmbedding*)input;

  for (int i = 0; i < n_embeddings; i++) {
    BitVectorVertexEmbedding embedding = embeddings[i];
    std::vector<BitVectorVertexEmbedding> extensions = get_extensions_bitvector (embedding, csr);

    for (auto extension : extensions) {
      if (clique_filter (csr, &extension)) {
        clique_process_bit_vector (output, extension);
        next_step.push_back (extension);
      }
    }
  }
}

void run_single_step_initial_vector (void* input, int n_embeddings, CSR* csr,
                      std::vector<VectorVertexEmbedding<1>>& output,
                      std::vector<VectorVertexEmbedding<1>>& next_step)
{
  VectorVertexEmbedding<0>* embeddings = (VectorVertexEmbedding<0>*)input;

  for (int i = 0; i < n_embeddings; i++) {
    VectorVertexEmbedding<0> embedding = embeddings[i];
    std::vector<VectorVertexEmbedding<1>> extensions = get_extensions_vector (embedding, csr);
    std::cout << "extensions " << extensions.size () << std::endl;
    for (auto extension : extensions) {
      if (clique_filter_vector (csr, &extension)) {
        clique_process_vector (output, extension);
        next_step.push_back (extension);
      }
    }
  }
}

__device__
void printf_embedding (BitVectorVertexEmbedding* embedding)
{
  printf ("[");
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      printf ("%d, ", u);
    }
  }

  printf ("]\n");
}

/*__global__
void run_single_step_bitvector_embedding (void* input, int n_embeddings, CSR* csr,
                      void* output_ptr,
                      int* n_output,
                      void* next_step, int* n_next_step,
                      int* n_output_1, int* n_next_step_1)
{
  int id;

#ifdef USE_CSR_IN_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread,
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread,
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#else
#ifdef USE_CONSTANT_MEM
  csr = (CSR*) csr_constant_buff;
#endif

#endif

  BitVectorVertexEmbedding* embeddings = (BitVectorVertexEmbedding*)input;
  BitVectorVertexEmbedding* new_embeddings = (BitVectorVertexEmbedding*)next_step;
  BitVectorVertexEmbedding* output = ((BitVectorVertexEmbedding*)output_ptr);
#ifdef USE_EMBEDDING_IN_LOCAL_MEM
  unsigned char temp_buffer [sizeof(BitVectorVertexEmbedding)];
#endif
  id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
#ifdef USE_FIXED_THREADS
  if (n_embeddings >= MAX_CUDA_THREADS) {
    int embeddings_per_thread = n_embeddings/MAX_CUDA_THREADS+1;

    start = id*embeddings_per_thread;
    end = (id+1)*embeddings_per_thread < n_embeddings ? (id+1)*embeddings_per_thread : n_embeddings;
  } else {
    if (id >= n_embeddings)
      return;

    start = id;
    end = id+1;
  }
#else
  if (id >= n_embeddings)
      return;

  start = id;
  end = id+1;
#endif

  int q[1000] = {0};

#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #if 0
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[per_thread_shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[0];
  #else
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
  #endif
#endif

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int lane_id = threadIdx.x % WARP_SIZE;
  for (int i = start; i < end; i++) {
    #ifdef USE_EMBEDDING_IN_SHARED_MEM
      #ifdef SHARED_MEM_NON_COALESCING
        memcpy (local_shared_buff, &embeddings[i], sizeof(BitVectorVertexEmbedding));

        BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*) local_shared_buff;
      #else
        const int thread_block_size = WARP_SIZE;
        const int last_emb = WARP_SIZE*(warp_id+1);
        if (blockIdx.x*blockDim.x + (warp_id+1)*WARP_SIZE > n_embeddings) {
          assert (false);
          //thread_block_size = n_embeddings - blockIdx.x*blockDim.x -
          //                    warp_id*WARP_SIZE;
          //last_emb = warp_id*WARP_SIZE + thread_block_size;
        }

        for (int emb = WARP_SIZE*warp_id; emb < last_emb; emb++) {
          SharedMemElem* embedding_buff = (SharedMemElem*) &embeddings[emb+blockIdx.x*blockDim.x];

          for (int j = 0; j < sizeof (BitVectorVertexEmbedding)/sizeof (SharedMemElem);
               j += thread_block_size) {
            int idx = per_thread_shared_mem_size/sizeof(SharedMemElem)*emb;
            if (true or j + lane_id  < sizeof (BitVectorVertexEmbedding)/sizeof (SharedMemElem)) { //TODO: Remove this if by doing padding with VertexEmbedding
              shared_buff[idx + j + lane_id] = embedding_buff[j + lane_id];
            }
          }
        }

        BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*) local_shared_buff;
      #endif
      //embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_LOCAL_MEM)
      //memcpy (&temp[0], &embeddings[i], sizeof (VertexEmbedding));
      memcpy (&temp_buffer[0], &embeddings[i], sizeof(BitVectorVertexEmbedding));
      BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*)&temp_buffer[0];
      //VertexEmbedding* embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_GLOBAL_MEM)
      BitVectorVertexEmbedding* embedding = &embeddings[i];
    #else
      #error "None of USE_EMBEDDING_IN_*_MEM option defined"
    #endif

  #if 1
    typedef uint32_t VertexEmbeddingChange;
    const uint32_t max_changes = 32; //max changes per warp
    //__shared__ int32_t shared_mem [max_changes*THREAD_BLOCK_SIZE/WARP_SIZE+THREAD_BLOCK_SIZE/WARP_SIZE+1];
    __shared__ VertexEmbeddingChange changes[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t changed_thread_ids[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    //For each warp record new embeddings (or changes) using atomic operations
    __shared__ int32_t n_extensions[THREAD_BLOCK_SIZE/WARP_SIZE];
    n_extensions[warp_id] = 0;
    __shared__ uint32_t prev_n_outputs [THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t prev_n_next_steps [THREAD_BLOCK_SIZE/WARP_SIZE];
    
    const uint32_t mask = ~0U;
    for (int u = 0; u < N; u++) {
      int e = csr->get_start_edge_idx(u);
      
      while (true) {
        
        int predicate = 1;
        //n_extensions[warp_id] = 0;

        if (e <= csr->get_end_edge_idx(u)) {
          int v = csr->get_edges () [e];
          if (embedding->test(u) and embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              int prev_n_extensions = atomicAdd (&n_extensions[warp_id], 1);
              changes[warp_id*max_changes+prev_n_extensions] = v;
              changed_thread_ids[warp_id*max_changes+prev_n_extensions] = id;
              //memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (VertexEmbedding));
              //memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (VertexEmbedding));
            }

            extension->reset(v);
          }

          e++;

          if (e > csr->get_end_edge_idx(u)) {
            predicate = 0;
          }
        } else {
          predicate = 0;
        }

        int32_t n_changes = n_extensions[warp_id];

        if (n_changes > max_changes)
          assert (false);

        //assert (__activemask () == ~0U);
        uint32_t orig_prev_n_output = 0;
        uint32_t orig_prev_n_next_step = 0;
        
        if (lane_id == 0) {
          if (n_changes > 0) {
            n_extensions[warp_id] = 0;
            orig_prev_n_output = atomicAdd (n_output, n_changes);
            orig_prev_n_next_step = atomicAdd (n_next_step, n_changes);
            //if (warp_id == 2 && (threadIdx.x == 64 || threadIdx.x == 65) && blockIdx.x == 0) {
            //  printf ("prev_n_next_steps[warp_id]: %d\n", prev_n_next_steps[warp_id]);
            //}
          }
        }
        
        //__syncwarp ();
        
        if (n_changes > 0) {
          
          uint32_t prev_n_output = __shfl_sync (__activemask (), orig_prev_n_output, 0);
          uint32_t prev_n_next_step = __shfl_sync (__activemask (), orig_prev_n_next_step, 0);
          
          for (int i = 0; i < n_changes; i++) {
            
            int changes_idx = warp_id*max_changes + i;
            uint32_t expected_thread_id = changed_thread_ids[changes_idx];
            if (expected_thread_id == id) {
              int v = changes[changes_idx];
              BitVectorVertexEmbedding* extension = embedding;
              extension->set (v);
              
              memcpy (&output[prev_n_output + i], extension, 
                      sizeof(BitVectorVertexEmbedding));
              memcpy (&new_embeddings[prev_n_next_step + i], extension,
                      sizeof (BitVectorVertexEmbedding));
              extension->reset (v);
            }
          }
        }

        if (n_extensions[warp_id] != 0) {
          printf ("n_extensions[warp_id] not zero but is %d\n", n_extensions[warp_id]);
        }
        if (!__any_sync (__activemask (), predicate)) {
          break;
        }
        //n_extensions[warp_id] = 0;
      }
    }
  #else
    for (int u = 0; u < N; u++) {
      if (embedding->test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (BitVectorVertexEmbedding));
              memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (BitVectorVertexEmbedding));
            }
            extension->reset(v);
          }
        }
      }
    }
  #endif

  }

  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}*/


template <size_t size>
__host__ __device__
inline bool is_embedding_canonical (CSR* csr, VectorVertexEmbedding<size>* embedding, int v)
{
  if (embedding->get_vertex (0) > v)
    return false;
  
  if (size <= 2)
    return true;
  
  bool found_neighbor = false;
  for (int j = 0; j < embedding->get_n_vertices (); j++) {
    int v_j = embedding->get_vertex (j);
    if (found_neighbor == false && csr->has_edge (v_j, v)) {
      found_neighbor = true;
    } else if (found_neighbor == true && v_j > v) {
      return false;
    }
  }

  return true;
}

template <size_t embedding_size> 
__global__
void run_single_step_vectorvertex_embedding (void* input, int n_embeddings, CSR* csr,
                      void* output_ptr,
                      int* n_output,
                      void* next_step, int* n_next_step,
                      int* n_output_1, int* n_next_step_1,
                      int only_copy_change)
{
  int id;

#ifdef USE_CSR_IN_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread,
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread,
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#else
#ifdef USE_CONSTANT_MEM
  csr = (CSR*) csr_constant_buff;
#endif

#endif

  VectorVertexEmbedding<embedding_size>* embeddings = (VectorVertexEmbedding<embedding_size>*)input;
  
#ifdef USE_EMBEDDING_IN_LOCAL_MEM
  unsigned char temp_buffer [sizeof(VectorVertexEmbedding<embedding_size+1>)];
#endif
  id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
#ifdef USE_FIXED_THREADS
  if (n_embeddings >= MAX_CUDA_THREADS) {
    int embeddings_per_thread = n_embeddings/MAX_CUDA_THREADS+1;

    start = id*embeddings_per_thread;
    end = (id+1)*embeddings_per_thread < n_embeddings ? (id+1)*embeddings_per_thread : n_embeddings;
  } else {
    if (id >= n_embeddings)
      return;

    start = id;
    end = id+1;
  }
#else
  if (id >= n_embeddings)
      return;

  start = id;
  end = id+1;
#endif

  int q[1000] = {0};

#ifdef USE_EMBEDDING_IN_SHARED_MEM
//TODO: Support VectorVertexEmbedding
  #if 0
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    assert (per_thread_shared_mem_size >= sizeof (VectorVertexEmbedding<embedding_size>));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[per_thread_shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[0];
  #else
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
  #endif
#endif

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int lane_id = threadIdx.x % WARP_SIZE;
  for (int i = start; i < end; i++) {
    #ifdef USE_EMBEDDING_IN_SHARED_MEM
    //TODO: Support VectorVertexEmbedding, size+1
      #ifdef SHARED_MEM_NON_COALESCING
        memcpy (local_shared_buff, &embeddings[i], sizeof(VectorVertexEmbedding<embedding_size>));

        VectorVertexEmbedding<embedding_size>* embedding = (VectorVertexEmbedding<embedding_size>*) local_shared_buff;
      #else
        const int thread_block_size = WARP_SIZE;
        const int last_emb = WARP_SIZE*(warp_id+1);
        if (blockIdx.x*blockDim.x + (warp_id+1)*WARP_SIZE > n_embeddings) {
          assert (false);
          /*thread_block_size = n_embeddings - blockIdx.x*blockDim.x -
                              warp_id*WARP_SIZE;
          last_emb = warp_id*WARP_SIZE + thread_block_size;*/
        }

        for (int emb = WARP_SIZE*warp_id; emb < last_emb; emb++) {
          SharedMemElem* embedding_buff = (SharedMemElem*) &embeddings[emb+blockIdx.x*blockDim.x];

          for (int j = 0; j < sizeof (VectorVertexEmbedding<embedding_size>)/sizeof (SharedMemElem);
               j += thread_block_size) {
            int idx = per_thread_shared_mem_size/sizeof(SharedMemElem)*emb;
            if (true or j + lane_id  < sizeof (VectorVertexEmbedding<embedding_size>)/sizeof (SharedMemElem)) { //TODO: Remove this if by doing padding with VertexEmbedding
              shared_buff[idx + j + lane_id] = embedding_buff[j + lane_id];
            }
          }
        }

        VectorVertexEmbedding<embedding_size>* embedding = (VectorVertexEmbedding<embedding_size>*) local_shared_buff;
      #endif
      //embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_LOCAL_MEM)
      //memcpy (&temp[0], &embeddings[i], sizeof (VertexEmbedding));
      //memcpy (&temp_buffer[0], &embeddings[i], sizeof(VectorVertexEmbedding<embedding_size>));
      VectorVertexEmbedding<embedding_size+1>* embedding = (VectorVertexEmbedding<embedding_size+1>*)&temp_buffer[0];
      embedding->clear ();
      vector_embedding_from_one_less_size (embeddings[i], *embedding);
      //VertexEmbedding* embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_GLOBAL_MEM)
    //TODO: Support VectorVertexEmbedding with size + 1, below is wrong
      VectorVertexEmbedding<embedding_size+1>* embedding = &embeddings[i];
    #else
      #error "None of USE_EMBEDDING_IN_*_MEM option defined"
    #endif

  #if 0
  //TODO: Support VectorVertexEmbedding with size + 1.
    typedef uint32_t VertexEmbeddingChange;
    const uint32_t max_changes = 32; //max changes per warp
    //__shared__ int32_t shared_mem [max_changes*THREAD_BLOCK_SIZE/WARP_SIZE+THREAD_BLOCK_SIZE/WARP_SIZE+1];
    __shared__ VertexEmbeddingChange changes[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t changed_thread_ids[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    //For each warp record new embeddings (or changes) using atomic operations
    __shared__ int32_t n_extensions[THREAD_BLOCK_SIZE/WARP_SIZE];
    n_extensions[warp_id] = 0;
    __shared__ uint32_t prev_n_outputs [THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t prev_n_next_steps [THREAD_BLOCK_SIZE/WARP_SIZE];
    
    const uint32_t mask = ~0U;
    for (int u = 0; u < N; u++) {
      int e = csr->get_start_edge_idx(u);
      
      while (true) {
        
        int predicate = 1;
        //n_extensions[warp_id] = 0;

        if (e <= csr->get_end_edge_idx(u)) {
          int v = csr->get_edges () [e];
          if (embedding->test(u) and embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              int prev_n_extensions = atomicAdd (&n_extensions[warp_id], 1);
              changes[warp_id*max_changes+prev_n_extensions] = v;
              changed_thread_ids[warp_id*max_changes+prev_n_extensions] = id;
              //memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (VertexEmbedding));
              //memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (VertexEmbedding));
            }

            extension->reset(v);
          }

          e++;

          if (e > csr->get_end_edge_idx(u)) {
            predicate = 0;
          }
        } else {
          predicate = 0;
        }

        int32_t n_changes = n_extensions[warp_id];

        if (n_changes > max_changes)
          assert (false);

        //assert (__activemask () == ~0U);
        uint32_t orig_prev_n_output = 0;
        uint32_t orig_prev_n_next_step = 0;
        
        if (lane_id == 0) {
          if (n_changes > 0) {
            n_extensions[warp_id] = 0;
            orig_prev_n_output = atomicAdd (n_output, n_changes);
            orig_prev_n_next_step = atomicAdd (n_next_step, n_changes);
            //if (warp_id == 2 && (threadIdx.x == 64 || threadIdx.x == 65) && blockIdx.x == 0) {
            //  printf ("prev_n_next_steps[warp_id]: %d\n", prev_n_next_steps[warp_id]);
            //}
          }
        }
        
        //__syncwarp ();
        
        if (n_changes > 0) {
          
          uint32_t prev_n_output = __shfl_sync (__activemask (), orig_prev_n_output, 0);
          uint32_t prev_n_next_step = __shfl_sync (__activemask (), orig_prev_n_next_step, 0);
          
          for (int i = 0; i < n_changes; i++) {
            
            int changes_idx = warp_id*max_changes + i;
            uint32_t expected_thread_id = changed_thread_ids[changes_idx];
            if (expected_thread_id == id) {
              int v = changes[changes_idx];
              BitVectorVertexEmbedding* extension = embedding;
              extension->set (v);
              
              memcpy (&output[prev_n_output + i], extension, 
                      sizeof(BitVectorVertexEmbedding));
              memcpy (&new_embeddings[prev_n_next_step + i], extension,
                      sizeof (BitVectorVertexEmbedding));
              extension->reset (v);
            }
          }
        }

        if (n_extensions[warp_id] != 0) {
          printf ("n_extensions[warp_id] not zero but is %d\n", n_extensions[warp_id]);
        }
        if (!__any_sync (__activemask (), predicate)) {
          break;
        }
        //n_extensions[warp_id] = 0;
      }
    }
  #else
    for (int i = 0; i < embedding->get_n_vertices (); i++) {
      int u = embedding->get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        int v = csr->get_edges () [e];

        if (is_embedding_canonical<embedding_size+1> (csr, embedding, v) && embedding->has (v) == false) {
          VectorVertexEmbedding<embedding_size+1>* extension = embedding;
          extension->add_unsorted (v);
          
          if (clique_filter_vector (csr, extension)) {
            //VectorVertexEmbedding<embedding_size+1> extension = *embedding;
            //extension.add_last_in_sort_order ();
            int o = atomicAdd(n_output,1);
            int n = atomicAdd(n_next_step,1);
            
            if (only_copy_change) {
              int* new_embeddings = (int*) next_step;
              int* output = (int*) output_ptr;

              new_embeddings[2*n] = id;
              new_embeddings[2*n+1] = v;
              output[2*o] = id;
              output[2*o+1] = v;
            }
            else {
              VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step;
              VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
              memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
              memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
            }
            //output[o].add_last_in_sort_order ();
            //new_embeddings[n].add_last_in_sort_order ();
          }
          extension->remove_last ();
        }
      }
    }
  #endif
  }

  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}

void print_embedding (BitVectorVertexEmbedding embedding, std::ostream& os)
{
  os << "[";
  for (int u = 0; u < N; u++) {
    if (embedding.test(u)) {
      os << u << ", ";
    }
  }
  os << "]";
}

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

double_t convertTimeValToDouble (struct timeval _time)
{
  return ((double_t)_time.tv_sec) + ((double_t)_time.tv_usec)/1000000.0f;
}

struct timeval getTimeOfDay ()
{
  struct timeval _time;

  if (gettimeofday (&_time, NULL) == -1) {
    fprintf (stderr, "gettimeofday returned -1\n");
    perror ("");
    abort ();
  }

  return _time;
}

enum EmbeddingType {
  VectorVertex,
  BitVector,
};

int main (int argc, char* argv[])
{
  std::vector<Vertex> vertices;
  int n_edges = 0;

  if (argc < 2) {
    std::cout << "Arguments: graph-file" << std::endl;
    return -1;
  }

  char* graph_file = argv[1];
  FILE* fp = fopen (graph_file, "r+");
  if (fp == nullptr) {
    std::cout << "File '" << graph_file << "' not found" << std::endl;
    return 1;
  }

  while (true) {
    char line[LINE_SIZE];
    char num_str[LINE_SIZE];
    size_t line_size;

    if (fgets (line, LINE_SIZE, fp) == nullptr) {
      break;
    }

    int id, label;
    int bytes_read;

    bytes_read = sscanf (line, "%d %d", &id, &label);
    Vertex vertex (id, label);
    char* _line = line + chars_in_int (id) + chars_in_int (label);
    do {
      int num;

      bytes_read = sscanf (_line, "%d", &num);
      if (bytes_read > 0) {
        vertex.add_edge (num);
        _line += chars_in_int (num);
        n_edges++;
      }

    } while (bytes_read > 0);

    vertex.sort_edges ();

    vertices.push_back (vertex);
  }

  fclose (fp);

  std::cout << "n_edges "<<n_edges <<std::endl;
  std::cout << "vertices " << vertices.size () << std::endl; 
  Graph graph (vertices, n_edges);

  CSR* csr = new CSR(N, N_EDGES);
  std::cout << "sizeof(CSR)"<< sizeof(CSR)<<std::endl;
  std::cout <<"sizeof(VertexEmbedding)" << sizeof(BitVectorVertexEmbedding) << std::endl;
  csr_from_graph (csr, graph);
  
#ifdef USE_CONSTANT_MEM
  cudaMemcpyToSymbol (csr_constant_buff, csr, sizeof(CSR));
  //~ CSR* csr_constant = (CSR*) &csr_constant_buff[0];
  //~ csr_constant->n_vertices = csr->get_n_vertices ();
  //~ printf ("csr->get_n_vertices () = %d\n", csr->get_n_vertices ());
  //~ csr_constant->n_edges = csr->get_n_edges ();
  //~ csr_constant->copy_vertices (csr, 0, csr->get_n_vertices ());
  //~ csr_constant->copy_edges (csr, 0, csr->get_n_edges ());
#endif

  std::vector<VectorVertexEmbedding<0>> initial_embeddings = get_initial_embedding_vector (csr);
  std::vector<VectorVertexEmbedding<1>> output_1;
  std::vector<VectorVertexEmbedding<2>> output_2;
  std::vector<VectorVertexEmbedding<3>> output_3;
  std::vector<VectorVertexEmbedding<4>> output_4;
  std::vector<VectorVertexEmbedding<5>> output_5;
  std::vector<VectorVertexEmbedding<6>> output_6;
  std::vector<VectorVertexEmbedding<7>> output_7;
  std::vector<VectorVertexEmbedding<8>> output_8;
  std::vector<std::pair<void*, size_t>> embeddings;
  //filter = clique_filter;
  //process = clique_process;
  size_t new_embeddings_size = 0;
  int iter = 0;
  {
    std::vector<VectorVertexEmbedding<1>> new_embeddings;
    run_single_step_initial_vector (&initial_embeddings[0], 1, csr, 
                                    output_1, new_embeddings);
    new_embeddings_size = new_embeddings.size ();
    embeddings.push_back (std::make_pair (malloc (sizeof (VectorVertexEmbedding<1>)*new_embeddings_size), new_embeddings_size));
    for (int i = 0; i < new_embeddings_size; i++) {
      ((VectorVertexEmbedding<1>*)embeddings[0].first)[i] = new_embeddings[i];
      int v = ((VectorVertexEmbedding<1>*)embeddings[0].first)[i].get_vertex (0);
      assert (v >= 0);
    }
  }

  iter = 1;
  double total_stream_time = 0;
  size_t global_mem_size = 10*1024*1024*1024UL;
#define PINNED_MEMORY
#ifdef PINNED_MEMORY
  char* global_mem_ptr;
  hipError_t malloc_error = hipHostMalloc ((void**)&global_mem_ptr, global_mem_size, hipHostMallocDefault);
  assert (malloc_error == hipSuccess);
#else
  char* global_mem_ptr = new char[global_mem_size];
#endif

  const size_t max_embedding_size_per_iter = (2000000/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
  double_t kernelTotalTime = 0.0;
  for (iter; iter < 7 && new_embeddings_size > 0; iter++) {
    std::cout << "iter " << iter << " embeddings " << new_embeddings_size << std::endl;
    
    size_t remaining_embeddings = new_embeddings_size;
    size_t n_embeddings = new_embeddings_size;
    #ifdef DEBUG
      memset (global_mem_ptr, 0, global_mem_size);
    #endif

    //Copy all embeddings to global memory
    size_t embedding_size = 0;
    size_t new_embedding_size = 0;
    size_t global_mem_iter = 0;
    switch (iter) {
      case 1: {
        embedding_size = sizeof (VectorVertexEmbedding<1>);
        new_embedding_size = sizeof (VectorVertexEmbedding<2>);
        
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<1>*)global_mem_ptr)[global_mem_iter] = ((VectorVertexEmbedding<1>*) iter.first)[i];
            int v = ((VectorVertexEmbedding<1>*)global_mem_ptr)[global_mem_iter].get_vertex (0);
            global_mem_iter++;
            assert (v >= 0);
          }
        }
        break;
      }      
      case 2: {
        embedding_size = sizeof (VectorVertexEmbedding<2>);
        new_embedding_size = sizeof (VectorVertexEmbedding<3>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<2>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<2>*)iter.first)[i];
          }
        }
        break;
      }
      
      case 3: {
        embedding_size = sizeof (VectorVertexEmbedding<3>);
        new_embedding_size = sizeof (VectorVertexEmbedding<4>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<3>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<3>*)iter.first)[i];
          }
        }
        break;
      }
      
      case 4: {
          embedding_size = sizeof (VectorVertexEmbedding<4>);
          new_embedding_size = sizeof (VectorVertexEmbedding<5>);
          for (auto iter: embeddings) {
            for (int i = 0; i < iter.second; i++) {
              ((VectorVertexEmbedding<4>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<4>*)iter.first)[i];
            }
          }
        break;
      }
      case 5: {
        embedding_size = sizeof (VectorVertexEmbedding<5>);
        new_embedding_size = sizeof (VectorVertexEmbedding<6>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<5>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<5>*)iter.first)[i];
          }
        }
        break;
      }
      case 6: {
        embedding_size = sizeof (VectorVertexEmbedding<6>);
        new_embedding_size = sizeof (VectorVertexEmbedding<7>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<6>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<6>*)iter.first)[i];
          }
        }
        break;
      }
      case 7: {
        embedding_size = sizeof (VectorVertexEmbedding<7>);
        new_embedding_size = sizeof (VectorVertexEmbedding<8>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<7>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<7>*)iter.first)[i];
          }
        }
        break;
      }
      case 8: {
        embedding_size = sizeof (VectorVertexEmbedding<8>);
        new_embedding_size = sizeof (VectorVertexEmbedding<9>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<8>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<8>*)iter.first)[i];
          }
        }
        break;
      }
    }

    //delete embeddings too because there is a memory leak?
    if (iter > 1) {
      for (auto iter: embeddings) {
        free(iter.first);
      }
    }

    embeddings.clear ();
    std::cout << "Copying to global_mem_ptr done. global mem used " << global_mem_iter*embedding_size << std::endl;
    
    void* embeddings_ptr = global_mem_ptr;

    size_t n_next_step_embeddings = 0;
    n_embeddings = 0;

    void* orig_new_embeddings_ptr = ((char*)global_mem_ptr) + (global_mem_iter)*(new_embedding_size); //Size of next embedding will be one more
    size_t max_embeddings = 40000000; //There is something with this value which makes it perform better, may be alignment?
    printf ("new_embedding_size %ld\n", new_embedding_size);
    void* orig_output_ptr = (char*)orig_new_embeddings_ptr + (max_embeddings)*(new_embedding_size);

    
    double stream_time_1 = convertTimeValToDouble (getTimeOfDay ());

    while (remaining_embeddings != 0) {      
      n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
      std::cout << "iter " << iter << " n_embeddings " << new_embeddings_size << " remaining_embeddings " << remaining_embeddings << std::endl;
      embeddings_ptr = ((char*)global_mem_ptr) + embedding_size*(new_embeddings_size - remaining_embeddings);
      //printf ("embeddings_ptr %x\n", embeddings_ptr);
      n_embeddings = std::min (remaining_embeddings, max_embedding_size_per_iter);

      remaining_embeddings -= n_embeddings;
      //n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
      
      const int N_STREAMS = 1;
      int only_copy_change = 0;
      assert (only_copy_change == 0); //TODO: Streams with only copy change
      void* new_embeddings_ptr[N_STREAMS];
      assert (max_embeddings%N_STREAMS == 0);
      for (int i = 0; i < N_STREAMS; i++) {
        new_embeddings_ptr[i] = (char*)orig_new_embeddings_ptr + i*new_embedding_size*max_embeddings/N_STREAMS;
      }

      void* output_ptr[N_STREAMS];
      for (int i = 0; i < N_STREAMS; i++) {
        output_ptr[i] = (char*)orig_output_ptr + i*new_embedding_size*max_embeddings;
      }
      int n_new_embeddings[N_STREAMS] = {0};
      int n_new_embeddings_1[N_STREAMS] = {0};
      int n_output[N_STREAMS] = {0};
      int n_output_1[N_STREAMS] = {0};
      char* device_embeddings[N_STREAMS];
      char *device_new_embeddings[N_STREAMS];
      int* device_n_embeddings[N_STREAMS];
      int* device_n_embeddings_1[N_STREAMS];
      char *device_outputs[N_STREAMS];
      int* device_n_outputs[N_STREAMS];
      int* device_n_outputs_1[N_STREAMS];
      CSR* device_csr[N_STREAMS];
      
      assert (N_STREAMS >= 1);

      for (int i = 0; i < N_STREAMS; i++) {
        const bool unified_mem = false;
        if (unified_mem == true) {
          //cudaMallocManaged (embeddings_ptr, n_embeddings*embedding_size);
          //device_embeddings = (char*)embeddings_ptr;
          assert(false);
        } else {
          hipMalloc (&device_embeddings[i], n_embeddings/N_STREAMS*embedding_size);
          hipMemcpy (device_embeddings[i], (char*)embeddings_ptr + i*n_embeddings/N_STREAMS*embedding_size,
                      n_embeddings/N_STREAMS*embedding_size, hipMemcpyHostToDevice);
        }
        hipMalloc (&device_new_embeddings[i], max_embeddings/N_STREAMS*(new_embedding_size));
        hipMalloc (&device_outputs[i], max_embeddings/N_STREAMS*(new_embedding_size));
        hipMalloc (&device_n_embeddings[i], sizeof (0));
        hipMalloc (&device_n_embeddings_1[i], sizeof (0));
        hipMalloc (&device_n_outputs[i], sizeof (0));
        hipMalloc (&device_n_outputs_1[i], sizeof (0));
        hipMalloc (&device_csr[i], sizeof(CSR)); //TODO: Remove copying CSR graph again and again
        
        hipMemcpy (device_n_embeddings[i], &n_new_embeddings[i],
                    sizeof (n_new_embeddings[i]), hipMemcpyHostToDevice);
        hipMemcpy (device_n_outputs[i], &n_output[i], sizeof (n_output[i]),
                    hipMemcpyHostToDevice);

        hipMemcpy (device_n_embeddings_1[i], &n_new_embeddings_1[i],
                    sizeof (n_new_embeddings_1[i]), hipMemcpyHostToDevice);
        hipMemcpy (device_n_outputs_1[i], &n_output_1[i], sizeof (n_output_1[i]),
                    hipMemcpyHostToDevice);

        hipMemcpy (device_csr[i], csr, sizeof (CSR), hipMemcpyHostToDevice);
        
        hipError_t error = hipGetLastError ();
        if (error != hipSuccess) {
          const char* error_string = hipGetErrorString (error);
          std::cout << "Cuda host to device copy error " << error_string << std::endl;
        } else {
          std::cout << "Cuda host to device copy success " << std::endl;
        }

        std::cout << "starting kernel with n_embeddings: " << n_embeddings;
      
        double t1 = convertTimeValToDouble (getTimeOfDay ());
        
    #ifdef USE_FIXED_THREADS
        //std::cout << " threads: " << MAX_CUDA_THREADS/THREAD_BLOCK_SIZE << std::endl;
        int thread_blocks = MAX_CUDA_THREADS/THREAD_BLOCK_SIZE;
    #else
        int thread_blocks = (n_embeddings%THREAD_BLOCK_SIZE != 0) ? (n_embeddings/THREAD_BLOCK_SIZE+1) : n_embeddings/THREAD_BLOCK_SIZE;
    #endif
        std::cout << " threads: " << thread_blocks << std::endl;
        
        switch (iter) {
          case 1: {
            run_single_step_vectorvertex_embedding<1><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings[i], n_embeddings/N_STREAMS, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings[i], device_n_embeddings[i],
                                  device_n_outputs_1[i], device_n_embeddings_1[i], only_copy_change);
            break;
          }
          case 2: {
            run_single_step_vectorvertex_embedding<2><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings[i], n_embeddings/N_STREAMS, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings[i], device_n_embeddings[i],
                                  device_n_outputs_1[i], device_n_embeddings_1[i], only_copy_change);
            break;
          }
          case 3: {
            run_single_step_vectorvertex_embedding<3><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings[i], n_embeddings/N_STREAMS, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings[i], device_n_embeddings[i],
                                  device_n_outputs_1[i], device_n_embeddings_1[i], only_copy_change);
            break;
          }
          case 4: {
            run_single_step_vectorvertex_embedding<4><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings[i], n_embeddings/N_STREAMS, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings[i], device_n_embeddings[i],
                                  device_n_outputs_1[i], device_n_embeddings_1[i], only_copy_change);
            break;
          }
          case 5: {
            run_single_step_vectorvertex_embedding<5><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings[i], n_embeddings/N_STREAMS, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings[i], device_n_embeddings[i],
                                  device_n_outputs_1[i], device_n_embeddings_1[i], only_copy_change);
            break;
          }
          case 6: {
            run_single_step_vectorvertex_embedding<6><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings[i], n_embeddings/N_STREAMS, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings[i], device_n_embeddings[i],
                                  device_n_outputs_1[i], device_n_embeddings_1[i], only_copy_change);
            break;
          }
          case 7: {
            run_single_step_vectorvertex_embedding<7><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings[i], n_embeddings/N_STREAMS, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings[i], device_n_embeddings[i],
                                  device_n_outputs_1[i], device_n_embeddings_1[i], only_copy_change);
            break;
          }
          case 8: {
            run_single_step_vectorvertex_embedding<8><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings[i], n_embeddings/N_STREAMS, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings[i], device_n_embeddings[i],
                                  device_n_outputs_1[i], device_n_embeddings_1[i], only_copy_change);
            break;
          }
        }
        
        hipDeviceSynchronize ();

        double t2 = convertTimeValToDouble (getTimeOfDay ());

        std::cout << "Execution time " << (t2-t1) << " secs" << std::endl;
        kernelTotalTime += (t2-t1);

        error = hipGetLastError ();
        if (error != hipSuccess) {
          const char* error_string = hipGetErrorString (error);
          std::cout << "Cuda kernel error " << error_string << std::endl;
        } else {
          std::cout << "Cuda success " << std::endl;
        }

        hipMemcpy (&n_new_embeddings[i], device_n_embeddings[i], sizeof(0), hipMemcpyDeviceToHost);

        hipMemcpy (&n_output[i], device_n_outputs[i], sizeof(0), hipMemcpyDeviceToHost);
        if (only_copy_change) {
          assert (false);
          //TODO: Change this to make an array of such ptrs
          hipMemcpy (new_embeddings_ptr, device_new_embeddings[i], n_new_embeddings[i]*2*sizeof(int), hipMemcpyDeviceToHost);
          hipMemcpy (output_ptr, device_outputs[i], n_output[i]*2*sizeof(int), hipMemcpyDeviceToHost);
        }
        else {
          hipMemcpy (new_embeddings_ptr[i], device_new_embeddings[i], n_new_embeddings[i]*(new_embedding_size), hipMemcpyDeviceToHost);
          hipMemcpy (output_ptr[i], device_outputs[i], n_output[i]*(new_embedding_size), hipMemcpyDeviceToHost);
        }
        hipMemcpy (&n_new_embeddings_1[i], device_n_embeddings_1[i], sizeof(0), hipMemcpyDeviceToHost);
        hipMemcpy (&n_output_1[i], device_n_outputs_1[i], sizeof(0), hipMemcpyDeviceToHost);

        error = hipGetLastError ();
        if (error != hipSuccess) {
          const char* error_string = hipGetErrorString (error);
          std::cout << "Cuda device to host copy error " << error_string << std::endl;
        } else {
          std::cout << "Cuda device to host copy success " << std::endl;
        }

        std::cout << "Stream " << i << std::endl;
        std::cout << "n_new_embeddings "<<n_new_embeddings[i]<<std::endl;
        std::cout << "n_new_embeddings_1 "<<n_new_embeddings_1[i];
        std::cout << " n_output "<<n_output[i];
        std::cout << " n_output_1 "<<n_output_1[i]<<std::endl;
      }
      
      double stream_time_2 = convertTimeValToDouble (getTimeOfDay ());

      total_stream_time += (stream_time_2-stream_time_1);
      //TODO: wait for all kernels and data transfers to finish
      for (int i = 0; i < N_STREAMS; i++) {
        n_next_step_embeddings += n_new_embeddings[i];
      }
      switch (iter) {
        case 1: {
          VectorVertexEmbedding<2>* new_embeddings = (VectorVertexEmbedding<2>*)malloc (sizeof (VectorVertexEmbedding<2>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<2> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<1>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [j] = embedding;
                j++;
              }
              else {
                VectorVertexEmbedding<2> embedding = ((VectorVertexEmbedding<2>*)(new_embeddings_ptr[stream]))[i];
                new_embeddings [j] = embedding;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
                j++;
              }
            }
          }
          
          assert (j == n_next_step_embeddings);
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<2> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<1>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_2.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_2.push_back (((VectorVertexEmbedding<2>*)output_ptr[stream])[i]);
              }
            }
          }
          
          break;
        }
        
        case 2: {
          VectorVertexEmbedding<3>* new_embeddings = (VectorVertexEmbedding<3>*)malloc (sizeof (VectorVertexEmbedding<3>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<3> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<2>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                assert (false);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<3> embedding = ((VectorVertexEmbedding<3>*)(new_embeddings_ptr[stream]))[i];
                new_embeddings [j] = embedding;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %ld vertices\n", embedding.get_n_vertices ());
                }
                #endif
                j++;
              }
            }
          }

          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<3> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<2>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_3.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_3.push_back (((VectorVertexEmbedding<3>*)output_ptr[stream])[i]);
              }
            }
          }

          break;
        }
        
        case 3: {
          VectorVertexEmbedding<4>* new_embeddings = (VectorVertexEmbedding<4>*)malloc (sizeof (VectorVertexEmbedding<4>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<4> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<3>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<4> embedding = ((VectorVertexEmbedding<4>*)(new_embeddings_ptr[stream]))[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));

          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<4> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<3>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_4.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_4.push_back (((VectorVertexEmbedding<4>*)output_ptr[stream])[i]);
              }
            }
          }
          break;
        }
        
        case 4: {
          VectorVertexEmbedding<5>* new_embeddings = (VectorVertexEmbedding<5>*)malloc (sizeof (VectorVertexEmbedding<5>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<5> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<4>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<5> embedding = ((VectorVertexEmbedding<5>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<5> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<4>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_5.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_5.push_back (((VectorVertexEmbedding<5>*)output_ptr[stream])[i]);
              }
            }
          }

          break;
        }
        
        case 5: {
          VectorVertexEmbedding<6>* new_embeddings = (VectorVertexEmbedding<6>*)malloc (sizeof (VectorVertexEmbedding<6>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<6> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<5>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<6> embedding = ((VectorVertexEmbedding<6>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<6> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<5>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_6.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_6.push_back (((VectorVertexEmbedding<6>*)output_ptr[stream])[i]);
              }
            }
          }
          break;
        }
        
        case 6: {
          VectorVertexEmbedding<7>* new_embeddings = (VectorVertexEmbedding<7>*)malloc (sizeof (VectorVertexEmbedding<7>)*n_next_step_embeddings);
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<7> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<6>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<7> embedding = ((VectorVertexEmbedding<7>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<7> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<6>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_7.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_7.push_back (((VectorVertexEmbedding<7>*)output_ptr[stream])[i]);
              }
            }
          }
          break;
        }
        
        case 7: {
          VectorVertexEmbedding<8>* new_embeddings = (VectorVertexEmbedding<8>*)malloc (sizeof(VectorVertexEmbedding<8>)*n_next_step_embeddings);
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<8> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<7>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<8> embedding = ((VectorVertexEmbedding<8>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<8> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<7>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_8.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_8.push_back (((VectorVertexEmbedding<8>*)output_ptr[stream])[i]);
              }
            }
          }

          break;
        }
      }
      
      //embeddings = new_embeddings;

      for (int i = 0; i < N_STREAMS; i++) {
        hipFree (device_embeddings[i]);
        hipFree (device_new_embeddings[i]);
        hipFree (device_n_embeddings[i]);
        hipFree (device_outputs[i]);
        hipFree (device_n_outputs[i]);
        hipFree (device_csr[i]);
      }
    }
    new_embeddings_size = n_next_step_embeddings;
    
  }

#ifdef PINNED_MEMORY
  hipFree (global_mem_ptr);
#else
  delete[] global_mem_ptr;
#endif
  std::cout << "Number of embeddings found "<< (output_1.size () + output_2.size () + output_3.size () + output_4.size () + output_5.size () + output_6.size () + output_7.size () + output_8.size ()) << std::endl;
  std::cout << "Time spent in GPU kernel execution " << kernelTotalTime << std::endl;
  std::cout << "Time spent in Streams " << total_stream_time << std::endl;
  
  /* For BitVectorVertexEmbedding
   for (iter; iter < 10 && embeddings.size () > 0; iter++) {
    std::cout << "iter " << iter << " embeddings " << embeddings.size () << std::endl;
    size_t global_mem_size = 3*1024*1024*1024UL;
    char* global_mem_ptr = new char[global_mem_size];
  #ifdef DEBUG
    memset (global_mem_ptr, 0, global_mem_size);
  #endif
    int n_embeddings = embeddings.size ();
    //n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
    std::cout << "iter " << iter << " n_embeddings " << n_embeddings << std::endl;
  
    for (int i = 0; i < n_embeddings; i++) {
      ((BitVectorVertexEmbedding*)global_mem_ptr)[i] = embeddings[i];
    }
    void* embeddings_ptr = global_mem_ptr;

    int n_new_embeddings = 0;
    int n_new_embeddings_1 = 0;
    void* new_embeddings_ptr = (char*)embeddings_ptr + (n_embeddings)*sizeof(BitVectorVertexEmbedding);
    int max_embeddings = 1000000;
    void* output_ptr = (char*)new_embeddings_ptr + (max_embeddings)*sizeof(BitVectorVertexEmbedding);
    int n_output = 0;
    int n_output_1 = 0;
    char* device_embeddings;
    char *device_new_embeddings;
    int* device_n_embeddings;
    int* device_n_embeddings_1;
    char *device_outputs;
    int* device_n_outputs;
    int* device_n_outputs_1;
    CSR* device_csr;

    cudaMalloc (&device_embeddings, n_embeddings*sizeof(BitVectorVertexEmbedding));
    cudaMemcpy (device_embeddings, embeddings_ptr,
                n_embeddings*sizeof(BitVectorVertexEmbedding),
                cudaMemcpyHostToDevice);
    cudaMalloc (&device_new_embeddings, max_embeddings*sizeof (BitVectorVertexEmbedding));
    cudaMalloc (&device_outputs, max_embeddings*sizeof (BitVectorVertexEmbedding));
    cudaMalloc (&device_n_embeddings, sizeof (0));
    cudaMalloc (&device_n_embeddings_1, sizeof (0));
    cudaMalloc (&device_n_outputs, sizeof (0));
    cudaMalloc (&device_n_outputs_1, sizeof (0));
    cudaMalloc (&device_csr, sizeof(CSR));

    cudaMemcpy (device_n_embeddings, &n_new_embeddings,
                sizeof (n_new_embeddings), cudaMemcpyHostToDevice);
    cudaMemcpy (device_n_outputs, &n_output, sizeof (n_output),
                cudaMemcpyHostToDevice);

    cudaMemcpy (device_n_embeddings_1, &n_new_embeddings_1,
                sizeof (n_new_embeddings_1), cudaMemcpyHostToDevice);
    cudaMemcpy (device_n_outputs_1, &n_output_1, sizeof (n_output_1),
                cudaMemcpyHostToDevice);

    cudaMemcpy (device_csr, csr, sizeof (CSR), cudaMemcpyHostToDevice);

    std::cout << "starting kernel with n_embeddings: " << n_embeddings;

    double t1 = convertTimeValToDouble (getTimeOfDay ());
#ifdef USE_FIXED_THREADS
    std::cout << " threads: " << MAX_CUDA_THREADS/THREAD_BLOCK_SIZE << std::endl;
      run_single_step_bitvector_embedding<<<MAX_CUDA_THREADS/THREAD_BLOCK_SIZE,THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
#else
    int thread_blocks = (n_embeddings%THREAD_BLOCK_SIZE != 0) ? (n_embeddings/THREAD_BLOCK_SIZE+1) : n_embeddings/THREAD_BLOCK_SIZE;
    std::cout << " threads: " << n_embeddings/THREAD_BLOCK_SIZE << std::endl;
    run_single_step_bitvector_embedding<<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
#endif

    cudaDeviceSynchronize ();

    double t2 = convertTimeValToDouble (getTimeOfDay ());

    std::cout << "Execution time " << (t2-t1) << " secs" << std::endl;
    kernelTotalTime += (t2-t1);

    cudaError_t error = cudaGetLastError ();
    if (error != cudaSuccess) {
      const char* error_string = cudaGetErrorString (error);
      std::cout << error_string << std::endl;
    } else {
      std::cout << "Cuda success " << std::endl;
    }

    cudaMemcpy (new_embeddings_ptr, device_new_embeddings, max_embeddings*sizeof(BitVectorVertexEmbedding), cudaMemcpyDeviceToHost);
    cudaMemcpy (output_ptr, device_outputs, max_embeddings*sizeof(BitVectorVertexEmbedding), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_new_embeddings, device_n_embeddings, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_output, device_n_outputs, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_new_embeddings_1, device_n_embeddings_1, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_output_1, device_n_outputs_1, sizeof(0), cudaMemcpyDeviceToHost);

    std::cout << "n_new_embeddings "<<n_new_embeddings<<std::endl;
    std::cout << "n_new_embeddings_1 "<<n_new_embeddings_1;
    std::cout << " n_output "<<n_output;
    std::cout << " n_output_1 "<<n_output_1<<std::endl;
    std::vector<BitVectorVertexEmbedding> new_embeddings;
  
    for (int i = 0; i < n_new_embeddings; i++) {
      BitVectorVertexEmbedding embedding = ((BitVectorVertexEmbedding*)new_embeddings_ptr)[i];
      new_embeddings.push_back (embedding);
    #ifdef DEBUG
      if (embedding.get_n_vertices () != (iter + 1)) {
        printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
      }
    #endif
    }
    for (int i = 0; i < n_output; i++) {
      output.push_back (((BitVectorVertexEmbedding*)output_ptr)[i]);
    }
    embeddings = new_embeddings;

    cudaFree (device_embeddings);
    cudaFree (device_new_embeddings);
    cudaFree (device_n_embeddings);
    cudaFree (device_outputs);
    cudaFree (device_n_outputs);
    cudaFree (device_csr);
    delete[] global_mem_ptr;
    #endif
  } 
   */
}

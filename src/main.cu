#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <string.h>
#include <assert.h>
#include <tuple>

//citeseer.graph
//const int N = 3312;
//const int N_EDGES = 9074;
//micro.graph
const int N = 100000;
const int N_EDGES = 2160312;
typedef uint32_t VertexID;

#include "csr.hpp"
#include "utils.hpp"

//#define USE_FIXED_THREADS
#define MAX_CUDA_THREADS (96*96)
#define THREAD_BLOCK_SIZE 256
#define WARP_SIZE 32
//#define ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING false

#define ENABLE_GRAPH_PARTITION_IN_SHARED_MEM
#if defined (ALL_THREAD_BLOCK_EMBEDDINGS_IN_SHARED_MEM_PER_VERTEX)  && !defined (EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK) 
  #error "For ALL_THREAD_BLOCK_EMBEDDINGS_IN_SHARED_MEM_PER_VERTEX, EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK should be defined "
#endif
#define USE_EMBEDDING_IN_LOCAL_MEM
//#define PROCESS_EMBEDDINGS_PER_VERTEX
#define GPU_QUERY_WAIT_TIME 1000UL

//#define ADD_TO_OUTPUT
//#define SHARED_MEM_NON_COALESCING
/**
  * The commit performing better is 698368fa19d023e3cb09705d820d333f79d0bf46.
  */
#ifdef SHARED_MEM_NON_COALESCING
  #ifndef USE_EMBEDDING_IN_SHARED_MEM
    #error "USE_EMBEDDING_IN_SHARED_MEM must be enabled with SHARED_MEM_NON_COALESCING"
  #endif
#endif
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #ifdef USE_FIXED_THREADS
    #error "USE_FIXED_THREADS cannot be enabled with USE_EMBEDDING_IN_SHARED_MEM"
  #endif
#endif

#define NEW_EMBEDDING_BUFFER_SIZE 128*1024*1024 //Size in terms of Bytes //Setting it to 128 MB makes citeseer performs a lot better

#define GRAPH_PARTITION_SIZE (48*1024) //24 KB is the size of each partition of graph

//#define USE_CONSTANT_MEM

typedef uint8_t SharedMemElem;

#define ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING false


//#define ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING true

const int N_THREADS = 256;

class GlobalMemAllocator
{
  static uint64_t memory_length;
  static uint64_t bump_pointer;
  static char* global_mem_ptr;

  public:

    static void initialize (char* _global_mem_ptr, uint64_t _memory_length) {
      global_mem_ptr = _global_mem_ptr;
      memory_length = _memory_length;
      bump_pointer = 0;
    }

    static uint64_t alloc (size_t sz) {
      assert (bump_pointer + sz < memory_length);

      uint64_t to = bump_pointer;

      bump_pointer += sz;

      return to;
    }

    static uint64_t allocated () {
      return bump_pointer;
    }
    static uint64_t alloc_vertices_array (size_t n_vertices) {
      return alloc (sizeof (VertexID)*n_vertices);
    }
    
    static void* get_global_mem_ptr () {
      return global_mem_ptr;
    }

};

uint64_t GlobalMemAllocator::memory_length;
uint64_t GlobalMemAllocator::bump_pointer;
char* GlobalMemAllocator::global_mem_ptr;

class VectorVertexEmbedding
{
private:
  uint64_t array_start_idx;
  uint32_t filled_size;
  uint32_t size;
  VertexID* array;

public:
  __host__
  VectorVertexEmbedding (uint32_t _max_size, uint64_t _array_start_idx, bool filled = false)
  {
    size = _max_size;
    filled_size = filled ? size : 0;
    array_start_idx = _array_start_idx;
    array = (VertexID*)((char*) GlobalMemAllocator::get_global_mem_ptr () + array_start_idx);
  }

  __host__ 
  std::vector<VertexID> to_vector ()
  {
    std::vector<VertexID> v;
    
    for (int i = 0; i < get_n_vertices (); i++) {
      v.push_back (get_vertex (i));
    }

    return v;
  }

  __host__
  void* get_array () {return array;}
  __device__ __host__
  uint64_t get_array_start_idx () { return array_start_idx;}
  
  __host__ __device__
  void add (int v)
  {
  #if DEBUG
    if (!(size != 0 and filled_size < size)) {
      printf ("filled_size %d, size %d\n", filled_size, size);
      //assert (size != 0 and filled_size < size);
      assert (false);
    }
  #endif
  
    add_unsorted (v);
  }

  __host__ __device__
  void add_last_in_sort_order () 
  {
    int v = array[filled_size-1];
    remove_last ();
    add (v);
  }

  __host__ __device__
  void add_unsorted (int v) 
  {
    array[filled_size++] = v;
  }
  
  __host__ __device__
  void remove (int v)
  {
    printf ("Do not support remove\n");
    assert (false);
  }
  
  // __host__ __device__
  // const bool has_logn (int v)
  // {
  //   int l = 0;
  //   int r = filled_size-1;
    
  //   while (l <= r) {
  //     int m = l+(r-l)/2;
      
  //     if (array[m] == v)
  //       return true;
      
  //     if (array[m] < v)
  //       l = m + 1;
  //     else
  //       r = m - 1;
  //   }
    
  //   return false;
  // }
  
  __host__ __device__
  bool has (int v) const
  {
    for (int i = 0; i < filled_size; i++) {
      if (array[i] == v) {
        return true;
      }
    }
    
    return false;
  }
  
  __host__ __device__
  size_t get_n_vertices () const
  {
    return filled_size;
  }
  
  __host__ __device__
  int get_vertex (int index, void* global_storage_start) const
  {
    return ((VertexID*)((char*)global_storage_start + array_start_idx))[index];
  }

  __device__
  int get_vertex (int index, void* global_storage_start, uint64_t global_start_idx) const
  {
    assert (array_start_idx >= global_start_idx);
    return ((VertexID*)((char*)global_storage_start + (array_start_idx - global_start_idx)))[index];
  }

  __host__ 
  int get_vertex (int index) const
  {
    return array[index];
  }
  
  __host__ __device__
  int get_last_vertex () const
  {
    return array[filled_size-1];
  }
  
  __host__ __device__
  size_t get_max_size () const
  {
    return size;
  }
  
  __host__ __device__
  void clear ()
  {
    filled_size = 0;
  }
  
  __host__ __device__
  void remove_last () 
  {
    assert (filled_size > 0);
    filled_size--;
  }
  __host__ __device__
  ~VectorVertexEmbedding ()
  {
    //delete[] array;
  }

  void print () 
  {
    std::cout << "[";
    for (int i = 0; i < filled_size; i++) {
      std::cout << get_vertex (i) << ", ";
    }
    std::cout << "]";
  }
};

std::vector<VectorVertexEmbedding> get_initial_embedding_vector (CSR* csr)
{
  VectorVertexEmbedding embedding (0, 0UL);
  std::vector <VectorVertexEmbedding> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

__host__
void vector_embedding_from_one_less_size (VectorVertexEmbedding const & in,
                                          VectorVertexEmbedding& out)
{
  //TODO: Optimize here, filled_size++ in add is being called several times
  //but can be called only once too
  //if  (false and vec_emb1.get_n_vertices () != size) {
  //  printf ("vec_emb1.get_n_vertices () %ld != size %d\n", vec_emb1.get_n_vertices (), size);
  //  assert (false);
  //}
  assert (in.get_n_vertices () <= out.get_n_vertices ());
  for (int i = 0; i < in.get_n_vertices (); i++) {
    out.add (in.get_vertex (i));
  }
}

std::vector<VectorVertexEmbedding> get_extensions_vector (VectorVertexEmbedding& embedding, CSR* csr)
{
  std::vector<VectorVertexEmbedding> extensions;
  size_t size;
  
  size = embedding.get_n_vertices ();

  if (size == 0) {
    for (int u = 0; u < N; u++) {
      uint64_t ptr =  GlobalMemAllocator::alloc_vertices_array(1);
      VectorVertexEmbedding extension(1,ptr);
      extension.add(u);
      extensions.push_back(extension);
    }
  } else {
    for (int i = 0; i < size; i++) {
      int u = embedding.get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        int v = csr->get_edges () [e];
        if (embedding.has (v) == false) {
          VectorVertexEmbedding extension(1, GlobalMemAllocator::alloc_vertices_array(size + 1));
          vector_embedding_from_one_less_size (embedding, extension);
          extension.add(v);
          extensions.push_back(extension);
        }
      }
    }
  }

  return extensions;
}

void run_single_step_initial_vector (std::vector<VectorVertexEmbedding>& input_embeddings,
                                     CSR* csr,
                                     std::vector<VectorVertexEmbedding>& output_embeddings,
                                     std::vector<VectorVertexEmbedding>& next_step_embeddings)
{
  for (int i = 0; i < input_embeddings.size (); i++) {
    VectorVertexEmbedding& embedding = input_embeddings[i];
    std::vector<VectorVertexEmbedding> extensions = get_extensions_vector (embedding, csr);
    for (auto extension : extensions) {
        output_embeddings.push_back (extension);
        next_step_embeddings.push_back (extension);
      }
   }
}

std::vector <std::vector <VertexID>> n_hop_cpu (CSR* csr, const int N_HOPS)
{
  std::vector <std::vector <VertexID>> hops = std::vector<std::vector<VertexID>> (csr->get_n_vertices ());

  int hop = 0;

  for (int vertex = 0; vertex < csr->get_n_vertices (); vertex++) {
    int start_edge_idx = csr->get_start_edge_idx (vertex);
    int end_edge_idx = csr->get_end_edge_idx (vertex);
    if (start_edge_idx != -1) {
      for (int edge = start_edge_idx; edge <= end_edge_idx; edge++) {
        hops[vertex].push_back (csr->get_edges()[edge]);
      }
    }
  }

  for (int vertex = 0; vertex < csr->get_n_vertices (); vertex++) {
    int hop = 1;
    std::vector <VertexID> vertex_hops[N_HOPS + 1];
    vertex_hops[0].insert (vertex_hops[0].begin(), hops[vertex].begin (), hops[vertex].end ());
    while (hop < N_HOPS) {
      for (int hop_vertex : vertex_hops[hop - 1]) {
        int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
        int end_edge_idx = csr->get_end_edge_idx (hop_vertex);
        
        if (start_edge_idx != -1) {
          for (int edge = start_edge_idx; edge <= end_edge_idx; edge++) {
            int v = csr->get_edges()[edge];
            vertex_hops[hop].push_back (v);
          }
        }
      }

      hops[vertex].insert (hops[vertex].begin (), vertex_hops[hop].begin (), vertex_hops[hop].end ());
      hop++;
    }
  }

  return hops;
}

#define MAX_LOAD_PER_TB (N_THREADS)
#define MAX_VERTICES_PER_TB 10
#if MAX_VERTICES_PER_TB < 1
  #error "MAX_VERTICES_PER_TB should be greater than or equal to 1"
#endif

#define WARP_HOP

const uint FULL_MASK = 0xffffffff;

__device__ inline int get_warp_mask_and_participating_threads (int condition, int& participating_threads, int& first_active_thread)
{
  uint warp_mask = __ballot_sync(FULL_MASK, condition);
  first_active_thread = -1;
  participating_threads = 0;
  int qq = 0;
  while (qq < 32) {
    if ((warp_mask & (1U << qq)) == (1U << qq)) {
      if (first_active_thread == -1) {
        first_active_thread = qq;
      }
      participating_threads++;
    }
    qq++;
  }

  return warp_mask;
}

__device__ int n_edges_to_warp_size (const int n_edges) 
{
  //Different warp sizes gives different performance. 32 is worst. adapative is a litter better.
  //Best is 4.
  return 4;
  if (n_edges <= 4) 
    return 2;
  else if (n_edges > 4 && n_edges <= 8)
    return 4;
  else if (n_edges > 8 && n_edges <= 16)
    return 8;
  else if (n_edges > 16 && n_edges <= 32) 
    return 16;
  else
    return 32;
}

#define MAX_EDGES (2*MAX_LOAD_PER_TB)
#undef USE_PARTITION_FOR_SHMEM
#define MAX_HOP_VERTICES_IN_SH_MEM (MAX_VERTICES_PER_TB)
#define ENABLE_GRAPH_PARTITION_FOR_GLOBAL_MEM

__global__ void get_max_lengths_for_vertices_first_iter (CSRPartition* void_csr,
                                                          int start_vertex, int end_vertex,
                                                          unsigned long long int* embeddings_additions_iter,
                                                          void* void_map_orig_embedding_to_additions)
{
  CSRPartition* csr = (CSRPartition*)void_csr;

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  VertexID vertex = thread_idx + start_vertex;
  if (vertex > end_vertex) {
    return;
  }

  int* map_orig_embedding_to_additions = (int*) void_map_orig_embedding_to_additions;
  unsigned long long int new_edges = 0;

  /*Perform a single hop for all vertices in the input embedding*/
  const int start_edge_idx = csr->get_start_edge_idx (vertex);
  const int end_edge_idx = csr->get_end_edge_idx (vertex);

  if (end_edge_idx != -1) {
    int e = (end_edge_idx - start_edge_idx) + 1;
    if (e < 0) {
      printf ("v %d s %d e %d\n", vertex, start_edge_idx, end_edge_idx);
    }
    assert (e >= 0);
    new_edges += e;
  }
  

  unsigned long long int additions_start_iter = atomicAdd (embeddings_additions_iter, new_edges);
  map_orig_embedding_to_additions[2*thread_idx] = additions_start_iter;
  map_orig_embedding_to_additions[2*thread_idx+1] = new_edges;
}

__global__ void get_max_lengths_for_vertices_single_step (CSRPartition* void_csr,
                                                          int start_vertex, int end_vertex,
                                                          unsigned long long int* void_embeddings_additions_iter,
                                                          void* void_map_orig_embedding_to_additions_prev_iter,
                                                          void* void_map_orig_embedding_to_additions_next_iter,
                                                          void* void_map_orig_embedding_to_additions_first_iter,
                                                          int* edges_to_prev_iter_additions,
                                                          int common_vertex_with_previous_partition,
                                                          int common_vertex_with_next_partition)
{
  CSRPartition* csr = (CSRPartition*)void_csr;

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  VertexID vertex = thread_idx + start_vertex;
  if (vertex > end_vertex) {
    return;
  }

  //VertexID* embedding_storage = (VertexID embedding_storage;
  unsigned long long int* embeddings_additions_iter = void_embeddings_additions_iter;
  int* map_orig_embedding_to_additions_next_iter = (int*)void_map_orig_embedding_to_additions_next_iter;
  int* map_orig_embedding_to_additions_prev_iter = (int*)void_map_orig_embedding_to_additions_prev_iter;
  int* map_orig_embedding_to_additions_first_iter = (int*) void_map_orig_embedding_to_additions_first_iter;
  unsigned long long int new_edges = 0;//map_orig_embedding_to_additions_first_iter[2*thread_idx + 1];
  // printf ("thread idx %d array_start_idx %ld\n", thread_idx, input_embedding->get_array_start_idx ());
  /*Perform a single hop for all vertices in the input embedding*/
  
  int start_edge_idx = csr->get_start_edge_idx (vertex);
  const int end_edge_idx = csr->get_end_edge_idx (vertex);
  if (end_edge_idx != -1) {
    while (start_edge_idx <= end_edge_idx) {
      int v = csr->get_edge (start_edge_idx);
      if (csr->is_vertex_in_partition (v) and v != common_vertex_with_previous_partition and v != common_vertex_with_next_partition) {
        assert (v-start_vertex >= 0);
        new_edges += map_orig_embedding_to_additions_prev_iter [2*(v-start_vertex)+1];
      }
      else
        new_edges += edges_to_prev_iter_additions[start_edge_idx - csr->first_edge_idx];

      start_edge_idx++;
    }
  }

  //printf ("new_edges %ld\n", new_edges);
  unsigned long long int additions_start_iter = atomicAdd (embeddings_additions_iter, new_edges);
  map_orig_embedding_to_additions_next_iter[2*thread_idx] = additions_start_iter;
  map_orig_embedding_to_additions_next_iter[2*thread_idx+1] = new_edges;
}

__global__ void run_hop_parallel_single_step (int N_HOPS, int hop, CSR* void_csr,
  void* void_embeddings_additions, 
  size_t num_neighbors,
  void* void_embeddings_additions_prev_hop,
  int* map_orig_embedding_to_additions,
  int* previous_stage_filled_range,
  int* hop_vertex_to_roots,
  int* map_vertex_to_hop_vertex_data,
  int* source_vertex_idx,
  unsigned long long int* profile_branch_1, unsigned long long int* profile_branch_2)
{
  CSR* csr = (CSR*)void_csr;
  __shared__ int vertices[MAX_VERTICES_PER_TB];
  __shared__ int previous_step_end[MAX_VERTICES_PER_TB];
  __shared__ int n_vertex_load;
  __shared__ int thread_idx_to_load[2*MAX_LOAD_PER_TB];
  __shared__ int last_hop_vertex_id;
  __shared__ int last_hop_vertex_roots_remaining;
  __shared__ int last_hop_vertex_roots_done;

#ifdef USE_PARTITION_FOR_SHMEM
  __shared__ VertexID shmem_csr_edges[MAX_EDGES];
  __shared__ int hop_vertex_in_shared_mem[MAX_HOP_VERTICES_IN_SH_MEM];
  __shared__ int hop_vertices_in_shared_mem_start_edge_idx[MAX_HOP_VERTICES_IN_SH_MEM];
  __shared__ int hop_vertices_in_shared_mem_end_edge_idx[MAX_HOP_VERTICES_IN_SH_MEM];
  __shared__ int hop_vertices_in_shared_mem_size;
  __shared__ int shmem_csr_edges_size;
#endif 

  int laneid = threadIdx.x%warpSize;
  int warpid = threadIdx.x/warpSize;

  VertexID* embeddings_additions = (VertexID*)void_embeddings_additions;
  VertexID* embeddings_additions_prev_hop = (VertexID*)void_embeddings_additions_prev_hop;
  int thread_idx = blockIdx.x*blockDim.x + threadIdx.x;

  if (hop != 0) {
    thread_idx_to_load [2*threadIdx.x] = -1;
    thread_idx_to_load [2*threadIdx.x + 1] = -1;

    __syncthreads ();

    if (threadIdx.x == 0) {
      last_hop_vertex_id = -1;
      last_hop_vertex_roots_remaining = -1;
      int load = 0;
      n_vertex_load = 0;
      int load_assigned_index = 0;
      int warp_assigned = 0;

#ifdef USE_PARTITION_FOR_SHMEM
      hop_vertices_in_shared_mem_size = 0;
      int edges_in_shared_mem = 0;
      shmem_csr_edges_size = 0;
#endif

      while (n_vertex_load < MAX_VERTICES_PER_TB && load < MAX_LOAD_PER_TB) {
        vertices[n_vertex_load] = atomicAdd(source_vertex_idx, 1);
        if (vertices[n_vertex_load] >= gridDim.x) {
          break;
        }
        
        int start_edge_idx = csr->get_start_edge_idx (vertices[n_vertex_load]);
        const int end_edge_idx = csr->get_end_edge_idx (vertices[n_vertex_load]);
        const int n_edges = (end_edge_idx != -1) ? (end_edge_idx - start_edge_idx + 1) : 0;
#ifdef USE_PARTITION_FOR_SHMEM
        if (hop_vertices_in_shared_mem_size < MAX_HOP_VERTICES_IN_SH_MEM && n_edges != 0 && 
            n_edges + edges_in_shared_mem < MAX_EDGES) {
          int v = vertices[n_vertex_load];
          hop_vertex_in_shared_mem[hop_vertices_in_shared_mem_size] = v;
          edges_in_shared_mem += n_edges;
          hop_vertices_in_shared_mem_size++;
        }
#endif
        int shfl_warp_size = n_edges_to_warp_size(n_edges);
        int root_vertices = map_vertex_to_hop_vertex_data[2*vertices[n_vertex_load] + 1];

        if (root_vertices != 0 and n_edges != 0) {
          int root_vertex_idx;
          for (root_vertex_idx = 0; root_vertex_idx < root_vertices && warp_assigned < MAX_LOAD_PER_TB; root_vertex_idx++) {
            for (int ii = warp_assigned; ii < min (warp_assigned + shfl_warp_size, MAX_LOAD_PER_TB); ii++) {
              thread_idx_to_load[2*ii] = n_vertex_load;
              thread_idx_to_load[2*ii+1] = root_vertex_idx;
            }
            warp_assigned += shfl_warp_size;
            load_assigned_index += 1;
          }

          if (warp_assigned >= MAX_LOAD_PER_TB) {
            last_hop_vertex_roots_remaining = root_vertices - root_vertex_idx;
            last_hop_vertex_roots_done = root_vertex_idx;
            last_hop_vertex_id = n_vertex_load;
          }

          load += root_vertices*shfl_warp_size;
          n_vertex_load++;
        }
      }
    }
    
    __syncthreads ();

#ifdef USE_PARTITION_FOR_SHMEM
    for (int __hop = 0; __hop < hop_vertices_in_shared_mem_size/(blockDim.x/warpSize) + 1; __hop++) {
      int hop = __hop * (blockDim.x/warpSize) + warpid;
      if (hop >= hop_vertices_in_shared_mem_size) {
        continue;
      }

      int start_edge_idx = csr->get_start_edge_idx (hop_vertex_in_shared_mem[hop]);
      const int end_edge_idx = csr->get_end_edge_idx (hop_vertex_in_shared_mem[hop]);
      const int n_edges = (end_edge_idx != -1) ? (end_edge_idx - start_edge_idx + 1) : 0;
      assert (n_edges > 0);
      int _shmem_start = -1;
      if (laneid == 0) {
        _shmem_start = atomicAdd (&shmem_csr_edges_size, n_edges);
      }

      int shmem_start = __shfl_sync (FULL_MASK, _shmem_start, 0, warpSize);
      assert (shmem_start != -1);
      for (int e = 0; e < n_edges/warpSize + 1; e++) {
        int edge_idx = e*warpSize + laneid;
        if (edge_idx < n_edges) {
          shmem_csr_edges[shmem_start + edge_idx] = csr->get_edges ()[start_edge_idx + edge_idx];
        }
      }
      __syncwarp ();
      if (laneid == 0) {
        hop_vertices_in_shared_mem_start_edge_idx[hop] = shmem_start;
        hop_vertices_in_shared_mem_end_edge_idx[hop] = shmem_start + n_edges - 1;
      }

      __syncwarp ();
    }
#endif

    __syncthreads ();

    assert (n_vertex_load <= MAX_VERTICES_PER_TB);
    int _curr_vertex_id = thread_idx_to_load[2*threadIdx.x];
    int root_vertex_idx = thread_idx_to_load[2*threadIdx.x + 1];
    
    int hop_vertex_start_idx = -1;
    int n_root_vertices = -1;
    int root_vertex = -1;
    int hop_idx = -1;
    int first_active_thread = -1;
    int participating_threads = 0;

    if (_curr_vertex_id != -1 && root_vertex_idx != -1 && vertices[_curr_vertex_id] < gridDim.x) {
      hop_vertex_start_idx = map_vertex_to_hop_vertex_data[2*vertices[_curr_vertex_id]];
      n_root_vertices = map_vertex_to_hop_vertex_data[2*vertices[_curr_vertex_id] + 1];
      root_vertex = hop_vertex_to_roots[hop_vertex_start_idx + 2*root_vertex_idx];

      hop_idx = hop_vertex_to_roots[hop_vertex_start_idx + 2*root_vertex_idx + 1];

      if (root_vertex != -1 && root_vertex < gridDim.x) {
        int vertex = root_vertex;
        int start = map_orig_embedding_to_additions[2*vertex];
      }
    }

    __syncthreads ();

    uint warp_hop_mask = get_warp_mask_and_participating_threads (_curr_vertex_id != -1 && 
      vertices[_curr_vertex_id] < gridDim.x && root_vertex_idx != -1 && root_vertex != -1 && root_vertex < gridDim.x, participating_threads, first_active_thread);
      //__syncthreads ();
    if (_curr_vertex_id != -1 && root_vertex_idx != -1 && vertices[_curr_vertex_id] < gridDim.x) {
      if (root_vertex != -1 && root_vertex < gridDim.x) {
        int vertex = root_vertex;
        int start = map_orig_embedding_to_additions[2*vertex];
        int hop_vertex = embeddings_additions_prev_hop[hop_idx];
       
        assert (hop_vertex == vertices[_curr_vertex_id]);
        int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
        const int end_edge_idx = csr->get_end_edge_idx (hop_vertex);
        __syncwarp (warp_hop_mask);


        int* end = &previous_stage_filled_range[2*vertex + 1];
        if (end_edge_idx != -1) {
          __syncwarp (warp_hop_mask);
          int e = -1;
          const int n_edges = end_edge_idx - start_edge_idx + 1;
          int shfl_warp_size = n_edges_to_warp_size(n_edges);
          if (laneid%shfl_warp_size == 0) {
            e = atomicAdd (end, n_edges);
          }
          //TODO: Add synchronization point
          int _e = __shfl_sync (warp_hop_mask, e, 0, shfl_warp_size);  
          assert (_e != -1);
#ifdef USE_PARTITION_FOR_SHMEM
          if (_curr_vertex_id >= hop_vertices_in_shared_mem_size) {
            int iter = 0;
            while (start_edge_idx + laneid%shfl_warp_size <= end_edge_idx) {
              VertexID edge = csr->get_edges ()[start_edge_idx + laneid%shfl_warp_size];
              embeddings_additions[start + _e + iter*shfl_warp_size + laneid%shfl_warp_size] = edge;
              start_edge_idx += shfl_warp_size;
              iter++;
            }
          } else {
            int iter = 0;
            int _start_edge_idx = hop_vertices_in_shared_mem_start_edge_idx[_curr_vertex_id];
            int _end_edge_idx = hop_vertices_in_shared_mem_end_edge_idx[_curr_vertex_id];
            assert (hop_vertex == hop_vertex_in_shared_mem[_curr_vertex_id]);
            assert (n_edges == (_end_edge_idx - _start_edge_idx) + 1);
            while (_start_edge_idx + laneid%shfl_warp_size <= _end_edge_idx) {
              VertexID edge = shmem_csr_edges[_start_edge_idx + laneid%shfl_warp_size];
              embeddings_additions[start + _e + iter*shfl_warp_size + laneid%shfl_warp_size] = edge;
              _start_edge_idx += shfl_warp_size;
              iter++;
            }
          }
#else
          int iter = 0;
          while (start_edge_idx + laneid%shfl_warp_size <= end_edge_idx) {
            VertexID edge = csr->get_edges ()[start_edge_idx + laneid%shfl_warp_size];
            int addr = start + _e + iter*shfl_warp_size + laneid%shfl_warp_size;
            //assert (addr < ) //TODO: Add asserts.
            embeddings_additions[addr] = edge;
            start_edge_idx += shfl_warp_size;
            iter++;
          }
#endif
        }

        __syncwarp (warp_hop_mask);

      }
    }

    __syncwarp ();
    
    if (last_hop_vertex_id != -1 && last_hop_vertex_roots_remaining != -1) {
      int hop_vertex = vertices[last_hop_vertex_id];
      int hop_vertex_start_idx = map_vertex_to_hop_vertex_data[2*hop_vertex];
      int n_root_vertices = map_vertex_to_hop_vertex_data[2*hop_vertex + 1];
      
      __syncthreads ();
      
      for (int i = 0; i < last_hop_vertex_roots_remaining/blockDim.x + 1; i++) {
        int root_idx = i*blockDim.x + threadIdx.x;
        if (root_idx >= last_hop_vertex_roots_remaining) {
          continue;
        }
        
        int root_vertex = hop_vertex_to_roots[hop_vertex_start_idx + 2*(root_idx + last_hop_vertex_roots_done)];
        int hop_idx = hop_vertex_to_roots[hop_vertex_start_idx + 2*(root_idx + last_hop_vertex_roots_done) + 1];
        int start = map_orig_embedding_to_additions[2*root_vertex];
        int* end = &previous_stage_filled_range[2*root_vertex + 1];
        int hop_vertex = embeddings_additions_prev_hop[hop_idx];
        int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
        const int end_edge_idx = csr->get_end_edge_idx (hop_vertex);

        if (end_edge_idx != -1) {
          int e = atomicAdd (end, end_edge_idx - start_edge_idx + 1);
          int iter = 0;
          
          while (start_edge_idx <= end_edge_idx) {
            VertexID edge = csr->get_edges ()[start_edge_idx];
            if (root_vertex == 3030 and edge == 3111) {
              assert (false);
            }
            embeddings_additions[start + e + iter] = edge;
            start_edge_idx++;
            iter++;
          }
        }
      }
      __syncthreads ();
    }
  } else {
    int source_vertex = blockIdx.x;
    assert (source_vertex < csr->get_n_vertices ());
    int start = map_orig_embedding_to_additions[2*source_vertex];
    
    int start_edge_idx = csr->get_start_edge_idx (source_vertex);
    const int end_edge_idx = csr->get_end_edge_idx (source_vertex);
    const int n_edges = end_edge_idx - start_edge_idx + 1;

    if (end_edge_idx != -1) {
      int* end = &previous_stage_filled_range[2*source_vertex + 1];

      for (int i = 0; i < n_edges/blockDim.x + 1; i++) {
        int edge_idx = i*blockDim.x + threadIdx.x;
        if (edge_idx < n_edges) {
          VertexID edge = csr->get_edges()[start_edge_idx + edge_idx];
          int e = atomicAdd (end, 1);
          assert (start + e < num_neighbors);
          embeddings_additions[start + e] = edge;    
        }
      }
    }
  
    __syncthreads ();
    previous_stage_filled_range[2*source_vertex] = start;
  }
}

__global__ void update_filled_ranges (int n_vertices, int* previous_stage_filled_range)
{
  int thread_idx = threadIdx.x + blockDim.x*blockIdx.x;

  if (thread_idx >= n_vertices) 
    return;
  
  previous_stage_filled_range[2*thread_idx] = previous_stage_filled_range[2*thread_idx] + previous_stage_filled_range[2*thread_idx+1];
}

std::vector <std::unordered_set <VertexID>> n_hop_cpu_distinct (CSR* csr, const int N_HOPS)
{
  std::vector <std::unordered_set <VertexID>> hops = std::vector<std::unordered_set<VertexID>> (csr->get_n_vertices ());

  int hop = 0;

  for (int vertex = 0; vertex < csr->get_n_vertices (); vertex++) {
    int start_edge_idx = csr->get_start_edge_idx (vertex);
    int end_edge_idx = csr->get_end_edge_idx (vertex);
    if (start_edge_idx != -1) {
      for (int edge = start_edge_idx; edge <= end_edge_idx; edge++) {
        hops[vertex].insert (csr->get_edges()[edge]);
      }
    }
  }

  for (int vertex = 0; vertex < csr->get_n_vertices (); vertex++) {
    int hop = 1;
    std::unordered_set <VertexID> vertex_hops[N_HOPS + 1];
    vertex_hops[0].insert (hops[vertex].begin (), hops[vertex].end ());
    while (hop < N_HOPS) {
      for (int hop_vertex : vertex_hops[hop - 1]) {
        int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
        int end_edge_idx = csr->get_end_edge_idx (hop_vertex);
        
        if (start_edge_idx != -1) {
          for (int edge = start_edge_idx; edge <= end_edge_idx; edge++) {
            int v = csr->get_edges()[edge];
            if (hops[vertex].count (v) == 0)
              vertex_hops[hop].insert (v);
          }
        }
      }

      hops[vertex].insert (vertex_hops[hop].begin (), vertex_hops[hop].end ());
      hop++;
    }
  }

  return hops;
}

void copy_partition_to_gpu (CSRPartition& partition, CSRPartition*& device_csr, CSR::Vertex*& device_vertex_array, CSR::Edge*& device_edge_array)
{
  EXECUTE_CUDA_FUNC (hipMalloc (&device_csr, sizeof(CSRPartition)));
  EXECUTE_CUDA_FUNC (hipMalloc (&device_vertex_array, sizeof(CSR::Vertex)*partition.get_n_vertices ()));
  EXECUTE_CUDA_FUNC (hipMalloc (&device_edge_array, sizeof(CSR::Edge)*partition.get_n_edges ()));
  EXECUTE_CUDA_FUNC (hipMemcpy (device_vertex_array, partition.vertices, sizeof (CSR::Vertex)*partition.get_n_vertices (), hipMemcpyHostToDevice));
  EXECUTE_CUDA_FUNC (hipMemcpy (device_edge_array, partition.edges, sizeof (CSR::Edge)*partition.get_n_edges (), hipMemcpyHostToDevice));

  CSRPartition device_csr_partition_value = CSRPartition (partition.first_vertex_id, partition.last_vertex_id, 
                                                          partition.first_edge_idx, partition.last_edge_idx, 
                                                          device_vertex_array, device_edge_array);
  EXECUTE_CUDA_FUNC (hipMemcpy (device_csr, &device_csr_partition_value, sizeof(CSRPartition), hipMemcpyHostToDevice));
}

int get_common_vertex_with_previous_partition (std::vector<CSRPartition> csr_partitions, int partition_idx)
{
  if (partition_idx <= 0)
    return -1;

  if (csr_partitions[partition_idx].first_vertex_id == csr_partitions[partition_idx - 1].last_vertex_id) {
    return csr_partitions[partition_idx].first_vertex_id;
  }

  return -1;
}


int get_common_vertex_with_next_partition (std::vector<CSRPartition> csr_partitions, int partition_idx)
{
  if (partition_idx >= csr_partitions.size () - 1)
    return -1;

  if (csr_partitions[partition_idx].last_vertex_id == csr_partitions[partition_idx + 1].first_vertex_id) {
    return csr_partitions[partition_idx].last_vertex_id;
  }

  return -1;
}

void create_csr_partitions (CSR* csr, std::vector<CSRPartition>& csr_partitions, const size_t effective_partition_size)
{
  std::vector<std::tuple<VertexID, VertexID, int, int>> vertex_partition_positions_vector;

  //Create Partitions.
  int u = 0;
  int partition_edge_start_idx = 0;

  while (u < csr->get_n_vertices ()) {
    int n_edges = 0;
    int u_start = u;
    int end_edge_idx = 0;
    int u_end = csr->get_n_vertices () - 1;
    int edges = 0;
    int partial_edges = 0;
    for (int v = u; v < csr->get_n_vertices (); v++) {
      int start = csr->get_start_edge_idx (v);
      const int end = csr->get_end_edge_idx (v);
      if (end != -1) {
        if (v == u) {
          //std::cout << "1829: " << " partition_edge_start_idx " << partition_edge_start_idx << " u " << u << " start " << start << " end " << end << std::endl;
        }
        if (v == u && partition_edge_start_idx >= start) {
          start = partition_edge_start_idx;
        }
        edges = end - start + 1;
        assert (edges >= 0);
      } else {
        edges = 0;
      }
      if ((n_edges + edges) * sizeof (CSR::Edge) + (v-u_start + 1)*sizeof(CSR::Vertex) >= effective_partition_size) {
        end_edge_idx = (effective_partition_size - (v-u_start + 1)*sizeof(CSR::Vertex))/sizeof (CSR::Edge) - n_edges;
        //std::cout << " v " << v << " n_edges " << n_edges << " u " << u_start  << "  sizeof (CSR::Edge) " << sizeof (CSR::Edge) <<  " sizeof(CSR::Vertex) " << sizeof(CSR::Vertex) << " end_edge_idx " << end_edge_idx << " effective_partition_size " << effective_partition_size << " start " << start << " end " << end << std::endl;
        if (end_edge_idx <= 0) {
          u = v;
          u_end = v - 1;
          partial_edges = 0;
          end_edge_idx = start - 1;
        } else if (end_edge_idx < edges) {
          u = v;
          u_end = v;
          partial_edges = end_edge_idx;
          end_edge_idx += start - 1; //Including last edge
        } else {
          u_end = v;
          u = v + 1;
          partial_edges = 0;
          end_edge_idx += start - 1; //Including last edge
        }

        if (u_end < u_start) 
        {
          std::cout << "u_end : " << u_end << " u_start: "  << u_start  << std::endl;
          std::cout << "ERROR: Cannot create partition " << std::endl;
          assert (false);
        }

        break;
      }

      n_edges += edges;
    }

    vertex_partition_positions_vector.push_back (std::make_tuple (u_start, u_end, partition_edge_start_idx, (end_edge_idx == 0) ? csr->get_end_edge_idx (u_end) : end_edge_idx));
    //Vertex partition: [u_start, u_end]. Edge partition is all edges from u_start to u_end if end_edge_idx = 0. otherwise all edges of vertices from u_start to u_end - 1 and edges of u_end u_end.start_edge_idx to end_edge_idx.
    
    partition_edge_start_idx = end_edge_idx + 1;

    if (u_end == csr->get_n_vertices () - 1) {
      break;
    }
  }

  std::cout << __LINE__ << ": " << partition_edge_start_idx << " " << csr->get_n_edges () - 1 << std::endl;


  //Create remaining partitions if last vertex's edges are remaining
  if (partition_edge_start_idx != 1 && partition_edge_start_idx < csr->get_n_edges ()) {
    assert ((csr->get_n_edges () - partition_edge_start_idx) * sizeof (CSR::Edge) + (1)*sizeof(CSR::Vertex) <= effective_partition_size);
    vertex_partition_positions_vector.push_back (std::make_tuple (csr->get_n_vertices () - 1, csr->get_n_vertices () - 1, partition_edge_start_idx, csr->get_n_edges ()- 1));
  }

  //Create partitions
  for (auto p : vertex_partition_positions_vector) {
    int u = std::get<0> (p);
    int v = std::get<1> (p);
    int start = std::get<2> (p);
    int end = std::get<3> (p);

    CSR::Vertex* vertex_array = new CSR::Vertex[v - u + 1];
    memcpy (vertex_array, &csr->get_vertices ()[u], (v-u + 1)*sizeof(CSR::Vertex));
    vertex_array[0].set_start_edge_id (start);
    vertex_array[v-u].set_end_edge_id (end);

    CSR::Edge* edge_array = new CSR::Edge[end - start + 1];
    memcpy (edge_array, &csr->get_edges ()[start], (end - start + 1)*sizeof (CSR::Edge));
    CSRPartition part = CSRPartition (u, v, start, end, vertex_array, edge_array);
    csr_partitions.push_back (part);
  }

  /** Check if partitions created are correct**/
  //Sum of edges of all partitions is equal to N_EDGES
  int sum_partition_edges = 0;

  for (int id = 0; id < csr_partitions.size (); id++) {
    auto part = csr_partitions[id];
    std::cout << id << " " << part.last_edge_idx << " " << part.first_edge_idx << " " << part.first_vertex_id << " " << part.last_vertex_id << std::endl;
    if (part.last_edge_idx != -1) {
      sum_partition_edges += part.last_edge_idx - part.first_edge_idx + 1;
    }
  }

  if (!(sum_partition_edges == N_EDGES)) {
    std::cout << __LINE__ <<": "<<sum_partition_edges  << " " << N_EDGES << std::endl;
  }
  assert (sum_partition_edges == N_EDGES);

  int sum_vertices = 0;
  for (int p = 0; p < csr_partitions.size (); p++) {
    if (p > 0 && csr_partitions[p].first_vertex_id == csr_partitions[p-1].last_vertex_id) {
      sum_vertices += csr_partitions[p].last_vertex_id - (csr_partitions[p].first_vertex_id);
    } else {
      sum_vertices += csr_partitions[p].last_vertex_id - csr_partitions[p].first_vertex_id + 1;
    }
  }

  assert (sum_vertices == N);

  int equal_edges = 0;

  /*Check if union of all partitions is equal to the graph*/
  for (int p = 0; p < csr_partitions.size (); p++) {
    int u = csr_partitions[p].first_vertex_id;
    int v = csr_partitions[p].last_vertex_id;
    int end = csr_partitions[p].last_edge_idx;
    int start = csr_partitions[p].first_edge_idx;
    for (int vertex = u; vertex <= v; vertex++) {
      int _start = csr->get_start_edge_idx (vertex);
      if (p > 0 && vertex == csr_partitions[p-1].last_vertex_id) {
        _start = start;
      }
      int _end = csr->get_end_edge_idx (vertex);
      int part_start = csr_partitions[p].get_start_edge_idx (vertex);
      int part_end = csr_partitions[p].get_end_edge_idx (vertex);
      
      if (_end != -1 && part_end != -1) {
        while (_start <= _end && _start <= end && part_start <= part_end) {
          if (!(csr->get_edges ()[_start] == csr_partitions[p].get_edge (part_start))) {
            std::cout << "part_start " << part_start << " part_end " << 
            part_end << " _start " << _start << " _end " << _end << " vertex " 
            << vertex << std::endl;  
            abort ();
          }
          
          equal_edges++;
          part_start++;
          _start++;
        }
      }
    }
  }

  assert (equal_edges == N_EDGES);  
  /********Checking DONE*******/
}


size_t compute_source_to_root_data (std::vector<std::vector<std::pair <VertexID, int>>>& host_src_to_roots,
                                    CSR* csr, int hop, int*** final_map_vertex_to_additions, int** additions_sizes, 
                                    VertexID** neighbors, int* neighbors_sizes,
                                    int*& device_src_to_roots, int*& device_src_to_root_positions)
{
  int* host_src_to_roots_positions = nullptr;
  int *host_src_to_roots_linear = nullptr;

  double t1 = convertTimeValToDouble(getTimeOfDay ());
  host_src_to_roots.clear ();
  //Create per hop vertex data
  for (int v = 0; v < csr->get_n_vertices (); v++) {
    host_src_to_roots.push_back (std::vector<std::pair <VertexID, int> > ());
  }
  for (int v = 0; v < csr->get_n_vertices (); v++) {
    int start = final_map_vertex_to_additions[hop-1][0][2*v];
    int end   = additions_sizes[hop-1][2*v + 1];
    for (int i = 0; i < end; i++) {
      int src = neighbors[hop-1][start + i];
      assert (start + i < neighbors_sizes[hop-1]/sizeof(VertexID));
      assert (src >= 0 && src < N);
      host_src_to_roots[src].push_back (std::make_pair (v, start + i));
    }
  }

  int host_hop_vertex_data_size = 0;

  for (int v = 0; v < csr->get_n_vertices (); v++) {
    host_hop_vertex_data_size += host_src_to_roots[v].size ();
  }

  host_src_to_roots_linear = new int [2*host_hop_vertex_data_size];
  host_src_to_roots_positions = new int[2*csr->get_n_vertices ()];
  int iter = 0;

  for (int v = 0; v < csr->get_n_vertices (); v++) {
    for (int i = 0; i < host_src_to_roots[v].size (); i++) {
      host_src_to_roots_linear[iter + 2*i] = std::get<0> (host_src_to_roots[v][i]);
      host_src_to_roots_linear[iter + 2*i + 1] = std::get<1> (host_src_to_roots[v][i]);
    }

    host_src_to_roots_positions [2*v] = iter;
    host_src_to_roots_positions [2*v + 1] = host_src_to_roots[v].size ();
    iter += 2*host_src_to_roots[v].size ();
  }

  double t2 = convertTimeValToDouble(getTimeOfDay ());
        
  std::cout << "Time taken to create hop vertex data: " << (t2 - t1) << " secs " << std::endl;
  EXECUTE_CUDA_FUNC (hipMalloc (&device_src_to_roots, 
                                  2*host_hop_vertex_data_size*sizeof (int)));
  EXECUTE_CUDA_FUNC (hipMemcpy (device_src_to_roots, 
                                  host_src_to_roots_linear, 
                                  2*host_hop_vertex_data_size*sizeof (int), 
                                  hipMemcpyHostToDevice));
  EXECUTE_CUDA_FUNC (hipMalloc (&device_src_to_root_positions, 
                                  2*csr->get_n_vertices()*sizeof (int)));
  EXECUTE_CUDA_FUNC (hipMemcpy (device_src_to_root_positions,  
                                  host_src_to_roots_positions, 
                                  2*csr->get_n_vertices()*sizeof (int), 
                                  hipMemcpyHostToDevice));
  delete host_src_to_roots_linear;
  delete host_src_to_roots_positions;

  return host_hop_vertex_data_size;
}

int main (int argc, char* argv[])
{
  std::vector<Vertex> vertices;
  int n_edges = 0;

  if (argc < 2) {
    std::cout << "Arguments: graph-file" << std::endl;
    return -1;
  }

  char* graph_file = argv[1];
  FILE* fp = fopen (graph_file, "r+");
  if (fp == nullptr) {
    std::cout << "File '" << graph_file << "' not found" << std::endl;
    return 1;
  }

  Graph graph (fp);

  fclose (fp);

  std::cout << "n_edges "<<graph.get_n_edges () <<std::endl;
  std::cout << "vertices " << graph.get_vertices ().size () << std::endl; 


  CSR* csr = new CSR(N, N_EDGES);
  std::cout << "sizeof(CSR)"<< sizeof(CSR)<<std::endl;
  csr_from_graph (csr, graph);

  size_t global_mem_size = 15*1024*1024*1024UL;
  #define PINNED_MEMORY
  #ifdef PINNED_MEMORY
    char* global_mem_ptr;
    hipError_t malloc_error = hipHostMalloc ((void**)&global_mem_ptr, global_mem_size);
    std::cout << "Malloc error: " << hipGetErrorString (malloc_error) << std::endl;
    assert (malloc_error == hipSuccess);
  #else
    char* global_mem_ptr = new char[global_mem_size];
  #endif

  std::cout << "Pinned Memory Allocated" << std::endl;
  GlobalMemAllocator::initialize (global_mem_ptr, global_mem_size);

  std::vector<VectorVertexEmbedding> initial_embeddings = get_initial_embedding_vector (csr);
  std::vector<VectorVertexEmbedding> output;
  size_t new_embeddings_size = 0;

  std::vector<VectorVertexEmbedding>& input_embeddings = initial_embeddings;
  std::vector<VectorVertexEmbedding> iter_1_embeddings;
  {
    run_single_step_initial_vector (input_embeddings, csr, output, iter_1_embeddings);
    input_embeddings = iter_1_embeddings;
  }

  double total_stream_time = 0;

  const size_t max_embedding_size_per_iter = (12000000/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
  double_t kernelTotalTime = 0.0;
  std::vector<CSRPartition> csr_partitions;

#ifdef ENABLE_GRAPH_PARTITION_FOR_GLOBAL_MEM 
  create_csr_partitions (csr, csr_partitions, GRAPH_PARTITION_SIZE - sizeof (CSRPartition));
#else
  CSRPartition full_partition = CSRPartition (0, csr->get_n_vertices () - 1, 0, csr->get_n_edges () - 1, 
                                              csr->get_vertices (), csr->get_edges ());
  csr_partitions.push_back (full_partition);
#endif

  const int N_HOPS = 2;
  
  //Graph on GPU
  CSRPartition* device_csr;
  CSR::Vertex* device_vertex_array;
  CSR::Edge* device_edge_array;
  int* device_vertex_partition_positions;

#if 0
  EXECUTE_CUDA_FUNC (hipMalloc (&device_vertex_partition_positions, n_partitions*sizeof(int)*2));
  EXECUTE_CUDA_FUNC (hipMemcpy (device_vertex_partition_positions, vertex_partition_positions, n_partitions*sizeof(int)*2, hipMemcpyHostToDevice));
#endif

  double gpu_time = 0;
  
  std::cout << "Generating additions" << std::endl;
  int* device_additions_sizes;
  EXECUTE_CUDA_FUNC (hipMalloc (&device_additions_sizes, sizeof(VertexID)*csr->get_n_vertices ()*2));
  EXECUTE_CUDA_FUNC (hipMemset (device_additions_sizes, 0, sizeof(VertexID)*csr->get_n_vertices ()*2));
  void* device_additions; //Storage to store inputs added to each embedding
  void* device_additions_prev_hop = nullptr;

  int* device_filled_ranges;
  EXECUTE_CUDA_FUNC (hipMalloc (&device_filled_ranges, sizeof (int)*csr->get_n_vertices ()));
  int* device_prev_thread_idx_to_edge_in_additions = nullptr;

  std::vector<std::vector <std::pair <VertexID, int>>> host_src_to_roots;

  VertexID** neighbors = new VertexID* [N_HOPS];
  int* neighbors_sizes = new int [N_HOPS];
  int** additions_sizes = new int* [N_HOPS];
  int*** final_map_vertex_to_additions = new int**[N_HOPS];
  //Map of idx of embedding to the start of how many inputs are added and number of new embeddings
  int*** device_map_vertex_to_additions = new int**[N_HOPS]; new int*[csr_partitions.size ()]; 

  for (int hop = 0; hop < N_HOPS; hop++) {
    int* source_vertex_idx;
    int* device_src_to_roots;
    int* device_src_to_root_positions;
    unsigned long long* device_max_neighbors_iter;
    unsigned long long int* device_profile_branch_1;
    unsigned long long int* device_profile_branch_2;
    size_t num_neighbors = 0;
    final_map_vertex_to_additions[hop] = new int*[csr_partitions.size ()];
    //size_t map_vertex_to_additions_size;
    const size_t map_vertex_to_additions_size = csr->get_n_vertices () * sizeof (VertexID) * 2;
    final_map_vertex_to_additions[hop][0] = (int*)new char[map_vertex_to_additions_size];
    size_t final_map_vertex_to_additions_iter = 0;
    device_map_vertex_to_additions[hop] = new int*[csr_partitions.size ()];
    
    for (int p = 0; p < csr_partitions.size (); p++) {
      device_map_vertex_to_additions[hop][p] = nullptr;
    }
    /********************Get the output additions lengths*******************/
    for (int partition_idx = 0; partition_idx < csr_partitions.size (); partition_idx++) {
      unsigned long long num_neighbors_iter = 0;
      CSRPartition& partition = csr_partitions[partition_idx];

      copy_partition_to_gpu (partition, device_csr, device_vertex_array, device_edge_array);

      num_neighbors_iter = 0;
      const int partition_map_vertex_to_additions_size = partition.get_n_vertices ()* 2;
      
      EXECUTE_CUDA_FUNC (hipMalloc (&device_max_neighbors_iter, 
                                     sizeof(unsigned long long)));
      EXECUTE_CUDA_FUNC (hipMemset (device_max_neighbors_iter, 0, 
                                     sizeof (unsigned long long)));
      EXECUTE_CUDA_FUNC (hipMalloc (&device_map_vertex_to_additions[hop][partition_idx], 
                                     partition_map_vertex_to_additions_size*sizeof (VertexID)));

      std::cout << "Calling cuda kernel for hop: " << hop << " partition: " << partition_idx << " vertex = [" << csr_partitions[partition_idx].first_vertex_id << ", "<< csr_partitions[partition_idx].last_vertex_id << "]" << std::endl;

      int N_THREADS = 128;
      int N_BLOCKS = (partition.get_n_vertices ()%128 == 0) ? partition.get_n_vertices ()/128 : partition.get_n_vertices ()/128 + 1;
      double t1 = convertTimeValToDouble(getTimeOfDay ());

      int* device_edges_to_prev_iter_additions;
      const int vertex_with_next_partition = get_common_vertex_with_next_partition (csr_partitions, partition_idx);
      const int vertex_with_prev_partition = get_common_vertex_with_previous_partition (csr_partitions, partition_idx);
                                                                            
      if (hop > 0) {
        int* edges_to_prev_iter_additions;
        edges_to_prev_iter_additions = new int[partition.get_n_edges ()];
        for (int e = partition.first_edge_idx; e <= partition.last_edge_idx; e++) {
          VertexID v = partition.get_edge (e);
          if (!partition.is_vertex_in_partition(v) || 
              v == vertex_with_next_partition || v == vertex_with_prev_partition) {
            edges_to_prev_iter_additions[e - partition.first_edge_idx] = final_map_vertex_to_additions[hop-1][0][2*v + 1];
          }
        }

        EXECUTE_CUDA_FUNC (hipMalloc (&device_edges_to_prev_iter_additions, 
                                       partition.get_n_edges ()*sizeof(VertexID)));
        EXECUTE_CUDA_FUNC (hipMemcpy (device_edges_to_prev_iter_additions, 
                                       edges_to_prev_iter_additions, 
                                       partition.get_n_edges ()*sizeof(VertexID), 
                                       hipMemcpyHostToDevice));
      }

      if (hop == 0) {
        get_max_lengths_for_vertices_first_iter <<<N_BLOCKS, N_THREADS>>> (device_csr, partition.first_vertex_id, 
                                                                           partition.last_vertex_id,
                                                                           device_max_neighbors_iter,
                                                                           device_map_vertex_to_additions[hop][partition_idx]);
      } else {
        get_max_lengths_for_vertices_single_step <<<N_BLOCKS, N_THREADS>>> (device_csr, partition.first_vertex_id, 
                                                                            partition.last_vertex_id,
                                                                            device_max_neighbors_iter,
                                                                            device_map_vertex_to_additions[hop-1][partition_idx],
                                                                            device_map_vertex_to_additions[hop][partition_idx],
                                                                            device_map_vertex_to_additions[0][partition_idx],
                                                                            device_edges_to_prev_iter_additions,
                                                                            vertex_with_prev_partition,
                                                                            vertex_with_next_partition);
      }
  
      EXECUTE_CUDA_FUNC (hipDeviceSynchronize ());
      double t2 = convertTimeValToDouble(getTimeOfDay ());
  
      gpu_time += t2 - t1;
  
      std::cout << "Cuda Kernel Done " << std::endl;
      is_cuda_error (hipGetLastError ());
      EXECUTE_CUDA_FUNC (hipMemcpy (&num_neighbors_iter, device_max_neighbors_iter, sizeof(unsigned long long), hipMemcpyDeviceToHost));
      std::cout << "New Neighbors " << num_neighbors_iter << std::endl;
      
      int *partition_map_vertex_to_additions = new int[partition_map_vertex_to_additions_size];
      EXECUTE_CUDA_FUNC (hipMemcpy (partition_map_vertex_to_additions, 
                                     device_map_vertex_to_additions[hop][partition_idx], 
                                     partition_map_vertex_to_additions_size*sizeof (int), 
                                     hipMemcpyDeviceToHost));
      
      if (partition_idx == 0) {
        memcpy (&final_map_vertex_to_additions[hop][0][final_map_vertex_to_additions_iter],
                partition_map_vertex_to_additions,
                partition_map_vertex_to_additions_size*sizeof (int));
        final_map_vertex_to_additions_iter += partition_map_vertex_to_additions_size;
      } else if (vertex_with_prev_partition != -1) {
        int common_vertex = vertex_with_prev_partition;
        int common_vertex_new_additions = partition_map_vertex_to_additions[2*(common_vertex - common_vertex) + 1];
        int common_vertex_start_pos = final_map_vertex_to_additions[hop][0][2*common_vertex];

        for (int v = csr_partitions[partition_idx - 1].first_vertex_id; 
             v < csr_partitions[partition_idx - 1].last_vertex_id; v++) {
          if (final_map_vertex_to_additions[hop][0][2*v] > common_vertex_start_pos) {
            final_map_vertex_to_additions[hop][0][2*v] += common_vertex_new_additions;
          }
        }
        final_map_vertex_to_additions[hop][0][2*common_vertex + 1] += common_vertex_new_additions;
        int start_pos = 0;
        //TODO: start_pos is sum of all embedding additions so far
        int max_v = 0;
        for (int v = csr_partitions[partition_idx - 1].first_vertex_id; 
             v <= csr_partitions[partition_idx - 1].last_vertex_id; v++) {
          int p = final_map_vertex_to_additions[hop][0][2*v] + final_map_vertex_to_additions[hop][0][2*v + 1];
          if (p > start_pos) {
            start_pos = p;
            max_v = v;
          }
        }
        
        assert (start_pos <= (num_neighbors + common_vertex_new_additions));
        for (int v = 1; v < csr_partitions[partition_idx].get_n_vertices (); v++) {
          int vertex = csr_partitions[partition_idx].first_vertex_id + v;
          assert (partition_map_vertex_to_additions[2*v] >= 0);
          int vertex_start_pos = partition_map_vertex_to_additions[2*v];
          if (vertex_start_pos > partition_map_vertex_to_additions[0]) {
            vertex_start_pos -= common_vertex_new_additions;
          }

          final_map_vertex_to_additions[hop][0][2*vertex] = start_pos + vertex_start_pos;
          final_map_vertex_to_additions[hop][0][2*vertex + 1] = partition_map_vertex_to_additions[2*v + 1];
        }
        
        final_map_vertex_to_additions_iter += partition_map_vertex_to_additions_size - 2;
      } else {
        int start_pos = 0;
        for (int v = csr_partitions[partition_idx - 1].first_vertex_id; 
             v <= csr_partitions[partition_idx - 1].last_vertex_id; v++) {
          int pos = final_map_vertex_to_additions[hop][0][2*v] + final_map_vertex_to_additions[hop][0][2*v + 1];
          start_pos = max (start_pos, pos);
        }
        assert (start_pos <= num_neighbors);
        for (int v = 0; v < csr_partitions[partition_idx].get_n_vertices (); v++) {
          int vertex = csr_partitions[partition_idx].first_vertex_id + v;
          final_map_vertex_to_additions[hop][0][2*vertex] = start_pos + partition_map_vertex_to_additions[2*v];
          final_map_vertex_to_additions[hop][0][2*vertex + 1] = partition_map_vertex_to_additions[2*v + 1];
        }

        final_map_vertex_to_additions_iter += partition_map_vertex_to_additions_size;
      }

      num_neighbors += num_neighbors_iter;
    }

    num_neighbors = num_neighbors * sizeof (VertexID);
    EXECUTE_CUDA_FUNC (hipFree (device_max_neighbors_iter));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_additions, num_neighbors));
    EXECUTE_CUDA_FUNC (hipMemset (device_additions, -1, num_neighbors));
    /**************************DONE**********************/

    EXECUTE_CUDA_FUNC (hipMalloc (&source_vertex_idx, sizeof(int)));
    EXECUTE_CUDA_FUNC (hipMemset (source_vertex_idx, 0,  sizeof (int)));
    
    EXECUTE_CUDA_FUNC (hipMalloc (&device_profile_branch_1, sizeof (unsigned long)));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_profile_branch_2, sizeof (unsigned long)));
    EXECUTE_CUDA_FUNC (hipMemset (device_profile_branch_1, 0, sizeof (unsigned long)));
    EXECUTE_CUDA_FUNC (hipMemset (device_profile_branch_2, 0, sizeof (unsigned long)));

    int N_BLOCKS = csr->get_n_vertices ();
    
    neighbors[hop] = (VertexID*) new char[num_neighbors];
    neighbors_sizes[hop] = num_neighbors;
    
    if (hop > 0) {
      compute_source_to_root_data (host_src_to_roots, csr, hop, 
                                   final_map_vertex_to_additions, 
                                   additions_sizes, neighbors, 
                                   neighbors_sizes, device_src_to_roots, 
                                   device_src_to_root_positions);
    }

    CSR* device_csr1;
    EXECUTE_CUDA_FUNC (hipMalloc (&device_csr1, sizeof (CSR)));
    EXECUTE_CUDA_FUNC (hipMemcpy (device_csr1, csr, sizeof (CSR), hipMemcpyHostToDevice));

    int* device_final_map_vertex_to_additions;

    EXECUTE_CUDA_FUNC (hipMemset (device_additions_sizes, 0, sizeof(VertexID)*csr->get_n_vertices ()*2));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_final_map_vertex_to_additions, map_vertex_to_additions_size));
    EXECUTE_CUDA_FUNC (hipMemcpy (device_final_map_vertex_to_additions, 
                                   &final_map_vertex_to_additions[hop][0][0],
                                   map_vertex_to_additions_size,
                                   hipMemcpyHostToDevice));

    double t1 = convertTimeValToDouble(getTimeOfDay ());
    run_hop_parallel_single_step <<<N_BLOCKS, N_THREADS>>> (N_HOPS, hop, device_csr1,  
                                                            device_additions,
                                                            num_neighbors,
                                                            device_additions_prev_hop,
                                                            device_final_map_vertex_to_additions,
                                                            device_additions_sizes,
                                                            device_src_to_roots,
                                                            device_src_to_root_positions,
                                                            source_vertex_idx,
                                                            device_profile_branch_1,
                                                            device_profile_branch_2);
    EXECUTE_CUDA_FUNC (hipDeviceSynchronize ());
    double t2 = convertTimeValToDouble(getTimeOfDay ());
    gpu_time += t2 - t1;
    additions_sizes[hop] = new int[csr->get_n_vertices () * 2];
    EXECUTE_CUDA_FUNC (hipMemcpy (neighbors[hop], device_additions, neighbors_sizes[hop], hipMemcpyDeviceToHost));
    EXECUTE_CUDA_FUNC (hipMemcpy (additions_sizes[hop], device_additions_sizes, csr->get_n_vertices ()*sizeof(VertexID)*2, hipMemcpyDeviceToHost));
    
#ifdef PROFILE
    unsigned long profile_branch_1, profile_branch_2;
    EXECUTE_CUDA_FUNC (hipMemcpy (&profile_branch_1, device_profile_branch_1, sizeof(profile_branch_1), hipMemcpyDeviceToHost));
    EXECUTE_CUDA_FUNC (hipMemcpy (&profile_branch_2, device_profile_branch_2, sizeof(profile_branch_1), hipMemcpyDeviceToHost));

    std::cout << "profile_branch_1 " << profile_branch_1 << std::endl;
    std::cout << "profile_branch_2 " << profile_branch_2 << std::endl;
#endif
    device_additions_prev_hop = device_additions;
  }
  
  std::cout << "Getting embeddings from GPU" << std::endl;
  std::vector <VectorVertexEmbedding> produced_embeddings;
  for (int input_embedding_idx = 0; input_embedding_idx < csr->get_n_vertices (); input_embedding_idx++) {
    size_t produced_embedding_size = 0;
    for (int hop = 0; hop < N_HOPS; hop++) {
      VectorVertexEmbedding& input_embedding = input_embeddings[input_embedding_idx];
      int n_additions = additions_sizes[hop][2*input_embedding_idx + 1];
      produced_embedding_size += n_additions;
    }
    //std::cout << " input_embedding_idx " << input_embedding_idx << std::endl;
    int copied = 0;
    size_t global_mem_idx = GlobalMemAllocator::alloc_vertices_array (produced_embedding_size);
    for (int hop = 0; hop < N_HOPS; hop++) {
      VectorVertexEmbedding& input_embedding = input_embeddings[input_embedding_idx];
      int start_idx = final_map_vertex_to_additions[hop][0][2*input_embedding_idx];
      int n_additions = additions_sizes[hop][2*input_embedding_idx + 1];
      //std::cout << "i " << input_embedding_idx << " produced_embedding_size " << produced_embedding_size << " global_mem_idx " << global_mem_idx << std::endl;
      VertexID* ptr = (VertexID*) ((char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_idx);
      memcpy (ptr + copied, &neighbors[hop][start_idx], sizeof(VertexID)*n_additions);

      if (input_embedding_idx == 3030) {
        for (int ii = start_idx; ii < n_additions + start_idx; ii++) {
          std::cout << neighbors[hop][ii] << std::endl;
        }
      }
      copied += n_additions;
    }

    VectorVertexEmbedding embedding = VectorVertexEmbedding ((uint32_t)produced_embedding_size, global_mem_idx, true);
    produced_embeddings.push_back (embedding);
  }

  hipFree (device_csr);

  std::cout << "Generating CPU Embeddings:" << std::endl;
  double cpu_t1 = convertTimeValToDouble (getTimeOfDay ());
  std::vector<std::vector<VertexID>> hops = n_hop_cpu (csr, N_HOPS);
  double cpu_t2 = convertTimeValToDouble (getTimeOfDay ());

  std::cout << "CPU Time: " << (cpu_t2 - cpu_t1) << " secs" << std::endl;
  std::cout << "GPU Time: " << gpu_time << " secs" << std::endl;
  assert (produced_embeddings.size () == hops.size ());
  for (int idx = 0; idx < produced_embeddings.size (); idx++) {
    if (idx == 3030) {
      std::cout << "vertices " << hops[idx].size () << std::endl;
    }
    std::unordered_set<VertexID> cpu_set = std::unordered_set<VertexID> (hops[idx].begin (), hops[idx].end ());
    std::vector<VertexID> vector_hops;
    vector_hops.insert (vector_hops.begin (), cpu_set.begin(), cpu_set.end ());
    std::sort (vector_hops.begin (), vector_hops.end ());
    std::vector<VertexID> gpu_vector = produced_embeddings [idx].to_vector ();
    std::unordered_set<VertexID> gpu_vector_set = std::unordered_set<VertexID> (gpu_vector.begin (), gpu_vector.end ());
    gpu_vector = std::vector<VertexID> (gpu_vector_set.begin (), gpu_vector_set.end ());
    std::sort (gpu_vector.begin (), gpu_vector.end ());

    if (vector_hops != gpu_vector) {
      std::cout << "checking for vertex " << idx << " start " << final_map_vertex_to_additions[1][0][2*idx] << " " << additions_sizes[1][2*idx+1] << std::endl;
      std::cout << "size " << vector_hops.size () << " " << gpu_vector.size () << std::endl;
      #if 1
      for (int i = 0; i < max (vector_hops.size (), gpu_vector.size ()); i++) {
        if (i < min (vector_hops.size (), gpu_vector.size ()))
          std::cout << vector_hops[i] << "  " << gpu_vector[i] << std::endl;
        else if (i < vector_hops.size ()) 
          std::cout << vector_hops[i] << std::endl;
        else if (i < gpu_vector.size ()) 
          std::cout << "     " << gpu_vector[i] << std::endl;
      }
      #endif
    }
    assert (vector_hops == gpu_vector);
  }

#ifdef PINNED_MEMORY
  // hipFree (global_mem_ptr);
#else
  delete[] global_mem_ptr;
#endif
  std::cout << "Number of embeddings found "<< input_embeddings.size () << std::endl;
  std::cout << "Time spent in GPU kernel execution " << kernelTotalTime << std::endl;
  std::cout << "Time spent in Streams " << total_stream_time << std::endl;
}
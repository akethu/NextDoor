
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <string.h>
#include <assert.h>
#include <tuple>

#define LINE_SIZE 1024*1024
//#define USE_FIXED_THREADS
#define MAX_CUDA_THREADS (96*96)
#define THREAD_BLOCK_SIZE 256
#define WARP_SIZE 32
#define ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING false //TODO: there is bug with citeseer.graph when this is enabled
//#define USE_CSR_IN_SHARED
//#define USE_EMBEDDING_IN_SHARED_MEM
//#define USE_EMBEDDING_IN_GLOBAL_MEM
#define USE_EMBEDDING_IN_LOCAL_MEM
#define PROCESS_EMBEDDINGS_PER_VERTEX
//#define SHARED_MEM_NON_COALESCING
/**
  * The commit performing better is 698368fa19d023e3cb09705d820d333f79d0bf46.
  */
#ifdef SHARED_MEM_NON_COALESCING
  #ifndef USE_EMBEDDING_IN_SHARED_MEM
    #error "USE_EMBEDDING_IN_SHARED_MEM must be enabled with SHARED_MEM_NON_COALESCING"
  #endif
#endif
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #ifdef USE_FIXED_THREADS
    #error "USE_FIXED_THREADS cannot be enabled with USE_EMBEDDING_IN_SHARED_MEM"
  #endif
#endif

#define NEW_EMBEDDING_BUFFER_SIZE 128*1024*1024 //Size in terms of Bytes

//#define USE_CONSTANT_MEM

typedef uint8_t SharedMemElem;

//citeseer.graph
const int N = 3312;
const int N_EDGES = 9074;

//micro.graph
//const int N = 100000;
//const int N_EDGES = 2160312;

enum BUFFER_STATUS {
  GPU_USING,
  CPU_COPYING,
  FREE,
};

class Vertex
{
private:
  int id;
  int label;
  std::vector <int> edges;

public:
  Vertex (int _id, int _label) : label(_label), id (_id)
  {
  }

  int set_id (int _id) {id = _id;}
  int get_id () {return id;}
  int get_label () {return label;}
  void add_edge (int vertexID) {edges.push_back (vertexID);}
  void sort_edges () {std::sort (edges.begin(), edges.end ());}
  std::vector <int>& get_edges () {return edges;}
  void print (std::ostream& os)
  {
    os << id << " " << label << " ";
    for (auto edge : edges) {
      os << edge << " ";
    }

    os << std::endl;
  }

  static bool compare_vertex (Vertex& v1, Vertex& v2) 
  {
    return v1.edges.size () > v2.edges.size ();
  }
};

int chars_in_int (int num)
{
  if (num == 0) return sizeof(char);
  return (int)((ceil(log10(num))+1)*sizeof(char));
}

class Graph
{
private:
  std::vector<Vertex> vertices;
  int n_edges;

public:
  Graph (std::vector<Vertex> _vertices, int _n_edges) :
    vertices (_vertices), n_edges(_n_edges)
  {}

  const std::vector<Vertex>& get_vertices () {return vertices;}
  int get_n_edges () {return n_edges;}
};

class CSR
{
public:
  struct Vertex
  {
    int id;
    int label;
    int start_edge_id;
    int end_edge_id;
    __host__ __device__
    Vertex ()
    {
      id = -1;
      label = -1;
      start_edge_id = -1;
      end_edge_id = -1;
    }

    void set_from_graph_vertex (::Vertex& vertex)
    {
      id = vertex.get_id ();
      label = vertex.get_label ();
    }

    void set_start_edge_id (int start) {start_edge_id = start;}
    void set_end_edge_id (int end) {end_edge_id = end;}
  };

  typedef int Edge;

public:
  CSR::Vertex vertices[N];
  CSR::Edge edges[N_EDGES];
  int n_vertices;
  int n_edges;

public:
  CSR (int _n_vertices, int _n_edges)
  {
    n_vertices = _n_vertices;
    n_edges = _n_edges;
  }

  __host__ __device__
  CSR ()
  {
    n_vertices = N;
    n_edges = N_EDGES;
  }

  void print (std::ostream& os)
  {
    for (int i = 0; i < n_vertices; i++) {
      os << vertices[i].id << " " << vertices[i].label << " ";
      for (int edge_iter = vertices[i].start_edge_id;
           edge_iter <= vertices[i].end_edge_id; edge_iter++) {
        os << edges[edge_iter] << " ";
      }
      os << std::endl;
    }
  }

  __host__ __device__
  int get_start_edge_idx (int vertex_id)
  {
    if (!(vertex_id < n_vertices && 0 <= vertex_id)) {
      printf ("vertex_id %d, n_vertices %d\n", vertex_id, n_vertices);
      assert (false);
    }
    return vertices[vertex_id].start_edge_id;
  }

  __host__ __device__
  int get_end_edge_idx (int vertex_id)
  {
    assert (vertex_id < n_vertices && 0 <= vertex_id);
    return vertices[vertex_id].end_edge_id;
  }

  __host__ __device__
  bool has_edge (int u, int v)
  {
    //TODO: Since graph is sorted, do this using binary search
    for (int e = get_start_edge_idx (u); e <= get_end_edge_idx (u); e++) {
      if (edges[e] == v) {
        return true;
      }
    }

    return false;
  }

  __host__ __device__
  const CSR::Edge* get_edges () {return &edges[0];}

  __host__ __device__
  const CSR::Vertex* get_vertices () {return &vertices[0];}

  __host__ __device__
  int get_n_vertices () {return n_vertices;}

  __host__ __device__
  void copy_vertices (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      vertices[i] = src->get_vertices()[i];
    }
  }

  __host__ __device__
  void copy_edges (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      edges[i] = src->get_edges ()[i];
    }
  }

  __host__ __device__
  int get_n_edges () {return n_edges;}
};

#ifdef USE_CONSTANT_MEM
  __constant__ unsigned char csr_constant_buff[sizeof(CSR)];
#endif

void csr_from_graph (CSR* csr, Graph& graph)
{
  int edge_iterator = 0;
  auto graph_vertices = graph.get_vertices ();
  for (int i = 0; i < graph_vertices.size (); i++) {
    ::Vertex& vertex = graph_vertices[i];
    csr->vertices[i].set_from_graph_vertex (graph_vertices[i]);
    csr->vertices[i].set_start_edge_id (edge_iterator);
    for (auto edge : vertex.get_edges ()) {
      csr->edges[edge_iterator] = edge;
      edge_iterator++;
    }

    csr->vertices[i].set_end_edge_id (edge_iterator-1);
  }
}

//template <size_t N> using VertexEmbedding = std::bitset<N>;

#define CVT_TO_NEXT_MULTIPLE(n,k) ((n) %(k) ==0 ? (n) : ((n)/(k)+1)*(k))
class BitVectorVertexEmbedding
{
private:
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  unsigned char array[CVT_TO_NEXT_MULTIPLE(N/8, 32*sizeof(SharedMemElem))];
#else
  unsigned char array[(N/8)];
#endif

public:
  __device__ __host__
  BitVectorVertexEmbedding ()
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    assert (array != nullptr);
    reset ();
    assert (all_false () == true);
  }

  __host__ __device__
  size_t convert_to_bytes_multiple (size_t n)
  {
    return (n/8)*8;
  }

  __host__ __device__
  BitVectorVertexEmbedding (const BitVectorVertexEmbedding& embedding)
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    for (int i = 0; i <  convert_to_bytes_multiple(N)/8; i++) {
      array[i] = embedding.array[i];
    }
  }

  __host__ __device__
  void set (int index)
  {
    assert (index >= 0 and index < N);
    array[index/8] = array[index/8] | (1 << (index %8));
  }

  __host__ __device__
  void set ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = (unsigned char) (~(0UL));
    }
  }

  __host__ __device__
  void reset ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = 0;
    }
  }

  __host__ __device__
  void reset (int index)
  {
    assert (index >= 0 and index < N);
    array[index/8] = array[index/8] & (~(1UL << (index %8)));
  }

  __host__ __device__
  bool test (int index)
  {
    return (bool) ((array[index/8] >> (index % 8))&1);
  }

  __host__ __device__
  bool all_false ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      if (array[i] != 0UL) {
        return false;
      }
    }

    return true;
  }
  
  __host__ __device__
  int get_n_vertices () 
  {
    int n_vertices = 0;
    for (int i = 0; i < N; i++) {
      if (test(i) == true) {
        n_vertices++;
      }
    }
    
    return n_vertices;
  }
  
  __host__ __device__
  ~BitVectorVertexEmbedding ()
  {
    //delete[] array;
  }
};

//typedef BitVectorVertexEmbedding VertexEmbedding;

template <uint32_t size> 
class VectorVertexEmbedding
{
private:
  uint32_t array[size];
  uint32_t filled_size;
  
public:
  __device__ __host__
  VectorVertexEmbedding ()
  {
    filled_size = 0;
  }

  __host__ __device__
  VectorVertexEmbedding (const VectorVertexEmbedding<size>& embedding)
  {
  #if DEBUG
    assert (embedding.get_max_size () <= get_max_size ());
  #endif
    filled_size = 0;
    for (int i = 0; i < embedding.get_n_vertices (); i++) {
      add (embedding.get_vertex (i));
    }
  }
  
  __host__ __device__
  void add (int v)
  {
  #if DEBUG
    if (!(size != 0 and filled_size < size)) {
      printf ("filled_size %d, size %d\n", filled_size, size);
      //assert (size != 0 and filled_size < size);
      assert (false);
    }
  #endif
  
    add_unsorted (v);
    return;
    int pos = 0;
    
    for (int i = 0; i < filled_size; i++) {
      if (array[i] > v) {
        pos = i;
        break;
      }
    }
    
    for (int i = filled_size-1; i >= pos ; i--) {
      array[i+1] = array[i];
    }
    
    array[pos] = v;
    filled_size++;
  }

  __host__ __device__
  void add_last_in_sort_order () 
  {
    int v = array[filled_size-1];
    remove_last ();
    add (v);
  }

  __host__ __device__
  void add_unsorted (int v) 
  {
    array[filled_size++] = v;
  }
  
  __host__ __device__
  void remove (int v)
  {
    printf ("Do not support remove\n");
    assert (false);
  }
  
  __host__ __device__
  bool has_logn (int v)
  {
    int l = 0;
    int r = filled_size-1;
    
    while (l <= r) {
      int m = l+(r-l)/2;
      
      if (array[m] == v)
        return true;
      
      if (array[m] < v)
        l = m + 1;
      else
        r = m - 1;
    }
    
    return false;
  }
  
  __host__ __device__
  bool has (int v)
  {
    for (int i = 0; i < filled_size; i++) {
      if (array[i] == v) {
        return true;
      }
    }
    
    return false;
  }
  
  __host__ __device__
  size_t get_n_vertices () const
  {
    return filled_size;
  }
  
  __host__ __device__
  int get_vertex (int index) const
  {
    return array[index];
  }
  
  __host__ __device__
  int get_last_vertex () const
  {
    return array[filled_size-1];
  }
  
  __host__ __device__
  size_t get_max_size () const
  {
    return size;
  }
  
  __host__ __device__
  void clear ()
  {
    filled_size = 0;
  }
  
  __host__ __device__
  void remove_last () 
  {
    assert (filled_size > 0);
    filled_size--;
  }
  __host__ __device__
  ~VectorVertexEmbedding ()
  {
    //delete[] array;
  }
};

template <uint32_t size>
__host__ __device__
void vector_embedding_from_one_less_size (VectorVertexEmbedding<size>& vec_emb1,
                                          VectorVertexEmbedding<size+1>& vec_emb2)
{
  //TODO: Optimize here, filled_size++ in add is being called several times
  //but can be called only once too
  //if  (false and vec_emb1.get_n_vertices () != size) {
  //  printf ("vec_emb1.get_n_vertices () %ld != size %d\n", vec_emb1.get_n_vertices (), size);
  //  assert (false);
  //}
  for (int i = 0; i < vec_emb1.get_n_vertices (); i++) {
    vec_emb2.add (vec_emb1.get_vertex (i));
  }
}

template <uint32_t size> 
void bitvector_to_vector_embedding (BitVectorVertexEmbedding& bit_emb, 
                                    VectorVertexEmbedding<size>& vec_emb)
{
  for (int u = 0; u < N; u++) {
    if (bit_emb.test(u)) {
      vec_emb.add (u);
    }
  }
}

void print_embedding (BitVectorVertexEmbedding embedding, std::ostream& os);

std::vector<BitVectorVertexEmbedding> get_extensions_bitvector (BitVectorVertexEmbedding& embedding, CSR* csr)
{
  std::vector<BitVectorVertexEmbedding> extensions;

  if (embedding.all_false ()) {
    for (int u = 0; u < N; u++) {
      BitVectorVertexEmbedding extension;
      extension.set(u);
      //print_embedding (extension, std::cout);
      //std::cout << " " << u << std::endl;
      extensions.push_back (extension);
    }
  } else {
    for (int u = 0; u < N; u++) {
      if (embedding.test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding.test (v) == false) {
            BitVectorVertexEmbedding extension = BitVectorVertexEmbedding(embedding);
            extension.set(v);
            extensions.push_back(extension);
          }
        }
      }
    }
  }

  return extensions;
}

template <uint32_t size>
std::vector<VectorVertexEmbedding<size+1>> get_extensions_vector (VectorVertexEmbedding<size>& embedding, CSR* csr)
{
  std::vector<VectorVertexEmbedding<size+1>> extensions;

  if (embedding.get_n_vertices () == 0) {
    for (int u = 0; u < N; u++) {
      VectorVertexEmbedding<size+1> extension;
      extension.add(u);
      //print_embedding (extension, std::cout);
      //std::cout << " " << u << std::endl;
      extensions.push_back (extension);
    }
  } else {
    for (int i = 0; i < embedding.get_n_vertices (); i++) {
      int u = embedding.get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        int v = csr->get_edges () [e];
        if (embedding.has (v) == false) {
          VectorVertexEmbedding<size+1> extension;
          vector_embedding_from_one_less_size (embedding, extension);
          extension.add(v);
          extensions.push_back(extension);
        }
      }
    }
  }

  return extensions;
}

std::vector<BitVectorVertexEmbedding> get_initial_embedding_bitvector (CSR* csr)
{
  BitVectorVertexEmbedding embedding;
  std::vector <BitVectorVertexEmbedding> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

std::vector<VectorVertexEmbedding<0>> get_initial_embedding_vector (CSR* csr)
{
  VectorVertexEmbedding<0> embedding;
  std::vector <VectorVertexEmbedding<0>> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

bool (*filter) (CSR* csr, BitVectorVertexEmbedding& embedding);
void (*process) (std::vector<BitVectorVertexEmbedding>& output, BitVectorVertexEmbedding& embedding);

__host__ __device__
bool clique_filter (CSR* csr, BitVectorVertexEmbedding* embedding)
{
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      for (int v = 0; v < N; v++) {
        if (u != v and embedding->test(v)) {
          if (!csr->has_edge (u, v)) {
            return false;
          }
        }
      }
    }
  }

  return true;
}


template <uint32_t size>
__host__ __device__
bool clique_filter_vector (CSR* csr, VectorVertexEmbedding<size>* embedding)
{
  for (int i = 0; i < embedding->get_n_vertices (); i++) {
    int u = embedding->get_vertex (i);
    for (int j = 0; j < embedding->get_n_vertices (); j++) {
      int v = embedding->get_vertex (j);
      if (u != v) {
        if (!csr->has_edge (u, v)) {
          return false;
        }
      }
    }
  }

  return true;
}

void clique_process_bit_vector (std::vector<BitVectorVertexEmbedding>& output, BitVectorVertexEmbedding& embedding)
{
  output.push_back (embedding);
}

template <uint32_t size>
void clique_process_vector (std::vector<VectorVertexEmbedding<size>>& output, VectorVertexEmbedding<size>& embedding)
{
  output.push_back (embedding);
}

void run_single_step_initial_bitvector (void* input, int n_embeddings, CSR* csr,
                      std::vector<BitVectorVertexEmbedding>& output,
                      std::vector<BitVectorVertexEmbedding>& next_step)
{
  BitVectorVertexEmbedding* embeddings = (BitVectorVertexEmbedding*)input;

  for (int i = 0; i < n_embeddings; i++) {
    BitVectorVertexEmbedding embedding = embeddings[i];
    std::vector<BitVectorVertexEmbedding> extensions = get_extensions_bitvector (embedding, csr);

    for (auto extension : extensions) {
      if (clique_filter (csr, &extension)) {
        clique_process_bit_vector (output, extension);
        next_step.push_back (extension);
      }
    }
  }
}

void run_single_step_initial_vector (void* input, int n_embeddings, CSR* csr,
                      std::vector<VectorVertexEmbedding<1>>& output,
                      std::vector<VectorVertexEmbedding<1>>& next_step)
{
  VectorVertexEmbedding<0>* embeddings = (VectorVertexEmbedding<0>*)input;

  for (int i = 0; i < n_embeddings; i++) {
    VectorVertexEmbedding<0> embedding = embeddings[i];
    std::vector<VectorVertexEmbedding<1>> extensions = get_extensions_vector (embedding, csr);
    std::cout << "extensions " << extensions.size () << std::endl;
    for (auto extension : extensions) {
      if (clique_filter_vector (csr, &extension)) {
        clique_process_vector (output, extension);
        next_step.push_back (extension);
      }
    }
  }
}

__device__
void printf_embedding (BitVectorVertexEmbedding* embedding)
{
  printf ("[");
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      printf ("%d, ", u);
    }
  }

  printf ("]\n");
}

/*__global__
void run_single_step_bitvector_embedding (void* input, int n_embeddings, CSR* csr,
                      void* output_ptr,
                      int* n_output,
                      void* next_step, int* n_next_step,
                      int* n_output_1, int* n_next_step_1)
{
  int id;

#ifdef USE_CSR_IN_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread,
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread,
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#else
#ifdef USE_CONSTANT_MEM
  csr = (CSR*) csr_constant_buff;
#endif

#endif

  BitVectorVertexEmbedding* embeddings = (BitVectorVertexEmbedding*)input;
  BitVectorVertexEmbedding* new_embeddings = (BitVectorVertexEmbedding*)next_step;
  BitVectorVertexEmbedding* output = ((BitVectorVertexEmbedding*)output_ptr);
#ifdef USE_EMBEDDING_IN_LOCAL_MEM
  unsigned char temp_buffer [sizeof(BitVectorVertexEmbedding)];
#endif
  id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
#ifdef USE_FIXED_THREADS
  if (n_embeddings >= MAX_CUDA_THREADS) {
    int embeddings_per_thread = n_embeddings/MAX_CUDA_THREADS+1;

    start = id*embeddings_per_thread;
    end = (id+1)*embeddings_per_thread < n_embeddings ? (id+1)*embeddings_per_thread : n_embeddings;
  } else {
    if (id >= n_embeddings)
      return;

    start = id;
    end = id+1;
  }
#else
  if (id >= n_embeddings)
      return;

  start = id;
  end = id+1;
#endif

  int q[1000] = {0};

#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #if 0
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[per_thread_shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[0];
  #else
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
  #endif
#endif

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int lane_id = threadIdx.x % WARP_SIZE;
  for (int i = start; i < end; i++) {
    #ifdef USE_EMBEDDING_IN_SHARED_MEM
      #ifdef SHARED_MEM_NON_COALESCING
        memcpy (local_shared_buff, &embeddings[i], sizeof(BitVectorVertexEmbedding));

        BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*) local_shared_buff;
      #else
        const int thread_block_size = WARP_SIZE;
        const int last_emb = WARP_SIZE*(warp_id+1);
        if (blockIdx.x*blockDim.x + (warp_id+1)*WARP_SIZE > n_embeddings) {
          assert (false);
          //thread_block_size = n_embeddings - blockIdx.x*blockDim.x -
          //                    warp_id*WARP_SIZE;
          //last_emb = warp_id*WARP_SIZE + thread_block_size;
        }

        for (int emb = WARP_SIZE*warp_id; emb < last_emb; emb++) {
          SharedMemElem* embedding_buff = (SharedMemElem*) &embeddings[emb+blockIdx.x*blockDim.x];

          for (int j = 0; j < sizeof (BitVectorVertexEmbedding)/sizeof (SharedMemElem);
               j += thread_block_size) {
            int idx = per_thread_shared_mem_size/sizeof(SharedMemElem)*emb;
            if (true or j + lane_id  < sizeof (BitVectorVertexEmbedding)/sizeof (SharedMemElem)) { //TODO: Remove this if by doing padding with VertexEmbedding
              shared_buff[idx + j + lane_id] = embedding_buff[j + lane_id];
            }
          }
        }

        BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*) local_shared_buff;
      #endif
      //embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_LOCAL_MEM)
      //memcpy (&temp[0], &embeddings[i], sizeof (VertexEmbedding));
      memcpy (&temp_buffer[0], &embeddings[i], sizeof(BitVectorVertexEmbedding));
      BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*)&temp_buffer[0];
      //VertexEmbedding* embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_GLOBAL_MEM)
      BitVectorVertexEmbedding* embedding = &embeddings[i];
    #else
      #error "None of USE_EMBEDDING_IN_*_MEM option defined"
    #endif

  #if 1
    typedef uint32_t VertexEmbeddingChange;
    const uint32_t max_changes = 32; //max changes per warp
    //__shared__ int32_t shared_mem [max_changes*THREAD_BLOCK_SIZE/WARP_SIZE+THREAD_BLOCK_SIZE/WARP_SIZE+1];
    __shared__ VertexEmbeddingChange changes[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t changed_thread_ids[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    //For each warp record new embeddings (or changes) using atomic operations
    __shared__ int32_t n_extensions[THREAD_BLOCK_SIZE/WARP_SIZE];
    n_extensions[warp_id] = 0;
    __shared__ uint32_t prev_n_outputs [THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t prev_n_next_steps [THREAD_BLOCK_SIZE/WARP_SIZE];
    
    const uint32_t mask = ~0U;
    for (int u = 0; u < N; u++) {
      int e = csr->get_start_edge_idx(u);
      
      while (true) {
        
        int predicate = 1;
        //n_extensions[warp_id] = 0;

        if (e <= csr->get_end_edge_idx(u)) {
          int v = csr->get_edges () [e];
          if (embedding->test(u) and embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              int prev_n_extensions = atomicAdd (&n_extensions[warp_id], 1);
              changes[warp_id*max_changes+prev_n_extensions] = v;
              changed_thread_ids[warp_id*max_changes+prev_n_extensions] = id;
              //memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (VertexEmbedding));
              //memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (VertexEmbedding));
            }

            extension->reset(v);
          }

          e++;

          if (e > csr->get_end_edge_idx(u)) {
            predicate = 0;
          }
        } else {
          predicate = 0;
        }

        int32_t n_changes = n_extensions[warp_id];

        if (n_changes > max_changes)
          assert (false);

        //assert (__activemask () == ~0U);
        uint32_t orig_prev_n_output = 0;
        uint32_t orig_prev_n_next_step = 0;
        
        if (lane_id == 0) {
          if (n_changes > 0) {
            n_extensions[warp_id] = 0;
            orig_prev_n_output = atomicAdd (n_output, n_changes);
            orig_prev_n_next_step = atomicAdd (n_next_step, n_changes);
            //if (warp_id == 2 && (threadIdx.x == 64 || threadIdx.x == 65) && blockIdx.x == 0) {
            //  printf ("prev_n_next_steps[warp_id]: %d\n", prev_n_next_steps[warp_id]);
            //}
          }
        }
        
        //__syncwarp ();
        
        if (n_changes > 0) {
          
          uint32_t prev_n_output = __shfl_sync (__activemask (), orig_prev_n_output, 0);
          uint32_t prev_n_next_step = __shfl_sync (__activemask (), orig_prev_n_next_step, 0);
          
          for (int i = 0; i < n_changes; i++) {
            
            int changes_idx = warp_id*max_changes + i;
            uint32_t expected_thread_id = changed_thread_ids[changes_idx];
            if (expected_thread_id == id) {
              int v = changes[changes_idx];
              BitVectorVertexEmbedding* extension = embedding;
              extension->set (v);
              
              memcpy (&output[prev_n_output + i], extension, 
                      sizeof(BitVectorVertexEmbedding));
              memcpy (&new_embeddings[prev_n_next_step + i], extension,
                      sizeof (BitVectorVertexEmbedding));
              extension->reset (v);
            }
          }
        }

        if (n_extensions[warp_id] != 0) {
          printf ("n_extensions[warp_id] not zero but is %d\n", n_extensions[warp_id]);
        }
        if (!__any_sync (__activemask (), predicate)) {
          break;
        }
        //n_extensions[warp_id] = 0;
      }
    }
  #else
    for (int u = 0; u < N; u++) {
      if (embedding->test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (BitVectorVertexEmbedding));
              memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (BitVectorVertexEmbedding));
            }
            extension->reset(v);
          }
        }
      }
    }
  #endif

  }

  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}*/


template <size_t size>
__host__ __device__
inline bool is_embedding_canonical (CSR* csr, VectorVertexEmbedding<size>* embedding, int v)
{
  if (embedding->get_vertex (0) > v)
    return false;
  
  if (size <= 2)
    return true;
  
  bool found_neighbor = false;
  for (int j = 0; j < embedding->get_n_vertices (); j++) {
    int v_j = embedding->get_vertex (j);
    if (found_neighbor == false && csr->has_edge (v_j, v)) {
      found_neighbor = true;
    } else if (found_neighbor == true && v_j > v) {
      return false;
    }
  }

  return true;
}

template <size_t embedding_size> 
__global__
void run_single_step_vectorvertex_embedding (void* input, int n_embeddings, CSR* csr,
                      void* output_ptr,
                      int* n_output,
                      void* next_step_1, int* n_next_step_1, volatile BUFFER_STATUS* buff_1_status, //Should be volatile because of the busy wait loop
                      void* next_step_2, int* n_next_step_2, volatile BUFFER_STATUS* buff_2_status, //Should be volatile because of the busy wait loop
                      volatile int* curr_step_storage_id, //Should be volatile because threads coordinate based on this value
                      int only_copy_change)
{
  int id;

#ifdef USE_CSR_IN_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread,
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread,
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#else
#ifdef USE_CONSTANT_MEM
  csr = (CSR*) csr_constant_buff;
#endif

#endif

  VectorVertexEmbedding<embedding_size>* embeddings = (VectorVertexEmbedding<embedding_size>*)input;
  
#ifdef USE_EMBEDDING_IN_LOCAL_MEM
  unsigned char temp_buffer [sizeof(VectorVertexEmbedding<embedding_size+1>)];
#endif
  id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
#ifdef USE_FIXED_THREADS
  if (n_embeddings >= MAX_CUDA_THREADS) {
    int embeddings_per_thread = n_embeddings/MAX_CUDA_THREADS+1;

    start = id*embeddings_per_thread;
    end = (id+1)*embeddings_per_thread < n_embeddings ? (id+1)*embeddings_per_thread : n_embeddings;
  } else {
    if (id >= n_embeddings)
      return;

    start = id;
    end = id+1;
  }
#else
  if (id >= n_embeddings)
      return;

  start = id;
  end = id+1;
#endif

  int q[1000] = {0};

#ifdef USE_EMBEDDING_IN_SHARED_MEM
//TODO: Support VectorVertexEmbedding
  #if 0
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    assert (per_thread_shared_mem_size >= sizeof (VectorVertexEmbedding<embedding_size>));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[per_thread_shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[0];
  #else
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
  #endif
#endif

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int lane_id = threadIdx.x % WARP_SIZE;
  for (int i = start; i < end; i++) {
    #ifdef USE_EMBEDDING_IN_SHARED_MEM
    //TODO: Support VectorVertexEmbedding, size+1
      #ifdef SHARED_MEM_NON_COALESCING
        memcpy (local_shared_buff, &embeddings[i], sizeof(VectorVertexEmbedding<embedding_size>));

        VectorVertexEmbedding<embedding_size>* embedding = (VectorVertexEmbedding<embedding_size>*) local_shared_buff;
      #else
        const int thread_block_size = WARP_SIZE;
        const int last_emb = WARP_SIZE*(warp_id+1);
        if (blockIdx.x*blockDim.x + (warp_id+1)*WARP_SIZE > n_embeddings) {
          assert (false);
          /*thread_block_size = n_embeddings - blockIdx.x*blockDim.x -
                              warp_id*WARP_SIZE;
          last_emb = warp_id*WARP_SIZE + thread_block_size;*/
        }

        for (int emb = WARP_SIZE*warp_id; emb < last_emb; emb++) {
          SharedMemElem* embedding_buff = (SharedMemElem*) &embeddings[emb+blockIdx.x*blockDim.x];

          for (int j = 0; j < sizeof (VectorVertexEmbedding<embedding_size>)/sizeof (SharedMemElem);
               j += thread_block_size) {
            int idx = per_thread_shared_mem_size/sizeof(SharedMemElem)*emb;
            if (true or j + lane_id  < sizeof (VectorVertexEmbedding<embedding_size>)/sizeof (SharedMemElem)) { //TODO: Remove this if by doing padding with VertexEmbedding
              shared_buff[idx + j + lane_id] = embedding_buff[j + lane_id];
            }
          }
        }

        VectorVertexEmbedding<embedding_size>* embedding = (VectorVertexEmbedding<embedding_size>*) local_shared_buff;
      #endif
      //embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_LOCAL_MEM)
      //memcpy (&temp[0], &embeddings[i], sizeof (VertexEmbedding));
      //memcpy (&temp_buffer[0], &embeddings[i], sizeof(VectorVertexEmbedding<embedding_size>));
      VectorVertexEmbedding<embedding_size+1>* embedding = (VectorVertexEmbedding<embedding_size+1>*)&temp_buffer[0];
      embedding->clear ();
      vector_embedding_from_one_less_size (embeddings[i], *embedding);
      //VertexEmbedding* embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_GLOBAL_MEM)
    //TODO: Support VectorVertexEmbedding with size + 1, below is wrong
      VectorVertexEmbedding<embedding_size+1>* embedding = &embeddings[i];
    #else
      #error "None of USE_EMBEDDING_IN_*_MEM option defined"
    #endif

  #if 0
  //TODO: Support VectorVertexEmbedding with size + 1.
    typedef uint32_t VertexEmbeddingChange;
    const uint32_t max_changes = 32; //max changes per warp
    //__shared__ int32_t shared_mem [max_changes*THREAD_BLOCK_SIZE/WARP_SIZE+THREAD_BLOCK_SIZE/WARP_SIZE+1];
    __shared__ VertexEmbeddingChange changes[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t changed_thread_ids[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    //For each warp record new embeddings (or changes) using atomic operations
    __shared__ int32_t n_extensions[THREAD_BLOCK_SIZE/WARP_SIZE];
    n_extensions[warp_id] = 0;
    __shared__ uint32_t prev_n_outputs [THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t prev_n_next_steps [THREAD_BLOCK_SIZE/WARP_SIZE];
    
    const uint32_t mask = ~0U;
    for (int u = 0; u < N; u++) {
      int e = csr->get_start_edge_idx(u);
      
      while (true) {
        
        int predicate = 1;
        //n_extensions[warp_id] = 0;

        if (e <= csr->get_end_edge_idx(u)) {
          int v = csr->get_edges () [e];
          if (embedding->test(u) and embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              int prev_n_extensions = atomicAdd (&n_extensions[warp_id], 1);
              changes[warp_id*max_changes+prev_n_extensions] = v;
              changed_thread_ids[warp_id*max_changes+prev_n_extensions] = id;
              //memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (VertexEmbedding));
              //memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (VertexEmbedding));
            }

            extension->reset(v);
          }

          e++;

          if (e > csr->get_end_edge_idx(u)) {
            predicate = 0;
          }
        } else {
          predicate = 0;
        }

        int32_t n_changes = n_extensions[warp_id];

        if (n_changes > max_changes)
          assert (false);

        //assert (__activemask () == ~0U);
        uint32_t orig_prev_n_output = 0;
        uint32_t orig_prev_n_next_step = 0;
        
        if (lane_id == 0) {
          if (n_changes > 0) {
            n_extensions[warp_id] = 0;
            orig_prev_n_output = atomicAdd (n_output, n_changes);
            orig_prev_n_next_step = atomicAdd (n_next_step, n_changes);
            //if (warp_id == 2 && (threadIdx.x == 64 || threadIdx.x == 65) && blockIdx.x == 0) {
            //  printf ("prev_n_next_steps[warp_id]: %d\n", prev_n_next_steps[warp_id]);
            //}
          }
        }
        
        //__syncwarp ();
        
        if (n_changes > 0) {
          
          uint32_t prev_n_output = __shfl_sync (__activemask (), orig_prev_n_output, 0);
          uint32_t prev_n_next_step = __shfl_sync (__activemask (), orig_prev_n_next_step, 0);
          
          for (int i = 0; i < n_changes; i++) {
            
            int changes_idx = warp_id*max_changes + i;
            uint32_t expected_thread_id = changed_thread_ids[changes_idx];
            if (expected_thread_id == id) {
              int v = changes[changes_idx];
              BitVectorVertexEmbedding* extension = embedding;
              extension->set (v);
              
              memcpy (&output[prev_n_output + i], extension, 
                      sizeof(BitVectorVertexEmbedding));
              memcpy (&new_embeddings[prev_n_next_step + i], extension,
                      sizeof (BitVectorVertexEmbedding));
              extension->reset (v);
            }
          }
        }

        if (n_extensions[warp_id] != 0) {
          printf ("n_extensions[warp_id] not zero but is %d\n", n_extensions[warp_id]);
        }
        if (!__any_sync (__activemask (), predicate)) {
          break;
        }
        //n_extensions[warp_id] = 0;
      }
    }
  #else
    for (int i = 0; i < embedding->get_n_vertices (); i++) {
      int u = embedding->get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        
        int v = csr->get_edges () [e];

        if (is_embedding_canonical<embedding_size+1> (csr, embedding, v) && embedding->has (v) == false) {
          VectorVertexEmbedding<embedding_size+1>* extension = embedding;
          extension->add_unsorted (v);
          
          if (clique_filter_vector (csr, extension)) {
            //VectorVertexEmbedding<embedding_size+1> extension = *embedding;
            //extension.add_last_in_sort_order ();
            //int o = atomicAdd(n_output,1);
            //int n = atomicAdd(n_next_step_1,1);
            
            if (only_copy_change) {
              int o = atomicAdd(n_output, 1);
              int n = atomicAdd(n_next_step_1, 1);
              int* new_embeddings = (int*) next_step_1;
              int* output = (int*) output_ptr;

              new_embeddings[2*n] = id;
              new_embeddings[2*n+1] = v;
              output[2*o] = id;
              output[2*o+1] = v;
            }
            else {
              int storage_id = *curr_step_storage_id;
              const size_t max_n_embeddings = NEW_EMBEDDING_BUFFER_SIZE/sizeof (VectorVertexEmbedding<embedding_size+1>);
              //const int storage_id = 0;
              int n = 0;
              switch (storage_id) {
                case 0: {
                  int o = atomicAdd(n_output, 1);
                  n = atomicAdd(n_next_step_1, 1);
                  //Switch from buff1 to buff2
                  while (n >= max_n_embeddings) {//TODO: change it to do-while 
                    if (*curr_step_storage_id == 0) {
                      n = atomicSub (n_next_step_1, 1); //TODO: can remove that
                      *curr_step_storage_id = 1;
                      *buff_1_status = BUFFER_STATUS::CPU_COPYING;
                      while (*buff_2_status == BUFFER_STATUS::CPU_COPYING) {
                        /*unsigned long i = 0;
                        while (i <= (1UL<<30)) {
                          i++;
                        }*/
                      }
                      *buff_2_status = BUFFER_STATUS::GPU_USING;
                      n = atomicAdd(n_next_step_2, 1);
                    } else {
                      n = atomicSub (n_next_step_2, 1); //TODO: can remove that
                      *curr_step_storage_id = 0;
                      *buff_2_status = BUFFER_STATUS::CPU_COPYING;
                      while (*buff_1_status == BUFFER_STATUS::CPU_COPYING) {
                        /*unsigned long i = 0;
                        while (i <= (1UL<<30)) {
                          i++;
                        }*/
                      }
                      *buff_1_status = BUFFER_STATUS::GPU_USING;
                      n = atomicAdd(n_next_step_1, 1);
                    }
                  }
                  
                  if (*curr_step_storage_id == 1) {
                    //n = atomicAdd (n_next_step_2, 1);
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_2;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  } else {
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_1;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  }
                  break;
                }

                case 1: {
                  int o = atomicAdd(n_output, 1);
                  n = atomicAdd(n_next_step_2, 1);
                  if (n >= max_n_embeddings) {
                    //Switch from buff2 to buff1
                    atomicSub (n_next_step_2, 1); //TODO: can remove that
                    *curr_step_storage_id = 0;
                    *buff_2_status = BUFFER_STATUS::CPU_COPYING;
                    while (*buff_1_status == BUFFER_STATUS::CPU_COPYING) {
                      /*unsigned long i = 0;
                      while (i <= (1UL<<30)) {
                        i++;
                      }*/
                    }
                    *buff_1_status = BUFFER_STATUS::GPU_USING;
                    n = atomicAdd (n_next_step_1, 1);
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_1;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  } else {
                    VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step_2;
                    VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
                    memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                    memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
                  }
                }
              }
            }
            //output[o].add_last_in_sort_order ();
            //new_embeddings[n].add_last_in_sort_order ();
          }
          extension->remove_last ();
        }
      }
    }
  #endif
  }

  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}

void print_embedding (BitVectorVertexEmbedding embedding, std::ostream& os)
{
  os << "[";
  for (int u = 0; u < N; u++) {
    if (embedding.test(u)) {
      os << u << ", ";
    }
  }
  os << "]";
}

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

double_t convertTimeValToDouble (struct timeval _time)
{
  return ((double_t)_time.tv_sec) + ((double_t)_time.tv_usec)/1000000.0f;
}

struct timeval getTimeOfDay ()
{
  struct timeval _time;

  if (gettimeofday (&_time, NULL) == -1) {
    fprintf (stderr, "gettimeofday returned -1\n");
    perror ("");
    abort ();
  }

  return _time;
}

enum EmbeddingType {
  VectorVertex,
  BitVector,
};

int main (int argc, char* argv[])
{
  std::vector<Vertex> vertices;
  int n_edges = 0;

  if (argc < 2) {
    std::cout << "Arguments: graph-file" << std::endl;
    return -1;
  }

  char* graph_file = argv[1];
  FILE* fp = fopen (graph_file, "r+");
  if (fp == nullptr) {
    std::cout << "File '" << graph_file << "' not found" << std::endl;
    return 1;
  }

  while (true) {
    char line[LINE_SIZE];
    char num_str[LINE_SIZE];
    size_t line_size;

    if (fgets (line, LINE_SIZE, fp) == nullptr) {
      break;
    }

    int id, label;
    int bytes_read;

    bytes_read = sscanf (line, "%d %d", &id, &label);
    Vertex vertex (id, label);
    char* _line = line + chars_in_int (id) + chars_in_int (label);
    do {
      int num;

      bytes_read = sscanf (_line, "%d", &num);
      if (bytes_read > 0) {
        vertex.add_edge (num);
        _line += chars_in_int (num);
        n_edges++;
      }

    } while (bytes_read > 0);

    vertex.sort_edges ();

    vertices.push_back (vertex);
  }

  fclose (fp);

  std::cout << "n_edges "<<n_edges <<std::endl;
  std::cout << "vertices " << vertices.size () << std::endl; 

#ifdef PROCESS_EMBEDDINGS_PER_VERTEX
  std::cout << "Sorting " << std::endl;
  std::vector <Vertex> old_vertices = vertices;
  std::vector <int> new_to_old_vertex_ids;
  std::vector <int> old_to_new_vertex_ids;
  for (size_t i = 0; i < vertices.size (); i++) {
    new_to_old_vertex_ids.push_back (vertices[i].get_id ());
    old_to_new_vertex_ids.push_back (vertices[i].get_id ());
  }

  std::sort (vertices.begin (), vertices.end (), Vertex::compare_vertex);
  assert (vertices[0].get_edges ().size () >= vertices[vertices.size () - 1].get_edges ().size ());
  for (size_t i = 0; i < vertices.size (); i++) {
    int old_id = vertices[i].get_id ();
    int new_id = i;
    vertices[i].set_id (new_id);
    new_to_old_vertex_ids[new_id] = old_id;
    old_to_new_vertex_ids[old_id] = new_id;
  }

  for (size_t i = 0; i < vertices.size (); i++) {
    std::vector <int>& edges = vertices[i].get_edges ();
    for (size_t j = 0; j < edges.size (); j++) {
      edges[j] = old_to_new_vertex_ids[edges[j]];
    }
  }
#endif 

  Graph graph (vertices, n_edges);

  CSR* csr = new CSR(N, N_EDGES);
  std::cout << "sizeof(CSR)"<< sizeof(CSR)<<std::endl;
  std::cout <<"sizeof(VertexEmbedding)" << sizeof(BitVectorVertexEmbedding) << std::endl;
  csr_from_graph (csr, graph);
  
#ifdef USE_CONSTANT_MEM
  cudaMemcpyToSymbol (csr_constant_buff, csr, sizeof(CSR));
  //~ CSR* csr_constant = (CSR*) &csr_constant_buff[0];
  //~ csr_constant->n_vertices = csr->get_n_vertices ();
  //~ printf ("csr->get_n_vertices () = %d\n", csr->get_n_vertices ());
  //~ csr_constant->n_edges = csr->get_n_edges ();
  //~ csr_constant->copy_vertices (csr, 0, csr->get_n_vertices ());
  //~ csr_constant->copy_edges (csr, 0, csr->get_n_edges ());
#endif

  std::vector<VectorVertexEmbedding<0>> initial_embeddings = get_initial_embedding_vector (csr);
  std::vector<VectorVertexEmbedding<1>> output_1;
  std::vector<VectorVertexEmbedding<2>> output_2;
  std::vector<VectorVertexEmbedding<3>> output_3;
  std::vector<VectorVertexEmbedding<4>> output_4;
  std::vector<VectorVertexEmbedding<5>> output_5;
  std::vector<VectorVertexEmbedding<6>> output_6;
  std::vector<VectorVertexEmbedding<7>> output_7;
  std::vector<VectorVertexEmbedding<8>> output_8;
  std::vector<std::pair<void*, size_t>> embeddings;
  //filter = clique_filter;
  //process = clique_process;
  size_t new_embeddings_size = 0;
  int iter = 0;
  {
    std::vector<VectorVertexEmbedding<1>> new_embeddings;
    run_single_step_initial_vector (&initial_embeddings[0], 1, csr, 
                                    output_1, new_embeddings);
    new_embeddings_size = new_embeddings.size ();
    embeddings.push_back (std::make_pair (malloc (sizeof (VectorVertexEmbedding<1>)*new_embeddings_size), new_embeddings_size));
    for (int i = 0; i < new_embeddings_size; i++) {
      ((VectorVertexEmbedding<1>*)embeddings[0].first)[i] = new_embeddings[i];
      int v = ((VectorVertexEmbedding<1>*)embeddings[0].first)[i].get_vertex (0);
      assert (v >= 0);
    }
  }

  iter = 1;
  double total_stream_time = 0;
  size_t global_mem_size = 15*1024*1024*1024UL;
#define PINNED_MEMORY
#ifdef PINNED_MEMORY
  char* global_mem_ptr;
  hipError_t malloc_error = hipHostMalloc ((void**)&global_mem_ptr, global_mem_size, hipHostMallocDefault);
  assert (malloc_error == hipSuccess);
#else
  char* global_mem_ptr = new char[global_mem_size];
#endif

  const size_t max_embedding_size_per_iter = (12000000/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
  double_t kernelTotalTime = 0.0;
  for (iter; iter < 8 && new_embeddings_size > 0; iter++) {
    std::cout << "iter " << iter << " embeddings " << new_embeddings_size << std::endl;
    
    size_t remaining_embeddings = new_embeddings_size;
    size_t n_embeddings = new_embeddings_size;
    #ifdef DEBUG
      memset (global_mem_ptr, 0, global_mem_size);
    #endif

    //Copy all embeddings to global memory
    size_t embedding_size = 0;
    size_t new_embedding_size = 0;
    size_t global_mem_iter = 0;
    switch (iter) {
      case 1: {
        embedding_size = sizeof (VectorVertexEmbedding<1>);
        new_embedding_size = sizeof (VectorVertexEmbedding<2>);
        
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<1>*)global_mem_ptr)[global_mem_iter] = ((VectorVertexEmbedding<1>*) iter.first)[i];
            int v = ((VectorVertexEmbedding<1>*)global_mem_ptr)[global_mem_iter].get_vertex (0);
            global_mem_iter++;
            assert (v >= 0);
          }
        }
        break;
      }      
      case 2: {
        embedding_size = sizeof (VectorVertexEmbedding<2>);
        new_embedding_size = sizeof (VectorVertexEmbedding<3>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<2>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<2>*)iter.first)[i];
          }
        }
        break;
      }
      
      case 3: {
        embedding_size = sizeof (VectorVertexEmbedding<3>);
        new_embedding_size = sizeof (VectorVertexEmbedding<4>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<3>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<3>*)iter.first)[i];
          }
        }
        break;
      }
      
      case 4: {
          embedding_size = sizeof (VectorVertexEmbedding<4>);
          new_embedding_size = sizeof (VectorVertexEmbedding<5>);
          for (auto iter: embeddings) {
            for (int i = 0; i < iter.second; i++) {
              ((VectorVertexEmbedding<4>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<4>*)iter.first)[i];
            }
          }
        break;
      }
      case 5: {
        embedding_size = sizeof (VectorVertexEmbedding<5>);
        new_embedding_size = sizeof (VectorVertexEmbedding<6>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<5>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<5>*)iter.first)[i];
          }
        }
        break;
      }
      case 6: {
        embedding_size = sizeof (VectorVertexEmbedding<6>);
        new_embedding_size = sizeof (VectorVertexEmbedding<7>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<6>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<6>*)iter.first)[i];
          }
        }
        break;
      }
      case 7: {
        embedding_size = sizeof (VectorVertexEmbedding<7>);
        new_embedding_size = sizeof (VectorVertexEmbedding<8>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<7>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<7>*)iter.first)[i];
          }
        }
        break;
      }
      case 8: {
        embedding_size = sizeof (VectorVertexEmbedding<8>);
        new_embedding_size = sizeof (VectorVertexEmbedding<9>);
        for (auto iter: embeddings) {
          for (int i = 0; i < iter.second; i++) {
            ((VectorVertexEmbedding<8>*)global_mem_ptr)[global_mem_iter++] = ((VectorVertexEmbedding<8>*)iter.first)[i];
          }
        }
        break;
      }
    }

    //delete embeddings too because there is a memory leak?
    if (iter > 1) {
      for (auto iter: embeddings) {
        free(iter.first);
      }
    }

    embeddings.clear ();
    std::cout << "Copying to global_mem_ptr done. global mem used " << global_mem_iter*embedding_size << std::endl;
    
    void* embeddings_ptr = global_mem_ptr;

    size_t n_next_step_embeddings = 0;
    n_embeddings = 0;

    void* orig_new_embeddings_ptr = ((char*)global_mem_ptr) + (global_mem_iter)*(new_embedding_size); //Size of next embedding will be one more
    //size_t max_embeddings = 40000000; //There is something with this value which makes it perform better, may be alignment?
    size_t max_embeddings = NEW_EMBEDDING_BUFFER_SIZE/(new_embedding_size);
    printf ("new_embedding_size %ld\n", new_embedding_size);
    void* orig_output_ptr = (char*)orig_new_embeddings_ptr + (max_embeddings)*(new_embedding_size);

    hipError_t error;
    double stream_time_1 = convertTimeValToDouble (getTimeOfDay ());
    
    while (remaining_embeddings != 0) {      
      n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
      std::cout << "iter " << iter << " n_embeddings " << new_embeddings_size << " remaining_embeddings " << remaining_embeddings << std::endl;
      embeddings_ptr = ((char*)global_mem_ptr) + embedding_size*(new_embeddings_size - remaining_embeddings);
      //printf ("embeddings_ptr %x\n", embeddings_ptr);
      n_embeddings = std::min (remaining_embeddings, max_embedding_size_per_iter);

      remaining_embeddings -= n_embeddings;
      //n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
      
      const int N_STREAMS = 1;
      //assert (max_embeddings/N_STREAMS >= 30000000);
      int only_copy_change = 0;
      assert (only_copy_change == 0); //TODO: Streams with only copy change
      void* new_embeddings_ptr[N_STREAMS];
      assert (max_embeddings%N_STREAMS == 0);
      for (int i = 0; i < N_STREAMS; i++) {
        new_embeddings_ptr[i] = (char*)orig_new_embeddings_ptr + i*new_embedding_size*max_embeddings/N_STREAMS;
      }

      void* output_ptr[N_STREAMS];
      for (int i = 0; i < N_STREAMS; i++) {
        output_ptr[i] = (char*)orig_output_ptr + i*new_embedding_size*max_embeddings/N_STREAMS;
      }
      int n_new_embeddings_1[N_STREAMS] = {0};
      int n_new_embeddings_2[N_STREAMS] = {0};
      int n_output[N_STREAMS] = {0};
      int n_output_1[N_STREAMS] = {0};
      int curr_new_embeddings_idx[N_STREAMS] = {0};
      char* device_embeddings[N_STREAMS];
      char *device_new_embeddings_1[N_STREAMS];
      char *device_new_embeddings_2[N_STREAMS];
      BUFFER_STATUS *device_new_embeddings_1_status[N_STREAMS];
      BUFFER_STATUS *device_new_embeddings_2_status[N_STREAMS];
      int* device_n_embeddings_1[N_STREAMS];
      int* device_n_embeddings_2[N_STREAMS];
      int* device_curr_new_embeddings_idx[N_STREAMS];
      char *device_outputs[N_STREAMS];
      int* device_n_outputs[N_STREAMS];
      int* device_n_outputs_1[N_STREAMS];
      BUFFER_STATUS new_embeddings_1_status[N_STREAMS];
      BUFFER_STATUS new_embeddings_2_status[N_STREAMS];
      for (int i = 0; i < N_STREAMS; i++) {
        new_embeddings_1_status[i] = BUFFER_STATUS::FREE;
        new_embeddings_2_status[i] = BUFFER_STATUS::FREE;
      }
      
      CSR* device_csr[N_STREAMS];
      
      assert (N_STREAMS >= 1);

      hipStream_t streams[N_STREAMS];
      for (int i = 0; i < N_STREAMS; i++) {
        hipError_t err = hipStreamCreate (&streams[i]);
        assert (err == hipSuccess);
      }

      const bool stream_synchronize = true;
      size_t per_stream_embeddings_done = 0;
      size_t per_stream_n_embeddings = n_embeddings/N_STREAMS;
      assert (N_STREAMS == 1);
      for (int i = 0; i < N_STREAMS; i++) {
        const bool unified_mem = false;
        if (false && iter == 2) {
          if (i == 0) {
            per_stream_n_embeddings = (n_embeddings*9)/10;
          } else {
            per_stream_n_embeddings = (n_embeddings*1)/10;
          }
        }
        if (unified_mem == true) {
          //cudaMallocManaged (embeddings_ptr, n_embeddings*embedding_size);
          //device_embeddings = (char*)embeddings_ptr;
          assert(false);
        } else {
          hipMalloc (&device_embeddings[i], per_stream_n_embeddings*embedding_size);
          hipMemcpyAsync (device_embeddings[i], (char*)embeddings_ptr + per_stream_embeddings_done*embedding_size,
                           per_stream_n_embeddings*embedding_size, hipMemcpyHostToDevice, streams[i]);
        }

        hipMalloc (&device_new_embeddings_1[i], max_embeddings/N_STREAMS*(new_embedding_size));
        hipMalloc (&device_new_embeddings_2[i], max_embeddings/N_STREAMS*(new_embedding_size));
        hipMalloc (&device_n_embeddings_1[i], sizeof(int));
        hipMalloc (&device_outputs[i], max_embeddings/N_STREAMS*(new_embedding_size));
        hipMalloc (&device_n_embeddings_1[i], sizeof (int));
        hipMalloc (&device_n_embeddings_2[i], sizeof (int));
        hipMalloc (&device_n_outputs[i], sizeof (int));
        hipMalloc (&device_n_outputs_1[i], sizeof (int));
        hipMalloc (&device_csr[i], sizeof(CSR)); //TODO: Remove copying CSR graph again and again
        hipMalloc (&device_curr_new_embeddings_idx[i], sizeof (int));
        hipMalloc (&device_new_embeddings_1_status[i], sizeof (BUFFER_STATUS));
        hipMalloc (&device_new_embeddings_2_status[i], sizeof (BUFFER_STATUS));
        hipMemcpyAsync (device_new_embeddings_1_status[i], &new_embeddings_1_status[i], 
                         sizeof (new_embeddings_1_status[i]), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync (device_new_embeddings_2_status[i], &new_embeddings_2_status[i], 
                         sizeof (new_embeddings_2_status[i]), hipMemcpyHostToDevice, streams[i]);
        {
          hipError_t error = hipGetLastError ();
          if (error != hipSuccess) {
            const char* error_string = hipGetErrorString (error);
            std::cout << "Cuda host to device copy error " << error_string << std::endl;
          } else {
            std::cout << "Cuda host to device copy success " << std::endl;
          }
        }
        hipMemcpyAsync (device_n_embeddings_1[i], &n_new_embeddings_1[i],
                         sizeof (n_new_embeddings_1[i]), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync (device_n_outputs[i], &n_output[i], sizeof (n_output[i]),
                         hipMemcpyHostToDevice, streams[i]);
        
        {
          hipError_t error = hipGetLastError ();
          if (error != hipSuccess) {
            const char* error_string = hipGetErrorString (error);
            std::cout << "Cuda host to device copy error " << error_string << std::endl;
          } else {
            std::cout << "Cuda host to device copy success " << std::endl;
          }
        }                         
        
        assert (hipMemcpyAsync (device_n_embeddings_2[i], &n_new_embeddings_2[i],
                         sizeof (n_new_embeddings_2[i]), hipMemcpyHostToDevice, streams[i]) == hipSuccess);
        hipMemcpyAsync (device_n_outputs_1[i], &n_output_1[i], sizeof (n_output_1[i]),
                         hipMemcpyHostToDevice, streams[i]);
        
        {
          hipError_t error = hipGetLastError ();
          if (error != hipSuccess) {
            const char* error_string = hipGetErrorString (error);
            std::cout << "Cuda host to device copy error " << error_string << std::endl;
          } else {
            std::cout << "Cuda host to device copy success " << std::endl;
          }
        }
        hipMemcpyAsync (device_csr[i], csr, sizeof (CSR), hipMemcpyHostToDevice, streams[i]);
        
        hipMemcpyAsync (device_curr_new_embeddings_idx[i], &curr_new_embeddings_idx[i], 
                         sizeof (curr_new_embeddings_idx[i]), hipMemcpyHostToDevice,  
                         streams[i]);
        
        hipError_t error = hipGetLastError ();
        if (error != hipSuccess) {
          const char* error_string = hipGetErrorString (error);
          std::cout << "Cuda host to device copy error " << error_string << std::endl;
        } else {
          std::cout << "Cuda host to device copy success " << std::endl;
        }

        per_stream_embeddings_done += per_stream_n_embeddings;
      }

      for (int i = 0; i < N_STREAMS; i++) {
        double t1 = convertTimeValToDouble (getTimeOfDay ());
        if (stream_synchronize)
          hipStreamSynchronize (streams[i]);
        if (false && iter == 2) {
          if (i == 0) {
            per_stream_n_embeddings = (n_embeddings*9)/10;
          } else {
            per_stream_n_embeddings = (n_embeddings*1)/10;
          }
        }
        std::cout << "starting kernel with n_embeddings: " << per_stream_n_embeddings ;
    #ifdef USE_FIXED_THREADS
        //std::cout << " threads: " << MAX_CUDA_THREADS/THREAD_BLOCK_SIZE << std::endl;
        int thread_blocks = MAX_CUDA_THREADS/THREAD_BLOCK_SIZE;
    #else
        int thread_blocks = (per_stream_n_embeddings%THREAD_BLOCK_SIZE != 0) ? (per_stream_n_embeddings/THREAD_BLOCK_SIZE+1) : per_stream_n_embeddings/THREAD_BLOCK_SIZE;
    #endif
        std::cout << " threads: " << thread_blocks << std::endl;
        
        switch (iter) {
          case 1: {
            run_single_step_vectorvertex_embedding<1><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i], 
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
          case 2: {
            run_single_step_vectorvertex_embedding<2><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
          case 3: {
            run_single_step_vectorvertex_embedding<3><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
          case 4: {
            run_single_step_vectorvertex_embedding<4><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
          case 5: {
            run_single_step_vectorvertex_embedding<5><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
          case 6: {
            run_single_step_vectorvertex_embedding<6><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
          case 7: {
            run_single_step_vectorvertex_embedding<7><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
          case 8: {
            run_single_step_vectorvertex_embedding<8><<<thread_blocks, THREAD_BLOCK_SIZE, 0, streams[i]>>> (device_embeddings[i], per_stream_n_embeddings, device_csr[i],
                                  device_outputs[i], device_n_outputs[i],
                                  device_new_embeddings_1[i], device_n_embeddings_1[i], device_new_embeddings_1_status[i],
                                  device_new_embeddings_2[i], device_n_embeddings_2[i], device_new_embeddings_2_status[i],
                                  device_curr_new_embeddings_idx[i], only_copy_change);
            break;
          }
        }
        
        //cudaDeviceSynchronize ();
        if (iter >= 2 && ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING) {
          int curr_step_storage_id = 0;
          std::cout << "copying to n_outputs_1" << std::endl; 
          hipStream_t t;
          int prev_curr_step_storage_id = 0;
          hipStreamCreate (&t);

          while (hipStreamQuery (streams[i]) == hipErrorNotReady) {
            do {
              assert (hipMemcpyAsync (&curr_step_storage_id, device_curr_new_embeddings_idx[i], sizeof (curr_step_storage_id), hipMemcpyDeviceToHost, t) == hipSuccess);
              hipStreamSynchronize (t);
              usleep (10000);
            } while (prev_curr_step_storage_id == curr_step_storage_id &&
                     hipStreamQuery (streams[i]) == hipErrorNotReady);
            
            std::cout << "prev_curr_step_storage_id = " << prev_curr_step_storage_id <<
                         " curr_step_storage_id = " << curr_step_storage_id << 
                         " hipStreamQuery (streams[i]) == hipErrorNotReady " << 
                         (hipStreamQuery (streams[i]) == hipErrorNotReady) << std::endl;
            
            //copy
            switch (prev_curr_step_storage_id) {
              case 0: {
                //new_embeddings_1_status[i] = BUFFER_STATUS::CPU_COPYING;
                //assert (cudaMemcpyAsync (&device_new_embeddings_1_status[i], &new_embeddings_1_status[i], 
                //        sizeof (new_embeddings_1_status[i]), cudaMemcpyHostToDevice, t) == 0);
                int p = 0;
                assert (hipMemcpyAsync (&p, device_n_embeddings_1[i], sizeof (p), hipMemcpyDeviceToHost, t) == hipSuccess);
                std::cout << "Copying " << p << " new embeddings each of size " << new_embedding_size << std::endl;
                n_new_embeddings_1[i] += p;
                if (p > max_embeddings/N_STREAMS) {
                  std::cout << "SYNCHRONIZATION ISSUE: device_n_embeddings_1[i] " << p << " > per_stream_n_embeddings " << max_embeddings/N_STREAMS << std::endl;
                } 
                if (true) {
                  hipError_t err = hipMemcpyAsync ((char*)new_embeddings_ptr[i] + (n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size, device_new_embeddings_1[i], max_embeddings/N_STREAMS*new_embedding_size, hipMemcpyDeviceToHost, t);
                  if (err != hipSuccess) {
                    std::cout << hipGetErrorString (err) << std::endl;
                    assert (false);
                  }
                }
                p = 0;
                assert (hipMemcpyAsync (device_n_embeddings_1[i], &p, sizeof (p), hipMemcpyHostToDevice, t) == hipSuccess);
                new_embeddings_1_status[i] = BUFFER_STATUS::FREE;
                hipError_t e = hipMemcpyAsync (device_new_embeddings_1_status[i], &new_embeddings_1_status[i], 
                                                 sizeof (new_embeddings_1_status[i]), hipMemcpyHostToDevice, t);
                //std::cout << cudaGetErrorString (e) << std::endl;
                assert (e == hipSuccess);
                break;
              }

              case 1: {
                int p = 0;
                //new_embeddings_2_status[i] = BUFFER_STATUS::CPU_COPYING;
                //assert (cudaMemcpyAsync (&device_new_embeddings_2_status[i], &new_embeddings_2_status[i], 
                //        sizeof (new_embeddings_2_status[i]), cudaMemcpyHostToDevice, t) == 0);
                assert (hipMemcpyAsync (&p, device_n_embeddings_2[i], sizeof (p), hipMemcpyDeviceToHost, t) == hipSuccess);
                std::cout << "Copying " << p << " new embeddings each of size " << new_embedding_size << std::endl;
                n_new_embeddings_2[i] += p;
                if (p > max_embeddings/N_STREAMS) {
                  std::cout << "SYNCHRONIZATION ISSUE: device_n_embeddings_1[i] " << p << " > per_stream_n_embeddings " << max_embeddings/N_STREAMS << std::endl;
                } 
                if (true) {
                  hipError_t err = hipMemcpyAsync ((char*)new_embeddings_ptr[i]+(n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size, device_new_embeddings_2[i], max_embeddings/N_STREAMS*new_embedding_size, hipMemcpyDeviceToHost, t);
                  if (err != hipSuccess) {
                    std::cout << hipGetErrorString (err) << std::endl;
                    assert (false);
                  }
                }
                p = 0;
                assert (hipMemcpyAsync (device_n_embeddings_2[i], &p, sizeof (p), hipMemcpyHostToDevice, t) == hipSuccess);
                new_embeddings_2_status[i] = BUFFER_STATUS::FREE;
                assert (hipMemcpyAsync (device_new_embeddings_2_status[i], &new_embeddings_2_status[i], 
                        sizeof (new_embeddings_2_status[i]), hipMemcpyHostToDevice, t) == 0);
                break;
              }
            }
            prev_curr_step_storage_id = curr_step_storage_id;
            hipStreamSynchronize (t);
            std::cout << "Copying done " << std::endl;
            //cudaStreamQuery (streams[i]);
          }

          hipStreamDestroy (t);
        }

        if (stream_synchronize)
          hipStreamSynchronize (streams[i]);

      double t2 = convertTimeValToDouble (getTimeOfDay ());

      std::cout << "Execution time " << (t2-t1) << " secs" << std::endl;
      kernelTotalTime += (t2-t1);

        error = hipGetLastError ();
        if (error != hipSuccess) {
          const char* error_string = hipGetErrorString (error);
          std::cout << "Cuda kernel error " << error_string << std::endl;
        } else {
          std::cout << "Cuda success " << std::endl;
        }
      }

      for (int i = 0; i < N_STREAMS; i++) 
      {
        //cudaStreamSynchronize (streams[i]);

        if (only_copy_change) {
          assert (false);
          //TODO: Change this to make an array of such ptrs
          hipMemcpyAsync (new_embeddings_ptr, device_new_embeddings_1[i], n_new_embeddings_1[i]*2*sizeof(int), hipMemcpyDeviceToHost, streams[i]);
          hipMemcpyAsync (output_ptr, device_outputs[i], n_output[i]*2*sizeof(int), hipMemcpyDeviceToHost, streams[i]);
        }
        else {
          std::cout << "n_output[" << i << "] = " << n_output[i] << std::endl;
          int _n_new = 0;
          assert (hipMemcpyAsync (&_n_new, device_n_embeddings_1[i], sizeof(0), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
          
          if (_n_new > 0 ) {
            assert (hipMemcpyAsync ((char*)new_embeddings_ptr[i] + (n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size,
                                     device_new_embeddings_1[i], _n_new*(new_embedding_size), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
          }
          n_new_embeddings_1[i] += _n_new;
          assert (hipMemcpyAsync (&_n_new, device_n_embeddings_2[i], sizeof(0), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
          
          if (_n_new > 0 ) {
            assert (hipMemcpyAsync ((char*)new_embeddings_ptr[i] + (n_new_embeddings_1[i] + n_new_embeddings_2[i])*new_embedding_size,
                                     device_new_embeddings_2[i], _n_new*(new_embedding_size), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
          }
          n_new_embeddings_2[i] += _n_new;
          assert (hipMemcpyAsync (&n_output[i], device_n_outputs[i], sizeof(0), hipMemcpyDeviceToHost, streams[i]) == hipSuccess);
          
          //assert (cudaMemcpyAsync (output_ptr[i], device_outputs[i], n_output[i]*(new_embedding_size), cudaMemcpyDeviceToHost, streams[i])== cudaSuccess);
          n_output[i] = 0;
        }
        //cudaMemcpyAsync (&n_new_embeddings_1[i], device_n_embeddings_1[i], sizeof(0), cudaMemcpyDeviceToHost, streams[i]);
        //cudaMemcpyAsync (&n_output_1[i], device_n_outputs_1[i], sizeof(0), cudaMemcpyDeviceToHost, streams[i]);

        error = hipGetLastError ();
        if (error != hipSuccess) {
          const char* error_string = hipGetErrorString (error);
          std::cout << "Cuda device to host copy error " << error_string << std::endl;
        } else {
          std::cout << "Cuda device to host copy success " << std::endl;
        }

        std::cout << "Stream " << i << std::endl;
        std::cout << "n_new_embeddings "<<n_new_embeddings_1[i] + n_new_embeddings_2[i]<<std::endl;
        std::cout << "n_new_embeddings_1 "<<n_new_embeddings_1[i];
        std::cout << " n_output "<<n_output[i];
        std::cout << " n_output_1 "<<n_output_1[i]<<std::endl;
      }
      
      for (int i = 0; i < N_STREAMS; i++) {
        hipStreamSynchronize (streams[i]);
        hipError_t err = hipStreamDestroy (streams[i]);
        assert (err == hipSuccess);
      }

      double stream_time_2 = convertTimeValToDouble (getTimeOfDay ());

      total_stream_time += (stream_time_2-stream_time_1);

      //TODO: wait for all kernels and data transfers to finish
      for (int i = 0; i < N_STREAMS; i++) {
        n_next_step_embeddings += n_new_embeddings_1[i]+ n_new_embeddings_2[i];
      }
      switch (iter) {
        case 1: {
          VectorVertexEmbedding<2>* new_embeddings = (VectorVertexEmbedding<2>*)malloc (sizeof (VectorVertexEmbedding<2>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<2> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<1>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [j] = embedding;
                j++;
              }
              else {
                VectorVertexEmbedding<2> embedding = ((VectorVertexEmbedding<2>*)(new_embeddings_ptr[stream]))[i];
                new_embeddings [j] = embedding;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
                j++;
              }
            }
          }
          
          assert (j == n_next_step_embeddings);
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<2> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<1>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_2.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_2.push_back (((VectorVertexEmbedding<2>*)output_ptr[stream])[i]);
              }
            }
          }
          
          break;
        }
        
        case 2: {
          VectorVertexEmbedding<3>* new_embeddings = (VectorVertexEmbedding<3>*)malloc (sizeof (VectorVertexEmbedding<3>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<3> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<2>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                assert (false);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<3> embedding = ((VectorVertexEmbedding<3>*)(new_embeddings_ptr[stream]))[i];
                new_embeddings [j] = embedding;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %ld vertices\n", embedding.get_n_vertices ());
                }
                #endif
                j++;
              }
            }
          }

          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<3> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<2>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_3.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_3.push_back (((VectorVertexEmbedding<3>*)output_ptr[stream])[i]);
              }
            }
          }

          break;
        }
        
        case 3: {
          VectorVertexEmbedding<4>* new_embeddings = (VectorVertexEmbedding<4>*)malloc (sizeof (VectorVertexEmbedding<4>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<4> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<3>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<4> embedding = ((VectorVertexEmbedding<4>*)(new_embeddings_ptr[stream]))[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));

          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<4> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<3>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_4.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_4.push_back (((VectorVertexEmbedding<4>*)output_ptr[stream])[i]);
              }
            }
          }
          break;
        }
        
        case 4: {
          VectorVertexEmbedding<5>* new_embeddings = (VectorVertexEmbedding<5>*)malloc (sizeof (VectorVertexEmbedding<5>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<5> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<4>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<5> embedding = ((VectorVertexEmbedding<5>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<5> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<4>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_5.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_5.push_back (((VectorVertexEmbedding<5>*)output_ptr[stream])[i]);
              }
            }
          }

          break;
        }
        
        case 5: {
          VectorVertexEmbedding<6>* new_embeddings = (VectorVertexEmbedding<6>*)malloc (sizeof (VectorVertexEmbedding<6>)*n_next_step_embeddings);
          
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<6> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<5>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<6> embedding = ((VectorVertexEmbedding<6>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<6> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<5>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_6.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_6.push_back (((VectorVertexEmbedding<6>*)output_ptr[stream])[i]);
              }
            }
          }
          break;
        }
        
        case 6: {
          VectorVertexEmbedding<7>* new_embeddings = (VectorVertexEmbedding<7>*)malloc (sizeof (VectorVertexEmbedding<7>)*n_next_step_embeddings);
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<7> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<6>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<7> embedding = ((VectorVertexEmbedding<7>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<7> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<6>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_7.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_7.push_back (((VectorVertexEmbedding<7>*)output_ptr[stream])[i]);
              }
            }
          }
          break;
        }
        
        case 7: {
          VectorVertexEmbedding<8>* new_embeddings = (VectorVertexEmbedding<8>*)malloc (sizeof(VectorVertexEmbedding<8>)*n_next_step_embeddings);
          size_t j = 0;
          for (int stream = 0; stream < N_STREAMS; stream++) {
            for (int i = 0; i < n_new_embeddings_1[stream]; i++) {
              if (only_copy_change) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<8> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<7>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                new_embeddings [i] = embedding;
              }
              else {
                VectorVertexEmbedding<8> embedding = ((VectorVertexEmbedding<8>*)new_embeddings_ptr[stream])[i];
                new_embeddings [j] = embedding;
                j++;
                #ifdef DEBUG
                if (embedding.get_n_vertices () != (iter + 1)) {
                  printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
                }
                #endif
              }
            }
          }
          
          embeddings.push_back (std::make_pair (&new_embeddings[0], n_next_step_embeddings));
          for (int stream = 0; stream < N_STREAMS; stream++) {
            if (only_copy_change) {
              for (int i = 0; i < n_output[stream]; i++) {
                int id = ((int*)new_embeddings_ptr)[2*i];
                int v = ((int*)new_embeddings_ptr)[2*i+1];
                VectorVertexEmbedding<8> embedding;
                vector_embedding_from_one_less_size (((VectorVertexEmbedding<7>*)embeddings_ptr)[id], embedding);
                embedding.add (v);
                output_8.push_back (embedding);
              }
            } else {
              for (int i = 0; i < n_output[stream]; i++) {
                output_8.push_back (((VectorVertexEmbedding<8>*)output_ptr[stream])[i]);
              }
            }
          }

          break;
        }
      }
      
      //embeddings = new_embeddings;

      for (int i = 0; i < N_STREAMS; i++) {
        hipFree (device_embeddings[i]);
        hipFree (device_new_embeddings_1[i]);
        hipFree (device_n_embeddings_1[i]);
        hipFree (device_outputs[i]);
        hipFree (device_n_outputs[i]);
        hipFree (device_csr[i]);
      }
    }

    new_embeddings_size = n_next_step_embeddings;
    
  }

#ifdef PINNED_MEMORY
  hipFree (global_mem_ptr);
#else
  delete[] global_mem_ptr;
#endif
  std::cout << "Number of embeddings found "<< (output_1.size () + output_2.size () + output_3.size () + output_4.size () + output_5.size () + output_6.size () + output_7.size () + output_8.size ()) << std::endl;
  std::cout << "Time spent in GPU kernel execution " << kernelTotalTime << std::endl;
  std::cout << "Time spent in Streams " << total_stream_time << std::endl;
  
  /* For BitVectorVertexEmbedding
   for (iter; iter < 10 && embeddings.size () > 0; iter++) {
    std::cout << "iter " << iter << " embeddings " << embeddings.size () << std::endl;
    size_t global_mem_size = 3*1024*1024*1024UL;
    char* global_mem_ptr = new char[global_mem_size];
  #ifdef DEBUG
    memset (global_mem_ptr, 0, global_mem_size);
  #endif
    int n_embeddings = embeddings.size ();
    //n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
    std::cout << "iter " << iter << " n_embeddings " << n_embeddings << std::endl;
  
    for (int i = 0; i < n_embeddings; i++) {
      ((BitVectorVertexEmbedding*)global_mem_ptr)[i] = embeddings[i];
    }
    void* embeddings_ptr = global_mem_ptr;

    int n_new_embeddings = 0;
    int n_new_embeddings_1 = 0;
    void* new_embeddings_ptr = (char*)embeddings_ptr + (n_embeddings)*sizeof(BitVectorVertexEmbedding);
    int max_embeddings = 1000000;
    void* output_ptr = (char*)new_embeddings_ptr + (max_embeddings)*sizeof(BitVectorVertexEmbedding);
    int n_output = 0;
    int n_output_1 = 0;
    char* device_embeddings;
    char *device_new_embeddings;
    int* device_n_embeddings;
    int* device_n_embeddings_1;
    char *device_outputs;
    int* device_n_outputs;
    int* device_n_outputs_1;
    CSR* device_csr;

    cudaMalloc (&device_embeddings, n_embeddings*sizeof(BitVectorVertexEmbedding));
    cudaMemcpy (device_embeddings, embeddings_ptr,
                n_embeddings*sizeof(BitVectorVertexEmbedding),
                cudaMemcpyHostToDevice);
    cudaMalloc (&device_new_embeddings, max_embeddings*sizeof (BitVectorVertexEmbedding));
    cudaMalloc (&device_outputs, max_embeddings*sizeof (BitVectorVertexEmbedding));
    cudaMalloc (&device_n_embeddings, sizeof (0));
    cudaMalloc (&device_n_embeddings_1, sizeof (0));
    cudaMalloc (&device_n_outputs, sizeof (0));
    cudaMalloc (&device_n_outputs_1, sizeof (0));
    cudaMalloc (&device_csr, sizeof(CSR));

    cudaMemcpy (device_n_embeddings, &n_new_embeddings,
                sizeof (n_new_embeddings), cudaMemcpyHostToDevice);
    cudaMemcpy (device_n_outputs, &n_output, sizeof (n_output),
                cudaMemcpyHostToDevice);

    cudaMemcpy (device_n_embeddings_1, &n_new_embeddings_1,
                sizeof (n_new_embeddings_1), cudaMemcpyHostToDevice);
    cudaMemcpy (device_n_outputs_1, &n_output_1, sizeof (n_output_1),
                cudaMemcpyHostToDevice);

    cudaMemcpy (device_csr, csr, sizeof (CSR), cudaMemcpyHostToDevice);

    std::cout << "starting kernel with n_embeddings: " << n_embeddings;

    double t1 = convertTimeValToDouble (getTimeOfDay ());
#ifdef USE_FIXED_THREADS
    std::cout << " threads: " << MAX_CUDA_THREADS/THREAD_BLOCK_SIZE << std::endl;
      run_single_step_bitvector_embedding<<<MAX_CUDA_THREADS/THREAD_BLOCK_SIZE,THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
#else
    int thread_blocks = (n_embeddings%THREAD_BLOCK_SIZE != 0) ? (n_embeddings/THREAD_BLOCK_SIZE+1) : n_embeddings/THREAD_BLOCK_SIZE;
    std::cout << " threads: " << n_embeddings/THREAD_BLOCK_SIZE << std::endl;
    run_single_step_bitvector_embedding<<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
#endif

    cudaDeviceSynchronize ();

    double t2 = convertTimeValToDouble (getTimeOfDay ());

    std::cout << "Execution time " << (t2-t1) << " secs" << std::endl;
    kernelTotalTime += (t2-t1);

    cudaError_t error = cudaGetLastError ();
    if (error != cudaSuccess) {
      const char* error_string = cudaGetErrorString (error);
      std::cout << error_string << std::endl;
    } else {
      std::cout << "Cuda success " << std::endl;
    }

    cudaMemcpy (new_embeddings_ptr, device_new_embeddings, max_embeddings*sizeof(BitVectorVertexEmbedding), cudaMemcpyDeviceToHost);
    cudaMemcpy (output_ptr, device_outputs, max_embeddings*sizeof(BitVectorVertexEmbedding), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_new_embeddings, device_n_embeddings, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_output, device_n_outputs, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_new_embeddings_1, device_n_embeddings_1, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_output_1, device_n_outputs_1, sizeof(0), cudaMemcpyDeviceToHost);

    std::cout << "n_new_embeddings "<<n_new_embeddings<<std::endl;
    std::cout << "n_new_embeddings_1 "<<n_new_embeddings_1;
    std::cout << " n_output "<<n_output;
    std::cout << " n_output_1 "<<n_output_1<<std::endl;
    std::vector<BitVectorVertexEmbedding> new_embeddings;
  
    for (int i = 0; i < n_new_embeddings; i++) {
      BitVectorVertexEmbedding embedding = ((BitVectorVertexEmbedding*)new_embeddings_ptr)[i];
      new_embeddings.push_back (embedding);
    #ifdef DEBUG
      if (embedding.get_n_vertices () != (iter + 1)) {
        printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
      }
    #endif
    }
    for (int i = 0; i < n_output; i++) {
      output.push_back (((BitVectorVertexEmbedding*)output_ptr)[i]);
    }
    embeddings = new_embeddings;

    cudaFree (device_embeddings);
    cudaFree (device_new_embeddings);
    cudaFree (device_n_embeddings);
    cudaFree (device_outputs);
    cudaFree (device_n_outputs);
    cudaFree (device_csr);
    delete[] global_mem_ptr;
    #endif
  } 
   */
}

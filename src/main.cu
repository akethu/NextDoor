
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>

#include <string.h>
#include <assert.h>

#define LINE_SIZE 1024*1024
#define MAX_CUDA_THREADS 65536
#define THREAD_BLOCK_SIZE 1024

const int N = 3312;
const int N_EDGES = 9074;
class Vertex 
{
private:
  int id;
  int label;
  std::vector <int> edges;
  
public:
  Vertex (int _id, int _label) : label(_label), id (_id)
  {
  }
  
  int get_id () {return id;}
  int get_label () {return label;}
  void add_edge (int vertexID) {edges.push_back (vertexID);}
  std::vector <int>& get_edges () {return edges;}
  void print (std::ostream& os) 
  {
    os << id << " " << label << " ";
    for (auto edge : edges) {
      os << edge << " ";
    }
    
    os << std::endl;
  }
};

int chars_in_int (int num)
{
  if (num == 0) return sizeof(char);
  return (int)((ceil(log10(num))+1)*sizeof(char));
}

class Graph 
{
private:
  std::vector<Vertex> vertices;
  int n_edges;

public:
  Graph (std::vector<Vertex> _vertices, int _n_edges) :
    vertices (_vertices), n_edges(_n_edges)
  {}
  
  const std::vector<Vertex>& get_vertices () {return vertices;}
  int get_n_edges () {return n_edges;}
};

class CSR 
{
public:
  struct Vertex
  {
    short int id;
    short int label;
    short int start_edge_id;
    short int end_edge_id;
    __host__ __device__
    Vertex ()
    {
      id = -1;
      label = -1;
      start_edge_id = -1;
      end_edge_id = -1;
    }
    
    void set_from_graph_vertex (::Vertex& vertex) 
    {
      id = vertex.get_id ();
      label = vertex.get_label ();
    }
    
    void set_start_edge_id (short int start) {start_edge_id = start;}
    void set_end_edge_id (short int end) {end_edge_id = end;}
  };
  
  typedef short int Edge;
  
public:
  CSR::Vertex vertices[N];
  CSR::Edge edges[N_EDGES];
  int n_vertices;
  int n_edges;
  
public:
  CSR (int _n_vertices, int _n_edges) 
  {
    n_vertices = _n_vertices;
    n_edges = _n_edges;
  }
  
  __host__ __device__
  CSR ()
  {
    n_vertices = N;
    n_edges = N_EDGES;
  }
  
  void print (std::ostream& os)
  {
    for (int i = 0; i < n_vertices; i++) {
      os << vertices[i].id << " " << vertices[i].label << " ";
      for (int edge_iter = vertices[i].start_edge_id; 
           edge_iter < vertices[i].end_edge_id; edge_iter++) {
        os << edges[edge_iter] << " ";
      }
      os << std::endl;
    }
  }
  
  __host__ __device__
  int get_start_edge_idx (int vertex_id)
  {
    assert (vertex_id < n_vertices && 0 <= vertex_id);
    return vertices[vertex_id].start_edge_id;
  }
  
  __host__ __device__
  int get_end_edge_idx (int vertex_id)
  {
    assert (vertex_id < n_vertices && 0 <= vertex_id);
    return vertices[vertex_id].end_edge_id;
  }
  
  __host__ __device__
  bool has_edge (int u, int v)
  {
    for (int e = get_start_edge_idx (u); e <= get_end_edge_idx (u); e++) {
      if (edges[e] == v) {
        return true;
      }
    }
    
    return false;
  }
  
  __host__ __device__
  const CSR::Edge* get_edges () {return &edges[0];}
  
  __host__ __device__
  const CSR::Vertex* get_vertices () {return &vertices[0];}
  
  __host__ __device__
  int get_n_vertices () {return n_vertices;}
  
  __host__ __device__
  void copy_vertices (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      vertices[i] = src->get_vertices()[i];
    }
  }
  
  __host__ __device__
  void copy_edges (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      edges[i] = src->get_edges ()[i];
    }
  }
  
  __host__ __device__
  int get_n_edges () {return n_edges;}
};

void csr_from_graph (CSR* csr, Graph& graph)
{
  int edge_iterator = 0;
  auto graph_vertices = graph.get_vertices ();
  for (int i = 0; i < graph_vertices.size (); i++) {
    ::Vertex& vertex = graph_vertices[i];
    csr->vertices[i].set_from_graph_vertex (graph_vertices[i]);
    csr->vertices[i].set_start_edge_id (edge_iterator);
    for (auto edge : vertex.get_edges ()) {
      csr->edges[edge_iterator] = edge;
      edge_iterator++;
    }
    
    csr->vertices[i].set_end_edge_id (edge_iterator-1);
  }
}
  
//template <size_t N> using VertexEmbedding = std::bitset<N>;

class VertexEmbedding
{
private:
  unsigned char array[((N/8)+1)];

public:
  __device__ __host__
  VertexEmbedding () 
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    assert (array != nullptr);
    reset ();
    assert (all_false () == true);
  }
  
  __host__ __device__
  size_t convert_to_bytes_multiple (size_t n)
  {
    return ((n/8)+1)*8;
  }
  
  __host__ __device__
  VertexEmbedding (const VertexEmbedding& embedding)
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    for (int i = 0; i <  convert_to_bytes_multiple(N)/8; i++) {
      array[i] = embedding.array[i];
    }
  }
  
  __host__ __device__
  void set (int index)
  {
    array[index/8] = array[index/8] | (1 << (index %8));
  }
  
  __host__ __device__
  void set ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = (unsigned char) (~(0UL));
    }
  }
  
  __host__ __device__
  void reset ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = 0;
    }
  }
  
  __host__ __device__
  void reset (int index)
  {
    array[index/8] = array[index/8] & (~(1UL << (index %8)));
  }
  
  __host__ __device__
  bool test (int index)
  {
    return (bool) ((array[index/8] >> (index % 8))&1);
  }
  
  __host__ __device__
  bool all_false ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      if (array[i] != 0UL) {
        return false;
      }
    }
    
    return true;
  }
  __host__ __device__
  ~VertexEmbedding ()
  {
    //delete[] array;
  }
};

void print_embedding (VertexEmbedding embedding, std::ostream& os);


std::vector<VertexEmbedding> get_extensions (VertexEmbedding& embedding, CSR* csr)
{
  std::vector<VertexEmbedding> extensions;
  
  if (embedding.all_false ()) {
    for (int u = 0; u < N; u++) {
      VertexEmbedding extension;
      extension.set(u);
      //print_embedding (extension, std::cout);
      //std::cout << " " << u << std::endl;
      extensions.push_back (extension);
    }
  } else {
    for (int u = 0; u < N; u++) {
      if (embedding.test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding.test (v) == false) {
            VertexEmbedding extension = VertexEmbedding(embedding);
            extension.set(v);
            extensions.push_back(extension);
          }
        }
      }
    }
  }
  
  return extensions;
}

std::vector<VertexEmbedding> get_initial_embedding (CSR* csr)
{
  VertexEmbedding embedding;
  std::vector <VertexEmbedding> embeddings;

  embeddings.push_back (embedding);
  
  return embeddings;
}

bool (*filter) (CSR* csr, VertexEmbedding& embedding);
void (*process) (std::vector<VertexEmbedding>& output, VertexEmbedding& embedding);

__host__ __device__
bool clique_filter (CSR* csr, VertexEmbedding* embedding)
{
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      for (int v = 0; v < N; v++) {
        if (u != v and embedding->test(v)) {
          if (!csr->has_edge (u, v)) {
            return false;
          }
        }
      }
    }
  }
  
  return true;
}

void clique_process (std::vector<VertexEmbedding>& output, VertexEmbedding& embedding)
{
  output.push_back (embedding);
}

void run_single_step_initial (void* input, int n_embeddings, CSR* csr,
                      std::vector<VertexEmbedding>& output,
                      std::vector<VertexEmbedding>& next_step)
{  
  VertexEmbedding* embeddings = (VertexEmbedding*)input;
  
  for (int i = 0; i < n_embeddings; i++) {
    VertexEmbedding embedding = embeddings[i];
    std::vector<VertexEmbedding> extensions = get_extensions (embedding, csr);
    
    for (auto extension : extensions) {
      if (clique_filter (csr, &extension)) {
        clique_process (output, extension);
        next_step.push_back (extension);
      }
    }
  }
}

__device__ 
void printf_embedding (VertexEmbedding* embedding) 
{
  printf ("[");
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      printf ("%d, ", u);
    }
  }
  
  printf ("]\n");
}

//#define USE_SHARED

__global__
void run_single_step (void* input, int n_embeddings, CSR* csr,
                      void* output_ptr, 
                      int* n_output,
                      void* next_step, int* n_next_step)
{
  int id;

#ifdef USE_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread, 
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread, 
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#endif
  
  VertexEmbedding* embeddings = (VertexEmbedding*)input;
  VertexEmbedding* new_embeddings = (VertexEmbedding*)next_step;
  VertexEmbedding* output = ((VertexEmbedding*)output_ptr);
  unsigned char temp [sizeof (VertexEmbedding)];
  id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
  if (n_embeddings >= MAX_CUDA_THREADS) {
    int embeddings_per_thread = n_embeddings/MAX_CUDA_THREADS+1;
  
    start = id*embeddings_per_thread;
    end = (id+1)*embeddings_per_thread < n_embeddings ? (id+1)*embeddings_per_thread : n_embeddings;
  } else {
    if (id >= n_embeddings) 
      return;
    
    start = id;
    end = id+1;
  }
  
  int q[1000] = {0};
  
  for (int i = start; i < end; i++) {
    VertexEmbedding& embedding = embeddings[i];
    
    for (int u = 0; u < N; u++) {
      if (embedding.test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding.test (v) == false) {
            memcpy (&temp[0], &embedding, sizeof (VertexEmbedding));
             
            VertexEmbedding* extension = (VertexEmbedding*)(&temp[0]);
            extension->set(v);
            if (clique_filter (csr, extension)) {
              memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (VertexEmbedding));
              memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (VertexEmbedding));
            }
          }
        }
      }
    }
  }
  
  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}

void print_embedding (VertexEmbedding embedding, std::ostream& os)
{
  os << "[";
  for (int u = 0; u < N; u++) {
    if (embedding.test(u)) {
      os << u << ", ";
    }
  }
  os << "]";
}

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main (int argc, char* argv[])
{
  std::vector<Vertex> vertices;
  int n_edges = 0;
  
  if (argc < 2) {
    std::cout << "Arguments: graph-file" << std::endl;
    return -1;
  }
  
  char* graph_file = argv[1];
  FILE* fp = fopen (graph_file, "r+");
  if (fp == nullptr) {
    std::cout << "File '" << graph_file << "' not found" << std::endl;
    return 1;
  }
  
  print_kernel<<<1,10>>> ();
  hipDeviceSynchronize ();
  
  while (true) {
    char line[LINE_SIZE];
    char num_str[LINE_SIZE];
    size_t line_size;
    
    if (fgets (line, LINE_SIZE, fp) == nullptr) {
      break;
    }
    
    int id, label;
    int bytes_read;
    
    bytes_read = sscanf (line, "%d %d", &id, &label);
    Vertex vertex (id, label);
    char* _line = line + chars_in_int (id) + chars_in_int (label);
    do {
      int num;
      
      bytes_read = sscanf (_line, "%d", &num);
      if (bytes_read > 0) {
        vertex.add_edge (num);
        _line += chars_in_int (num);
        n_edges++;
      }
        
    } while (bytes_read > 0);
    
    vertices.push_back (vertex);
  }
  
  fclose (fp);
  
  //std::cout << "n_edges "<<n_edges <<std::endl;
  Graph graph (vertices, n_edges);
  
  CSR* csr = new CSR(N, N_EDGES); 
  std::cout << "sizeof(CSR)"<< sizeof(CSR)<<std::endl;
  csr_from_graph (csr, graph);

  std::vector<VertexEmbedding> initial_embeddings = get_initial_embedding (csr);
  std::vector<VertexEmbedding> output;
  std::vector<VertexEmbedding> embeddings = initial_embeddings;
  //filter = clique_filter;
  //process = clique_process;
  int iter = 0;
  {
    std::vector<VertexEmbedding> new_embeddings;
    run_single_step_initial (&initial_embeddings[0], 1, csr, output, new_embeddings);
    
    embeddings = new_embeddings;
  }
  
  iter = 1;
  
  
  
  for (iter; iter < 10 && embeddings.size () > 0; iter++) {
    std::cout << "iter " << iter << " embeddings " << embeddings.size () << std::endl;
    size_t global_mem_size = 3*1024*1024*1024UL;
    char* global_mem_ptr = new char[global_mem_size];
    int n_embeddings = embeddings.size ();
    for (int i = 0; i < n_embeddings; i++) {
      ((VertexEmbedding*)global_mem_ptr)[i] = embeddings[i];
    }
    
    void* embeddings_ptr = global_mem_ptr;
    
    int n_new_embeddings = 0;
    void* new_embeddings_ptr = (char*)embeddings_ptr + (n_embeddings)*sizeof(VertexEmbedding);
    int max_embeddings = 1000000;
    void* output_ptr = (char*)new_embeddings_ptr + (max_embeddings)*sizeof(VertexEmbedding);
    int n_output = 0;
    
    char* device_embeddings;
    char *device_new_embeddings;
    int* device_n_embeddings;
    char *device_outputs;
    int* device_n_outputs;
    CSR* device_csr;
    
    hipMalloc (&device_embeddings, n_embeddings*sizeof(VertexEmbedding));
    hipMemcpy (device_embeddings, embeddings_ptr, 
                n_embeddings*sizeof(VertexEmbedding), 
                hipMemcpyHostToDevice);
    hipMalloc (&device_new_embeddings, max_embeddings*sizeof (VertexEmbedding));
    hipMalloc (&device_outputs, max_embeddings*sizeof (VertexEmbedding));
    hipMalloc (&device_n_embeddings, sizeof (0));
    hipMalloc (&device_n_outputs, sizeof (0));
    hipMalloc (&device_csr, sizeof(CSR));
    
    hipMemcpy (device_n_embeddings, &n_new_embeddings, 
                sizeof (n_new_embeddings), hipMemcpyHostToDevice);
    hipMemcpy (device_n_outputs, &n_output, sizeof (n_output), 
                hipMemcpyHostToDevice);
    
    hipMemcpy (device_csr, csr, sizeof (CSR), hipMemcpyHostToDevice);
    
    std::cout << "starting kernel with n_embeddings: " << n_embeddings;
    
    if (false and n_embeddings < MAX_CUDA_THREADS) {
      std::cout << " threads: " << n_embeddings/256 << std::endl;
      run_single_step<<<n_embeddings/256+1,256>>> (device_embeddings, n_embeddings, device_csr, 
                              device_outputs, device_n_outputs, 
                              device_new_embeddings, device_n_embeddings);
    } else {
      std::cout << " threads: " << MAX_CUDA_THREADS/THREAD_BLOCK_SIZE << std::endl;
      run_single_step<<<MAX_CUDA_THREADS/THREAD_BLOCK_SIZE,THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr, 
                              device_outputs, device_n_outputs, 
                              device_new_embeddings, device_n_embeddings);
    }
    
    hipDeviceSynchronize ();
    
    hipError_t error = hipGetLastError ();
    if (error != hipSuccess) {
      const char* error_string = hipGetErrorString (error);
      std::cout << error_string << std::endl;
    } else {
      std::cout << "Cuda success " << std::endl;
    }
    
    hipMemcpy (new_embeddings_ptr, device_new_embeddings, max_embeddings*sizeof(VertexEmbedding), hipMemcpyDeviceToHost);
    hipMemcpy (output_ptr, device_outputs, max_embeddings*sizeof(VertexEmbedding), hipMemcpyDeviceToHost);
    hipMemcpy (&n_new_embeddings, device_n_embeddings, sizeof(0), hipMemcpyDeviceToHost);
    hipMemcpy (&n_output, device_n_outputs, sizeof(0), hipMemcpyDeviceToHost);
    
    std::cout << "n_new_embeddings "<<n_new_embeddings<<std::endl;
    std::vector<VertexEmbedding> new_embeddings;
    for (int i = 0; i < n_new_embeddings; i++) {
      new_embeddings.push_back (((VertexEmbedding*)new_embeddings_ptr)[i]);
    }
    for (int i = 0; i < n_output; i++) {
      output.push_back (((VertexEmbedding*)output_ptr)[i]);
    }
    embeddings = new_embeddings;
    
    hipFree (device_embeddings);
    hipFree (device_new_embeddings);
    hipFree (device_n_embeddings);
    hipFree (device_outputs);
    hipFree (device_n_outputs);
    hipFree (device_csr);
    delete[] global_mem_ptr;
  }
  
  std::cout << "Number of embeddings found "<< output.size () << std::endl;
  
  /*for (auto embedding : output) {
    print_embedding (embedding, std::cout);
    std::cout << std::endl;
  }*/
}

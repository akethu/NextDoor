#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <string.h>
#include <assert.h>
#include <tuple>

#define LINE_SIZE 1024*1024
//#define USE_FIXED_THREADS
#define MAX_CUDA_THREADS (96*96)
#define THREAD_BLOCK_SIZE 256
#define WARP_SIZE 32
//#define ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING false

#define ENABLE_GRAPH_PARTITION_IN_SHARED_MEM
#if defined (ALL_THREAD_BLOCK_EMBEDDINGS_IN_SHARED_MEM_PER_VERTEX)  && !defined (EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK) 
  #error "For ALL_THREAD_BLOCK_EMBEDDINGS_IN_SHARED_MEM_PER_VERTEX, EMBEDDING_PER_PARTITIONS_IN_THREADBLOCK should be defined "
#endif
#define USE_EMBEDDING_IN_LOCAL_MEM
//#define PROCESS_EMBEDDINGS_PER_VERTEX
#define GPU_QUERY_WAIT_TIME 1000UL

//#define ADD_TO_OUTPUT
//#define SHARED_MEM_NON_COALESCING
/**
  * The commit performing better is 698368fa19d023e3cb09705d820d333f79d0bf46.
  */
#ifdef SHARED_MEM_NON_COALESCING
  #ifndef USE_EMBEDDING_IN_SHARED_MEM
    #error "USE_EMBEDDING_IN_SHARED_MEM must be enabled with SHARED_MEM_NON_COALESCING"
  #endif
#endif
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #ifdef USE_FIXED_THREADS
    #error "USE_FIXED_THREADS cannot be enabled with USE_EMBEDDING_IN_SHARED_MEM"
  #endif
#endif

#define NEW_EMBEDDING_BUFFER_SIZE 128*1024*1024 //Size in terms of Bytes //Setting it to 128 MB makes citeseer performs a lot better

#define GRAPH_PARTITION_SIZE (48 * 1024) //24 KB is the size of each partition of graph

//#define USE_CONSTANT_MEM

typedef uint8_t SharedMemElem;
typedef uint32_t VertexID;
//citeseer.graph
//const int N = 3312;
//const int N_EDGES = 9074;
#define ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING false

//micro.graph
const int N = 100000;
const int N_EDGES = 2160312;
//#define ENABLE_NEW_EMBEDDINGS_ON_THE_FLY_COPYING true

const int N_THREADS = 1024;

double_t convertTimeValToDouble (struct timeval _time)
{
  return ((double_t)_time.tv_sec) + ((double_t)_time.tv_usec)/1000000.0f;
}


struct timeval getTimeOfDay ()
{
  struct timeval _time;

  if (gettimeofday (&_time, NULL) == -1) {
    fprintf (stderr, "gettimeofday returned -1\n");
    perror ("");
    abort ();
  }

  return _time;
}

enum BUFFER_STATUS {
  GPU_USING,
  CPU_COPYING,
  READY_CPU_COPYING,
  FREE,
};

class Vertex
{
private:
  int id;
  int label;
  std::vector <int> edges;

public:
  Vertex (int _id, int _label) : label(_label), id (_id)
  {
  }

  void set_id (int _id) {id = _id;}
  int get_id () {return id;}
  int get_label () {return label;}
  void add_edge (int vertexID) {edges.push_back (vertexID);}
  void sort_edges () {std::sort (edges.begin(), edges.end ());}
  std::vector <int>& get_edges () {return edges;}
  void print (std::ostream& os)
  {
    os << id << " " << label << " ";
    for (auto edge : edges) {
      os << edge << " ";
    }

    os << std::endl;
  }

  static bool compare_vertex (Vertex& v1, Vertex& v2) 
  {
    return v1.edges.size () < v2.edges.size ();
  }
};

int chars_in_int (int num)
{
  if (num == 0) return sizeof(char);
  return (int)((ceil(log10(num))+1)*sizeof(char));
}

class Graph
{
private:
  std::vector<Vertex> vertices;
  int n_edges;

public:
  Graph (std::vector<Vertex> _vertices, int _n_edges) :
    vertices (_vertices), n_edges(_n_edges)
  {}

  const std::vector<Vertex>& get_vertices () {return vertices;}
  int get_n_edges () {return n_edges;}
};



class CSR
{
public:
  struct Vertex
{
  int id;
  int label;
  int start_edge_id;
  int end_edge_id;
  __host__ __device__
  Vertex ()
  {
    id = -1;
    label = -1;
    start_edge_id = -1;
    end_edge_id = -1;
  }

  void set_from_graph_vertex (::Vertex& vertex)
  {
    id = vertex.get_id ();
    label = vertex.get_label ();
  }

  __host__ __device__ int get_start_edge_idx () {return start_edge_id;}
  __host__ __device__ int get_end_edge_idx () {return end_edge_id;}
  __host__ __device__ VertexID get_id () {return id;}
  __host__ __device__ void set_start_edge_id (int start) {start_edge_id = start;}
  __host__ __device__ void set_end_edge_id (int end) {end_edge_id = end;}
};

typedef int Edge;

  CSR::Vertex vertices[N];
  CSR::Edge edges[N_EDGES];
  int n_vertices;
  int n_edges;

public:
  CSR (int _n_vertices, int _n_edges)
  {
    n_vertices = _n_vertices;
    n_edges = _n_edges;
  }

  __host__ __device__
  CSR ()
  {
    n_vertices = N;
    n_edges = N_EDGES;
  }

  void print (std::ostream& os)
  {
    for (int i = 0; i < n_vertices; i++) {
      os << vertices[i].id << " " << vertices[i].label << " ";
      for (int edge_iter = vertices[i].start_edge_id;
           edge_iter <= vertices[i].end_edge_id; edge_iter++) {
        os << edges[edge_iter] << " ";
      }
      os << std::endl;
    }
  }

  __host__ __device__
  int get_start_edge_idx (int vertex_id)
  {
    if (!(vertex_id < n_vertices && 0 <= vertex_id)) {
      printf ("vertex_id %d, n_vertices %d\n", vertex_id, n_vertices);
      assert (false);
    }
    return vertices[vertex_id].start_edge_id;
  }

  __host__ __device__
  int get_end_edge_idx (int vertex_id)
  {
    assert (vertex_id < n_vertices && 0 <= vertex_id);
    return vertices[vertex_id].end_edge_id;
  }

  __host__ __device__
  bool has_edge (int u, int v)
  {
    //TODO: Since graph is sorted, do this using binary search
    for (int e = get_start_edge_idx (u); e <= get_end_edge_idx (u); e++) {
      if (edges[e] == v) {
        return true;
      }
    }

    return false;
  }

  __host__ __device__
  const CSR::Edge* get_edges () {return &edges[0];}

  __host__ __device__
  const CSR::Vertex* get_vertices () {return &vertices[0];}

  __host__ __device__
  int get_n_vertices () {return n_vertices;}

  __host__ __device__
  void copy_vertices (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      vertices[i] = src->get_vertices()[i];
    }
  }

  __host__ __device__
  void copy_edges (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      edges[i] = src->get_edges ()[i];
    }
  }

  __host__ __device__
  int get_n_edges () {return n_edges;}
};

class CSRPartition
{
public:
  int start_vertex_id;
  int end_vertex_id;
  int edge_start_idx;
  int edge_end_idx;
  CSR::Vertex *vertices;
  CSR::Edge *edges;

  __device__
  CSRPartition () 
  {

  }

  __device__
  void initialize (int _start, int _end, int _edge_start_idx, int _edge_end_idx, CSR::Vertex* _vertices, CSR::Edge* _edges)
  {
    start_vertex_id = _start;
    end_vertex_id = _end;
    vertices = _vertices;
    edges = _edges;
    edge_start_idx = _edge_start_idx;
    edge_end_idx = _edge_end_idx;
  }

  CSRPartition (int _start, int _end, int _edge_start_idx, int _edge_end_idx, CSR::Vertex* _vertices, CSR::Edge* _edges)
  {
    start_vertex_id = _start;
    end_vertex_id = _end;
    vertices = _vertices;
    edges = _edges;
    edge_start_idx = _edge_start_idx;
    edge_end_idx = _edge_end_idx;
  }

  __host__ __device__
  int get_start_edge_idx (int vertex_id) {
    if (!(vertex_id <= end_vertex_id && start_vertex_id <= vertex_id)) {
      printf ("vertex_id %d, end_vertex %d, start_vertex %d\n", vertex_id, end_vertex_id, start_vertex_id);
      assert (false);
    }
    return vertices[vertex_id - start_vertex_id].start_edge_id;
  }

  __host__ __device__
  int get_end_edge_idx (int vertex_id)
  {
    assert (vertex_id <= end_vertex_id && start_vertex_id <= vertex_id);
    return vertices[vertex_id - start_vertex_id].end_edge_id;
  }
  
  __host__ __device__
  CSR::Edge get_edge (int idx) 
  {
    assert (idx >= edge_start_idx && idx <= edge_end_idx);
    return edges[idx - edge_start_idx];
  }
};

#ifdef USE_CONSTANT_MEM
  __constant__ unsigned char csr_constant_buff[sizeof(CSR)];
#endif

void csr_from_graph (CSR* csr, Graph& graph)
{
  int edge_iterator = 0;
  auto graph_vertices = graph.get_vertices ();
  for (int i = 0; i < graph_vertices.size (); i++) {
    ::Vertex& vertex = graph_vertices[i];
    csr->vertices[i].set_from_graph_vertex (graph_vertices[i]);
    csr->vertices[i].set_start_edge_id (edge_iterator);
    for (auto edge : vertex.get_edges ()) {
      csr->edges[edge_iterator] = edge;
      edge_iterator++;
    }

    csr->vertices[i].set_end_edge_id (edge_iterator-1);
  }
}

class GlobalMemAllocator
{
  static uint64_t memory_length;
  static uint64_t bump_pointer;
  static char* global_mem_ptr;

  public:

    static void initialize (char* _global_mem_ptr, uint64_t _memory_length) {
      global_mem_ptr = _global_mem_ptr;
      memory_length = _memory_length;
      bump_pointer = 0;
    }

    static uint64_t alloc (size_t sz) {
      assert (bump_pointer + sz < memory_length);

      uint64_t to = bump_pointer;

      bump_pointer += sz;

      return to;
    }

    static uint64_t allocated () {
      return bump_pointer;
    }
    static uint64_t alloc_vertices_array (size_t n_vertices) {
      return alloc (sizeof (VertexID)*n_vertices);
    }
    
    static void* get_global_mem_ptr () {
      return global_mem_ptr;
    }

};

uint64_t GlobalMemAllocator::memory_length;
uint64_t GlobalMemAllocator::bump_pointer;
char* GlobalMemAllocator::global_mem_ptr;

class VectorVertexEmbedding
{
private:
  uint64_t array_start_idx;
  uint32_t filled_size;
  uint32_t size;
  VertexID* array;

public:
  __host__
  VectorVertexEmbedding (uint32_t _max_size, uint64_t _array_start_idx, bool filled = false)
  {
    size = _max_size;
    filled_size = filled ? size : 0;
    array_start_idx = _array_start_idx;
    array = (VertexID*)((char*) GlobalMemAllocator::get_global_mem_ptr () + array_start_idx);
  }

  __host__ 
  std::vector<VertexID> to_vector ()
  {
    std::vector<VertexID> v;

    for (int i = 0; i < get_n_vertices (); i++) {
      v.push_back (get_vertex (i));
    }

    return v;
  }

  __host__
  void* get_array () {return array;}
  __device__ __host__
  uint64_t get_array_start_idx () { return array_start_idx;}
  // __host__ __device__
  // VectorVertexEmbedding (const VectorVertexEmbedding& embedding)
  // {
  // #if DEBUG
  //   assert (embedding.get_max_size () <= get_max_size ());
  // #endif
  //   filled_size = 0;
  //   for (int i = 0; i < embedding.get_n_vertices (); i++) {
  //     add (embedding.get_vertex (i));
  //   }
  // }
  
  __host__ __device__
  void add (int v)
  {
  #if DEBUG
    if (!(size != 0 and filled_size < size)) {
      printf ("filled_size %d, size %d\n", filled_size, size);
      //assert (size != 0 and filled_size < size);
      assert (false);
    }
  #endif
  
    add_unsorted (v);
  }

  __host__ __device__
  void add_last_in_sort_order () 
  {
    int v = array[filled_size-1];
    remove_last ();
    add (v);
  }

  __host__ __device__
  void add_unsorted (int v) 
  {
    array[filled_size++] = v;
  }
  
  __host__ __device__
  void remove (int v)
  {
    printf ("Do not support remove\n");
    assert (false);
  }
  
  // __host__ __device__
  // const bool has_logn (int v)
  // {
  //   int l = 0;
  //   int r = filled_size-1;
    
  //   while (l <= r) {
  //     int m = l+(r-l)/2;
      
  //     if (array[m] == v)
  //       return true;
      
  //     if (array[m] < v)
  //       l = m + 1;
  //     else
  //       r = m - 1;
  //   }
    
  //   return false;
  // }
  
  __host__ __device__
  bool has (int v) const
  {
    for (int i = 0; i < filled_size; i++) {
      if (array[i] == v) {
        return true;
      }
    }
    
    return false;
  }
  
  __host__ __device__
  size_t get_n_vertices () const
  {
    return filled_size;
  }
  
  __host__ __device__
  int get_vertex (int index, void* global_storage_start) const
  {
    return ((VertexID*)((char*)global_storage_start + array_start_idx))[index];
  }

  __device__
  int get_vertex (int index, void* global_storage_start, uint64_t global_start_idx) const
  {
    assert (array_start_idx >= global_start_idx);
    return ((VertexID*)((char*)global_storage_start + (array_start_idx - global_start_idx)))[index];
  }

  __host__ 
  int get_vertex (int index) const
  {
    return array[index];
  }
  
  __host__ __device__
  int get_last_vertex () const
  {
    return array[filled_size-1];
  }
  
  __host__ __device__
  size_t get_max_size () const
  {
    return size;
  }
  
  __host__ __device__
  void clear ()
  {
    filled_size = 0;
  }
  
  __host__ __device__
  void remove_last () 
  {
    assert (filled_size > 0);
    filled_size--;
  }
  __host__ __device__
  ~VectorVertexEmbedding ()
  {
    //delete[] array;
  }

  void print () 
  {
    std::cout << "[";
    for (int i = 0; i < filled_size; i++) {
      std::cout << get_vertex (i) << ", ";
    }
    std::cout << "]";
  }
};

std::vector<VectorVertexEmbedding> get_initial_embedding_vector (CSR* csr)
{
  VectorVertexEmbedding embedding (0, 0UL);
  std::vector <VectorVertexEmbedding> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

__host__
void vector_embedding_from_one_less_size (VectorVertexEmbedding const & in,
                                          VectorVertexEmbedding& out)
{
  //TODO: Optimize here, filled_size++ in add is being called several times
  //but can be called only once too
  //if  (false and vec_emb1.get_n_vertices () != size) {
  //  printf ("vec_emb1.get_n_vertices () %ld != size %d\n", vec_emb1.get_n_vertices (), size);
  //  assert (false);
  //}
  assert (in.get_n_vertices () <= out.get_n_vertices ());
  for (int i = 0; i < in.get_n_vertices (); i++) {
    out.add (in.get_vertex (i));
  }
}

std::vector<VectorVertexEmbedding> get_extensions_vector (VectorVertexEmbedding& embedding, CSR* csr)
{
  std::vector<VectorVertexEmbedding> extensions;
  size_t size;
  
  size = embedding.get_n_vertices ();

  if (size == 0) {
    for (int u = 0; u < N; u++) {
      uint64_t ptr =  GlobalMemAllocator::alloc_vertices_array(1);
      VectorVertexEmbedding extension(1,ptr);
      extension.add(u);
      extensions.push_back(extension);
    }
  } else {
    for (int i = 0; i < size; i++) {
      int u = embedding.get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        int v = csr->get_edges () [e];
        if (embedding.has (v) == false) {
          VectorVertexEmbedding extension(1, GlobalMemAllocator::alloc_vertices_array(size + 1));
          vector_embedding_from_one_less_size (embedding, extension);
          extension.add(v);
          extensions.push_back(extension);
        }
      }
    }
  }

  return extensions;
}

void run_single_step_initial_vector (std::vector<VectorVertexEmbedding>& input_embeddings,
                                     CSR* csr,
                                     std::vector<VectorVertexEmbedding>& output_embeddings,
                                     std::vector<VectorVertexEmbedding>& next_step_embeddings)
{
  for (int i = 0; i < input_embeddings.size (); i++) {
    VectorVertexEmbedding& embedding = input_embeddings[i];
    std::vector<VectorVertexEmbedding> extensions = get_extensions_vector (embedding, csr);
    for (auto extension : extensions) {
        output_embeddings.push_back (extension);
        next_step_embeddings.push_back (extension);
      }
   }
}

bool is_cuda_error (hipError_t error) 
{
  //hipError_t error = hipGetLastError ();
  if (error != hipSuccess) {
    const char* error_string = hipGetErrorString (error);
    std::cout << "Cuda Error: " << error_string << std::endl;
    return true;
  }

  return false;
}

#define EXECUTE_CUDA_FUNC(x) assert (is_cuda_error (x) == false);

__global__ void get_max_lengths_for_embeddings_first_iter (void* void_csr, void* input, size_t n_embeddings,
                                                          void* void_embedding_storage,
                                                          int global_mem_start_idx,
                                                          unsigned long long int* embeddings_additions_iter,
                                                          void* void_map_orig_embedding_to_additions)
{
  CSR* csr = (CSR*)void_csr;
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_idx >= n_embeddings) {
    return;
  }

  VectorVertexEmbedding* input_embeddings = (VectorVertexEmbedding*) input;
  VectorVertexEmbedding* input_embedding = &input_embeddings[thread_idx];
  //VertexID* embedding_storage = (VertexID embedding_storage;
  int* map_orig_embedding_to_additions = (int*) void_map_orig_embedding_to_additions;
  unsigned long long int new_edges = 0;
  // printf ("thread idx %d array_start_idx %ld\n", thread_idx, input_embedding->get_array_start_idx ());
  /*Perform a single hop for all vertices in the input embedding*/
  for (int vertex_idx = 0; vertex_idx < input_embedding->get_n_vertices (); vertex_idx++) {
    VertexID vertex = input_embedding->get_vertex (vertex_idx, void_embedding_storage, global_mem_start_idx);
    int start_edge_idx = csr->get_start_edge_idx (vertex);
    const int end_edge_idx = csr->get_end_edge_idx (vertex);

    if (start_edge_idx != -1) {
      int e = (end_edge_idx - start_edge_idx) + 1;
      assert (e >= 0);
      new_edges += e;
    }

    assert (thread_idx == vertex);
  }

  unsigned long long int additions_start_iter = atomicAdd (embeddings_additions_iter, new_edges);
  map_orig_embedding_to_additions[2*thread_idx] = additions_start_iter;
  map_orig_embedding_to_additions[2*thread_idx+1] = new_edges;
  //printf ("thread_idx %d additions %d\n", thread_idx, map_orig_embedding_to_additions[2*thread_idx+1]);
}

__global__ void get_max_lengths_for_embeddings_single_step (void* void_csr, void* input, 
                                                            size_t n_embeddings, 
                                                            void* void_embedding_storage,
                                                            int global_mem_start_idx,  
                                                            unsigned long long int* void_embeddings_additions_iter,
                                                            void* void_map_orig_embedding_to_additions_prev_iter,
                                                            void* void_map_orig_embedding_to_additions_next_iter,
                                                            void* void_map_orig_embedding_to_additions_first_iter)
{
  CSR* csr = (CSR*)void_csr;
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_idx >= n_embeddings) {
    return;
  }

  VectorVertexEmbedding* input_embeddings = (VectorVertexEmbedding*) input;
  VectorVertexEmbedding* input_embedding = &input_embeddings[thread_idx];
  //VertexID* embedding_storage = (VertexID embedding_storage;
  unsigned long long int* embeddings_additions_iter = void_embeddings_additions_iter;
  int* map_orig_embedding_to_additions_next_iter = (int*)void_map_orig_embedding_to_additions_next_iter;
  int* map_orig_embedding_to_additions_prev_iter = (int*)void_map_orig_embedding_to_additions_prev_iter;
  int* map_orig_embedding_to_additions_first_iter = (int*) void_map_orig_embedding_to_additions_first_iter;
  unsigned long long int new_edges = map_orig_embedding_to_additions_first_iter[2*thread_idx + 1];
  // printf ("thread idx %d array_start_idx %ld\n", thread_idx, input_embedding->get_array_start_idx ());
  /*Perform a single hop for all vertices in the input embedding*/
  for (int vertex_idx = 0; vertex_idx < input_embedding->get_n_vertices (); vertex_idx++) {
    VertexID vertex = input_embedding->get_vertex (vertex_idx, void_embedding_storage, global_mem_start_idx);
    int start_edge_idx = csr->get_start_edge_idx (vertex);
    const int end_edge_idx = csr->get_end_edge_idx (vertex);
    if (start_edge_idx != -1) {
      while (start_edge_idx <= end_edge_idx) {
        int v = csr->get_edges()[start_edge_idx];
        new_edges += map_orig_embedding_to_additions_prev_iter [2*v+1];
        start_edge_idx++;
      }
    }

    assert (thread_idx == vertex);
  }

  //printf ("new_edges %ld\n", new_edges);
  unsigned long long int additions_start_iter = atomicAdd (embeddings_additions_iter, new_edges);
  map_orig_embedding_to_additions_next_iter[2*thread_idx] = additions_start_iter;
  map_orig_embedding_to_additions_next_iter[2*thread_idx+1] = new_edges;
}

__global__ void run_single_step_embedding (int N_HOPS, void* void_csr, int* partition_range, int n_partitions, void* input, size_t n_embeddings, void* void_embedding_storage, uint64_t global_mem_start_idx,
                                           void* void_embeddings_additions, 
                                           size_t embeddings_additions_size,
                                           void* void_map_orig_embedding_to_additions, 
                                           size_t map_orig_embedding_to_additions_size,
                                           int* additions_sizes)
{
  CSR* csr = (CSR*)void_csr;
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  
#ifdef ENABLE_GRAPH_PARTITION_IN_SHARED_MEM
  __shared__ CSRPartition csr_partition;
  __shared__ char partition_vertex_edges [GRAPH_PARTITION_SIZE - sizeof (CSRPartition)];
  int partition_idx = n_partitions - 1;  

  for (int i = 0; i < n_partitions; i++) {
    if (thread_idx >= partition_range[2*i] && thread_idx <= partition_range[2*i + 1]) {
      partition_idx = i;
      break;
    }
  }
  int partition_start_vertex = partition_range[2*partition_idx];
  int partition_end_vertex = partition_range[2*partition_idx + 1];
  int partition_n_vertices = partition_end_vertex - partition_start_vertex;
  CSR::Vertex* vertex_array = (CSR::Vertex*)&partition_vertex_edges[0];
  int vertex_array_size = (partition_end_vertex - partition_start_vertex + 1)*sizeof(CSR::Vertex);
  CSR::Edge* edge_array = (CSR::Edge*)&partition_vertex_edges[vertex_array_size];
  
  int end_edge = csr->get_end_edge_idx (partition_end_vertex);
  if (end_edge == -1)
    end_edge = csr->get_start_edge_idx (partition_end_vertex);
  int start_edge = csr->get_start_edge_idx (partition_start_vertex);
  int partition_n_edges = end_edge - start_edge + 1;
  //if (!(sizeof (partition_vertex_edges) >= vertex_array_size + partition_n_edges*sizeof (CSR::Edge))) 
   // printf ("sizeof (partition_vertex_edges) %d vertex_array_size %d partition_n_edges*sizeof (CSR::Edge) %d \n", (int)sizeof (partition_vertex_edges), vertex_array_size, (int) partition_n_edges*sizeof (CSR::Edge));
  assert (sizeof (partition_vertex_edges) >= vertex_array_size + partition_n_edges*sizeof (CSR::Edge));
  csr_partition.initialize (partition_start_vertex, partition_end_vertex, start_edge, end_edge, vertex_array, edge_array);
  for (int i = 0; i < partition_n_vertices; i+=blockDim.x) {
    if (i + threadIdx.x <= partition_n_vertices) {
      vertex_array[i + threadIdx.x] = csr->get_vertices () [partition_start_vertex + i + threadIdx.x];
    }
  }

  for (int i = 0; i < partition_n_edges; i+=blockDim.x) {
    if (i + threadIdx.x <= partition_n_edges) {
      edge_array[i + threadIdx.x] = csr->get_edges () [start_edge + i + threadIdx.x];
    }
  }
  
  __syncthreads ();
#endif

  if (thread_idx >= n_embeddings) {
    return;
  }

  VectorVertexEmbedding* input_embeddings = (VectorVertexEmbedding*) input;
  VectorVertexEmbedding* input_embedding = &input_embeddings[thread_idx];
  VertexID* embeddings_additions = (VertexID*)void_embeddings_additions;

  int* map_orig_embedding_to_additions = (int*) void_map_orig_embedding_to_additions;

  unsigned long long int new_edges = 0;
  // printf ("thread idx %d array_start_idx %ld\n", thread_idx, input_embedding->get_array_start_idx ());
  /*Perform a single hop for all vertices in the input embedding*/
  int additions_filled = map_orig_embedding_to_additions[2*thread_idx];
  int start = map_orig_embedding_to_additions[2*thread_idx];
  int size = map_orig_embedding_to_additions[2*thread_idx+1];

  for (int vertex_idx = 0; vertex_idx < input_embedding->get_n_vertices (); vertex_idx++) {
    VertexID vertex = input_embedding->get_vertex (vertex_idx, void_embedding_storage, global_mem_start_idx);
  #ifdef ENABLE_GRAPH_PARTITION_IN_SHARED_MEM
    int start_edge_idx = csr_partition.get_start_edge_idx (vertex);
    const int end_edge_idx = csr_partition.get_end_edge_idx (vertex);
  #else
    int start_edge_idx = csr->get_start_edge_idx (vertex);
    const int end_edge_idx = csr->get_end_edge_idx (vertex);
  #endif
    if (start_edge_idx != -1) {
      while (start_edge_idx <= end_edge_idx) {
        VertexID edge = csr->get_edges ()[start_edge_idx];
        embeddings_additions[additions_filled++] = edge;
        start_edge_idx++;
      }
    }
  }

  int additions_end_idx = additions_filled;
  int additions_start_idx = start;
  int hop = 1;

  while (hop < N_HOPS) {    
    for (int vertex_idx = additions_start_idx; vertex_idx < additions_end_idx; vertex_idx++) {
      int vertex = embeddings_additions [vertex_idx];
#ifdef ENABLE_GRAPH_PARTITION_IN_SHARED_MEM
      int start_edge_idx = (vertex >= partition_start_vertex && vertex <= partition_end_vertex) ? csr_partition.get_start_edge_idx (vertex) : csr->get_start_edge_idx (vertex);
      const int end_edge_idx = (vertex >= partition_start_vertex && vertex <= partition_end_vertex) ? csr_partition.get_end_edge_idx (vertex) : csr->get_end_edge_idx (vertex);
#else
      int start_edge_idx = csr->get_start_edge_idx (vertex);
      const int end_edge_idx = csr->get_end_edge_idx (vertex);
#endif

      if (start_edge_idx != -1) {
        while (start_edge_idx <= end_edge_idx) {
          VertexID edge = csr->get_edges ()[start_edge_idx];
          // bool present = false;
          // for (int i = start; i < additions_filled; i++) {
          //   if (embeddings_additions[i] == edge) {
          //     present = true;
          //     break;
          //   }
          // }
          // if (present == false)
          embeddings_additions[additions_filled++] = edge;
          start_edge_idx++;
        }
      }
    }

    additions_start_idx = additions_end_idx;
    additions_end_idx = additions_filled;

    hop++;
  }

  //if (thread_idx == 0) {
    //printf ("additions_filled %d start %d\n", additions_filled, start);
  //}
  additions_sizes[thread_idx] = additions_filled - start;
}

std::vector <std::vector <VertexID>> n_hop_cpu (CSR* csr, const int N_HOPS)
{
  std::vector <std::vector <VertexID>> hops = std::vector<std::vector<VertexID>> (csr->get_n_vertices ());

  int hop = 0;

  for (int vertex = 0; vertex < csr->get_n_vertices (); vertex++) {
    int start_edge_idx = csr->get_start_edge_idx (vertex);
    int end_edge_idx = csr->get_end_edge_idx (vertex);
    if (start_edge_idx != -1) {
      for (int edge = start_edge_idx; edge <= end_edge_idx; edge++) {
        hops[vertex].push_back (csr->get_edges()[edge]);
      }
    }
  }

  for (int vertex = 0; vertex < csr->get_n_vertices (); vertex++) {
    int hop = 1;
    std::vector <VertexID> vertex_hops[N_HOPS + 1];
    vertex_hops[0].insert (vertex_hops[0].begin(), hops[vertex].begin (), hops[vertex].end ());
    while (hop < N_HOPS) {
      for (int hop_vertex : vertex_hops[hop - 1]) {
        int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
        int end_edge_idx = csr->get_end_edge_idx (hop_vertex);
        
        if (start_edge_idx != -1) {
          for (int edge = start_edge_idx; edge <= end_edge_idx; edge++) {
            int v = csr->get_edges()[edge];
            vertex_hops[hop].push_back (v);
          }
        }
      }

      hops[vertex].insert (hops[vertex].begin (), vertex_hops[hop].begin (), vertex_hops[hop].end ());
      hop++;
    }
  }

  return hops;
}

#define MAX_LOAD_PER_TB (N_THREADS)
#define MAX_VERTICES_PER_TB 10
#if MAX_VERTICES_PER_TB < 1
  #error "MAX_VERTICES_PER_TB should be greater than or equal to 1"
#endif

#define WARP_HOP

const uint FULL_MASK = 0xffffffff;

__device__ inline int get_warp_mask_and_participating_threads (int condition, int& participating_threads, int& first_active_thread)
{
  uint warp_mask = __ballot_sync(FULL_MASK, condition);
  first_active_thread = -1;
  participating_threads = 0;
  int qq = 0;
  while (qq < 32) {
    if ((warp_mask & (1U << qq)) == (1U << qq)) {
      if (first_active_thread == -1) {
        first_active_thread = qq;
      }
      participating_threads++;
    }
    qq++;
  }

  return warp_mask;
}

// __global__ void run_think_hybrid_single_step_embedding (int N_HOPS, int hop, void* void_csr,
//   void* void_embeddings_additions, 
//   size_t embeddings_additions_size,
//   int* map_orig_embedding_to_additions,
//   int* previous_stage_filled_range,
//   int* map_vertex_to_hop_vertex_data,
//   int* global_index)
// {
//   CSR* csr = (CSR*)void_csr;
//   __shared__ int vertices[MAX_VERTICES_PER_TB];
//   __shared__ int previous_step_end[MAX_VERTICES_PER_TB];
//   __shared__ int n_vertex_load;
//   __shared__ int thread_idx_to_load[2*MAX_LOAD_PER_TB];
//   __shared__ int last_vertex_id;
//   __shared__ int last_vertex_id_hops_remaining;
//   __shared__ int last_vertex_id_hops_done;
//   __shared__ int last_vertex_previous_step_start, last_vertex_previous_step_end;
  
//   int laneid = threadIdx.x%warpSize;
//   int warpid = threadIdx.x/warpSize;

//   VertexID* embeddings_additions = (VertexID*)void_embeddings_additions;
//   int thread_idx = blockIdx.x*blockDim.x + threadIdx.x;

//   if (hop != 0) {
//     thread_idx_to_load [2*threadIdx.x] = -1;
//     thread_idx_to_load [2*threadIdx.x + 1] = -1;

//     __syncthreads ();

//     if (threadIdx.x == 0) {
//       last_vertex_id = -1;
//       last_vertex_id_hops_remaining = -1;
//       int load = 0;
//       n_vertex_load = 0;
//       int load_assigned_index = 0;

//       while (n_vertex_load < MAX_VERTICES_PER_TB && load < MAX_LOAD_PER_TB) {
//         vertices[n_vertex_load] = atomicAdd(global_index, 1);
//         if (vertices[n_vertex_load] >= gridDim.x) {
//           break;
//         }
        
//         int hops_so_far = previous_stage_filled_range[2*vertices[n_vertex_load] + 1] - previous_stage_filled_range[2*vertices[n_vertex_load]];
        
//         int hop_idx;
//         for (hop_idx = 0; hop_idx < hops_so_far && load_assigned_index < MAX_LOAD_PER_TB; hop_idx++) {
//           thread_idx_to_load[2*load_assigned_index] = n_vertex_load;
//           thread_idx_to_load[2*load_assigned_index+1] = hop_idx;
//           load_assigned_index++;
//         }

//         if (load + hops_so_far > MAX_LOAD_PER_TB) {
//           last_vertex_id_hops_remaining = hops_so_far - hop_idx;
//           last_vertex_id_hops_done = hop_idx;
//           last_vertex_id = n_vertex_load;
//           last_vertex_previous_step_start = previous_stage_filled_range[2*vertices[n_vertex_load]];
//           last_vertex_previous_step_end = previous_stage_filled_range[2*vertices[n_vertex_load] + 1];
//         }

//         load += hops_so_far;
//         n_vertex_load++;
//       }
//     }
    
//     __syncthreads ();

//     assert (n_vertex_load <= MAX_VERTICES_PER_TB);
//     int _curr_vertex_id = thread_idx_to_load[2*threadIdx.x];
//     int hop_idx = thread_idx_to_load[2*threadIdx.x+1];
//     int first_active_thread = -1;
//     int participating_threads = 0;
//     uint warp_hop_mask = get_warp_mask_and_participating_threads (_curr_vertex_id != -1 && vertices[_curr_vertex_id] < gridDim.x, participating_threads, first_active_thread);

//     assert (first_active_thread != -1 || (first_active_thread == -1 && warp_hop_mask == 0));
//     if (_curr_vertex_id != -1 && vertices[_curr_vertex_id] < gridDim.x) {
//       int vertex = vertices[_curr_vertex_id];
//       int start = map_orig_embedding_to_additions[2*vertex];
//       previous_step_end[_curr_vertex_id] = previous_stage_filled_range[2*vertex+1];
//     }

//     __syncthreads ();

//     if (_curr_vertex_id != -1 && vertices[_curr_vertex_id] < gridDim.x) {
//       int vertex = vertices[_curr_vertex_id];
//       int start = map_orig_embedding_to_additions[2*vertex];
//       int previous_step_start = previous_stage_filled_range[2*vertex];
//       int hops_so_far = previous_step_end [_curr_vertex_id] - previous_step_start;

//       int hop_vertex = embeddings_additions[start + previous_step_start + hop_idx];
//       int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
//       const int end_edge_idx = csr->get_end_edge_idx (hop_vertex);
    
// #ifdef WARP_HOP
//       __syncwarp (warp_hop_mask);

//       for (int th = 0; th < participating_threads; th++) {
//         int target_thread_id = warpid*warpSize + th;
//         int _hop_vertex = __shfl_sync (warp_hop_mask, hop_vertex, th, warpSize);
//         assert (_hop_vertex != -1);
//         int _start_edge_idx = __shfl_sync (warp_hop_mask, start_edge_idx, th, warpSize);
//         const int _end_edge_idx = __shfl_sync (warp_hop_mask, end_edge_idx, th, warpSize);
//         int l = thread_idx_to_load[2*target_thread_id];
//         if (_end_edge_idx != -1 && l != -1) {
//           int _vertex = vertices[l];
//           int* end = &previous_stage_filled_range[2*_vertex + 1];
//           int _start = __shfl_sync (warp_hop_mask, start, th, warpSize);;
//           int e = -1;
//           if (laneid == first_active_thread)
//             e = atomicAdd (end, _end_edge_idx - _start_edge_idx + 1);
//           int _e = __shfl_sync (warp_hop_mask, e, first_active_thread, warpSize);
//           assert (_e != -1);
//           int iter = 0;
//           while (_start_edge_idx + laneid <= _end_edge_idx) {
//             VertexID edge = csr->get_edges ()[_start_edge_idx + laneid];
//             embeddings_additions[_start + _e + iter*participating_threads + laneid] = edge;
//             _start_edge_idx += participating_threads;
//             iter++;
//           }
//         }

//         __syncwarp (warp_hop_mask);
//       }
// #else
//       int* end = &previous_stage_filled_range[2*vertex + 1];
//       if (end_edge_idx != -1) {
//         while (start_edge_idx <= end_edge_idx) {
//           VertexID edge = csr->get_edges ()[start_edge_idx];
//           int e = atomicAdd (end, 1);
//           embeddings_additions[start + e] = edge;
//           start_edge_idx++;
//         }
//       }
// #endif
//     }
    
//     __syncthreads ();
//     _curr_vertex_id = thread_idx_to_load[2*threadIdx.x];
//     if (_curr_vertex_id != -1 && vertices[_curr_vertex_id] < gridDim.x) {
//       int v = vertices[_curr_vertex_id];
//       previous_stage_filled_range[2*v] = previous_step_end[_curr_vertex_id];
//     }
//     __syncthreads ();

//     warp_hop_mask = get_warp_mask_and_participating_threads (last_vertex_id != -1 && last_vertex_id_hops_remaining != -1, participating_threads, first_active_thread);
    
//     if (last_vertex_id != -1 && last_vertex_id_hops_remaining != -1) {
//       int vertex = vertices[last_vertex_id];
//       int start = map_orig_embedding_to_additions[2*vertex];

//       __syncthreads ();

//       int previous_step_start = last_vertex_previous_step_start;
//       int hops_so_far = last_vertex_previous_step_end - previous_step_start;
//       int* end = &previous_stage_filled_range[2*vertex + 1];
//       int hops_done = last_vertex_id_hops_done;
      
//       assert (last_vertex_id_hops_done + last_vertex_id_hops_remaining == hops_so_far);

// #ifdef WARP_HOP
//       for (int i = 0; i < last_vertex_id_hops_remaining/(blockDim.x/warpSize) + 1; i++) {
//         int hop_idx = i*(blockDim.x/warpSize) + warpid;
//         if (hop_idx >= last_vertex_id_hops_remaining) {
//           continue;
//         }
        
//         int hop_vertex = embeddings_additions[start + previous_step_start + hops_done + hop_idx];
//         int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
//         const int end_edge_idx = csr->get_end_edge_idx (hop_vertex);

//         __syncwarp (warp_hop_mask);

//         int e = -1;
//         if (end_edge_idx != -1) {
//           if (laneid == first_active_thread) {
//             e = atomicAdd (end, end_edge_idx - start_edge_idx + 1);
//           }
//         }

//         for (int th = 0; th < participating_threads; th++) {
//           int target_thread_id = warpid*warpSize + th;
          
//           if (end_edge_idx != -1) {
//             int _e = __shfl_sync (warp_hop_mask, e, first_active_thread, warpSize);
//             assert (_e != -1);
//             int iter = 0;
//             while (start_edge_idx + laneid <= end_edge_idx) {
//               VertexID edge = csr->get_edges ()[start_edge_idx + laneid];
//               embeddings_additions[start + _e + iter*participating_threads + laneid] = edge;
//               start_edge_idx += participating_threads;
//               iter++;
//             }
//           }
//         }
//       }
// #else
//       for (int i = 0; i < last_vertex_id_hops_remaining/blockDim.x + 1; i++) {
//         int hop_idx = i*blockDim.x + threadIdx.x;
//         if (hop_idx >= last_vertex_id_hops_remaining) {
//           continue;
//         }
        
//         int hop_vertex = embeddings_additions[start + previous_step_start + hops_done + hop_idx];
//         int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
//         const int end_edge_idx = csr->get_end_edge_idx (hop_vertex);
//         if (end_edge_idx != -1) {
//           int e = atomicAdd (end, end_edge_idx - start_edge_idx + 1);
//           int iter = 0;
//           while (start_edge_idx <= end_edge_idx) {
//             VertexID edge = csr->get_edges ()[start_edge_idx];
//             embeddings_additions[start + e + iter] = edge;
//             start_edge_idx++;
//             iter++;
//           }
//         }
//       }
// #endif

//       __syncthreads ();
//     }
//   } else {
//     int source_vertex = blockIdx.x;

//     int start = map_orig_embedding_to_additions[2*source_vertex];
//     int start_edge_idx = csr->get_start_edge_idx (source_vertex);
//     const int end_edge_idx = csr->get_end_edge_idx (source_vertex);
//     const int n_edges = end_edge_idx - start_edge_idx + 1;

//     if (end_edge_idx == -1) {
//       return;
//     }

//     int* end = &previous_stage_filled_range[2*source_vertex + 1];

//     for (int i = 0; i < n_edges/blockDim.x + 1; i++) {
//       int edge_idx = i*blockDim.x + threadIdx.x;
//       if (edge_idx >= n_edges) {
//         return;
//       }

//       VertexID edge = csr->get_edges ()[start_edge_idx + edge_idx];
//       int e = atomicAdd (end, 1);
//       embeddings_additions[start + e] = edge;    
//     }

//     previous_stage_filled_range[2*source_vertex] = 0;
//   }
// }

__device__ int n_edges_to_warp_size (const int n_edges) 
{
  //Different warp sizes gives different performance. 32 is worst. adapative is a litter better.
  //Best is 4.
  return 4;
  if (n_edges <= 4) 
    return 2;
  else if (n_edges > 4 && n_edges <= 8)
    return 4;
  else if (n_edges > 8 && n_edges <= 16)
    return 8;
  else if (n_edges > 16 && n_edges <= 32) 
    return 16;
  else
    return 32;
}

#define MAX_EDGES (2*MAX_LOAD_PER_TB)
#define USE_PARTITION_FOR_SHMEM
#define MAX_HOP_VERTICES_IN_SH_MEM (MAX_VERTICES_PER_TB)

__global__ void run_hop_parallel_single_step (int N_HOPS, int hop, void* void_csr,
  void* void_embeddings_additions, 
  size_t embeddings_additions_size,
  int* map_orig_embedding_to_additions,
  int* previous_stage_filled_range,
  int* hop_vertex_to_roots,
  int* map_vertex_to_hop_vertex_data,
  int* global_index,
  unsigned long long int* profile_branch_1, unsigned long long int* profile_branch_2)
{
  CSR* csr = (CSR*)void_csr;
  __shared__ int vertices[MAX_VERTICES_PER_TB];
  __shared__ int previous_step_end[MAX_VERTICES_PER_TB];
  __shared__ int n_vertex_load;
  __shared__ int thread_idx_to_load[2*MAX_LOAD_PER_TB];
  __shared__ int last_hop_vertex_id;
  __shared__ int last_hop_vertex_roots_remaining;
  __shared__ int last_hop_vertex_roots_done;

#ifdef USE_PARTITION_FOR_SHMEM
  __shared__ VertexID shmem_csr_edges[MAX_EDGES];
  __shared__ int hop_vertex_in_shared_mem[MAX_HOP_VERTICES_IN_SH_MEM];
  __shared__ int hop_vertices_in_shared_mem_start_edge_idx[MAX_HOP_VERTICES_IN_SH_MEM];
  __shared__ int hop_vertices_in_shared_mem_end_edge_idx[MAX_HOP_VERTICES_IN_SH_MEM];
  __shared__ int hop_vertices_in_shared_mem_size;
  __shared__ int shmem_csr_edges_size;
#endif 

  int laneid = threadIdx.x%warpSize;
  int warpid = threadIdx.x/warpSize;

  VertexID* embeddings_additions = (VertexID*)void_embeddings_additions;
  int thread_idx = blockIdx.x*blockDim.x + threadIdx.x;

  if (hop != 0) {
    thread_idx_to_load [2*threadIdx.x] = -1;
    thread_idx_to_load [2*threadIdx.x + 1] = -1;

    __syncthreads ();

    if (threadIdx.x == 0) {
      last_hop_vertex_id = -1;
      last_hop_vertex_roots_remaining = -1;
      int load = 0;
      n_vertex_load = 0;
      int load_assigned_index = 0;
      int warp_assigned = 0;

#ifdef USE_PARTITION_FOR_SHMEM
      hop_vertices_in_shared_mem_size = 0;
      int edges_in_shared_mem = 0;
      shmem_csr_edges_size = 0;
#endif

      while (n_vertex_load < MAX_VERTICES_PER_TB && load < MAX_LOAD_PER_TB) {
        vertices[n_vertex_load] = atomicAdd(global_index, 1);
        if (vertices[n_vertex_load] >= gridDim.x) {
          break;
        }
        
        int start_edge_idx = csr->get_start_edge_idx (vertices[n_vertex_load]);
        const int end_edge_idx = csr->get_end_edge_idx (vertices[n_vertex_load]);
        const int n_edges = (end_edge_idx != -1) ? (end_edge_idx - start_edge_idx + 1) : 0;
#ifdef USE_PARTITION_FOR_SHMEM
        if (hop_vertices_in_shared_mem_size < MAX_HOP_VERTICES_IN_SH_MEM && n_edges != 0 && 
            n_edges + edges_in_shared_mem < MAX_EDGES) {
          int v = vertices[n_vertex_load];
          hop_vertex_in_shared_mem[hop_vertices_in_shared_mem_size] = v;
          //hop_vertices_in_shared_mem_start_edge_idx[hop_vertices_in_shared_mem_size] = csr->get_start_edge_idx (v);
          //hop_vertices_in_shared_mem_end_edge_idx[hop_vertices_in_shared_mem_size] = csr->get_end_edge_idx (v);
          edges_in_shared_mem += n_edges;
          hop_vertices_in_shared_mem_size++;
        }
#endif
        int shfl_warp_size = n_edges_to_warp_size(n_edges);
        int root_vertices = map_vertex_to_hop_vertex_data[2*vertices[n_vertex_load] + 1];

        if (root_vertices != 0 and n_edges != 0) {
          int root_vertex_idx;
          for (root_vertex_idx = 0; root_vertex_idx < root_vertices && warp_assigned < MAX_LOAD_PER_TB; root_vertex_idx++) {
            for (int ii = warp_assigned; ii < min (warp_assigned + shfl_warp_size, MAX_LOAD_PER_TB); ii++) {
              thread_idx_to_load[2*ii] = n_vertex_load;
              thread_idx_to_load[2*ii+1] = root_vertex_idx;
            }
            warp_assigned += shfl_warp_size;
            load_assigned_index += 1;
          }

          if (warp_assigned >= MAX_LOAD_PER_TB) {
            last_hop_vertex_roots_remaining = root_vertices - root_vertex_idx;
            last_hop_vertex_roots_done = root_vertex_idx;
            last_hop_vertex_id = n_vertex_load;
          }

          load += root_vertices*shfl_warp_size;
          n_vertex_load++;
        }
      }
    }
    
    __syncthreads ();

#ifdef USE_PARTITION_FOR_SHMEM
    for (int i = 0; i < hop_vertices_in_shared_mem_size/(blockDim.x/warpSize) + 1; i++) {
      int hop = i * warpSize + warpid;
      if (hop >= hop_vertices_in_shared_mem_size) {
        continue;
      }

      int start_edge_idx = csr->get_start_edge_idx (hop_vertex_in_shared_mem[hop]);
      const int end_edge_idx = csr->get_end_edge_idx (hop_vertex_in_shared_mem[hop]);
      const int n_edges = (end_edge_idx != -1) ? (end_edge_idx - start_edge_idx + 1) : 0;
      int _shmem_start = -1;
      if (laneid == 0) {
        _shmem_start = atomicAdd (&shmem_csr_edges_size, n_edges);
      }

      int shmem_start = __shfl_sync (FULL_MASK, _shmem_start, 0, warpSize);
      assert (shmem_start != -1);
      for (int e = 0; e < n_edges/warpSize + 1; e++) {
        int edge_idx = e*warpSize + laneid;
        if (edge_idx < n_edges) {
          shmem_csr_edges[shmem_start + edge_idx] = csr->get_edges ()[start_edge_idx + edge_idx];
        }
      }
      __syncwarp ();
      if (laneid == 0) {
        hop_vertices_in_shared_mem_start_edge_idx[hop] = _shmem_start;
        hop_vertices_in_shared_mem_end_edge_idx[hop] = _shmem_start + n_edges - 1;
      }

      __syncwarp ();
    }
#endif

    __syncthreads ();

    assert (n_vertex_load <= MAX_VERTICES_PER_TB);
    int _curr_vertex_id = thread_idx_to_load[2*threadIdx.x];
    int root_vertex_idx = thread_idx_to_load[2*threadIdx.x + 1];
    
    int hop_vertex_start_idx = -1;
    int n_root_vertices = -1;
    int root_vertex = -1;
    int hop_idx = -1;
    int first_active_thread = -1;
    int participating_threads = 0;

    if (_curr_vertex_id != -1 && root_vertex_idx != -1 && vertices[_curr_vertex_id] < gridDim.x) {
      hop_vertex_start_idx = map_vertex_to_hop_vertex_data[2*vertices[_curr_vertex_id]];
      n_root_vertices = map_vertex_to_hop_vertex_data[2*vertices[_curr_vertex_id] + 1];
      root_vertex = hop_vertex_to_roots[hop_vertex_start_idx + 2*root_vertex_idx];

      hop_idx = hop_vertex_to_roots[hop_vertex_start_idx + 2*root_vertex_idx + 1];
      
      if (root_vertex != -1 && root_vertex < gridDim.x) {
        int vertex = root_vertex;
        int start = map_orig_embedding_to_additions[2*vertex];
        //previous_step_end[_curr_vertex_id] = previous_stage_filled_range[2*vertex+1];
      }
    }

    __syncthreads ();

    uint warp_hop_mask = get_warp_mask_and_participating_threads (_curr_vertex_id != -1 && 
      vertices[_curr_vertex_id] < gridDim.x && root_vertex_idx != -1 && root_vertex != -1 && root_vertex < gridDim.x, participating_threads, first_active_thread);
      //__syncthreads ();
    if (_curr_vertex_id != -1 && root_vertex_idx != -1 && vertices[_curr_vertex_id] < gridDim.x) {
      if (root_vertex != -1 && root_vertex < gridDim.x) {
        int vertex = root_vertex;
        int start = map_orig_embedding_to_additions[2*vertex];
        int hop_vertex = embeddings_additions[hop_idx];
        int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
        const int end_edge_idx = csr->get_end_edge_idx (hop_vertex);
        __syncwarp (warp_hop_mask);

#if 0
        __syncwarp (warp_hop_mask);

        for (int th = 0; th < participating_threads; th++) {
          int target_thread_id = warpid*warpSize + th;
          int _hop_vertex = __shfl_sync (warp_hop_mask, hop_vertex, th, warpSize);
          assert (_hop_vertex != -1);
          int _start_edge_idx = __shfl_sync (warp_hop_mask, start_edge_idx, th, warpSize);
          const int _end_edge_idx = __shfl_sync (warp_hop_mask, end_edge_idx, th, warpSize);
          int l = thread_idx_to_load[2*target_thread_id];
          if (_end_edge_idx != -1 && l != -1) {
            int _vertex = vertices[l];
            int* end = &previous_stage_filled_range[2*_vertex + 1];
            int _start = __shfl_sync (warp_hop_mask, start, th, warpSize);;
            int e = -1;
            if (laneid == first_active_thread)
              e = atomicAdd (end, _end_edge_idx - _start_edge_idx + 1);
            int _e = __shfl_sync (warp_hop_mask, e, first_active_thread, warpSize);
            assert (_e != -1);
            int iter = 0;
            while (_start_edge_idx + laneid <= _end_edge_idx) {
              VertexID edge = csr->get_edges ()[_start_edge_idx + laneid];
              embeddings_additions[_start + _e + iter*participating_threads + laneid] = edge;
              _start_edge_idx += participating_threads;
              iter++;
            }
          }

          __syncwarp (warp_hop_mask);
        }
#else
        int* end = &previous_stage_filled_range[2*vertex + 1];
        if (end_edge_idx != -1) {
          __syncwarp (warp_hop_mask);
          int e = -1;
          // if (vertex > 3700 && vertex < 3850)
          //   printf ("vertex %d laneid %d first_active_thread %d\n", vertex, laneid, first_active_thread);
          const int n_edges = end_edge_idx - start_edge_idx + 1;
          int shfl_warp_size = n_edges_to_warp_size(n_edges);
          if (laneid%shfl_warp_size == 0) {
            e = atomicAdd (end, n_edges);
          }
          //TODO: Add synchronization point
          //printf ("first_active_threads[threadIdx.x] %d shfl_warp_size %d\n", first_active_threads[threadIdx.x], shfl_warp_size);
          int _e = __shfl_sync (warp_hop_mask, e, 0, shfl_warp_size);  
          assert (_e != -1);
#ifdef USE_PARTITION_FOR_SHMEM
          if (_curr_vertex_id >= hop_vertices_in_shared_mem_size) {
            int iter = 0;
            while (start_edge_idx + laneid%shfl_warp_size <= end_edge_idx) {
              VertexID edge = csr->get_edges ()[start_edge_idx + laneid%shfl_warp_size];
              embeddings_additions[start + _e + iter*shfl_warp_size + laneid%shfl_warp_size] = edge;
              start_edge_idx += shfl_warp_size;
              iter++;
            }
          } else {
            int iter = 0;
            int start_edge_idx = hop_vertices_in_shared_mem_start_edge_idx[_curr_vertex_id];
            int end_edge_idx = hop_vertices_in_shared_mem_end_edge_idx[_curr_vertex_id];
            while (start_edge_idx + laneid%shfl_warp_size <= end_edge_idx) {
              VertexID edge = shmem_csr_edges[start_edge_idx + laneid%shfl_warp_size];
              embeddings_additions[start + _e + iter*shfl_warp_size + laneid%shfl_warp_size] = edge;
              start_edge_idx += shfl_warp_size;
              iter++;
            }
          }
#else
          int iter = 0;
          while (start_edge_idx + laneid%shfl_warp_size <= end_edge_idx) {
            VertexID edge = csr->get_edges ()[start_edge_idx + laneid%shfl_warp_size];
            embeddings_additions[start + _e + iter*shfl_warp_size + laneid%shfl_warp_size] = edge;
            start_edge_idx += shfl_warp_size;
            iter++;
          }
#endif
        }
  

        __syncwarp (warp_hop_mask);
#endif
      }
    }

    __syncwarp ();
    __syncthreads ();
    // TODO: if working on last hop of the  vertex then set start to end
    // _curr_vertex_id = thread_idx_to_load[2*threadIdx.x];
    // if (_curr_vertex_id != -1 && vertices[_curr_vertex_id] < gridDim.x) {
    //   int v = vertices[_curr_vertex_id];
    //   previous_stage_filled_range[2*v] = previous_step_end[_curr_vertex_id];
    // }
    __syncthreads ();

    //warp_hop_mask = get_warp_mask_and_participating_threads (last_vertex_id != -1 && last_vertex_id_hops_remaining != -1, participating_threads, first_active_thread);
    
    if (last_hop_vertex_id != -1 && last_hop_vertex_roots_remaining != -1) {
      int hop_vertex = vertices[last_hop_vertex_id];
      int hop_vertex_start_idx = map_vertex_to_hop_vertex_data[2*hop_vertex];
      int n_root_vertices = map_vertex_to_hop_vertex_data[2*hop_vertex + 1];
      
      __syncthreads ();
      
      //assert (last_vertex_id_hops_done + last_vertex_id_hops_remaining == hops_so_far);

#if 0
      for (int i = 0; i < last_vertex_id_hops_remaining/(blockDim.x/warpSize) + 1; i++) {
        int hop_idx = i*(blockDim.x/warpSize) + warpid;
        if (hop_idx >= last_vertex_id_hops_remaining) {
          continue;
        }
        
        int hop_vertex = embeddings_additions[start + previous_step_start + hops_done + hop_idx];
        int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
        const int end_edge_idx = csr->get_end_edge_idx (hop_vertex);

        __syncwarp (warp_hop_mask);

        int e = -1;
        if (end_edge_idx != -1) {
          if (laneid == first_active_thread) {
            e = atomicAdd (end, end_edge_idx - start_edge_idx + 1);
          }
        }

        for (int th = 0; th < participating_threads; th++) {
          int target_thread_id = warpid*warpSize + th;
          
          if (end_edge_idx != -1) {
            int _e = __shfl_sync (warp_hop_mask, e, first_active_thread, warpSize);
            assert (_e != -1);
            int iter = 0;
            while (start_edge_idx + laneid <= end_edge_idx) {
              VertexID edge = csr->get_edges ()[start_edge_idx + laneid];
              embeddings_additions[start + _e + iter*participating_threads + laneid] = edge;
              start_edge_idx += participating_threads;
              iter++;
            }
          }
        }
      }
#else
      for (int i = 0; i < last_hop_vertex_roots_remaining/blockDim.x + 1; i++) {
        int root_idx = i*blockDim.x + threadIdx.x;
        if (root_idx >= last_hop_vertex_roots_remaining) {
          continue;
        }
        
        int root_vertex = hop_vertex_to_roots[hop_vertex_start_idx + 2*(root_idx + last_hop_vertex_roots_done)];
        int hop_idx = hop_vertex_to_roots[hop_vertex_start_idx + 2*(root_idx + last_hop_vertex_roots_done) + 1];
        int start = map_orig_embedding_to_additions[2*root_vertex];
        int* end = &previous_stage_filled_range[2*root_vertex + 1];
        int hop_vertex = embeddings_additions[hop_idx];
        int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
        const int end_edge_idx = csr->get_end_edge_idx (hop_vertex);

        if (end_edge_idx != -1) {
          int e = atomicAdd (end, end_edge_idx - start_edge_idx + 1);
          int iter = 0;
          while (start_edge_idx <= end_edge_idx) {
            VertexID edge = csr->get_edges ()[start_edge_idx];
            embeddings_additions[start + e + iter] = edge;
            start_edge_idx++;
            iter++;
          }
        }
      }
#endif

      __syncthreads ();
    }
  } else {
    int source_vertex = blockIdx.x;

    int start = map_orig_embedding_to_additions[2*source_vertex];
    int start_edge_idx = csr->get_start_edge_idx (source_vertex);
    const int end_edge_idx = csr->get_end_edge_idx (source_vertex);
    const int n_edges = end_edge_idx - start_edge_idx + 1;

    if (end_edge_idx != -1) {
      int* end = &previous_stage_filled_range[2*source_vertex + 1];

      for (int i = 0; i < n_edges/blockDim.x + 1; i++) {
        int edge_idx = i*blockDim.x + threadIdx.x;
        if (edge_idx < n_edges) {
          VertexID edge = csr->get_edges ()[start_edge_idx + edge_idx];
          int e = atomicAdd (end, 1);
          embeddings_additions[start + e] = edge;    
        }
      }
    }
  
    __syncthreads ();
    previous_stage_filled_range[2*source_vertex] = start;
  }
}

__global__ void run_think_like_an_edge_single_step_embedding (int N_HOPS, int hop, void* void_csr,
  void* void_embeddings_additions, 
  size_t embeddings_additions_size,
  int* map_orig_embedding_to_additions,
  int* previous_stage_filled_range,
  size_t n_edges,
  int* prev_thread_idx_to_edge_in_additions,
  int* thread_idx_to_edge_in_additions,
  int* thread_idx_to_edge_in_additions_size)
{
  CSR* csr = (CSR*)void_csr;
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx >= n_edges)
  return;

  VertexID* embeddings_additions = (VertexID*)void_embeddings_additions;

  if (hop != 0) {
    int edge_idx = prev_thread_idx_to_edge_in_additions [2*thread_idx];
    int source_vertex = prev_thread_idx_to_edge_in_additions [2*thread_idx + 1];

    int start = map_orig_embedding_to_additions[2*source_vertex];
    int* end = &previous_stage_filled_range[source_vertex];
  
    int vertex = embeddings_additions[edge_idx];

    int start_edge_idx = csr->get_start_edge_idx (vertex);
    const int end_edge_idx = csr->get_end_edge_idx (vertex);

    if (end_edge_idx != -1) {
      while (start_edge_idx <= end_edge_idx) {
        VertexID edge = csr->get_edges ()[start_edge_idx];
        int e = atomicAdd (end, 1);
        embeddings_additions[start + e] = edge;
        if (hop < N_HOPS) {
          int q = atomicAdd (thread_idx_to_edge_in_additions_size, 2);
          thread_idx_to_edge_in_additions [q] = start + e;
          thread_idx_to_edge_in_additions [q + 1] = source_vertex;
        }

        start_edge_idx++;
      }
    }
  } else {
    int source_vertex = thread_idx;

    int start = map_orig_embedding_to_additions[2*thread_idx];
    int prev_end = 0;
    int* end = &previous_stage_filled_range[source_vertex];

    int vertex = thread_idx;

    int start_edge_idx = csr->get_start_edge_idx (vertex);
    const int end_edge_idx = csr->get_end_edge_idx (vertex);

    if (start_edge_idx != -1) {
      while (start_edge_idx <= end_edge_idx) {
        VertexID edge = csr->get_edges ()[start_edge_idx];
        int e = atomicAdd (end, 1);
        embeddings_additions[start + e] = edge;
        if (hop < N_HOPS) {
          int q = atomicAdd (thread_idx_to_edge_in_additions_size, 2);
          thread_idx_to_edge_in_additions [q] = start + e;
          thread_idx_to_edge_in_additions [q + 1] = vertex;
        }

        start_edge_idx++;
      }
    }
  }
}

std::vector <std::unordered_set <VertexID>> n_hop_cpu_distinct (CSR* csr, const int N_HOPS)
{
  std::vector <std::unordered_set <VertexID>> hops = std::vector<std::unordered_set<VertexID>> (csr->get_n_vertices ());

  int hop = 0;

  for (int vertex = 0; vertex < csr->get_n_vertices (); vertex++) {
    int start_edge_idx = csr->get_start_edge_idx (vertex);
    int end_edge_idx = csr->get_end_edge_idx (vertex);
    if (start_edge_idx != -1) {
      for (int edge = start_edge_idx; edge <= end_edge_idx; edge++) {
        hops[vertex].insert (csr->get_edges()[edge]);
      }
    }
  }

  for (int vertex = 0; vertex < csr->get_n_vertices (); vertex++) {
    int hop = 1;
    std::unordered_set <VertexID> vertex_hops[N_HOPS + 1];
    vertex_hops[0].insert (hops[vertex].begin (), hops[vertex].end ());
    while (hop < N_HOPS) {
      for (int hop_vertex : vertex_hops[hop - 1]) {
        int start_edge_idx = csr->get_start_edge_idx (hop_vertex);
        int end_edge_idx = csr->get_end_edge_idx (hop_vertex);
        
        if (start_edge_idx != -1) {
          for (int edge = start_edge_idx; edge <= end_edge_idx; edge++) {
            int v = csr->get_edges()[edge];
            if (hops[vertex].count (v) == 0)
              vertex_hops[hop].insert (v);
          }
        }
      }

      hops[vertex].insert (vertex_hops[hop].begin (), vertex_hops[hop].end ());
      hop++;
    }
  }

  return hops;
}

int main (int argc, char* argv[])
{
  std::vector<Vertex> vertices;
  int n_edges = 0;

  if (argc < 2) {
    std::cout << "Arguments: graph-file" << std::endl;
    return -1;
  }

  char* graph_file = argv[1];
  FILE* fp = fopen (graph_file, "r+");
  if (fp == nullptr) {
    std::cout << "File '" << graph_file << "' not found" << std::endl;
    return 1;
  }

  while (true) {
    char line[LINE_SIZE];
    char num_str[LINE_SIZE];
    size_t line_size;

    if (fgets (line, LINE_SIZE, fp) == nullptr) {
      break;
    }

    int id, label;
    int bytes_read;

    bytes_read = sscanf (line, "%d %d", &id, &label);
    Vertex vertex (id, label);
    char* _line = line + chars_in_int (id) + chars_in_int (label);
    do {
      int num;

      bytes_read = sscanf (_line, "%d", &num);
      if (bytes_read > 0) {
        vertex.add_edge (num);
        _line += chars_in_int (num);
        n_edges++;
      }

    } while (bytes_read > 0);

    vertex.sort_edges ();

    vertices.push_back (vertex);
  }

  fclose (fp);

  std::cout << "n_edges "<<n_edges <<std::endl;
  std::cout << "vertices " << vertices.size () << std::endl; 

  Graph graph (vertices, n_edges);

  CSR* csr = new CSR(N, N_EDGES);
  std::cout << "sizeof(CSR)"<< sizeof(CSR)<<std::endl;
  csr_from_graph (csr, graph);
  
#ifdef USE_CONSTANT_MEM
  hipMemcpyToSymbol(HIP_SYMBOL(csr_constant_buff), csr, sizeof(CSR));
  //~ CSR* csr_constant = (CSR*) &csr_constant_buff[0];
  //~ csr_constant->n_vertices = csr->get_n_vertices ();
  //~ printf ("csr->get_n_vertices () = %d\n", csr->get_n_vertices ());
  //~ csr_constant->n_edges = csr->get_n_edges ();
  //~ csr_constant->copy_vertices (csr, 0, csr->get_n_vertices ());
  //~ csr_constant->copy_edges (csr, 0, csr->get_n_edges ());
#endif
  size_t global_mem_size = 15*1024*1024*1024UL;
  #define PINNED_MEMORY
  #ifdef PINNED_MEMORY
    char* global_mem_ptr;
    hipError_t malloc_error = hipHostMalloc ((void**)&global_mem_ptr, global_mem_size);
    std::cout << "Malloc error: " << hipGetErrorString (malloc_error) << std::endl;
    assert (malloc_error == hipSuccess);
  #else
    char* global_mem_ptr = new char[global_mem_size];
  #endif

  std::cout << "Pinned Memory Allocated" << std::endl;
  GlobalMemAllocator::initialize (global_mem_ptr, global_mem_size);

  std::vector<VectorVertexEmbedding> initial_embeddings = get_initial_embedding_vector (csr);
  std::vector<VectorVertexEmbedding> output;
  size_t new_embeddings_size = 0;
  int iter = 0;
  std::vector<VectorVertexEmbedding>& input_embeddings = initial_embeddings;
  std::vector<VectorVertexEmbedding> iter_1_embeddings;
  {
    run_single_step_initial_vector (input_embeddings, csr, output, iter_1_embeddings);
    input_embeddings = iter_1_embeddings;
  }

  iter = 0;
  double total_stream_time = 0;

  const size_t max_embedding_size_per_iter = (12000000/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
  double_t kernelTotalTime = 0.0;
  std::vector <VectorVertexEmbedding> produced_embeddings;

  void* device_map_orig_embedding_to_additions_prev = nullptr; //Previous iterations map
  int* final_map_orig_embedding_to_additions;

  std::vector<std::tuple<VertexID, VertexID, int, int>> vertex_partition_positions_vector;
  
  //Create Partitions.
  int u = 0;
  const size_t effective_partition_size = GRAPH_PARTITION_SIZE - sizeof (CSRPartition);
  int partition_edge_start_idx = 0;

  while (u < csr->get_n_vertices ()) {
    int n_edges = 0;
    int u_start = u;
    int end_edge_idx = 0;
    int u_end = csr->get_n_vertices () - 1;
    int edges = 0;
    int partial_edges = 0;
    for (int v = u; v < csr->get_n_vertices (); v++) {
      int start = csr->get_start_edge_idx (v);
      const int end = csr->get_end_edge_idx (v);
      if (end != -1) {
        if (v == u) {
          std::cout << "1829: " << " partition_edge_start_idx " << partition_edge_start_idx << " u " << u << " start " << start << " end " << end << std::endl;
        }
        if (v == u && partition_edge_start_idx >= start) {
          start = partition_edge_start_idx;
        }
        edges = end - start + 1;
        assert (edges >= 0);
      } else {
        edges = 0;
      }
      if ((n_edges + edges) * sizeof (CSR::Edge) + (v-u_start + 1)*sizeof(CSR::Vertex) >= effective_partition_size) {
        end_edge_idx = (effective_partition_size - (v-u_start + 1)*sizeof(CSR::Vertex))/sizeof (CSR::Edge) - n_edges;
        std::cout << " v " << v << " n_edges " << n_edges << " u " << u_start  << "  sizeof (CSR::Edge) " << sizeof (CSR::Edge) <<  " sizeof(CSR::Vertex) " << sizeof(CSR::Vertex) << " end_edge_idx " << end_edge_idx << " effective_partition_size " << effective_partition_size << " start " << start << " end " << end << std::endl;
        if (end_edge_idx <= 0) {
          u = v;
          u_end = v - 1;
          partial_edges = 0;
          end_edge_idx = start - 1;
        } else if (end_edge_idx < edges) {
          u = v;
          u_end = v;
          partial_edges = end_edge_idx;
          end_edge_idx += start - 1; //Including last edge
        } else {
          u_end = v;
          u = v + 1;
          partial_edges = 0;
          end_edge_idx += start - 1; //Including last edge
        }

        if (u_end < u_start) 
        {
          std::cout << "u_end : " << u_end << " u_start: "  << u_start  << std::endl;
          std::cout << "ERROR: Cannot create partition " << std::endl;
          assert (false);
        }

        break;
      }

      n_edges += edges;
    }

    std::cout << "Creating partition start: " << u_start << " end: " << u_end << " partition_edge_start_idx " << partition_edge_start_idx << " end_edge_idx " << end_edge_idx << " n_edges " << n_edges << " partial_edges " << partial_edges << std::endl;


    //assert (end_edge_idx == 0 || (end_edge_idx != 0 && end_edge_idx - partition_edge_start_idx + 1 == n_edges + partial_edges));

    vertex_partition_positions_vector.push_back (std::make_tuple (u_start, u_end, partition_edge_start_idx, (end_edge_idx == 0) ? csr->get_end_edge_idx (u_end) : end_edge_idx));
    //Vertex partition: [u_start, u_end]. Edge partition is all edges from u_start to u_end if end_edge_idx = 0. otherwise all edges of vertices from u_start to u_end - 1 and edges of u_end u_end.start_edge_idx to end_edge_idx.
    
    partition_edge_start_idx = end_edge_idx + 1;

    if (u_end == csr->get_n_vertices () - 1) {
      break;
    }
    //std::cout << "u " << u <<  std::endl;
  }

  std::vector<CSRPartition> csr_partitions;
  /** Check if partitions created are correct**/
  for (auto p : vertex_partition_positions_vector) {
    int u = std::get<0> (p);
    int v = std::get<1> (p);
    int start = std::get<2> (p);
    int end = std::get<3> (p);

    CSR::Vertex* vertex_array = new CSR::Vertex[v - u + 1];
    memcpy (vertex_array, &csr->get_vertices ()[u], (v-u + 1)*sizeof(CSR::Vertex));
    vertex_array[0].set_start_edge_id (start);
    vertex_array[v-u].set_end_edge_id (end);

    std::cout << "P " << "u " << u << " v " << v << "[" << start << ", " << end << "]" << std::endl;
    CSR::Edge* edge_array = new CSR::Edge[end - start + 1];
    memcpy (edge_array, &csr->get_edges ()[start], (end - start + 1)*sizeof (CSR::Edge));
    // std::cout << "E " << sizeof (int)*(end_edge - start_edge + 1) << " V " << (p.second - p.first + 1)*sizeof (CSR::Vertex) << std::endl;
    CSRPartition part = CSRPartition (u, v, start, end, vertex_array, edge_array);
    csr_partitions.push_back (part);
  }

  //Sum of edges of all partitions is equal to N_EDGES
  int sum_partition_edges = 0;

  for (auto part : csr_partitions) {
    if (part.edge_end_idx != -1) {
      sum_partition_edges += part.edge_end_idx - part.edge_start_idx + 1;
    }
  }

  assert (sum_partition_edges == N_EDGES);

  int sum_vertices = 0;
  for (int p = 0; p < csr_partitions.size (); p++) {
    if (p > 0 && csr_partitions[p].start_vertex_id == csr_partitions[p-1].end_vertex_id) {
      sum_vertices += csr_partitions[p].end_vertex_id - (csr_partitions[p].start_vertex_id);
    } else {
      sum_vertices += csr_partitions[p].end_vertex_id - csr_partitions[p].start_vertex_id + 1;
    }
  }

  assert (sum_vertices == N);

  int equal_edges = 0;

  /*Check if union of all partitions is equal to the graph*/
  for (int p = 0; p < csr_partitions.size (); p++) {
    int u = csr_partitions[p].start_vertex_id;
    int v = csr_partitions[p].end_vertex_id;
    int end = csr_partitions[p].edge_end_idx;
    int start = csr_partitions[p].edge_start_idx;
    for (int vertex = u; vertex <= v; vertex++) {
      int _start = csr->get_start_edge_idx (vertex);
      if (p > 0 && vertex == csr_partitions[p-1].end_vertex_id) {
        _start = start;
      }
      int _end = csr->get_end_edge_idx (vertex);
      int part_start = csr_partitions[p].get_start_edge_idx (vertex);
      int part_end = csr_partitions[p].get_end_edge_idx (vertex);
      
      if (_end != -1 && part_end != -1) {
        while (_start <= _end && _start <= end && part_start <= part_end) {
          if (!(csr->get_edges ()[_start] == csr_partitions[p].get_edge (part_start))) {
            std::cout << "part_start " << part_start << " part_end " << 
            part_end << " _start " << _start << " _end " << _end << " vertex " 
            << vertex << std::endl;  
            abort ();
          }
          equal_edges++;
          part_start++;
          _start++;
        }
      }
    }
  }

  assert (equal_edges == N_EDGES);

  /********Checking DONE*******/

  /*Code for preparing additions kernels*/
  uint64_t vertices_in_embedding = input_embeddings[0].get_n_vertices ();
  uint64_t global_mem_start_idx = input_embeddings[0].get_array_start_idx ();
  uint64_t global_mem_end_idx = input_embeddings[input_embeddings.size () - 1].get_array_start_idx () + input_embeddings[input_embeddings.size () - 1].get_n_vertices ()*sizeof (VertexID);

  const int N_HOPS = 2;
  // std::cout << "-2   " << input_embeddings[input_embeddings.size () - 2].get_array_start_idx () + input_embeddings[input_embeddings.size () - 2].get_n_vertices ()*sizeof (VertexID) << std::endl;
  std::cout << "Number of input embeddings " << input_embeddings.size() << std::endl;
  std::cout << "global_mem_start_idx " << global_mem_start_idx << " global_mem_end_idx " << global_mem_end_idx << " allocated " << GlobalMemAllocator::allocated () << std::endl;
  std::cout << "vertices_in_embedding " << vertices_in_embedding << std::endl;
  assert (global_mem_end_idx == GlobalMemAllocator::allocated ());
  void* device_csr; //Graph on GPU
  int* device_map_orig_embedding_to_additions; //Map of idx of embedding to the start of how many inputs are added and number of new embeddings
  void* device_map_orig_embedding_to_additions_first;
  void* device_input_embeddings_storage; //Input embeddings copied to GPU from CPU
  void* device_input_embeddings;
  int* device_vertex_partition_positions;
  unsigned long long* device_embeddings_addition_iter;
  size_t map_orig_embedding_to_additions_size = input_embeddings.size () * sizeof (VertexID) * 2;

  // std::cout << "Preparing iteration " << iter << std::endl;
  EXECUTE_CUDA_FUNC (hipMalloc (&device_csr, sizeof(CSR)));
  EXECUTE_CUDA_FUNC (hipMemcpy (device_csr, csr, sizeof(CSR), hipMemcpyHostToDevice));
  EXECUTE_CUDA_FUNC (hipMalloc (&device_input_embeddings_storage, global_mem_end_idx - global_mem_start_idx));
  EXECUTE_CUDA_FUNC (hipMemcpy (device_input_embeddings_storage, (char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx, 
                                  global_mem_end_idx - global_mem_start_idx, hipMemcpyHostToDevice));
  EXECUTE_CUDA_FUNC (hipMalloc (&device_input_embeddings,input_embeddings.size()*sizeof(VectorVertexEmbedding)));
  EXECUTE_CUDA_FUNC (hipMemcpy (device_input_embeddings ,&input_embeddings[0], input_embeddings.size()*sizeof(VectorVertexEmbedding), hipMemcpyHostToDevice));
#if 0
  EXECUTE_CUDA_FUNC (hipMalloc (&device_vertex_partition_positions, n_partitions*sizeof(int)*2));
  EXECUTE_CUDA_FUNC (hipMemcpy (device_vertex_partition_positions, vertex_partition_positions, n_partitions*sizeof(int)*2, hipMemcpyHostToDevice));
#endif
  unsigned long long embeddings_addition_iter = 0;

  double gpu_time = 0;

  for (iter; iter < N_HOPS; iter++) {
    embeddings_addition_iter = 0;
    EXECUTE_CUDA_FUNC (hipMalloc (&device_embeddings_addition_iter, sizeof(unsigned long long)));
    EXECUTE_CUDA_FUNC (hipMemcpy (device_embeddings_addition_iter, &embeddings_addition_iter,  sizeof (unsigned long long), hipMemcpyHostToDevice));

    EXECUTE_CUDA_FUNC (hipMalloc (&device_map_orig_embedding_to_additions, map_orig_embedding_to_additions_size));
    if (false) {
      VectorVertexEmbedding* __m = (VectorVertexEmbedding*)malloc (input_embeddings.size()*sizeof(VectorVertexEmbedding));
      hipMemcpy (__m, device_input_embeddings, input_embeddings.size()*sizeof(VectorVertexEmbedding), hipMemcpyDeviceToHost);
      assert ((char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx + __m[0].get_array_start_idx () == __m[0].get_array());
      std::cout << "sizeof (VectorVertexEmbedding) " << sizeof(VectorVertexEmbedding) << std::endl;
      for (int i = 0; i < input_embeddings.size (); i++) {
        std::cout << "s " << (void*)((char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx + __m[i].get_array_start_idx ()) << " d " <<  __m[i].get_array() << std::endl;
        
        assert ((char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx + __m[i].get_array_start_idx () == __m[i].get_array());

        std::cout << "i "<< i << " v : " << __m[i].get_vertex (0, (char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx) << " sstart id: " <<  __m[i].get_array_start_idx () << std::endl;
      }

      break;
    }

    std::cout << "Calling cuda kernel for iteration " << iter << std::endl;

    int N_THREADS = 128;
    int N_BLOCKS = (input_embeddings.size()%128 == 0) ? input_embeddings.size()/128 : input_embeddings.size()/128 + 1;
    
    double t1 = convertTimeValToDouble(getTimeOfDay ());
    if (iter == 0) {
      get_max_lengths_for_embeddings_first_iter <<<N_BLOCKS, N_THREADS>>> (device_csr,
                                                    device_input_embeddings, input_embeddings.size(), device_input_embeddings_storage,
                                                    global_mem_start_idx,
                                                    device_embeddings_addition_iter,
                                                    device_map_orig_embedding_to_additions);
    } else {
      get_max_lengths_for_embeddings_single_step <<<N_BLOCKS, N_THREADS>>> (device_csr,
                                                  device_input_embeddings, 
                                                  input_embeddings.size (), 
                                                  device_input_embeddings_storage, 
                                                  global_mem_start_idx,
                                                  device_embeddings_addition_iter,
                                                  device_map_orig_embedding_to_additions_prev,
                                                  device_map_orig_embedding_to_additions,
                                                  device_map_orig_embedding_to_additions_first);
    }
    
    EXECUTE_CUDA_FUNC (hipDeviceSynchronize ());
    double t2 = convertTimeValToDouble(getTimeOfDay ());

    gpu_time += t2 - t1;
    if (device_map_orig_embedding_to_additions_prev != nullptr) {
      hipFree (device_map_orig_embedding_to_additions_prev);
    }

    EXECUTE_CUDA_FUNC (hipDeviceSynchronize ());
    std::cout << "Cuda Kernel Done " << std::endl;
    is_cuda_error (hipGetLastError ());    
    EXECUTE_CUDA_FUNC (hipMemcpy (&embeddings_addition_iter, device_embeddings_addition_iter, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    std::cout << "Embedding Additions " << embeddings_addition_iter << std::endl;

    if (iter == N_HOPS - 1) {
      //In Last iteration perform some cleanup and record the map_orig_embedding_to_additions_size
      final_map_orig_embedding_to_additions = (int*)new char[map_orig_embedding_to_additions_size];
      EXECUTE_CUDA_FUNC (hipMemcpy (final_map_orig_embedding_to_additions, device_map_orig_embedding_to_additions, map_orig_embedding_to_additions_size, hipMemcpyDeviceToHost));
      //hipFree (device_map_orig_embedding_to_additions);
      device_map_orig_embedding_to_additions_prev = nullptr;
    } else {
      device_map_orig_embedding_to_additions_prev = device_map_orig_embedding_to_additions;
    }
    if (iter == 0) {
      device_map_orig_embedding_to_additions_first = device_map_orig_embedding_to_additions;
    }

    hipFree (device_embeddings_addition_iter);
  }
  
  std::cout << "Generating additions" << std::endl;
  int* device_additions_sizes;
  EXECUTE_CUDA_FUNC (hipMalloc (&device_additions_sizes, sizeof(VertexID)*input_embeddings.size ()*2));
  EXECUTE_CUDA_FUNC (hipMemset (device_additions_sizes, 0, sizeof(VertexID)*input_embeddings.size ()*2));
  void* device_embeddings_additions; //Storage to store inputs added to each embedding
  size_t embeddings_additions_size = (embeddings_addition_iter+1)*sizeof(VertexID);
  EXECUTE_CUDA_FUNC (hipMalloc (&device_embeddings_additions, embeddings_additions_size));
  EXECUTE_CUDA_FUNC (hipMemset (device_embeddings_additions, -1, embeddings_additions_size));
  //Now generate all the next hop neighbours
  
  // run_single_step_embedding<<<N_BLOCKS, N_THREADS>>> (N_HOPS, device_csr, device_vertex_partition_positions, n_partitions, device_input_embeddings, input_embeddings.size (), device_input_embeddings_storage, global_mem_start_idx,
  // device_embeddings_additions, 
  //   embeddings_additions_size, 
  //   device_map_orig_embedding_to_additions, 
  //   map_orig_embedding_to_additions_size,
  //   device_additions_sizes);
  int* device_filled_ranges;
  EXECUTE_CUDA_FUNC (hipMalloc (&device_filled_ranges, sizeof (int)*input_embeddings.size ()));
  n_edges = input_embeddings.size ();
  int* device_prev_thread_idx_to_edge_in_additions = nullptr;

  VertexID* embedding_additions = new VertexID[embeddings_additions_size];
  int* additions_sizes = new int[input_embeddings.size ()*2];
  std::vector<std::vector <std::pair <VertexID, int>>> hop_vertex_data;// = std::vector <std::vector <std::pair <VertexID, int>>> (csr->get_n_vertices ());

  int *host_hop_vertex_data = nullptr;

  for (auto v : hop_vertex_data) {
    v.clear ();
  }

  for (int hop = 0; hop < N_HOPS; hop++) {
    int* device_thread_idx_to_edge_in_additions;
    int* device_thread_idx_to_edge_in_additions_size;
    int* global_index;
    int* device_hop_vertex_data;
    int* device_map_vertex_to_hop_vertex_data;
    unsigned long long int* device_profile_branch_1;
    unsigned long long int* device_profile_branch_2;

    EXECUTE_CUDA_FUNC (hipMalloc (&global_index, sizeof(int)));
    EXECUTE_CUDA_FUNC (hipMemset (global_index, 0,  sizeof (int)));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_thread_idx_to_edge_in_additions, embeddings_additions_size*2));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_thread_idx_to_edge_in_additions_size, sizeof (int)));
    EXECUTE_CUDA_FUNC (hipMemset (device_thread_idx_to_edge_in_additions_size, 0,  sizeof (int)));
    EXECUTE_CUDA_FUNC (hipMemset (global_index, 0,  sizeof (int)));
    
    EXECUTE_CUDA_FUNC (hipMalloc (&device_profile_branch_1, sizeof (unsigned long)));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_profile_branch_2, sizeof (unsigned long)));
    EXECUTE_CUDA_FUNC (hipMemset (device_profile_branch_1, 0, sizeof (unsigned long)));
    EXECUTE_CUDA_FUNC (hipMemset (device_profile_branch_2, 0, sizeof (unsigned long)));

    int N_BLOCKS = input_embeddings.size ();

    if (hop > 0) {
      int* host_map_vertex_to_hop_vertex_data = nullptr;
      double t1 = convertTimeValToDouble(getTimeOfDay ());
      //Create per hop vertex data
      for (int v = 0; v < csr->get_n_vertices (); v++) {
        hop_vertex_data.push_back (std::vector<std::pair <VertexID, int> > ());
      }
      for (int v = 0; v < csr->get_n_vertices (); v++) {
        int start = additions_sizes[2*v];
        int end   = additions_sizes[2*v + 1];
        for (int i = 0; i < end; i++) {
          int src = embedding_additions[start + i];
          hop_vertex_data[src].push_back (std::make_pair (v, start + i));
        }
      }

      int host_hop_vertex_data_size = 0;

      for (int v = 0; v < csr->get_n_vertices (); v++) {
        host_hop_vertex_data_size += hop_vertex_data[v].size ();
      }

      host_hop_vertex_data = new int [2*host_hop_vertex_data_size];
      host_map_vertex_to_hop_vertex_data = new int[2*csr->get_n_vertices ()];
      int host_hop_vertex_data_iter = 0;

      for (int v = 0; v < csr->get_n_vertices (); v++) {
        for (int i = 0; i < hop_vertex_data[v].size (); i++) {
          host_hop_vertex_data[host_hop_vertex_data_iter + 2*i] = std::get<0> (hop_vertex_data[v][i]);
          host_hop_vertex_data[host_hop_vertex_data_iter + 2*i + 1] = std::get<1> (hop_vertex_data[v][i]);
          if (v == 0) {
            printf ("v %d i %d s %d\n", v, std::get<1> (hop_vertex_data[v][i]), hop_vertex_data[v].size ());
          }
        }
        //memcpy (&host_hop_vertex_data[host_hop_vertex_data_iter], &hop_vertex_data[v][0], hop_vertex_data[v].size()*sizeof(hop_vertex_data[v][0]));
        host_map_vertex_to_hop_vertex_data [2*v] = host_hop_vertex_data_iter;
        host_map_vertex_to_hop_vertex_data [2*v + 1] = hop_vertex_data[v].size ();
        host_hop_vertex_data_iter += 2*hop_vertex_data[v].size ();
      }

      double t2 = convertTimeValToDouble(getTimeOfDay ());
            
      std::cout << "Time taken to create hop vertex data: " << (t2 - t1) << " secs " << std::endl;
      EXECUTE_CUDA_FUNC (hipMalloc (&device_hop_vertex_data, 2*host_hop_vertex_data_size*sizeof (int)));
      EXECUTE_CUDA_FUNC (hipMemcpy (device_hop_vertex_data, host_hop_vertex_data, 2*host_hop_vertex_data_size*sizeof (int), hipMemcpyHostToDevice));
      EXECUTE_CUDA_FUNC (hipMalloc (&device_map_vertex_to_hop_vertex_data, 2*csr->get_n_vertices()*sizeof (int)));
      EXECUTE_CUDA_FUNC (hipMemcpy (device_map_vertex_to_hop_vertex_data, host_map_vertex_to_hop_vertex_data, 2*csr->get_n_vertices()*sizeof (int), hipMemcpyHostToDevice));
    }

    double t1 = convertTimeValToDouble(getTimeOfDay ());
    run_hop_parallel_single_step <<<N_BLOCKS, N_THREADS>>> (N_HOPS, hop, device_csr,  
      device_embeddings_additions,
      embeddings_additions_size,
      device_map_orig_embedding_to_additions,
      device_additions_sizes,
      device_hop_vertex_data,
      device_map_vertex_to_hop_vertex_data,
      global_index,
      device_profile_branch_1,
      device_profile_branch_2);
    EXECUTE_CUDA_FUNC (hipDeviceSynchronize ());
    double t2 = convertTimeValToDouble(getTimeOfDay ());
    gpu_time += t2 - t1;

    EXECUTE_CUDA_FUNC (hipMemcpy (&n_edges, device_thread_idx_to_edge_in_additions_size, sizeof (int), hipMemcpyDeviceToHost));
    n_edges = n_edges/2;
    device_prev_thread_idx_to_edge_in_additions = device_thread_idx_to_edge_in_additions;
    EXECUTE_CUDA_FUNC (hipMemcpy (embedding_additions, device_embeddings_additions, embeddings_additions_size, hipMemcpyDeviceToHost));
    EXECUTE_CUDA_FUNC (hipMemcpy (additions_sizes, device_additions_sizes, input_embeddings.size ()*sizeof(int)*2, hipMemcpyDeviceToHost));

#ifdef PROFILE
    unsigned long profile_branch_1, profile_branch_2;
    EXECUTE_CUDA_FUNC (hipMemcpy (&profile_branch_1, device_profile_branch_1, sizeof(profile_branch_1), hipMemcpyDeviceToHost));
    EXECUTE_CUDA_FUNC (hipMemcpy (&profile_branch_2, device_profile_branch_2, sizeof(profile_branch_1), hipMemcpyDeviceToHost));

    std::cout << "profile_branch_1 " << profile_branch_1 << std::endl;
    std::cout << "profile_branch_2 " << profile_branch_2 << std::endl;
#endif
  }
  
  for (int input_embedding_idx = 0; input_embedding_idx < input_embeddings.size (); input_embedding_idx++) {
    VectorVertexEmbedding& input_embedding = input_embeddings[input_embedding_idx];
    int n_additions = additions_sizes[2*input_embedding_idx + 1];
    int start_idx = final_map_orig_embedding_to_additions[2*input_embedding_idx];
    size_t produced_embedding_size = n_additions;
    if (input_embedding_idx == 48) {
      std::cout << "n_additions " << n_additions << std::endl;
    }
    size_t global_mem_idx = GlobalMemAllocator::alloc_vertices_array (produced_embedding_size);
    //std::cout << "i " << input_embedding_idx << " produced_embedding_size " << produced_embedding_size << " global_mem_idx " << global_mem_idx << std::endl;
    VertexID* ptr = (VertexID*) ((char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_idx);
    ((VertexID*)ptr)[0] = ((VertexID*)input_embedding.get_array ())[0];
    // memcpy (ptr, input_embedding.get_array (), sizeof(VertexID)*input_embedding.get_n_vertices ());
    memcpy (ptr, &embedding_additions[start_idx], sizeof(VertexID)*n_additions);
    
    VectorVertexEmbedding embedding = VectorVertexEmbedding ((uint32_t)produced_embedding_size, global_mem_idx, true);
    produced_embeddings.push_back (embedding);
    // embedding.print ();
  }

  hipFree (device_csr);
  hipFree (device_input_embeddings_storage); 
  hipFree (device_input_embeddings);
  hipFree (device_embeddings_additions);

  std::cout << "Generating CPU Embeddings:" << std::endl;
  double cpu_t1 = convertTimeValToDouble (getTimeOfDay ());
  std::vector<std::vector<VertexID>> hops = n_hop_cpu (csr, N_HOPS);
  double cpu_t2 = convertTimeValToDouble (getTimeOfDay ());

  std::cout << "CPU Time: " << (cpu_t2 - cpu_t1) << " secs" << std::endl;
  std::cout << "GPU Time: " << gpu_time << " secs" << std::endl;
  assert (produced_embeddings.size () == hops.size ());
  for (int idx = 0; idx < produced_embeddings.size (); idx++) {
    std::unordered_set<VertexID> cpu_set = std::unordered_set<VertexID> (hops[idx].begin (), hops[idx].end ());
    std::vector<VertexID> vector_hops;
    vector_hops.insert (vector_hops.begin (), cpu_set.begin(), cpu_set.end ());
    std::sort (vector_hops.begin (), vector_hops.end ());
    std::vector<VertexID> gpu_vector = produced_embeddings [idx].to_vector ();
    std::unordered_set<VertexID> gpu_vector_set = std::unordered_set<VertexID> (gpu_vector.begin (), gpu_vector.end ());
    gpu_vector = std::vector<VertexID> (gpu_vector_set.begin (), gpu_vector_set.end ());
    std::sort (gpu_vector.begin (), gpu_vector.end ());

    if (vector_hops != gpu_vector) {
      std::cout << "checking for vertex " << idx << std::endl;
      std::cout << "size " << vector_hops.size () << " " << gpu_vector.size () << std::endl;
      for (int i = 0; i < min (vector_hops.size (), gpu_vector.size ()); i++) {
        std::cout << vector_hops[i] << "  " << gpu_vector[i] << std::endl;
      }
    }
    assert (vector_hops == gpu_vector);
  }

#if 0
  //Code for single kernel
  for (iter; iter < 5; iter++) {
    uint64_t vertices_in_embedding = input_embeddings[0].get_n_vertices ();
    uint64_t global_mem_start_idx = input_embeddings[0].get_array_start_idx ();
    uint64_t global_mem_end_idx = input_embeddings[input_embeddings.size () - 1].get_array_start_idx () + input_embeddings[input_embeddings.size () - 1].get_n_vertices ()*sizeof (VertexID);

    // std::cout << "-2   " << input_embeddings[input_embeddings.size () - 2].get_array_start_idx () + input_embeddings[input_embeddings.size () - 2].get_n_vertices ()*sizeof (VertexID) << std::endl;
    std::cout << "Number of input embeddings " << input_embeddings.size() << std::endl;
    std::cout << "global_mem_start_idx " << global_mem_start_idx << " global_mem_end_idx " << global_mem_end_idx << " allocated " << GlobalMemAllocator::allocated () << std::endl;
    assert (global_mem_end_idx == GlobalMemAllocator::allocated ());
    unsigned long long embeddings_addition_iter = 0;
    void* device_csr; //Graph on GPU
    void* device_embeddings_additions; //Storage to store inputs added to each embedding
    void* device_map_orig_embedding_to_additions; //Map of idx of embedding to the start of how many inputs are added and number of new embeddings
    void* device_input_embeddings_storage; //Input embeddings copied to GPU from CPU
    void* device_input_embeddings;
    unsigned long long* device_embeddings_addition_iter;

    size_t embedding_additions_size = sizeof(VertexID);
    for (int j = 0; j < iter; j++) 
      embedding_additions_size *= csr->get_n_edges ();
    size_t map_orig_embedding_to_additions_size = input_embeddings.size () * sizeof (VertexID) * 2;

    std::cout << "Preparing iteration " << iter << std::endl;
    EXECUTE_CUDA_FUNC (hipMalloc (&device_csr, sizeof(CSR)));
    EXECUTE_CUDA_FUNC (hipMemcpy (device_csr, csr, sizeof(CSR), hipMemcpyHostToDevice));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_embeddings_additions, embedding_additions_size));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_map_orig_embedding_to_additions, map_orig_embedding_to_additions_size));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_input_embeddings_storage, global_mem_end_idx - global_mem_start_idx));
    EXECUTE_CUDA_FUNC (hipMemcpy (device_input_embeddings_storage, (char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx, 
                                   global_mem_end_idx - global_mem_start_idx, hipMemcpyHostToDevice));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_input_embeddings,input_embeddings.size()*sizeof(VectorVertexEmbedding)));
    EXECUTE_CUDA_FUNC (hipMemcpy (device_input_embeddings ,&input_embeddings[0], input_embeddings.size()*sizeof(VectorVertexEmbedding), hipMemcpyHostToDevice));
    EXECUTE_CUDA_FUNC (hipMalloc (&device_embeddings_addition_iter, sizeof(unsigned long long)));

    EXECUTE_CUDA_FUNC (hipMemcpy (device_embeddings_addition_iter, &embeddings_addition_iter, sizeof (size_t), hipMemcpyHostToDevice));

    if (false) {
      VectorVertexEmbedding* __m = (VectorVertexEmbedding*)malloc (input_embeddings.size()*sizeof(VectorVertexEmbedding));
      hipMemcpy (__m, device_input_embeddings, input_embeddings.size()*sizeof(VectorVertexEmbedding), hipMemcpyDeviceToHost);
      assert ((char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx + __m[0].get_array_start_idx () == __m[0].get_array());
      std::cout << "sizeof (VectorVertexEmbedding) " << sizeof(VectorVertexEmbedding) << std::endl;
      for (int i = 0; i < input_embeddings.size (); i++) {
        std::cout << "s " << (void*)((char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx + __m[i].get_array_start_idx ()) << " d " <<  __m[i].get_array() << std::endl;
        
        assert ((char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx + __m[i].get_array_start_idx () == __m[i].get_array());

        std::cout << "i "<< i << " v : " << __m[i].get_vertex (0, (char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_start_idx) << " sstart id: " <<  __m[i].get_array_start_idx () << std::endl;
      }

      break;
    }

    std::cout << "Calling cuda kernel" << std::endl;

    int N_THREADS = 128;
    int N_BLOCKS = (input_embeddings.size()%128 == 0) ? input_embeddings.size()/128 : input_embeddings.size()/128 + 1;
    
    run_single_step_embedding <<<N_BLOCKS, N_THREADS>>> (device_csr, 
                                          device_input_embeddings, input_embeddings.size(), device_input_embeddings_storage,
                                          global_mem_start_idx, 
                                          device_embeddings_additions, embedding_additions_size, 
                                          device_map_orig_embedding_to_additions, map_orig_embedding_to_additions_size,
                                          device_embeddings_addition_iter);
    EXECUTE_CUDA_FUNC (hipDeviceSynchronize ());
    std::cout << "Cuda Kernel Done " << std::endl;
    is_cuda_error (hipGetLastError ());
    VertexID* embedding_additions;
    embedding_additions = (VertexID*)new char[embedding_additions_size];
    int* map_orig_embedding_to_additions;
    map_orig_embedding_to_additions = (int*)new char[map_orig_embedding_to_additions_size];
    
    EXECUTE_CUDA_FUNC (hipMemcpy (&embeddings_addition_iter, device_embeddings_addition_iter, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    std::cout << "Embedding Additions " << embeddings_addition_iter << std::endl;
    EXECUTE_CUDA_FUNC (hipMemcpy (embedding_additions, device_embeddings_additions, embedding_additions_size, hipMemcpyDeviceToHost));
    EXECUTE_CUDA_FUNC (hipMemcpy (map_orig_embedding_to_additions, device_map_orig_embedding_to_additions, map_orig_embedding_to_additions_size, hipMemcpyDeviceToHost));
    
    //Create new embeddings from the received additions
    for (int input_embedding_idx = 0; input_embedding_idx < input_embeddings.size (); input_embedding_idx++) {
      VectorVertexEmbedding& input_embedding = input_embeddings[input_embedding_idx];
      int n_additions = map_orig_embedding_to_additions[2*input_embedding_idx+1];
      int start_idx = map_orig_embedding_to_additions[2*input_embedding_idx];
      size_t produced_embedding_size = input_embedding.get_n_vertices () + n_additions;
      if (input_embedding_idx == 48) {
        std::cout << "n_additions " << n_additions << std::endl;
      }
      size_t global_mem_idx = GlobalMemAllocator::alloc_vertices_array (produced_embedding_size);
      // std::cout << "i " << input_embedding_idx << " produced_embedding_size " << produced_embedding_size << " global_mem_idx " << global_mem_idx << std::endl;
      VertexID* ptr = (VertexID*) ((char*)GlobalMemAllocator::get_global_mem_ptr () + global_mem_idx);
      ((VertexID*)ptr)[0] = ((VertexID*)input_embedding.get_array ())[0];
      memcpy (ptr, input_embedding.get_array (), sizeof(VertexID)*input_embedding.get_n_vertices ());
      memcpy (ptr + input_embedding.get_n_vertices (), &embedding_additions[start_idx], sizeof(VertexID)*n_additions);
      
      produced_embeddings.push_back (VectorVertexEmbedding ((uint32_t)produced_embedding_size, global_mem_idx, true));
    }

    input_embeddings = produced_embeddings;

    hipFree (device_csr);
    hipFree (device_embeddings_additions);
    hipFree (device_map_orig_embedding_to_additions);
    hipFree (device_input_embeddings_storage); 
    hipFree (device_input_embeddings);
    hipFree (device_embeddings_addition_iter);
  }
#endif

#ifdef PINNED_MEMORY
  // hipFree (global_mem_ptr);
#else
  delete[] global_mem_ptr;
#endif
  std::cout << "Number of embeddings found "<< input_embeddings.size () << std::endl;
  std::cout << "Time spent in GPU kernel execution " << kernelTotalTime << std::endl;
  std::cout << "Time spent in Streams " << total_stream_time << std::endl;
}
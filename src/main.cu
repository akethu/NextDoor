
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>

#include <string.h>
#include <assert.h>

#define LINE_SIZE 1024*1024
//#define USE_FIXED_THREADS
#define MAX_CUDA_THREADS (96*96)
#define THREAD_BLOCK_SIZE 256
#define WARP_SIZE 32
//#define USE_CSR_IN_SHARED
//#define USE_EMBEDDING_IN_SHARED_MEM
//#define USE_EMBEDDING_IN_GLOBAL_MEM
#define USE_EMBEDDING_IN_LOCAL_MEM
//#define SHARED_MEM_NON_COALESCING

#ifdef SHARED_MEM_NON_COALESCING
  #ifndef USE_EMBEDDING_IN_SHARED_MEM
    #error "USE_EMBEDDING_IN_SHARED_MEM must be enabled with SHARED_MEM_NON_COALESCING"
  #endif
#endif
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #ifdef USE_FIXED_THREADS
    #error "USE_FIXED_THREADS cannot be enabled with USE_EMBEDDING_IN_SHARED_MEM"
  #endif
#endif

//#define USE_CONSTANT_MEM

typedef uint8_t SharedMemElem;

//citeseer.graph
const int N = 3312;
const int N_EDGES = 9074;

//micro.graph
//const int N = 100000;
//const int N_EDGES = 2160312;

class Vertex
{
private:
  int id;
  int label;
  std::vector <int> edges;

public:
  Vertex (int _id, int _label) : label(_label), id (_id)
  {
  }

  int get_id () {return id;}
  int get_label () {return label;}
  void add_edge (int vertexID) {edges.push_back (vertexID);}
  std::vector <int>& get_edges () {return edges;}
  void print (std::ostream& os)
  {
    os << id << " " << label << " ";
    for (auto edge : edges) {
      os << edge << " ";
    }

    os << std::endl;
  }
};

int chars_in_int (int num)
{
  if (num == 0) return sizeof(char);
  return (int)((ceil(log10(num))+1)*sizeof(char));
}

class Graph
{
private:
  std::vector<Vertex> vertices;
  int n_edges;

public:
  Graph (std::vector<Vertex> _vertices, int _n_edges) :
    vertices (_vertices), n_edges(_n_edges)
  {}

  const std::vector<Vertex>& get_vertices () {return vertices;}
  int get_n_edges () {return n_edges;}
};

class CSR
{
public:
  struct Vertex
  {
    int id;
    int label;
    int start_edge_id;
    int end_edge_id;
    __host__ __device__
    Vertex ()
    {
      id = -1;
      label = -1;
      start_edge_id = -1;
      end_edge_id = -1;
    }

    void set_from_graph_vertex (::Vertex& vertex)
    {
      id = vertex.get_id ();
      label = vertex.get_label ();
    }

    void set_start_edge_id (int start) {start_edge_id = start;}
    void set_end_edge_id (int end) {end_edge_id = end;}
  };

  typedef int Edge;

public:
  CSR::Vertex vertices[N];
  CSR::Edge edges[N_EDGES];
  int n_vertices;
  int n_edges;

public:
  CSR (int _n_vertices, int _n_edges)
  {
    n_vertices = _n_vertices;
    n_edges = _n_edges;
  }

  __host__ __device__
  CSR ()
  {
    n_vertices = N;
    n_edges = N_EDGES;
  }

  void print (std::ostream& os)
  {
    for (int i = 0; i < n_vertices; i++) {
      os << vertices[i].id << " " << vertices[i].label << " ";
      for (int edge_iter = vertices[i].start_edge_id;
           edge_iter <= vertices[i].end_edge_id; edge_iter++) {
        os << edges[edge_iter] << " ";
      }
      os << std::endl;
    }
  }

  __host__ __device__
  int get_start_edge_idx (int vertex_id)
  {
    if (!(vertex_id < n_vertices && 0 <= vertex_id)) {
      printf ("vertex_id %d, n_vertices %d\n", vertex_id, n_vertices);
      assert (false);
    }
    return vertices[vertex_id].start_edge_id;
  }

  __host__ __device__
  int get_end_edge_idx (int vertex_id)
  {
    assert (vertex_id < n_vertices && 0 <= vertex_id);
    return vertices[vertex_id].end_edge_id;
  }

  __host__ __device__
  bool has_edge (int u, int v)
  {
    for (int e = get_start_edge_idx (u); e <= get_end_edge_idx (u); e++) {
      if (edges[e] == v) {
        return true;
      }
    }

    return false;
  }

  __host__ __device__
  const CSR::Edge* get_edges () {return &edges[0];}

  __host__ __device__
  const CSR::Vertex* get_vertices () {return &vertices[0];}

  __host__ __device__
  int get_n_vertices () {return n_vertices;}

  __host__ __device__
  void copy_vertices (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      vertices[i] = src->get_vertices()[i];
    }
  }

  __host__ __device__
  void copy_edges (CSR* src, int start, int end)
  {
    for (int i = start; i < end; i++) {
      edges[i] = src->get_edges ()[i];
    }
  }

  __host__ __device__
  int get_n_edges () {return n_edges;}
};

#ifdef USE_CONSTANT_MEM
  __constant__ unsigned char csr_constant_buff[sizeof(CSR)];
#endif

void csr_from_graph (CSR* csr, Graph& graph)
{
  int edge_iterator = 0;
  auto graph_vertices = graph.get_vertices ();
  for (int i = 0; i < graph_vertices.size (); i++) {
    ::Vertex& vertex = graph_vertices[i];
    csr->vertices[i].set_from_graph_vertex (graph_vertices[i]);
    csr->vertices[i].set_start_edge_id (edge_iterator);
    for (auto edge : vertex.get_edges ()) {
      csr->edges[edge_iterator] = edge;
      edge_iterator++;
    }

    csr->vertices[i].set_end_edge_id (edge_iterator-1);
  }
}

//template <size_t N> using VertexEmbedding = std::bitset<N>;

#define CVT_TO_NEXT_MULTIPLE(n,k) ((n) %(k) ==0 ? (n) : ((n)/(k)+1)*(k))
class BitVectorVertexEmbedding
{
private:
#ifdef USE_EMBEDDING_IN_SHARED_MEM
  unsigned char array[CVT_TO_NEXT_MULTIPLE(N/8, 32*sizeof(SharedMemElem))];
#else
  unsigned char array[(N/8)];
#endif

public:
  __device__ __host__
  BitVectorVertexEmbedding ()
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    assert (array != nullptr);
    reset ();
    assert (all_false () == true);
  }

  __host__ __device__
  size_t convert_to_bytes_multiple (size_t n)
  {
    return (n/8)*8;
  }

  __host__ __device__
  BitVectorVertexEmbedding (const BitVectorVertexEmbedding& embedding)
  {
    //array = new unsigned char[convert_to_bytes_multiple(N)/8];
    for (int i = 0; i <  convert_to_bytes_multiple(N)/8; i++) {
      array[i] = embedding.array[i];
    }
  }

  __host__ __device__
  void set (int index)
  {
    assert (index >= 0 and index < N);
    array[index/8] = array[index/8] | (1 << (index %8));
  }

  __host__ __device__
  void set ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = (unsigned char) (~(0UL));
    }
  }

  __host__ __device__
  void reset ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      array[i] = 0;
    }
  }

  __host__ __device__
  void reset (int index)
  {
    assert (index >= 0 and index < N);
    array[index/8] = array[index/8] & (~(1UL << (index %8)));
  }

  __host__ __device__
  bool test (int index)
  {
    return (bool) ((array[index/8] >> (index % 8))&1);
  }

  __host__ __device__
  bool all_false ()
  {
    for (int i = 0; i < convert_to_bytes_multiple(N)/8; i++) {
      if (array[i] != 0UL) {
        return false;
      }
    }

    return true;
  }
  
  __host__ __device__
  int get_n_vertices () 
  {
    int n_vertices = 0;
    for (int i = 0; i < N; i++) {
      if (test(i) == true) {
        n_vertices++;
      }
    }
    
    return n_vertices;
  }
  
  __host__ __device__
  ~BitVectorVertexEmbedding ()
  {
    //delete[] array;
  }
};

//typedef BitVectorVertexEmbedding VertexEmbedding;

template <uint32_t size> 
class VectorVertexEmbedding
{
private:
  uint32_t array[size];
  uint32_t filled_size;
  
public:
  __device__ __host__
  VectorVertexEmbedding ()
  {
    filled_size = 0;
  }

  __host__ __device__
  VectorVertexEmbedding (const VectorVertexEmbedding<size>& embedding)
  {
  #if DEBUG
    assert (embedding.get_max_size () <= get_max_size ());
  #endif
    filled_size = 0;
    for (int i = 0; i < embedding.get_n_vertices (); i++) {
      add (embedding.get_vertex (i));
    }
  }
  
  __host__ __device__
  void add (int v)
  {
  #if DEBUG
    if (!(size != 0 and filled_size < size)) {
      printf ("filled_size %d, size %d\n", filled_size, size);
      //assert (size != 0 and filled_size < size);
      assert (false);
    }
  #endif
    
    int pos = 0;
    
    for (int i = 0; i < filled_size; i++) {
      if (array[i] > v) {
        pos = i;
        break;
      }
    }
    
    for (int i = filled_size-1; i >= pos ; i--) {
      array[i+1] = array[i];
    }
    
    array[pos] = v;
    filled_size++;
  }

  __host__ __device__
  void add_last_in_sort_order () 
  {
    int v = array[filled_size-1];
    remove_last ();
    add (v);
  }

  __host__ __device__
  void add_unsorted (int v) 
  {
    array[filled_size++] = v;
  }
  
  __host__ __device__
  void remove (int v)
  {
    printf ("Do not support remove\n");
    assert (false);
  }
  
  __host__ __device__
  bool has_logn (int v)
  {
    int l = 0;
    int r = filled_size-1;
    
    while (l <= r) {
      int m = l+(r-l)/2;
      
      if (array[m] == v)
        return true;
      
      if (array[m] < v)
        l = m + 1;
      else
        r = m - 1;
    }
    
    return false;
  }
  
  __host__ __device__
  bool has (int v)
  {
    for (int i = 0; i < filled_size; i++) {
      if (array[i] == v) {
        return true;
      }
    }
    
    return false;
  }
  
  __host__ __device__
  size_t get_n_vertices () const
  {
    return filled_size;
  }
  
  __host__ __device__
  int get_vertex (int index) const
  {
    return array[index];
  }
  
  __host__ __device__
  int get_last_vertex () const
  {
    return array[filled_size-1];
  }
  
  __host__ __device__
  size_t get_max_size () const
  {
    return size;
  }
  
  __host__ __device__
  void clear ()
  {
    filled_size = 0;
  }
  
  __host__ __device__
  void remove_last () 
  {
    assert (filled_size > 0);
    filled_size--;
  }
  __host__ __device__
  ~VectorVertexEmbedding ()
  {
    //delete[] array;
  }
};

template <uint32_t size>
__host__ __device__
void vector_embedding_from_one_less_size (VectorVertexEmbedding<size>& vec_emb1,
                                          VectorVertexEmbedding<size+1>& vec_emb2)
{
  //TODO: Optimize here, filled_size++ in add is being called several times
  //but can be called only once too  
  for (int i = 0; i < vec_emb1.get_n_vertices (); i++) {
    vec_emb2.add (vec_emb1.get_vertex (i));
  }
}

template <uint32_t size> 
void bitvector_to_vector_embedding (BitVectorVertexEmbedding& bit_emb, 
                                    VectorVertexEmbedding<size>& vec_emb)
{
  for (int u = 0; u < N; u++) {
    if (bit_emb.test(u)) {
      vec_emb.add (u);
    }
  }
}

void print_embedding (BitVectorVertexEmbedding embedding, std::ostream& os);

std::vector<BitVectorVertexEmbedding> get_extensions_bitvector (BitVectorVertexEmbedding& embedding, CSR* csr)
{
  std::vector<BitVectorVertexEmbedding> extensions;

  if (embedding.all_false ()) {
    for (int u = 0; u < N; u++) {
      BitVectorVertexEmbedding extension;
      extension.set(u);
      //print_embedding (extension, std::cout);
      //std::cout << " " << u << std::endl;
      extensions.push_back (extension);
    }
  } else {
    for (int u = 0; u < N; u++) {
      if (embedding.test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding.test (v) == false) {
            BitVectorVertexEmbedding extension = BitVectorVertexEmbedding(embedding);
            extension.set(v);
            extensions.push_back(extension);
          }
        }
      }
    }
  }

  return extensions;
}

template <uint32_t size>
std::vector<VectorVertexEmbedding<size+1>> get_extensions_vector (VectorVertexEmbedding<size>& embedding, CSR* csr)
{
  std::vector<VectorVertexEmbedding<size+1>> extensions;

  if (embedding.get_n_vertices () == 0) {
    for (int u = 0; u < N; u++) {
      VectorVertexEmbedding<size+1> extension;
      extension.add(u);
      //print_embedding (extension, std::cout);
      //std::cout << " " << u << std::endl;
      extensions.push_back (extension);
    }
  } else {
    for (int i = 0; i < embedding.get_n_vertices (); i++) {
      int u = embedding.get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        int v = csr->get_edges () [e];
        if (embedding.has (v) == false) {
          VectorVertexEmbedding<size+1> extension;
          vector_embedding_from_one_less_size (embedding, extension);
          extension.add(v);
          extensions.push_back(extension);
        }
      }
    }
  }

  return extensions;
}

std::vector<BitVectorVertexEmbedding> get_initial_embedding_bitvector (CSR* csr)
{
  BitVectorVertexEmbedding embedding;
  std::vector <BitVectorVertexEmbedding> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

std::vector<VectorVertexEmbedding<0>> get_initial_embedding_vector (CSR* csr)
{
  VectorVertexEmbedding<0> embedding;
  std::vector <VectorVertexEmbedding<0>> embeddings;

  embeddings.push_back (embedding);

  return embeddings;
}

bool (*filter) (CSR* csr, BitVectorVertexEmbedding& embedding);
void (*process) (std::vector<BitVectorVertexEmbedding>& output, BitVectorVertexEmbedding& embedding);

__host__ __device__
bool clique_filter (CSR* csr, BitVectorVertexEmbedding* embedding)
{
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      for (int v = 0; v < N; v++) {
        if (u != v and embedding->test(v)) {
          if (!csr->has_edge (u, v)) {
            return false;
          }
        }
      }
    }
  }

  return true;
}


template <uint32_t size>
__host__ __device__
bool clique_filter_vector (CSR* csr, VectorVertexEmbedding<size>* embedding)
{
  for (int i = 0; i < embedding->get_n_vertices (); i++) {
    int u = embedding->get_vertex (i);
    for (int j = 0; j < embedding->get_n_vertices (); j++) {
      int v = embedding->get_vertex (j);
      if (u != v and embedding->has (v)) {
        if (!csr->has_edge (u, v)) {
          return false;
        }
      }
    }
  }

  return true;
}

void clique_process_bit_vector (std::vector<BitVectorVertexEmbedding>& output, BitVectorVertexEmbedding& embedding)
{
  output.push_back (embedding);
}

template <uint32_t size>
void clique_process_vector (std::vector<VectorVertexEmbedding<size>>& output, VectorVertexEmbedding<size>& embedding)
{
  output.push_back (embedding);
}

void run_single_step_initial_bitvector (void* input, int n_embeddings, CSR* csr,
                      std::vector<BitVectorVertexEmbedding>& output,
                      std::vector<BitVectorVertexEmbedding>& next_step)
{
  BitVectorVertexEmbedding* embeddings = (BitVectorVertexEmbedding*)input;

  for (int i = 0; i < n_embeddings; i++) {
    BitVectorVertexEmbedding embedding = embeddings[i];
    std::vector<BitVectorVertexEmbedding> extensions = get_extensions_bitvector (embedding, csr);

    for (auto extension : extensions) {
      if (clique_filter (csr, &extension)) {
        clique_process_bit_vector (output, extension);
        next_step.push_back (extension);
      }
    }
  }
}

void run_single_step_initial_vector (void* input, int n_embeddings, CSR* csr,
                      std::vector<VectorVertexEmbedding<1>>& output,
                      std::vector<VectorVertexEmbedding<1>>& next_step)
{
  VectorVertexEmbedding<0>* embeddings = (VectorVertexEmbedding<0>*)input;

  for (int i = 0; i < n_embeddings; i++) {
    VectorVertexEmbedding<0> embedding = embeddings[i];
    std::vector<VectorVertexEmbedding<1>> extensions = get_extensions_vector (embedding, csr);
    std::cout << "extensions " << extensions.size () << std::endl;
    for (auto extension : extensions) {
      if (clique_filter_vector (csr, &extension)) {
        clique_process_vector (output, extension);
        next_step.push_back (extension);
      }
    }
  }
}

__device__
void printf_embedding (BitVectorVertexEmbedding* embedding)
{
  printf ("[");
  for (int u = 0; u < N; u++) {
    if (embedding->test(u)) {
      printf ("%d, ", u);
    }
  }

  printf ("]\n");
}

/*__global__
void run_single_step_bitvector_embedding (void* input, int n_embeddings, CSR* csr,
                      void* output_ptr,
                      int* n_output,
                      void* next_step, int* n_next_step,
                      int* n_output_1, int* n_next_step_1)
{
  int id;

#ifdef USE_CSR_IN_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread,
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread,
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#else
#ifdef USE_CONSTANT_MEM
  csr = (CSR*) csr_constant_buff;
#endif

#endif

  BitVectorVertexEmbedding* embeddings = (BitVectorVertexEmbedding*)input;
  BitVectorVertexEmbedding* new_embeddings = (BitVectorVertexEmbedding*)next_step;
  BitVectorVertexEmbedding* output = ((BitVectorVertexEmbedding*)output_ptr);
#ifdef USE_EMBEDDING_IN_LOCAL_MEM
  unsigned char temp_buffer [sizeof(BitVectorVertexEmbedding)];
#endif
  id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
#ifdef USE_FIXED_THREADS
  if (n_embeddings >= MAX_CUDA_THREADS) {
    int embeddings_per_thread = n_embeddings/MAX_CUDA_THREADS+1;

    start = id*embeddings_per_thread;
    end = (id+1)*embeddings_per_thread < n_embeddings ? (id+1)*embeddings_per_thread : n_embeddings;
  } else {
    if (id >= n_embeddings)
      return;

    start = id;
    end = id+1;
  }
#else
  if (id >= n_embeddings)
      return;

  start = id;
  end = id+1;
#endif

  int q[1000] = {0};

#ifdef USE_EMBEDDING_IN_SHARED_MEM
  #if 0
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[per_thread_shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[0];
  #else
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
  #endif
#endif

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int lane_id = threadIdx.x % WARP_SIZE;
  for (int i = start; i < end; i++) {
    #ifdef USE_EMBEDDING_IN_SHARED_MEM
      #ifdef SHARED_MEM_NON_COALESCING
        memcpy (local_shared_buff, &embeddings[i], sizeof(BitVectorVertexEmbedding));

        BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*) local_shared_buff;
      #else
        const int thread_block_size = WARP_SIZE;
        const int last_emb = WARP_SIZE*(warp_id+1);
        if (blockIdx.x*blockDim.x + (warp_id+1)*WARP_SIZE > n_embeddings) {
          assert (false);
          //thread_block_size = n_embeddings - blockIdx.x*blockDim.x -
          //                    warp_id*WARP_SIZE;
          //last_emb = warp_id*WARP_SIZE + thread_block_size;
        }

        for (int emb = WARP_SIZE*warp_id; emb < last_emb; emb++) {
          SharedMemElem* embedding_buff = (SharedMemElem*) &embeddings[emb+blockIdx.x*blockDim.x];

          for (int j = 0; j < sizeof (BitVectorVertexEmbedding)/sizeof (SharedMemElem);
               j += thread_block_size) {
            int idx = per_thread_shared_mem_size/sizeof(SharedMemElem)*emb;
            if (true or j + lane_id  < sizeof (BitVectorVertexEmbedding)/sizeof (SharedMemElem)) { //TODO: Remove this if by doing padding with VertexEmbedding
              shared_buff[idx + j + lane_id] = embedding_buff[j + lane_id];
            }
          }
        }

        BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*) local_shared_buff;
      #endif
      //embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_LOCAL_MEM)
      //memcpy (&temp[0], &embeddings[i], sizeof (VertexEmbedding));
      memcpy (&temp_buffer[0], &embeddings[i], sizeof(BitVectorVertexEmbedding));
      BitVectorVertexEmbedding* embedding = (BitVectorVertexEmbedding*)&temp_buffer[0];
      //VertexEmbedding* embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_GLOBAL_MEM)
      BitVectorVertexEmbedding* embedding = &embeddings[i];
    #else
      #error "None of USE_EMBEDDING_IN_*_MEM option defined"
    #endif

  #if 1
    typedef uint32_t VertexEmbeddingChange;
    const uint32_t max_changes = 32; //max changes per warp
    //__shared__ int32_t shared_mem [max_changes*THREAD_BLOCK_SIZE/WARP_SIZE+THREAD_BLOCK_SIZE/WARP_SIZE+1];
    __shared__ VertexEmbeddingChange changes[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t changed_thread_ids[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    //For each warp record new embeddings (or changes) using atomic operations
    __shared__ int32_t n_extensions[THREAD_BLOCK_SIZE/WARP_SIZE];
    n_extensions[warp_id] = 0;
    __shared__ uint32_t prev_n_outputs [THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t prev_n_next_steps [THREAD_BLOCK_SIZE/WARP_SIZE];
    
    const uint32_t mask = ~0U;
    for (int u = 0; u < N; u++) {
      int e = csr->get_start_edge_idx(u);
      
      while (true) {
        
        int predicate = 1;
        //n_extensions[warp_id] = 0;

        if (e <= csr->get_end_edge_idx(u)) {
          int v = csr->get_edges () [e];
          if (embedding->test(u) and embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              int prev_n_extensions = atomicAdd (&n_extensions[warp_id], 1);
              changes[warp_id*max_changes+prev_n_extensions] = v;
              changed_thread_ids[warp_id*max_changes+prev_n_extensions] = id;
              //memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (VertexEmbedding));
              //memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (VertexEmbedding));
            }

            extension->reset(v);
          }

          e++;

          if (e > csr->get_end_edge_idx(u)) {
            predicate = 0;
          }
        } else {
          predicate = 0;
        }

        int32_t n_changes = n_extensions[warp_id];

        if (n_changes > max_changes)
          assert (false);

        //assert (__activemask () == ~0U);
        uint32_t orig_prev_n_output = 0;
        uint32_t orig_prev_n_next_step = 0;
        
        if (lane_id == 0) {
          if (n_changes > 0) {
            n_extensions[warp_id] = 0;
            orig_prev_n_output = atomicAdd (n_output, n_changes);
            orig_prev_n_next_step = atomicAdd (n_next_step, n_changes);
            //if (warp_id == 2 && (threadIdx.x == 64 || threadIdx.x == 65) && blockIdx.x == 0) {
            //  printf ("prev_n_next_steps[warp_id]: %d\n", prev_n_next_steps[warp_id]);
            //}
          }
        }
        
        //__syncwarp ();
        
        if (n_changes > 0) {
          
          uint32_t prev_n_output = __shfl_sync (__activemask (), orig_prev_n_output, 0);
          uint32_t prev_n_next_step = __shfl_sync (__activemask (), orig_prev_n_next_step, 0);
          
          for (int i = 0; i < n_changes; i++) {
            
            int changes_idx = warp_id*max_changes + i;
            uint32_t expected_thread_id = changed_thread_ids[changes_idx];
            if (expected_thread_id == id) {
              int v = changes[changes_idx];
              BitVectorVertexEmbedding* extension = embedding;
              extension->set (v);
              
              memcpy (&output[prev_n_output + i], extension, 
                      sizeof(BitVectorVertexEmbedding));
              memcpy (&new_embeddings[prev_n_next_step + i], extension,
                      sizeof (BitVectorVertexEmbedding));
              extension->reset (v);
            }
          }
        }

        if (n_extensions[warp_id] != 0) {
          printf ("n_extensions[warp_id] not zero but is %d\n", n_extensions[warp_id]);
        }
        if (!__any_sync (__activemask (), predicate)) {
          break;
        }
        //n_extensions[warp_id] = 0;
      }
    }
  #else
    for (int u = 0; u < N; u++) {
      if (embedding->test(u)) {
        for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
          int v = csr->get_edges () [e];
          if (embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (BitVectorVertexEmbedding));
              memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (BitVectorVertexEmbedding));
            }
            extension->reset(v);
          }
        }
      }
    }
  #endif

  }

  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}*/

template <size_t embedding_size> 
__global__
void run_single_step_vectorvertex_embedding (void* input, int n_embeddings, CSR* csr,
                      void* output_ptr,
                      int* n_output,
                      void* next_step, int* n_next_step,
                      int* n_output_1, int* n_next_step_1)
{
  int id;

#ifdef USE_CSR_IN_SHARED
  __shared__ unsigned char csr_shared_buff[sizeof (CSR)];
  id = threadIdx.x;
  CSR* csr_shared = (CSR*) csr_shared_buff;
  csr_shared->n_vertices = csr->get_n_vertices ();
  csr_shared->n_edges = csr->get_n_edges ();

  int vertices_per_thread = csr->get_n_vertices ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_vertices (csr, id*vertices_per_thread,
                             (id+1)*vertices_per_thread < csr->get_n_vertices () ? (id+1)*vertices_per_thread : csr->get_n_vertices ());

  int edges_per_thread = csr->get_n_edges ()/THREAD_BLOCK_SIZE + 1;
  csr_shared->copy_edges (csr, id*edges_per_thread,
                          (id+1)*edges_per_thread < csr->get_n_edges () ? (id+1)*edges_per_thread : csr->get_n_edges ());
  csr = csr_shared;
  __syncthreads ();
#else
#ifdef USE_CONSTANT_MEM
  csr = (CSR*) csr_constant_buff;
#endif

#endif

  VectorVertexEmbedding<embedding_size>* embeddings = (VectorVertexEmbedding<embedding_size>*)input;
  VectorVertexEmbedding<embedding_size+1>* new_embeddings = (VectorVertexEmbedding<embedding_size+1>*)next_step;
  VectorVertexEmbedding<embedding_size+1>* output = (VectorVertexEmbedding<embedding_size+1>*)output_ptr;
#ifdef USE_EMBEDDING_IN_LOCAL_MEM
  unsigned char temp_buffer [sizeof(VectorVertexEmbedding<embedding_size+1>)];
#endif
  id = blockIdx.x*blockDim.x + threadIdx.x;
  int start = id, end = id+1;
  //printf ("running id %d\n", id);
#ifdef USE_FIXED_THREADS
  if (n_embeddings >= MAX_CUDA_THREADS) {
    int embeddings_per_thread = n_embeddings/MAX_CUDA_THREADS+1;

    start = id*embeddings_per_thread;
    end = (id+1)*embeddings_per_thread < n_embeddings ? (id+1)*embeddings_per_thread : n_embeddings;
  } else {
    if (id >= n_embeddings)
      return;

    start = id;
    end = id+1;
  }
#else
  if (id >= n_embeddings)
      return;

  start = id;
  end = id+1;
#endif

  int q[1000] = {0};

#ifdef USE_EMBEDDING_IN_SHARED_MEM
//TODO: Support VectorVertexEmbedding
  #if 0
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    assert (per_thread_shared_mem_size >= sizeof (VectorVertexEmbedding<embedding_size>));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[per_thread_shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[0];
  #else
    const int shared_mem_size = 49152;
    const int per_thread_shared_mem_size = shared_mem_size/THREAD_BLOCK_SIZE;

    //assert (per_thread_shared_mem_size >= sizeof (VertexEmbedding));
    //per_thread_shared_mem_size = sizeof (VertexEmbedding);
    __shared__ SharedMemElem shared_buff[shared_mem_size/sizeof (SharedMemElem)];

    SharedMemElem* local_shared_buff = &shared_buff[per_thread_shared_mem_size/sizeof(SharedMemElem)*threadIdx.x];
  #endif
#endif

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int lane_id = threadIdx.x % WARP_SIZE;
  for (int i = start; i < end; i++) {
    #ifdef USE_EMBEDDING_IN_SHARED_MEM
    //TODO: Support VectorVertexEmbedding, size+1
      #ifdef SHARED_MEM_NON_COALESCING
        memcpy (local_shared_buff, &embeddings[i], sizeof(VectorVertexEmbedding<embedding_size>));

        VectorVertexEmbedding<embedding_size>* embedding = (VectorVertexEmbedding<embedding_size>*) local_shared_buff;
      #else
        const int thread_block_size = WARP_SIZE;
        const int last_emb = WARP_SIZE*(warp_id+1);
        if (blockIdx.x*blockDim.x + (warp_id+1)*WARP_SIZE > n_embeddings) {
          assert (false);
          /*thread_block_size = n_embeddings - blockIdx.x*blockDim.x -
                              warp_id*WARP_SIZE;
          last_emb = warp_id*WARP_SIZE + thread_block_size;*/
        }

        for (int emb = WARP_SIZE*warp_id; emb < last_emb; emb++) {
          SharedMemElem* embedding_buff = (SharedMemElem*) &embeddings[emb+blockIdx.x*blockDim.x];

          for (int j = 0; j < sizeof (VectorVertexEmbedding<embedding_size>)/sizeof (SharedMemElem);
               j += thread_block_size) {
            int idx = per_thread_shared_mem_size/sizeof(SharedMemElem)*emb;
            if (true or j + lane_id  < sizeof (VectorVertexEmbedding<embedding_size>)/sizeof (SharedMemElem)) { //TODO: Remove this if by doing padding with VertexEmbedding
              shared_buff[idx + j + lane_id] = embedding_buff[j + lane_id];
            }
          }
        }

        VectorVertexEmbedding<embedding_size>* embedding = (VectorVertexEmbedding<embedding_size>*) local_shared_buff;
      #endif
      //embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_LOCAL_MEM)
      //memcpy (&temp[0], &embeddings[i], sizeof (VertexEmbedding));
      //memcpy (&temp_buffer[0], &embeddings[i], sizeof(VectorVertexEmbedding<embedding_size>));
      VectorVertexEmbedding<embedding_size+1>* embedding = (VectorVertexEmbedding<embedding_size+1>*)&temp_buffer[0];
      embedding->clear ();
      vector_embedding_from_one_less_size (embeddings[i], *embedding);
      //VertexEmbedding* embedding = &embeddings[i];
    #elif defined(USE_EMBEDDING_IN_GLOBAL_MEM)
    //TODO: Support VectorVertexEmbedding with size + 1, below is wrong
      VectorVertexEmbedding<embedding_size+1>* embedding = &embeddings[i];
    #else
      #error "None of USE_EMBEDDING_IN_*_MEM option defined"
    #endif

  #if 0
  //TODO: Support VectorVertexEmbedding with size + 1.
    typedef uint32_t VertexEmbeddingChange;
    const uint32_t max_changes = 32; //max changes per warp
    //__shared__ int32_t shared_mem [max_changes*THREAD_BLOCK_SIZE/WARP_SIZE+THREAD_BLOCK_SIZE/WARP_SIZE+1];
    __shared__ VertexEmbeddingChange changes[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t changed_thread_ids[max_changes*THREAD_BLOCK_SIZE/WARP_SIZE];
    //For each warp record new embeddings (or changes) using atomic operations
    __shared__ int32_t n_extensions[THREAD_BLOCK_SIZE/WARP_SIZE];
    n_extensions[warp_id] = 0;
    __shared__ uint32_t prev_n_outputs [THREAD_BLOCK_SIZE/WARP_SIZE];
    __shared__ uint32_t prev_n_next_steps [THREAD_BLOCK_SIZE/WARP_SIZE];
    
    const uint32_t mask = ~0U;
    for (int u = 0; u < N; u++) {
      int e = csr->get_start_edge_idx(u);
      
      while (true) {
        
        int predicate = 1;
        //n_extensions[warp_id] = 0;

        if (e <= csr->get_end_edge_idx(u)) {
          int v = csr->get_edges () [e];
          if (embedding->test(u) and embedding->test (v) == false) {
            BitVectorVertexEmbedding* extension = embedding;
            extension->set(v);
            if (clique_filter (csr, extension)) {
              int prev_n_extensions = atomicAdd (&n_extensions[warp_id], 1);
              changes[warp_id*max_changes+prev_n_extensions] = v;
              changed_thread_ids[warp_id*max_changes+prev_n_extensions] = id;
              //memcpy (&output[atomicAdd(n_output,1)], extension, sizeof (VertexEmbedding));
              //memcpy (&new_embeddings[atomicAdd(n_next_step,1)], extension, sizeof (VertexEmbedding));
            }

            extension->reset(v);
          }

          e++;

          if (e > csr->get_end_edge_idx(u)) {
            predicate = 0;
          }
        } else {
          predicate = 0;
        }

        int32_t n_changes = n_extensions[warp_id];

        if (n_changes > max_changes)
          assert (false);

        //assert (__activemask () == ~0U);
        uint32_t orig_prev_n_output = 0;
        uint32_t orig_prev_n_next_step = 0;
        
        if (lane_id == 0) {
          if (n_changes > 0) {
            n_extensions[warp_id] = 0;
            orig_prev_n_output = atomicAdd (n_output, n_changes);
            orig_prev_n_next_step = atomicAdd (n_next_step, n_changes);
            //if (warp_id == 2 && (threadIdx.x == 64 || threadIdx.x == 65) && blockIdx.x == 0) {
            //  printf ("prev_n_next_steps[warp_id]: %d\n", prev_n_next_steps[warp_id]);
            //}
          }
        }
        
        //__syncwarp ();
        
        if (n_changes > 0) {
          
          uint32_t prev_n_output = __shfl_sync (__activemask (), orig_prev_n_output, 0);
          uint32_t prev_n_next_step = __shfl_sync (__activemask (), orig_prev_n_next_step, 0);
          
          for (int i = 0; i < n_changes; i++) {
            
            int changes_idx = warp_id*max_changes + i;
            uint32_t expected_thread_id = changed_thread_ids[changes_idx];
            if (expected_thread_id == id) {
              int v = changes[changes_idx];
              BitVectorVertexEmbedding* extension = embedding;
              extension->set (v);
              
              memcpy (&output[prev_n_output + i], extension, 
                      sizeof(BitVectorVertexEmbedding));
              memcpy (&new_embeddings[prev_n_next_step + i], extension,
                      sizeof (BitVectorVertexEmbedding));
              extension->reset (v);
            }
          }
        }

        if (n_extensions[warp_id] != 0) {
          printf ("n_extensions[warp_id] not zero but is %d\n", n_extensions[warp_id]);
        }
        if (!__any_sync (__activemask (), predicate)) {
          break;
        }
        //n_extensions[warp_id] = 0;
      }
    }
  #else
    for (int i = 0; i < embedding->get_n_vertices (); i++) {
      int u = embedding->get_vertex (i);
      for (int e = csr->get_start_edge_idx(u); e <= csr->get_end_edge_idx(u); e++) {
        int v = csr->get_edges () [e];
        bool is_canonical = true;
        is_canonical = !(embedding->get_vertex (0) > v);
        /*if (is_canonical) {
          bool found_neighbor = false;
          for (int j = 0; j < embedding->get_n_vertices (); j++) {
            int v_j = embedding->get_vertex (j);
            if (found_neighbor == false && csr->has_edge (v_j, v)) {
              found_neighbor = true;
            } else if (found_neighbor == true && v_j > v) {
              is_canonical = false;
              break;
            }
          }
        }*/
        if (is_canonical && embedding->has (v) == false) {
          VectorVertexEmbedding<embedding_size+1>* extension = embedding;
          extension->add_unsorted (v);
          
          if (clique_filter_vector (csr, extension)) {
            //VectorVertexEmbedding<embedding_size+1> extension = *embedding;
            //extension.add_last_in_sort_order ();
            int o = atomicAdd(n_output,1);
            int n = atomicAdd(n_next_step,1);
            memcpy (&output[o], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
            memcpy (&new_embeddings[n], extension, sizeof (VectorVertexEmbedding<embedding_size+1>));
            //output[o].add_last_in_sort_order ();
            new_embeddings[n].add_last_in_sort_order ();
          }
          extension->remove_last ();
        }
      }
    }
  #endif

  }

  //printf ("embeddings generated [1000, 2000)= %d and [2000, 3000) = %d\n", q[0], q[1]);
  //printf ("embeddings at i = 1500: %d\n", q[2]);
  //for (int i = 100; i < 1000; i++) {
  //  printf ("embeddings at i = %d %d\n", i, q[i]);
  //}
}

void print_embedding (BitVectorVertexEmbedding embedding, std::ostream& os)
{
  os << "[";
  for (int u = 0; u < N; u++) {
    if (embedding.test(u)) {
      os << u << ", ";
    }
  }
  os << "]";
}

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

double_t convertTimeValToDouble (struct timeval _time)
{
  return ((double_t)_time.tv_sec) + ((double_t)_time.tv_usec)/1000000.0f;
}

struct timeval getTimeOfDay ()
{
  struct timeval _time;

  if (gettimeofday (&_time, NULL) == -1) {
    fprintf (stderr, "gettimeofday returned -1\n");
    perror ("");
    abort ();
  }

  return _time;
}

enum EmbeddingType {
  VectorVertex,
  BitVector,
};

int main (int argc, char* argv[])
{
  std::vector<Vertex> vertices;
  int n_edges = 0;

  if (argc < 2) {
    std::cout << "Arguments: graph-file" << std::endl;
    return -1;
  }

  char* graph_file = argv[1];
  FILE* fp = fopen (graph_file, "r+");
  if (fp == nullptr) {
    std::cout << "File '" << graph_file << "' not found" << std::endl;
    return 1;
  }

  while (true) {
    char line[LINE_SIZE];
    char num_str[LINE_SIZE];
    size_t line_size;

    if (fgets (line, LINE_SIZE, fp) == nullptr) {
      break;
    }

    int id, label;
    int bytes_read;

    bytes_read = sscanf (line, "%d %d", &id, &label);
    Vertex vertex (id, label);
    char* _line = line + chars_in_int (id) + chars_in_int (label);
    do {
      int num;

      bytes_read = sscanf (_line, "%d", &num);
      if (bytes_read > 0) {
        vertex.add_edge (num);
        _line += chars_in_int (num);
        n_edges++;
      }

    } while (bytes_read > 0);

    vertices.push_back (vertex);
  }

  fclose (fp);

  std::cout << "n_edges "<<n_edges <<std::endl;
  std::cout << "vertices " << vertices.size () << std::endl; 
  Graph graph (vertices, n_edges);

  CSR* csr = new CSR(N, N_EDGES);
  std::cout << "sizeof(CSR)"<< sizeof(CSR)<<std::endl;
  std::cout <<"sizeof(VertexEmbedding)" << sizeof(BitVectorVertexEmbedding) << std::endl;
  csr_from_graph (csr, graph);
  
#ifdef USE_CONSTANT_MEM
  cudaMemcpyToSymbol (csr_constant_buff, csr, sizeof(CSR));
  //~ CSR* csr_constant = (CSR*) &csr_constant_buff[0];
  //~ csr_constant->n_vertices = csr->get_n_vertices ();
  //~ printf ("csr->get_n_vertices () = %d\n", csr->get_n_vertices ());
  //~ csr_constant->n_edges = csr->get_n_edges ();
  //~ csr_constant->copy_vertices (csr, 0, csr->get_n_vertices ());
  //~ csr_constant->copy_edges (csr, 0, csr->get_n_edges ());
#endif

  std::vector<VectorVertexEmbedding<0>> initial_embeddings = get_initial_embedding_vector (csr);
  std::vector<VectorVertexEmbedding<1>> output_1;
  std::vector<VectorVertexEmbedding<2>> output_2;
  std::vector<VectorVertexEmbedding<3>> output_3;
  std::vector<VectorVertexEmbedding<4>> output_4;
  std::vector<VectorVertexEmbedding<5>> output_5;
  std::vector<VectorVertexEmbedding<6>> output_6;
  std::vector<VectorVertexEmbedding<7>> output_7;
  std::vector<VectorVertexEmbedding<8>> output_8;
  void* embeddings;
  //filter = clique_filter;
  //process = clique_process;
  size_t new_embeddings_size = 0;
  int iter = 0;
  {
    std::vector<VectorVertexEmbedding<1>> new_embeddings;
    run_single_step_initial_vector (&initial_embeddings[0], 1, csr, 
                                    output_1, new_embeddings);
    new_embeddings_size = new_embeddings.size ();
    embeddings = malloc (sizeof (VectorVertexEmbedding<1>)*new_embeddings_size);
    for (int i = 0; i < new_embeddings_size; i++) {
      ((VectorVertexEmbedding<1>*)embeddings)[i] = new_embeddings[i];
      int v = ((VectorVertexEmbedding<1>*)embeddings)[i].get_vertex (0);
      assert (v >= 0);
    }
  }

  iter = 1;

  double_t kernelTotalTime = 0.0;
  for (iter; iter < 8 && new_embeddings_size > 0; iter++) {
    std::cout << "iter " << iter << " embeddings " << new_embeddings_size << std::endl;
    size_t global_mem_size = 10*1024*1024*1024UL;
    char* global_mem_ptr = new char[global_mem_size];
  #ifdef DEBUG
    memset (global_mem_ptr, 0, global_mem_size);
  #endif
    int n_embeddings = new_embeddings_size;
    //n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
    std::cout << "iter " << iter << " n_embeddings " << n_embeddings << std::endl;
    size_t embedding_size = 0;
    size_t new_embedding_size = 0;
    switch (iter) {
      case 1: {
        embedding_size = sizeof (VectorVertexEmbedding<1>);
        new_embedding_size = sizeof (VectorVertexEmbedding<2>);
        for (int i = 0; i < n_embeddings; i++) {
          ((VectorVertexEmbedding<1>*)global_mem_ptr)[i] = ((VectorVertexEmbedding<1>*) embeddings)[i];
          int v = ((VectorVertexEmbedding<1>*)global_mem_ptr)[i].get_vertex (0);
          assert (v >= 0);
        }
        break;
      }      
      case 2: {
        embedding_size = sizeof (VectorVertexEmbedding<2>);
        new_embedding_size = sizeof (VectorVertexEmbedding<3>);
        for (int i = 0; i < n_embeddings; i++) {
          ((VectorVertexEmbedding<2>*)global_mem_ptr)[i] = ((VectorVertexEmbedding<2>*)embeddings)[i];
        }
        break;
      }
      
      case 3: {
        embedding_size = sizeof (VectorVertexEmbedding<3>);
        new_embedding_size = sizeof (VectorVertexEmbedding<4>);
        for (int i = 0; i < n_embeddings; i++) {
          ((VectorVertexEmbedding<3>*)global_mem_ptr)[i] = ((VectorVertexEmbedding<3>*)embeddings)[i];
        }
        break;
      }
      
      case 4: {
          embedding_size = sizeof (VectorVertexEmbedding<4>);
          new_embedding_size = sizeof (VectorVertexEmbedding<5>);
          for (int i = 0; i < n_embeddings; i++) {
            ((VectorVertexEmbedding<4>*)global_mem_ptr)[i] = ((VectorVertexEmbedding<4>*)embeddings)[i];
        }
        break;
      }
      case 5: {
        embedding_size = sizeof (VectorVertexEmbedding<5>);
        new_embedding_size = sizeof (VectorVertexEmbedding<6>);
        for (int i = 0; i < n_embeddings; i++) {
          ((VectorVertexEmbedding<5>*)global_mem_ptr)[i] = ((VectorVertexEmbedding<5>*)embeddings)[i];
        }
        break;
      }
      case 6: {
        embedding_size = sizeof (VectorVertexEmbedding<6>);
        new_embedding_size = sizeof (VectorVertexEmbedding<7>);
        for (int i = 0; i < n_embeddings; i++) {
          ((VectorVertexEmbedding<6>*)global_mem_ptr)[i] = ((VectorVertexEmbedding<6>*)embeddings)[i];
        }
        break;
      }
      case 7: {
        embedding_size = sizeof (VectorVertexEmbedding<7>);
        new_embedding_size = sizeof (VectorVertexEmbedding<8>);
        for (int i = 0; i < n_embeddings; i++) {
          ((VectorVertexEmbedding<7>*)global_mem_ptr)[i] = ((VectorVertexEmbedding<7>*)embeddings)[i];
        }
        break;
      }
      case 8: {
        embedding_size = sizeof (VectorVertexEmbedding<8>);
        new_embedding_size = sizeof (VectorVertexEmbedding<9>);
        for (int i = 0; i < n_embeddings; i++) {
          ((VectorVertexEmbedding<8>*)global_mem_ptr)[i] = ((VectorVertexEmbedding<8>*)embeddings)[i];
        }
        break;
      }
    }
    
    //TODO: delete embeddings too because there is a memory leak?
    if (iter > 1) {
      free(embeddings);
    }
    
    void* embeddings_ptr = global_mem_ptr;

    int n_new_embeddings = 0;
    int n_new_embeddings_1 = 0;
    void* new_embeddings_ptr = (char*)embeddings_ptr + (n_embeddings)*(new_embedding_size); //Size of next embedding will be one more
    size_t max_embeddings = 40000000; 
    printf ("new_embedding_size %ld\n", new_embedding_size);
    void* output_ptr = (char*)new_embeddings_ptr + (max_embeddings)*(new_embedding_size);
    int n_output = 0;
    int n_output_1 = 0;
    char* device_embeddings;
    char *device_new_embeddings;
    int* device_n_embeddings;
    int* device_n_embeddings_1;
    char *device_outputs;
    int* device_n_outputs;
    int* device_n_outputs_1;
    CSR* device_csr;
    
    const bool unified_mem = false;
    if (unified_mem == true) {
      //cudaMallocManaged (embeddings_ptr, n_embeddings*embedding_size);
      //device_embeddings = (char*)embeddings_ptr;
      assert(false);
    } else {
      hipMalloc (&device_embeddings, n_embeddings*embedding_size);
      hipMemcpy (device_embeddings, embeddings_ptr,
                  n_embeddings*embedding_size, hipMemcpyHostToDevice);
    }
    hipMalloc (&device_new_embeddings, max_embeddings*(new_embedding_size));
    hipMalloc (&device_outputs, max_embeddings*(new_embedding_size));
    hipMalloc (&device_n_embeddings, sizeof (0));
    hipMalloc (&device_n_embeddings_1, sizeof (0));
    hipMalloc (&device_n_outputs, sizeof (0));
    hipMalloc (&device_n_outputs_1, sizeof (0));
    hipMalloc (&device_csr, sizeof(CSR));
    
    hipMemcpy (device_n_embeddings, &n_new_embeddings,
                sizeof (n_new_embeddings), hipMemcpyHostToDevice);
    hipMemcpy (device_n_outputs, &n_output, sizeof (n_output),
                hipMemcpyHostToDevice);

    hipMemcpy (device_n_embeddings_1, &n_new_embeddings_1,
                sizeof (n_new_embeddings_1), hipMemcpyHostToDevice);
    hipMemcpy (device_n_outputs_1, &n_output_1, sizeof (n_output_1),
                hipMemcpyHostToDevice);

    hipMemcpy (device_csr, csr, sizeof (CSR), hipMemcpyHostToDevice);
    
    std::cout << "starting kernel with n_embeddings: " << n_embeddings;
  
    double t1 = convertTimeValToDouble (getTimeOfDay ());
    
#ifdef USE_FIXED_THREADS
    //std::cout << " threads: " << MAX_CUDA_THREADS/THREAD_BLOCK_SIZE << std::endl;
    int thread_blocks = MAX_CUDA_THREADS/THREAD_BLOCK_SIZE;
#else
    int thread_blocks = (n_embeddings%THREAD_BLOCK_SIZE != 0) ? (n_embeddings/THREAD_BLOCK_SIZE+1) : n_embeddings/THREAD_BLOCK_SIZE;
#endif
    std::cout << " threads: " << thread_blocks << std::endl;
    switch (iter) {
      case 1: {
        run_single_step_vectorvertex_embedding<1><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
        break;
      }
      case 2: {
        run_single_step_vectorvertex_embedding<2><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
        break;
      }
      case 3: {
        run_single_step_vectorvertex_embedding<3><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
        break;
      }
      case 4: {
        run_single_step_vectorvertex_embedding<4><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
        break;
      }
      case 5: {
        run_single_step_vectorvertex_embedding<5><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
        break;
      }
      case 6: {
        run_single_step_vectorvertex_embedding<6><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
        break;
      }
      case 7: {
        run_single_step_vectorvertex_embedding<7><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
        break;
      }
      case 8: {
        run_single_step_vectorvertex_embedding<8><<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
        break;
      }
    }
    
    hipDeviceSynchronize ();

    double t2 = convertTimeValToDouble (getTimeOfDay ());

    std::cout << "Execution time " << (t2-t1) << " secs" << std::endl;
    kernelTotalTime += (t2-t1);

    hipError_t error = hipGetLastError ();
    if (error != hipSuccess) {
      const char* error_string = hipGetErrorString (error);
      std::cout << error_string << std::endl;
    } else {
      std::cout << "Cuda success " << std::endl;
    }

    hipMemcpy (new_embeddings_ptr, device_new_embeddings, max_embeddings*(new_embedding_size), hipMemcpyDeviceToHost);
    hipMemcpy (output_ptr, device_outputs, max_embeddings*(new_embedding_size), hipMemcpyDeviceToHost);
    hipMemcpy (&n_new_embeddings, device_n_embeddings, sizeof(0), hipMemcpyDeviceToHost);
    hipMemcpy (&n_output, device_n_outputs, sizeof(0), hipMemcpyDeviceToHost);
    hipMemcpy (&n_new_embeddings_1, device_n_embeddings_1, sizeof(0), hipMemcpyDeviceToHost);
    hipMemcpy (&n_output_1, device_n_outputs_1, sizeof(0), hipMemcpyDeviceToHost);

    std::cout << "n_new_embeddings "<<n_new_embeddings<<std::endl;
    std::cout << "n_new_embeddings_1 "<<n_new_embeddings_1;
    std::cout << " n_output "<<n_output;
    std::cout << " n_output_1 "<<n_output_1<<std::endl;
    new_embeddings_size = n_new_embeddings;
    switch (iter) {
      case 1: {
        VectorVertexEmbedding<2>* new_embeddings = (VectorVertexEmbedding<2>*)malloc (sizeof (VectorVertexEmbedding<2>)*n_new_embeddings);
        
        for (int i = 0; i < n_new_embeddings; i++) {
          VectorVertexEmbedding<2> embedding = ((VectorVertexEmbedding<2>*)new_embeddings_ptr)[i];
          new_embeddings [i] = embedding;
          #ifdef DEBUG
          if (embedding.get_n_vertices () != (iter + 1)) {
            printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
          }
          #endif
        }
        
        embeddings = &new_embeddings[0];
        for (int i = 0; i < n_output; i++) {
          output_2.push_back (((VectorVertexEmbedding<2>*)output_ptr)[i]);
        }
        
        break;
      }
      
      case 2: {
        VectorVertexEmbedding<3>* new_embeddings = (VectorVertexEmbedding<3>*)malloc (sizeof (VectorVertexEmbedding<3>)*n_new_embeddings);
        
        for (int i = 0; i < n_new_embeddings; i++) {
          VectorVertexEmbedding<3> embedding = ((VectorVertexEmbedding<3>*)new_embeddings_ptr)[i];
          new_embeddings [i] = embedding;
          #ifdef DEBUG
          if (embedding.get_n_vertices () != (iter + 1)) {
            printf ("embedding has %ld vertices\n", embedding.get_n_vertices ());
          }
          #endif
        }
        
        embeddings = &new_embeddings[0];
        for (int i = 0; i < n_output; i++) {
          output_3.push_back (((VectorVertexEmbedding<3>*)output_ptr)[i]);
        }
        break;
      }
      
      case 3: {
        VectorVertexEmbedding<4>* new_embeddings = (VectorVertexEmbedding<4>*)malloc (sizeof (VectorVertexEmbedding<4>)*n_new_embeddings);
        
        for (int i = 0; i < n_new_embeddings; i++) {
          VectorVertexEmbedding<4> embedding = ((VectorVertexEmbedding<4>*)new_embeddings_ptr)[i];
          new_embeddings [i] = embedding;
          #ifdef DEBUG
          if (embedding.get_n_vertices () != (iter + 1)) {
            printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
          }
          #endif
        }
        
        embeddings = &new_embeddings[0];
        for (int i = 0; i < n_output; i++) {
          output_4.push_back (((VectorVertexEmbedding<4>*)output_ptr)[i]);
        }
        break;
      }
      
      case 4: {
        VectorVertexEmbedding<5>* new_embeddings = (VectorVertexEmbedding<5>*)malloc (sizeof (VectorVertexEmbedding<5>)*n_new_embeddings);
        
        for (int i = 0; i < n_new_embeddings; i++) {
          VectorVertexEmbedding<5> embedding = ((VectorVertexEmbedding<5>*)new_embeddings_ptr)[i];
          new_embeddings [i] = embedding;
          #ifdef DEBUG
          if (embedding.get_n_vertices () != (iter + 1)) {
            printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
          }
          #endif
        }
        
        embeddings = &new_embeddings[0];
        for (int i = 0; i < n_output; i++) {
          output_5.push_back (((VectorVertexEmbedding<5>*)output_ptr)[i]);
        }
        break;
      }
      
      case 5: {
        VectorVertexEmbedding<6>* new_embeddings = (VectorVertexEmbedding<6>*)malloc (sizeof (VectorVertexEmbedding<6>)*n_new_embeddings);
        
        for (int i = 0; i < n_new_embeddings; i++) {
          VectorVertexEmbedding<6> embedding = ((VectorVertexEmbedding<6>*)new_embeddings_ptr)[i];
          new_embeddings [i] = embedding;
          #ifdef DEBUG
          if (embedding.get_n_vertices () != (iter + 1)) {
            printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
          }
          #endif
        }
        
        embeddings = &new_embeddings[0];
        for (int i = 0; i < n_output; i++) {
          output_6.push_back (((VectorVertexEmbedding<6>*)output_ptr)[i]);
        }
        break;
      }
      
      case 6: {
        VectorVertexEmbedding<7>* new_embeddings = (VectorVertexEmbedding<7>*)malloc (sizeof (VectorVertexEmbedding<7>)*n_new_embeddings);
        
        for (int i = 0; i < n_new_embeddings; i++) {
          VectorVertexEmbedding<7> embedding = ((VectorVertexEmbedding<7>*)new_embeddings_ptr)[i];
          new_embeddings [i] = embedding;
          #ifdef DEBUG
          if (embedding.get_n_vertices () != (iter + 1)) {
            printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
          }
          #endif
        }
        
        embeddings = &new_embeddings[0];
        for (int i = 0; i < n_output; i++) {
          output_7.push_back (((VectorVertexEmbedding<7>*)output_ptr)[i]);
        }
        break;
      }
      
      case 7: {
        VectorVertexEmbedding<8>* new_embeddings = (VectorVertexEmbedding<8>*)malloc (sizeof(VectorVertexEmbedding<8>)*n_new_embeddings);
        
        for (int i = 0; i < n_new_embeddings; i++) {
          VectorVertexEmbedding<8> embedding = ((VectorVertexEmbedding<8>*)new_embeddings_ptr)[i];
          new_embeddings [i] = embedding;
          #ifdef DEBUG
          if (embedding.get_n_vertices () != (iter + 1)) {
            printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
          }
          #endif
        }
        
        embeddings = &new_embeddings[0];
        for (int i = 0; i < n_output; i++) {
          output_8.push_back (((VectorVertexEmbedding<8>*)output_ptr)[i]);
        }
        break;
      }
    }
    
    //embeddings = new_embeddings;

    hipFree (device_embeddings);
    hipFree (device_new_embeddings);
    hipFree (device_n_embeddings);
    hipFree (device_outputs);
    hipFree (device_n_outputs);
    hipFree (device_csr);
    delete[] global_mem_ptr;
  }

  std::cout << "Number of embeddings found "<< (output_1.size () + output_2.size () + output_3.size () + output_4.size () + output_5.size () + output_6.size () + output_7.size () + output_8.size ()) << std::endl;
  std::cout << "Time spent in execution " << kernelTotalTime << std::endl;
  
  
  /* For BitVectorVertexEmbedding
   for (iter; iter < 10 && embeddings.size () > 0; iter++) {
    std::cout << "iter " << iter << " embeddings " << embeddings.size () << std::endl;
    size_t global_mem_size = 3*1024*1024*1024UL;
    char* global_mem_ptr = new char[global_mem_size];
  #ifdef DEBUG
    memset (global_mem_ptr, 0, global_mem_size);
  #endif
    int n_embeddings = embeddings.size ();
    //n_embeddings = (n_embeddings/THREAD_BLOCK_SIZE)*THREAD_BLOCK_SIZE;
    std::cout << "iter " << iter << " n_embeddings " << n_embeddings << std::endl;
  
    for (int i = 0; i < n_embeddings; i++) {
      ((BitVectorVertexEmbedding*)global_mem_ptr)[i] = embeddings[i];
    }
    void* embeddings_ptr = global_mem_ptr;

    int n_new_embeddings = 0;
    int n_new_embeddings_1 = 0;
    void* new_embeddings_ptr = (char*)embeddings_ptr + (n_embeddings)*sizeof(BitVectorVertexEmbedding);
    int max_embeddings = 1000000;
    void* output_ptr = (char*)new_embeddings_ptr + (max_embeddings)*sizeof(BitVectorVertexEmbedding);
    int n_output = 0;
    int n_output_1 = 0;
    char* device_embeddings;
    char *device_new_embeddings;
    int* device_n_embeddings;
    int* device_n_embeddings_1;
    char *device_outputs;
    int* device_n_outputs;
    int* device_n_outputs_1;
    CSR* device_csr;

    cudaMalloc (&device_embeddings, n_embeddings*sizeof(BitVectorVertexEmbedding));
    cudaMemcpy (device_embeddings, embeddings_ptr,
                n_embeddings*sizeof(BitVectorVertexEmbedding),
                cudaMemcpyHostToDevice);
    cudaMalloc (&device_new_embeddings, max_embeddings*sizeof (BitVectorVertexEmbedding));
    cudaMalloc (&device_outputs, max_embeddings*sizeof (BitVectorVertexEmbedding));
    cudaMalloc (&device_n_embeddings, sizeof (0));
    cudaMalloc (&device_n_embeddings_1, sizeof (0));
    cudaMalloc (&device_n_outputs, sizeof (0));
    cudaMalloc (&device_n_outputs_1, sizeof (0));
    cudaMalloc (&device_csr, sizeof(CSR));

    cudaMemcpy (device_n_embeddings, &n_new_embeddings,
                sizeof (n_new_embeddings), cudaMemcpyHostToDevice);
    cudaMemcpy (device_n_outputs, &n_output, sizeof (n_output),
                cudaMemcpyHostToDevice);

    cudaMemcpy (device_n_embeddings_1, &n_new_embeddings_1,
                sizeof (n_new_embeddings_1), cudaMemcpyHostToDevice);
    cudaMemcpy (device_n_outputs_1, &n_output_1, sizeof (n_output_1),
                cudaMemcpyHostToDevice);

    cudaMemcpy (device_csr, csr, sizeof (CSR), cudaMemcpyHostToDevice);

    std::cout << "starting kernel with n_embeddings: " << n_embeddings;

    double t1 = convertTimeValToDouble (getTimeOfDay ());
#ifdef USE_FIXED_THREADS
    std::cout << " threads: " << MAX_CUDA_THREADS/THREAD_BLOCK_SIZE << std::endl;
      run_single_step_bitvector_embedding<<<MAX_CUDA_THREADS/THREAD_BLOCK_SIZE,THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
#else
    int thread_blocks = (n_embeddings%THREAD_BLOCK_SIZE != 0) ? (n_embeddings/THREAD_BLOCK_SIZE+1) : n_embeddings/THREAD_BLOCK_SIZE;
    std::cout << " threads: " << n_embeddings/THREAD_BLOCK_SIZE << std::endl;
    run_single_step_bitvector_embedding<<<thread_blocks, THREAD_BLOCK_SIZE>>> (device_embeddings, n_embeddings, device_csr,
                              device_outputs, device_n_outputs,
                              device_new_embeddings, device_n_embeddings,
                              device_n_outputs_1, device_n_embeddings_1);
#endif

    cudaDeviceSynchronize ();

    double t2 = convertTimeValToDouble (getTimeOfDay ());

    std::cout << "Execution time " << (t2-t1) << " secs" << std::endl;
    kernelTotalTime += (t2-t1);

    cudaError_t error = cudaGetLastError ();
    if (error != cudaSuccess) {
      const char* error_string = cudaGetErrorString (error);
      std::cout << error_string << std::endl;
    } else {
      std::cout << "Cuda success " << std::endl;
    }

    cudaMemcpy (new_embeddings_ptr, device_new_embeddings, max_embeddings*sizeof(BitVectorVertexEmbedding), cudaMemcpyDeviceToHost);
    cudaMemcpy (output_ptr, device_outputs, max_embeddings*sizeof(BitVectorVertexEmbedding), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_new_embeddings, device_n_embeddings, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_output, device_n_outputs, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_new_embeddings_1, device_n_embeddings_1, sizeof(0), cudaMemcpyDeviceToHost);
    cudaMemcpy (&n_output_1, device_n_outputs_1, sizeof(0), cudaMemcpyDeviceToHost);

    std::cout << "n_new_embeddings "<<n_new_embeddings<<std::endl;
    std::cout << "n_new_embeddings_1 "<<n_new_embeddings_1;
    std::cout << " n_output "<<n_output;
    std::cout << " n_output_1 "<<n_output_1<<std::endl;
    std::vector<BitVectorVertexEmbedding> new_embeddings;
  
    for (int i = 0; i < n_new_embeddings; i++) {
      BitVectorVertexEmbedding embedding = ((BitVectorVertexEmbedding*)new_embeddings_ptr)[i];
      new_embeddings.push_back (embedding);
    #ifdef DEBUG
      if (embedding.get_n_vertices () != (iter + 1)) {
        printf ("embedding has %d vertices\n", embedding.get_n_vertices ());
      }
    #endif
    }
    for (int i = 0; i < n_output; i++) {
      output.push_back (((BitVectorVertexEmbedding*)output_ptr)[i]);
    }
    embeddings = new_embeddings;

    cudaFree (device_embeddings);
    cudaFree (device_new_embeddings);
    cudaFree (device_n_embeddings);
    cudaFree (device_outputs);
    cudaFree (device_n_outputs);
    cudaFree (device_csr);
    delete[] global_mem_ptr;
    #endif
  } 
   */
}

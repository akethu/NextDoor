#include "hip/hip_runtime.h"
#include <omp.h>
#include "libNextDoor.hpp"

int numNeighborsSampledAtStep(int step)
{
  int n = 1;

  if (step < 0) {
    return 0;
  }

  if (stepSize(0) == 1) {
    n = 1;
  } else {
    n = stepSize(0);
  }

  for(int i = 1; i <= step; i++) {
    if (stepSize(i) == 1) {
      n += stepSize(i);
    } else {
      n += n * stepSize(i);
    }
  }

  return n;
}

typedef std::unordered_map<VertexID, std::unordered_set<VertexID>> AdjMatrix;

void csrToAdjMatrix(CSR* csr, AdjMatrix& adjMatrix)
{
  for (VertexID v : csr->iterate_vertices()) {
    adjMatrix[v] = std::unordered_set<VertexID> ();
    for (EdgePos_t i = csr->get_start_edge_idx(v); 
         i <= csr->get_end_edge_idx(v); i++) {
      VertexID e = csr->get_edges()[i];
      adjMatrix[v].insert(e);
    }
  }
}

template<class SampleType>
bool checkAdjacencyMatrixResult(CSR* csr, const VertexID_t INVALID_VERTEX, std::vector<VertexID_t>& initialSamples, 
                                const size_t finalSampleSize, std::vector<VertexID_t>& hFinalSamples, 
                                std::vector<SampleType>& samples, int maxSteps)
{
  std::cout << "checking results" << std::endl;
  AdjMatrix adjMatrix;

  csrToAdjMatrix(csr, adjMatrix);
  size_t numNeighborsToSampleAtStep = 0;

  for (int step = 0; step < min(maxSteps, steps()); step++) {
    bool foundError = false;
    std::cout << "Step: "<< step << " finalSampleSize " << finalSampleSize << " numNeighborsToSampleAtStep " << numNeighborsToSampleAtStep << std::endl;
    const size_t startIdxForCurrStep = (step == 0) ? 0 : (numNeighborsToSampleAtStep + stepSize(step));
    for (size_t s = 0; s < hFinalSamples.size(); s += finalSampleSize) {
      const size_t sampleId = s/finalSampleSize;
      size_t contentsLength = 0;

      //Two kinds of check are performed here.
      //1. If there is an edge in sample's adjacency matrix then there is same edge in the Graph.
      //2. All edges that can exist between vertices of two layers in graph also exists between 
      // sample's adjacency matrix.

      //Check first condition
      for (EdgePos_t v = 0; v < samples[sampleId].adjacencyMatrixLen[step]; v++) {
        VertexID_t col = samples[sampleId].adjacencyMatrixCol[step][v];
        VertexID_t row = samples[sampleId].adjacencyMatrixRow[step][v];
        VertexID_t transit = hFinalSamples[s + startIdxForCurrStep + col];
        VertexID_t prevVertex = -1;
        
        if (step == 0) {
          prevVertex = initialSamples[sampleId * initialSampleSize(nullptr) + row];
        } else {
          prevVertex = hFinalSamples[s + numNeighborsToSampleAtStep + row];
        }
        contentsLength += (int)(transit != INVALID_VERTEX);

        if (!foundError && transit != INVALID_VERTEX &&
          adjMatrix[prevVertex].count(transit) == 0) {
          std::cout << "col: " << col << " row: " << row << std::endl;
          printf("%s:%d Invalid '%d' in Sample '%ld' at for previous step vertex '%d' Step '%d'\n", __FILE__, __LINE__, transit, sampleId, prevVertex, step);
          foundError = true;
        }
      }

      //Check second condition
      for (EdgePos_t v = 0; v < (EdgePos_t)stepSize(step); v++) {
        VertexID_t transit = hFinalSamples[s + startIdxForCurrStep + v];
        EdgePos_t prevSZ = (step == 0) ? initialSampleSize(nullptr) : stepSize(step - 1);
        for (EdgePos_t prevVertexIdx = 0; prevVertexIdx < prevSZ; prevVertexIdx++) {
          VertexID_t prevVertex = -1;
          if (step == 0) {
            prevVertex = initialSamples[sampleId * initialSampleSize(nullptr) + prevVertexIdx];
          } else {
            prevVertex = hFinalSamples[s + numNeighborsToSampleAtStep + prevVertexIdx];
          }
          if (adjMatrix[prevVertex].count(transit) == 1) {
            //Edge exist in graph. So, search for that there is an edge in the sample.
            bool foundEdge = false;
            for (int e = 0; e < samples[sampleId].adjacencyMatrixLen[step]; e++) {
              VertexID_t col = samples[sampleId].adjacencyMatrixCol[step][e];
              VertexID_t row = samples[sampleId].adjacencyMatrixRow[step][e];
              VertexID_t v1 = -1;
              if (step == 0) {
                v1 = initialSamples[sampleId * initialSampleSize(nullptr) + row];
              } else {
                v1 = hFinalSamples[s + numNeighborsToSampleAtStep + row];
              }
              VertexID_t v2 = hFinalSamples[s + startIdxForCurrStep + col];

              if (v1 == prevVertex && v2 == transit) {
                foundEdge = true;
                break;
              }
            }

            if (!foundError && !foundEdge) {
              printf("Edge '%d'->'%d' exists in Graph but not in sample %d at step %d\n", prevVertex, transit, sampleId, step);
              foundError = true;
            }
          }
        }
      }
      

      // if (!foundError && contentsLength == 0) {
      //   printf("Step %d: '%ld' vertices sampled for sample '%ld' but sum of edges of all vertices in sample is '%ld'\n", 
      //           step, contentsLength, sampleId, adjMatrix[initialVal].size());
      //   foundError = true;
      // }
    }

    if (foundError)
      return false;
    
    if (step >= 1)
      numNeighborsToSampleAtStep += stepSize(step);
  }

  return true;
}

bool checkSampledVerticesResult(CSR* csr, const VertexID_t INVALID_VERTEX, std::vector<VertexID_t>& initialSamples, 
                                const size_t finalSampleSize, std::vector<VertexID_t>& finalSamples, int maxSteps)
{
  //Check result by traversing all sampled neighbors and making
  //sure that if neighbors at kth-hop is an adjacent vertex of one
  //of the k-1th hop neighbors.

  //First create the adjacency matrix.
  std::cout << "checking results" << std::endl;
  AdjMatrix adj_matrix;

  csrToAdjMatrix(csr, adj_matrix);

  //Now check the correctness
  size_t numNeighborsToSampleAtStep = 0;
  
  for (int step = 0; step < min(maxSteps, steps()); step++) {
    if (step == 0) { 
      bool foundError = false;
      #pragma omp parallel for shared(foundError)
      for (size_t s = 0; s < finalSamples.size(); s += finalSampleSize) {
        std::unordered_set<VertexID_t> uniqueNeighbors;
        // printf("omp_get_num_threads() %d\n", omp_get_num_threads());
        const size_t sampleId = s/finalSampleSize;
        const VertexID_t initialVal = initialSamples[sampleId];
        size_t contentsLength = 0;
        if (stepSize(step) != ALL_NEIGHBORS) {
          for (size_t v = s + numNeighborsToSampleAtStep; v < s + stepSize(step); v++) {
            VertexID_t transit = finalSamples[v];
            uniqueNeighbors.insert(transit);
            contentsLength += (int)(transit != INVALID_VERTEX);

            if (!foundError && transit != INVALID_VERTEX &&
                adj_matrix[initialVal].count(transit) == 0) {
              printf("%s:%d Invalid '%d' in Sample '%ld' at Step '%d'\n", __FILE__, __LINE__, transit, sampleId, step);
              foundError = true;
            }
          }

          if (!foundError && contentsLength == 0 && adj_matrix[initialVal].size() > 0) {
            printf("Step %d: '%ld' vertices sampled for sample '%ld' but sum of edges of all vertices in sample is '%ld'\n", 
                    step, contentsLength, sampleId, adj_matrix[initialVal].size());
            foundError = true;
          }
        } else {
          for (size_t v = s + numNeighborsToSampleAtStep; v < s + adj_matrix[initialVal].size(); v++) {
            VertexID_t transit = finalSamples[v];
            uniqueNeighbors.insert(transit);
            contentsLength += (int)(transit != INVALID_VERTEX);

            if (!foundError && transit != INVALID_VERTEX &&
                adj_matrix[initialVal].count(transit) == 0) {
              printf("%s:%d Invalid '%d' in Sample '%ld' at Step '%d'\n", __FILE__, __LINE__, transit, sampleId, step);
              foundError = true;
            }
          }

          if (!foundError && adj_matrix[initialVal].size() != contentsLength) {
            printf("%s:%d Sample '%ld' has %ld neighbors but %ld are sampled at Step '%d'\n", __FILE__, __LINE__, sampleId, 
                   adj_matrix[initialVal].size(), contentsLength, step);
            foundError = true;
          }
        }
      }

      if (foundError) return false;
    } else {
      bool foundError = false;
      #pragma omp parallel for shared(foundError)
      for (size_t s = 0; s < finalSamples.size(); s += finalSampleSize) {
        const size_t sampleId = s/finalSampleSize;
        size_t contentsLength = 0;
        size_t sumEdgesOfNeighborsAtPrevStep = 0;
        
        for (size_t v = s + numNeighborsSampledAtStep(step-2); v < s + numNeighborsSampledAtStep(step-1); v++) {
          sumEdgesOfNeighborsAtPrevStep +=  adj_matrix[finalSamples[v]].size();
        }
        
        // if (sampleId == 48) {
        //   printf("step %d start %d end %d\n", step, numNeighborsSampledAtStep(step-1),
        //          ((step == steps() - 1) ? finalSampleSize : numNeighborsSampledAtStep(step)));
        // }
        for (size_t v = s + numNeighborsSampledAtStep(step-1); 
             v < s + ((step == steps() - 1) ? finalSampleSize : numNeighborsSampledAtStep(step)); v++) {
          VertexID_t transit = finalSamples[v];
          contentsLength += (int)(transit != INVALID_VERTEX);
          
          bool found = false;
          if (transit != INVALID_VERTEX) {

            for (size_t v1 = s + numNeighborsSampledAtStep(step-2); v1 < s + numNeighborsSampledAtStep(step-1); v1++) {
              if (adj_matrix[finalSamples[v1]].count(transit) > 0) {
                found = true;
                break;
              }
            }

            if (!foundError && found == false) {
              printf("%s:%d Invalid '%d' in Sample '%ld' at Step '%d'\n", __FILE__, __LINE__, transit, sampleId, step);
              std::cout << "Contents of sample : [";
              for (size_t v2 = s; v2 < s + finalSampleSize; v2++) {
                std::cout << finalSamples[v2] << ", ";
              }
              std::cout << "]" << std::endl;
              foundError = true;
            }
          }
        }

        if (!foundError && contentsLength == 0 && sumEdgesOfNeighborsAtPrevStep > 0) {
          printf("Step %d: '%ld' vertices sampled for sample '%ld' but sum of edges of all vertices in sample is '%ld'\n", 
                  step, contentsLength, sampleId, sumEdgesOfNeighborsAtPrevStep);
          std::cout << "Contents of sample : [";
          for (size_t v2 = s; v2 < s + finalSampleSize; v2++) {
            std::cout << finalSamples[v2] << ", ";
          }
          std::cout << "]" << std::endl;
          foundError = true;
        }
      }

      if (foundError) return false;
    }

    numNeighborsToSampleAtStep = stepSizeAtStep(step);
  }

  return true;
}
#include "hip/hip_runtime.h"
#include "testBase.h"

#include <stdlib.h>

#define NUM_ROOT_VERTICES 100

class MultiRWSample
{
public:
  int rootVertices[NUM_ROOT_VERTICES];
  int lastRootIdx;
};

struct MultiRWApp {
  __host__ __device__ int steps() {return 100;}

  __host__ __device__ 
  int stepSize(int k) {
    return 1;
  }

  template<class SampleType>
  __device__ inline
  VertexID next(int step,CSRPartition* csr, const VertexID* transit, const VertexID sampleIdx,
                SampleType* sample, 
                const float max_weight,
                const CSR::Edge* transitEdges, const float* transitEdgeWeights,
                const EdgePos_t numEdges, const EdgePos_t neighbrID, hiprandState* state)
  {
    if (numEdges == 1) {
      VertexID_t v = transitEdges[0];
      if (step > 0) {
        sample->rootVertices[sample->lastRootIdx] = v;
      }

      return transitEdges[0];
    }
    
    EdgePos_t x = RandNumGen::rand_int(state, numEdges);
    VertexID_t v = transitEdges[x];

    if (step > 0) {
      sample->rootVertices[sample->lastRootIdx] = v;
    }

    return v;
  }

  template<class SampleType, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool DECREASE_GM_LOADS>
  __device__ inline
  VertexID nextCached(int step, const VertexID transit, const VertexID sampleIdx, 
    SampleType* sample,
                const float max_weight,
                const CSR::Edge* transitEdges, const float* transitEdgeWeights,
                const EdgePos_t numEdges, const EdgePos_t neighbrID, 
                hiprandState* state, VertexID_t* cachedEdges, float* cachedWeights,
                bool* globalLoadBV)
  {
    if (numEdges == 1) {
      VertexID_t v;
      if (CACHE_EDGES)
        v = cacheAndGet<CACHE_SIZE, DECREASE_GM_LOADS>(0, transitEdges, cachedEdges, globalLoadBV);
      else 
        v = transitEdges[0];
      if (step > 0) {
        sample->rootVertices[sample->lastRootIdx] = v;
      }

      return v;
    }
    
    EdgePos_t x = RandNumGen::rand_int(state, numEdges);
    VertexID_t v;
    if (CACHE_EDGES)
      v = cacheAndGet<CACHE_SIZE, DECREASE_GM_LOADS>(x, transitEdges, cachedEdges, globalLoadBV);
    else
      v = transitEdges[x];
      
    if (step > 0) {
      sample->rootVertices[sample->lastRootIdx] = v;
    }

    return v;
  }

  __host__ __device__ int samplingType()
  {
    return SamplingType::IndividualNeighborhood;
  }

  __host__ __device__ OutputFormat outputFormat()
  {
    return SampledVertices;
  }

  __host__ __device__ EdgePos_t numSamples(CSR* graph)
  {
    return graph->get_n_vertices();
  }

  template<class SampleType>
  __host__ std::vector<VertexID_t> initialSample(int sampleIdx, CSR* graph, SampleType& sample)
  {
    std::vector<VertexID_t> initialValue;
    initialValue.push_back(sample.rootVertices[0]);

    return initialValue;
  }

  __host__ __device__ EdgePos_t initialSampleSize(CSR* graph)
  {
    return 1;
  }

  __host__ __device__ bool hasExplicitTransits()
  {
    return true;
  }

  template<class SampleType>
  __device__ VertexID_t stepTransits(int step, const VertexID_t sampleID, SampleType& sample, int transitIdx, hiprandState* randState)
  {
    CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];
    //Use rejection sampling to sample based on the degree of vertices.
    int x = RandNumGen::rand_int(randState, NUM_ROOT_VERTICES);
    //printf("x %d\n", x);
    sample.lastRootIdx = x;
    return sample.rootVertices[x];
  }

  template<class SampleType>
  __host__ SampleType initializeSample(CSR* graph, const VertexID_t sampleID)
  {
    SampleType sample;
    //printf("sample %d\n", sampleID);
    for (int i = 0; i < NUM_ROOT_VERTICES; i++) {
      sample.rootVertices[i] = i % graph->get_n_vertices();
      // if (sampleID + i < graph->get_n_vertices()) {
      //   sample.rootVertices[i] = sampleID + i;
      // } else {
      //   sample.rootVertices[i] = sampleID;
      // }
    }
    return sample;
  }
};

#define RUNS 1
#define CHECK_RESULTS false


template<class SampleType>
bool checkMultiRWResult(NextDoorData<SampleType>& nextDoorData)
{
  //Check result by traversing all sampled neighbors and making
  //sure that if neighbors at kth-hop is an adjacent vertex of one
  //of the k-1th hop neighbors.
  CSR* csr = nextDoorData.csr;
  auto& initialSamples = nextDoorData.initialContents;
  auto finalSampleSize = getFinalSampleSize<MultiRWApp>();
  auto& finalSamples = nextDoorData.hFinalSamples;
  auto INVALID_VERTEX = nextDoorData.INVALID_VERTEX;
  auto& samples = nextDoorData.samples;
  int maxSteps = 4;

  //First create the adjacency matrix.
  std::cout << "checking results" << std::endl;
  AdjMatrix adj_matrix;

  csrToAdjMatrix(csr, adj_matrix);

  //Now check the correctness
  size_t numNeighborsToSampleAtStep = 0;
  
  for (int step = 0; step < min(maxSteps, MultiRWApp().steps()); step++) {
    if (step == 0) { 
      bool foundError = false;
      #pragma omp parallel for shared(foundError)
      for (size_t s = 0; s < finalSamples.size(); s += finalSampleSize) {
        std::unordered_set<VertexID_t> uniqueNeighbors;
        // printf("omp_get_num_threads() %d\n", omp_get_num_threads());
        const size_t sampleId = s/finalSampleSize;
        size_t contentsLength = 0;
        if (MultiRWApp().stepSize(step) != ALL_NEIGHBORS) {
          for (size_t v = s + numNeighborsToSampleAtStep; v < s + MultiRWApp().stepSize(step); v++) {
            VertexID_t transit = finalSamples[v];
            uniqueNeighbors.insert(transit);
            contentsLength += (int)(transit != INVALID_VERTEX);

            bool foundInRootVertices = false;
            for (int ii = 0; ii < NUM_ROOT_VERTICES; ii++) {
              VertexID_t rv = samples[sampleId].rootVertices[ii];
              if (adj_matrix[rv].count(transit) == 1) {
                foundInRootVertices = true;
                break;
              }
            }

            if (!foundError && transit != INVALID_VERTEX &&
                !foundInRootVertices) {
              printf("%s:%d Invalid '%d' in Sample '%ld' at Step '%d'\n", __FILE__, __LINE__, transit, sampleId, step);
              foundError = true;
            }
          }

          if (!foundError && contentsLength == 0) {
            printf("Step %d: '%ld' vertices sampled for sample '%ld' but sum of edges of all vertices in sample is '%ld'\n", 
                    step, contentsLength, sampleId, adj_matrix[samples[sampleId].rootVertices[0]].size());
            foundError = true;
          }
        } 
      }

      if (foundError) return false;
    } else {
      bool foundError = false;
      #pragma omp parallel for shared(foundError)
      for (size_t s = 0; s < finalSamples.size(); s += finalSampleSize) {
        const size_t sampleId = s/finalSampleSize;
        size_t contentsLength = 0;
        size_t sumEdgesOfNeighborsAtPrevStep = 0;
        const VertexID_t initialVal = samples[sampleId].rootVertices[0];

        for (size_t v = s + numNeighborsSampledAtStep<MultiRWApp>(step-2); v < s + numNeighborsSampledAtStep<MultiRWApp>(step-1); v++) {
          sumEdgesOfNeighborsAtPrevStep +=  adj_matrix[finalSamples[v]].size();
        }
        
        // if (sampleId == 48) {
        //   printf("step %d start %d end %d\n", step, numNeighborsSampledAtStep(step-1),
        //          ((step == steps() - 1) ? finalSampleSize : numNeighborsSampledAtStep(step)));
        // }
        for (size_t v = s + numNeighborsSampledAtStep<MultiRWApp>(step-1); 
             v < s + ((step == MultiRWApp().steps() - 1) ? finalSampleSize : numNeighborsSampledAtStep<MultiRWApp>(step)); v++) {
          VertexID_t transit = finalSamples[v];
          contentsLength += (int)(transit != INVALID_VERTEX);
          
          bool foundInRootVertices = false;
          for (auto rv : samples[sampleId].rootVertices) {
            if (adj_matrix[rv].count(transit) == 1) {
              foundInRootVertices = true;
              break;
            }
          }

          if (transit != INVALID_VERTEX) {
            if (!foundError && !foundInRootVertices) {
              printf("%s:%d Invalid '%d' in Sample '%ld' at Step '%d'\n", __FILE__, __LINE__, transit, sampleId, step);
              std::cout << "Contents of sample : [";
              for (size_t v2 = s; v2 < s + finalSampleSize; v2++) {
                std::cout << finalSamples[v2] << ", ";
              }
              std::cout << "]" << std::endl;
              foundError = true;
            }
          }
        }

        if (!foundError && contentsLength == 0 && sumEdgesOfNeighborsAtPrevStep > 0) {
          printf("Step %d: '%ld' vertices sampled for sample '%ld' but sum of edges of all vertices in sample is '%ld'\n", 
                  step, contentsLength, sampleId, sumEdgesOfNeighborsAtPrevStep);
          std::cout << "Contents of sample : [";
          for (size_t v2 = s; v2 < s + finalSampleSize; v2++) {
            std::cout << finalSamples[v2] << ", ";
          }
          std::cout << "]" << std::endl;
          foundError = true;
        }
      }

      if (foundError) return false;
    }

    numNeighborsToSampleAtStep = stepSizeAtStep<MultiRWApp>(step);
  }

  return true;
}

// APP_TEST(DeepWalk, CiteseerTP, GRAPH_PATH"/citeseer-weighted.graph", 10, false, "TransitParallel") 
// APP_TEST(DeepWalk, CiteseerSP, GRAPH_PATH"/citeseer-weighted.graph", 10, false, "SampleParallel") 
// APP_TEST(DeepWalk, MicoTP, GRAPH_PATH"/micro-weighted.graph", 10, false, "TransitParallel")
// APP_TEST(DeepWalk, MicoSP, GRAPH_PATH"/micro-weighted.graph", 10, false, "SampleParallel") 
// APP_TEST(DeepWalk, PpiTP, GRAPH_PATH"/ppi_sampled_matrix", 10, false, "TransitParallel")
// APP_TEST(DeepWalk, PpiSP, GRAPH_PATH"/ppi_sampled_matrix", 10, false, "SampleParallel")
APP_TEST(MultiRWSample, MultiRW, MultiRWApp, RedditTP, GRAPH_PATH"/reddit_sampled_matrix", RUNS, CHECK_RESULTS, checkMultiRWResult, "TransitParallel", false)
APP_TEST(MultiRWSample, MultiRW, MultiRWApp, RedditLB, GRAPH_PATH"/reddit_sampled_matrix", RUNS, CHECK_RESULTS, checkMultiRWResult, "TransitParallel", true)
APP_TEST(MultiRWSample, MultiRW, MultiRWApp, RedditSP, GRAPH_PATH"/reddit_sampled_matrix", RUNS, CHECK_RESULTS, checkMultiRWResult, "SampleParallel", false)
//APP_TEST(MultiRW, DeepWalk, RedditLB, GRAPH_PATH"/reddit_sampled_matrix", RUNS, CHECK_RESULTS, checkSampledVerticesResult, "TransitParallel", true)
APP_TEST(MultiRWSample, MultiRW, MultiRWApp, LiveJournalTP, GRAPH_PATH"/soc-LiveJournal1-weighted.graph", RUNS, CHECK_RESULTS, checkMultiRWResult, "TransitParallel", false)
APP_TEST(MultiRWSample, MultiRW, MultiRWApp, LiveJournalLB, GRAPH_PATH"/soc-LiveJournal1-weighted.graph", RUNS, CHECK_RESULTS, checkMultiRWResult, "TransitParallel", true)
APP_TEST(MultiRWSample, MultiRW, MultiRWApp, LiveJournalSP, GRAPH_PATH"/soc-LiveJournal1-weighted.graph", RUNS, CHECK_RESULTS, checkMultiRWResult, "SampleParallel", false)
APP_TEST(MultiRWSample, MultiRW, MultiRWApp, OrkutTP, GRAPH_PATH"/com-orkut-weighted.graph", RUNS, CHECK_RESULTS, checkMultiRWResult, "TransitParallel", false)
APP_TEST(MultiRWSample, MultiRW, MultiRWApp, OrkutLB, GRAPH_PATH"/com-orkut-weighted.graph", RUNS, CHECK_RESULTS, checkMultiRWResult, "TransitParallel", true)
APP_TEST(MultiRWSample, MultiRW, MultiRWApp, OrkutSP, GRAPH_PATH"/com-orkut-weighted.graph", RUNS, CHECK_RESULTS, checkMultiRWResult, "SampleParallel", false)
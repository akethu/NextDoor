#include "testBase.h"

__host__ __device__ int steps() {return 1;}

__host__ __device__ 
int stepSize(int k) {
  return 1;
}

__device__ inline
VertexID next(int step, const VertexID transit, const VertexID sample, 
              const float max_weight,
              const CSR::Edge* transitEdges, const float* transitEdgeWeights,
              const EdgePos_t numEdges, const EdgePos_t neighbrID, hiprandState* state)
{
  EdgePos_t id = RandNumGen::rand_int(state, numEdges);
  return transitEdges[id];
}

template<int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool DECREASE_GM_LOADS>
__device__ inline
VertexID nextCached(int step, const VertexID transit, const VertexID sample, 
              const float max_weight,
              const CSR::Edge* transitEdges, const float* transitEdgeWeights,
              const EdgePos_t numEdges, const EdgePos_t neighbrID, 
              hiprandState* state, VertexID_t* cachedEdges, float* cachedWeights,
              bool* globalLoadBV)
{
  EdgePos_t id = RandNumGen::rand_int(state, numEdges);
  if (CACHE_EDGES)
    return cacheAndGet<CACHE_SIZE, DECREASE_GM_LOADS>(id, transitEdges, cachedEdges, globalLoadBV);
  return transitEdges[id];
}

__host__ __device__ int samplingType()
{
  return SamplingType::CollectiveNeighborhood;
}

__host__ __device__ OutputFormat outputFormat()
{
  return AdjacencyMatrix;
}

#define VERTICES_PER_SAMPLE 2

__host__ EdgePos_t numSamples(CSR* graph)
{
  return graph->get_n_vertices() / VERTICES_PER_SAMPLE;
}

__host__ std::vector<VertexID_t> initialSample(int sampleIdx, CSR* graph)
{
  std::vector<VertexID_t> initialValue;

  for (int i = 0; i < VERTICES_PER_SAMPLE; i++) {
    initialValue.push_back(sampleIdx * VERTICES_PER_SAMPLE + i);
  }
}

#define RUNS 1
#define CHECK_RESULTS true

//APP_TEST(KHop, RedditTP, GRAPH_PATH"/reddit_sampled_matrix", RUNS, CHECK_RESULTS, "TransitParallel", false)
APP_TEST(KHop, RedditSP, GRAPH_PATH"/reddit_sampled_matrix", RUNS, CHECK_RESULTS, "SampleParallel", false)
// APP_TEST(KHop, RedditLB, GRAPH_PATH"/reddit_sampled_matrix", RUNS, CHECK_RESULTS, "TransitParallel", true)
// APP_TEST(KHop, LiveJournalTP, GRAPH_PATH"/soc-LiveJournal1-weighted.graph", RUNS, CHECK_RESULTS, "TransitParallel", false)
// APP_TEST(KHop, LiveJournalLB, GRAPH_PATH"/soc-LiveJournal1-weighted.graph", RUNS, CHECK_RESULTS, "TransitParallel", true)
// APP_TEST(KHop, LiveJournalSP, GRAPH_PATH"/soc-LiveJournal1-weighted.graph", RUNS, CHECK_RESULTS, "SampleParallel", false)
// APP_TEST(KHop, OrkutTP, GRAPH_PATH"/com-orkut-weighted.graph", RUNS, CHECK_RESULTS, "TransitParallel", false)
// APP_TEST(KHop, OrkutLB, GRAPH_PATH"/com-orkut-weighted.graph", RUNS, CHECK_RESULTS, "TransitParallel", true)
// APP_TEST(KHop, OrkutSP, GRAPH_PATH"/com-orkut-weighted.graph", RUNS, CHECK_RESULTS, "SampleParallel", false)

// APP_TEST(KHop, Citeseer, GRAPH_PATH"/citeseer.graph", 1, true, "TransitParallel")
// APP_TEST(KHop, Mico, GRAPH_PATH"/micro.graph", 1, false, "TransitParallel")
// APP_TEST(KHop, Reddit, GRAPH_PATH"/reddit_sampled_matrix", 1, false, "TransitParallel")


// TEST(KHop, Citeseer) {
//   EXPECT_TRUE(nextdoor(GRAPH_PATH"/citeseer.graph", "adj-list", "text", CHECK_RESULTS, false));
// }

// TEST(KHop, Mico) {
//   EXPECT_TRUE(nextdoor(GRAPH_PATH"/micro.graph", "adj-list", "text", CHECK_RESULTS, false));
// }

// TEST(KHop, Reddit) {
//   EXPECT_TRUE(nextdoor(GRAPH_PATH"/reddit_sampled_matrix", "adj-list", "text", CHECK_RESULTS, false));
// }


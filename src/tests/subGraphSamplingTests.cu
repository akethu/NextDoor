#include "hip/hip_runtime.h"
#include "testBase.h"

#include <stdlib.h>

#define VERTICES_IN_CLUSTERS 16
#define CLUSTERS_IN_SAMPLE 2
#define VERTICES_PER_SAMPLE (VERTICES_IN_CLUSTERS*CLUSTERS_IN_SAMPLE)

class SubGraphSample 
{
public:
  int *adjMatrixTotalLen;

  int adjMatrixLength;
  int adjMatrixPos;
  VertexID_t vertices[VERTICES_IN_CLUSTERS*CLUSTERS_IN_SAMPLE];
  int adjacencyMatrixLen;
  int *adjacencyMatrixRow;
  int *adjacencyMatrixCol;
};

int * dRowStorage;
int * dColStorage;
int* dAdjMatrixTotalLen;
std::vector<std::vector<VertexID_t>> clusters;

struct SubGraphSamplingAppI {
  __host__ __device__ int steps() {return 2;}

  __host__ __device__ 
  int stepSize(int k) {
    if (k == 0) return 1;
    return VERTICES_PER_SAMPLE;
  }

  template<typename SampleType, typename EdgeArray, typename WeightArray>
  __device__ inline
  VertexID next(int step, CSRPartition* csr, const VertexID* transits, const VertexID sampleIdx,
                SampleType* sample, 
                const float max_weight,
                EdgeArray& transitEdges, WeightArray& transitEdgeWeights,
                const EdgePos_t numEdges, const VertexID_t neighbrID, hiprandState* state)
  {
    VertexID_t v1 = transits[0];
    if (step == 0) {
      ::atomicAdd(&sample->adjMatrixLength, numEdges);
      return v1;
    }

    int v2Idx = neighbrID; //for (int v2Idx = 0; v2Idx < VERTICES_PER_SAMPLE; v2Idx++) //
    {
      VertexID_t v2 = sample->vertices[v2Idx];
      bool hasEdge = utils::binarySearch(transitEdges, v2, numEdges);
      // if (sampleIdx == 1929) {
      //   printf("sampleIdx %d v1 %d v2 %d hasEdge %d v2Idx %d\n", sampleIdx, v1, v2, hasEdge, v2Idx);
      // }

      if (hasEdge) {
        int len = ::atomicAdd(&sample->adjacencyMatrixLen, 1) + sample->adjMatrixPos;
        //int cooIdx = step * NUM_SAMPLED_VERTICES + len;
        sample->adjacencyMatrixRow[len] = v1;
        sample->adjacencyMatrixCol[len] = v2;
        //sample->adjacencyMatrixVal[len] = 1.0f;
        // if (sampleIdx == 49 && v1==1569 && v2==1570) { 
        //   printf("sampleIdx %d v1 %d v2 %d hasEdge %d v2Idx %d len %d %d %d %d\n", sampleIdx, v1, v2, hasEdge, v2Idx, len, sample->adjacencyMatrixLen, 
        //          sample->adjMatrixLength, sample->adjMatrixPos);
        // }
        // if (sampleIdx == 1929 || (len >= 32765 && len <= 32766)) { //v1==76921 && v2==205491 && 
        //   printf("sampleIdx %d v1 %d v2 %d hasEdge %d v2Idx %d len %d %d %d\n", sampleIdx, v1, v2, hasEdge, v2Idx, len, sample->adjacencyMatrixLen, sample->adjMatrixLength);
        // }
      }

    }

    return -1;
  }

  __host__ __device__ int samplingType()
  {
    return SamplingType::IndividualNeighborhood;
  }

  __host__ __device__ OutputFormat outputFormat()
  {
    return AdjacencyMatrix;
  }

  __host__ EdgePos_t numSamples(CSR* graph)
  {
    return (graph->get_n_edges() > 100000000) ? 800000 : min(1500000, (graph->get_n_vertices()*8)/VERTICES_IN_CLUSTERS);
  }

  __host__ __device__ bool hasExplicitTransits()
  {
    return true;
  }

  template<class SampleType>
  __device__ VertexID_t stepTransits(int step, const VertexID_t sampleID, SampleType& sample, int transitIdx, hiprandState* randState)
  {
    if (transitIdx == 0 && step == 1) {
      sample.adjMatrixPos = ::atomicAdd(sample.adjMatrixTotalLen, sample.adjMatrixLength);
    }

    VertexID_t v = sample.vertices[transitIdx];

    return v;
  }

  template<class SampleType>
  __host__ std::vector<VertexID_t> initialSample(int sampleIdx, CSR* graph, SampleType& sample)
  {
    std::vector<VertexID_t> initialValue;
    int idx = 0;
    for (int i = 0; i < CLUSTERS_IN_SAMPLE; i++) {
      VertexID_t clusterIdx = (sampleIdx * CLUSTERS_IN_SAMPLE + i) % clusters.size();//rand() % graph->get_n_vertices();
      //initialValue.insert(initialValue.begin(), clusters[clusterIdx].begin(), clusters[clusterIdx].end());
      for (auto v : clusters[clusterIdx]) {
        sample.vertices[idx] = v;
        idx++;
        initialValue.push_back(v);
      }
    }

    return initialValue;
  }

  template<class SampleType>
  __host__ SampleType initializeSample(CSR* graph, const VertexID_t sampleID)
  {
    SampleType sample;
    sample.adjacencyMatrixLen = 0;
    sample.adjMatrixLength = 0;
    sample.adjMatrixTotalLen = dAdjMatrixTotalLen;
    sample.adjacencyMatrixRow = dRowStorage;
    sample.adjacencyMatrixCol = dColStorage;

    return sample;
  }

  __host__ __device__ EdgePos_t initialSampleSize(CSR* graph) { return VERTICES_PER_SAMPLE;}
};

#define RUNS 1
#define CHECK_RESULTS false
#include "../check_results.cu"

template<class SampleType, typename App>
bool checkSubGraphResult(NextDoorData<SampleType, App>& nextDoorData)
{
  //Check result by traversing all sampled neighbors and making
  //sure that if neighbors at kth-hop is an adjacent vertex of one
  //of the k-1th hop neighbors.
  CSR* csr = nextDoorData.csr;
  auto& initialSamples = nextDoorData.initialContents;
  auto finalSampleSize = getFinalSampleSize<SubGraphSamplingAppI>();
  auto& finalSamples = nextDoorData.hFinalSamples;
  auto INVALID_VERTEX = nextDoorData.INVALID_VERTEX;
  auto& samples = nextDoorData.samples;
  int maxSteps = 4;

  //First create the adjacency matrix.
  std::cout << "checking results" << std::endl;
  AdjMatrix adj_matrix(csr->get_n_vertices(), std::unordered_set<VertexID> ());

  csrToAdjMatrix(csr, adj_matrix);

  //Now check the correctness
  size_t numNeighborsToSampleAtStep = 0;
  bool foundError = false;
  int sampleIdx = 0;
  int* hRowStorage = new int[csr->get_n_edges()*DIVUP(App().numSamples(csr)*VERTICES_PER_SAMPLE, csr->get_n_vertices())];
  int* hColStorage = new int[csr->get_n_edges()*DIVUP(App().numSamples(csr)*VERTICES_PER_SAMPLE, csr->get_n_vertices())];

  CHK_CU(hipMemcpy(hRowStorage, dRowStorage, csr->get_n_edges()*DIVUP(App().numSamples(csr)*VERTICES_PER_SAMPLE, csr->get_n_vertices())*sizeof(CSR::Edge), hipMemcpyDeviceToHost));
  CHK_CU(hipMemcpy(hColStorage, dColStorage, csr->get_n_edges()*DIVUP(App().numSamples(csr)*VERTICES_PER_SAMPLE, csr->get_n_vertices())*sizeof(CSR::Edge), hipMemcpyDeviceToHost));

  #pragma omp parallel for shared(foundError)
  for (int sampleIdx = 0; sampleIdx < samples.size(); sampleIdx++) {
    auto sample = samples[sampleIdx];
    //Go through all edges between two vertices and see if they exist in the graph
    for (int e = 0; e < sample.adjacencyMatrixLen; e++) {
      VertexID_t v1 = hRowStorage[e + sample.adjMatrixPos];
      VertexID_t v2 = hColStorage[e + sample.adjMatrixPos];

      if (!foundError && adj_matrix[v1].count(v2) == 0) {
        printf("Sample '%d': no edge '%d' -> '%d' in graph\n", sampleIdx, v1, v2);
        foundError = true;
      }
    }

    // if (sampleIdx == 49) {
    // for (int e = 0; e < sample.adjacencyMatrixLen; e++) {
    //   VertexID_t v1 = hRowStorage[e + sample.adjMatrixPos];
    //   VertexID_t v2 = hColStorage[e + sample.adjMatrixPos];

    //   printf("Sample '%d': '%d' -> '%d' at '%d'\n", sampleIdx, v1, v2, e);
    // } 
    // }

    //Go through edges between two vertices in graph and see if they exist in sample
    for (int vidx1 = 0; vidx1 < VERTICES_PER_SAMPLE; vidx1++) {
      VertexID_t v1 = sample.vertices[vidx1];
      for (int vidx2 = 0; vidx2 < VERTICES_PER_SAMPLE; vidx2++) {
        VertexID_t v2 = sample.vertices[vidx2];

        if (adj_matrix[v1].count(v2) == 1 and v1 != v2) {
          bool foundEdge = false;
          //Edge in Graph. Check if it is in Sample.
          for (int e = 0; e < sample.adjacencyMatrixLen; e++) {
            if ( hRowStorage[e + sample.adjMatrixPos]== v1 && hColStorage[e + sample.adjMatrixPos] == v2) {
              foundEdge = true;
              break;
            }
          }

          if (!foundEdge) {
            if (!foundError) {
              printf("Sample '%d': Edge '%d'->'%d' exists in graph but not in sample of length '%d' sample.adjMatrixPos '%d'\n", sampleIdx, v1, v2, sample.adjacencyMatrixLen, sample.adjMatrixPos);
            }
            foundError = true;
          }
        }
      }
    }
  }

  printf("Results Checked? %d\n", !foundError);
  if (foundError) return false;
  return true;
}

bool foo(const char* graph_file, const char* graph_type, const char* graph_format, 
  const int nruns, const bool chk_results, const bool print_samples,
  const char* kernelType, const bool enableLoadBalancing,
  bool (*checkResultsFunc)(NextDoorData<SubGraphSample, SubGraphSamplingAppI>&))
{
  Graph graph; 
  CSR* csr;
  if ((csr = loadGraph(graph, (char*)graph_file, (char*)graph_type, (char*)graph_format)) == nullptr) {
    return false;
  }

  std::cout << "Graph has " <<graph.get_n_edges () << " edges and " << 
      graph.get_vertices ().size () << " vertices " << std::endl;
  
  // std::string parts_file = "/mnt/homes/abhinav/nextdoor-experiments/cluster_gcn/reddit-parts-txt";
  // std::ifstream partitionsFile(parts_file);
  // partitionsFile >> partitionsJson;
  // partitionsFile.close();
  // size_t maximumSize = 0;

  // for (auto& item : partitionsJson.items()) {
  //   maximumSize = std::max(item.value().size(), maximumSize);
  // }

  // std::cout << "maximumSize " << maximumSize << std::endl; 
  
  //Create Clusters
  clusters = std::vector<std::vector<VertexID_t>>(csr->get_n_vertices()/VERTICES_IN_CLUSTERS);
  for (int clusterIdx = 0; clusterIdx < csr->get_n_vertices()/VERTICES_IN_CLUSTERS; clusterIdx++) {
    for (int v = 0; v < VERTICES_IN_CLUSTERS; v++) {
      clusters[clusterIdx].push_back(clusterIdx * VERTICES_IN_CLUSTERS + v);
    }
  }
  printCudaMemInfo();
  GPUCSRPartition gpuCSRPartition = transferCSRToGPU(csr);
  
  NextDoorData<SubGraphSample, SubGraphSamplingAppI> nextDoorData;
  nextDoorData.csr = csr;
  nextDoorData.gpuCSRPartition = gpuCSRPartition;
  CHK_CU(hipMalloc(&dAdjMatrixTotalLen, sizeof(int)));
  CHK_CU(hipMemset(dAdjMatrixTotalLen, 0, sizeof(int)));
  CHK_CU(hipMalloc(&dRowStorage, sizeof(VertexID_t) * graph.get_n_edges()*DIVUP(SubGraphSamplingAppI().numSamples(csr)*VERTICES_PER_SAMPLE, csr->get_n_vertices())));
  CHK_CU(hipMalloc(&dColStorage, sizeof(VertexID_t) * graph.get_n_edges()*DIVUP(SubGraphSamplingAppI().numSamples(csr)*VERTICES_PER_SAMPLE, csr->get_n_vertices())));
  //CHK_CU(hipMalloc(&SubGraphSample::rowStorage, sizeof(VertexID_t) * graph.get_n_edges()));
  allocNextDoorDataOnGPU<SubGraphSample, SubGraphSamplingAppI>(csr, nextDoorData);
  
  for (int i = 0; i < RUNS; i++) {
    if (strcmp(kernelType, "TransitParallel") == 0)
      doTransitParallelSampling<SubGraphSample, SubGraphSamplingAppI>(csr, gpuCSRPartition, nextDoorData, enableLoadBalancing);
    else if (strcmp(kernelType, "SampleParallel") == 0)
      doSampleParallelSampling<SubGraphSample, SubGraphSamplingAppI>(csr, gpuCSRPartition, nextDoorData);
    else
      abort();
  }

  getFinalSamples(nextDoorData);

  // int hTotalLen = 0;

  // CHK_CU(hipMemcpy(&hTotalLen, dAdjMatrixTotalLen, sizeof(int), hipMemcpyDeviceToHost));
  // std::cout<<hTotalLen<<std::endl;
  bool toRet = false;
  if (chk_results) {
    toRet = checkResultsFunc(nextDoorData);
  }

  CHK_CU(hipFree(dRowStorage));
  CHK_CU(hipFree(dColStorage));
  CHK_CU(hipFree(dAdjMatrixTotalLen));
  freeDeviceData(nextDoorData);
  return true;
}

#define SubGraphAPP_TEST(TestName,Path,Runs,CheckResults,chkResultsFunc,KernelType,LoadBalancing) \
  TEST(ClusterGCNSampling, TestName) { \
    EXPECT_TRUE(foo(Path, (char*)"adj-list", (char*)"text", 1, CheckResults, false, KernelType, LoadBalancing, chkResultsFunc));\
  }

#define SubGraphAPP_TEST_BINARY(TestName,Path,Runs,CheckResults,chkResultsFunc,KernelType,LoadBalancing)\
  TEST(ClusterGCNSampling, TestName) { \
  bool b = foo(Path, "edge-list", "binary", Runs, CheckResults, false, KernelType, LoadBalancing, chkResultsFunc);\
  EXPECT_TRUE(b);\
}

SubGraphAPP_TEST_BINARY(LiveJournalSP, LJ1_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "SampleParallel", false)
SubGraphAPP_TEST_BINARY(LiveJournalLB, LJ1_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", true)
SubGraphAPP_TEST_BINARY(LiveJournalTP, LJ1_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", false)

SubGraphAPP_TEST_BINARY(OrkutSP, ORKUT_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "SampleParallel", false)
SubGraphAPP_TEST_BINARY(OrkutLB, ORKUT_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", true)
SubGraphAPP_TEST_BINARY(OrkutTP, ORKUT_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", false)

SubGraphAPP_TEST_BINARY(PatentsSP, PATENTS_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "SampleParallel", false)
SubGraphAPP_TEST_BINARY(PatentsLB, PATENTS_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", true)
SubGraphAPP_TEST_BINARY(PatentsTP, PATENTS_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", false)

SubGraphAPP_TEST_BINARY(RedditSP, REDDIT_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "SampleParallel", false)
SubGraphAPP_TEST_BINARY(RedditLB, REDDIT_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", true)
SubGraphAPP_TEST_BINARY(RedditTP, REDDIT_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", false)

SubGraphAPP_TEST_BINARY(PPISP, PPI_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "SampleParallel", false)
SubGraphAPP_TEST_BINARY(PPILB, PPI_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", true)
SubGraphAPP_TEST_BINARY(PPITP, PPI_PATH, RUNS, CHECK_RESULTS, checkSubGraphResult, "TransitParallel", false)
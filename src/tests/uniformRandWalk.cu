#include "testBase.h"

__host__ __device__ int steps() {return 10;}

__host__ __device__ 
int stepSize(int k) {
  return 1;
}

__device__ inline
VertexID next(int step, const VertexID transit, const VertexID sample, 
              const float max_weight,
              const CSR::Edge* transitEdges, const float* transitEdgeWeights,
              const EdgePos_t numEdges, const EdgePos_t neighbrID, hiprandState* state)
{
  EdgePos_t id = RandNumGen::rand_int(state, numEdges);
  return transitEdges[id];
}

//nvprof bin/test_rw_10.2_x86_64 by-pass --graph-file=/mnt/homes/abhinav/GPUesque-for-eval/input/reddit_sampled_matrix --walks-per-node=1 --walk-length=10 --walk-mode=0

APP_TEST(UniformRandWalk, CiteseerTP, GRAPH_PATH"/citeseer-weighted.graph", 10, "TransitParallel") 
APP_TEST(UniformRandWalk, CiteseerSP, GRAPH_PATH"/citeseer-weighted.graph", 10, "SampleParallel") 
APP_TEST(UniformRandWalk, MicoTP, GRAPH_PATH"/micro-weighted.graph", 10, "TransitParallel")
APP_TEST(UniformRandWalk, MicoSP, GRAPH_PATH"/micro-weighted.graph", 10, "SampleParallel") 
APP_TEST(UniformRandWalk, RedditTP, GRAPH_PATH"/reddit_sampled_matrix", 10, "TransitParallel")
APP_TEST(UniformRandWalk, RedditSP, GRAPH_PATH"/reddit_sampled_matrix", 10, "SampleParallel")
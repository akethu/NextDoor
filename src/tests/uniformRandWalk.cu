#include "testBase.h"

__host__ __device__ int steps() {return 10;}

__host__ __device__ 
int stepSize(int k) {
  return 1;
}

__device__ inline
VertexID next(int step, const VertexID transit, const VertexID sample, 
              const CSR::Edge* transitEdges, const EdgePos_t numEdges,
              const EdgePos_t neighbrID, 
              hiprandState* state)
{
  EdgePos_t id = RandNumGen::rand_int(state, numEdges);
  return transitEdges[id];
}

//nvprof bin/test_rw_10.2_x86_64 by-pass --graph-file=/mnt/homes/abhinav/GPUesque-for-eval/input/reddit_sampled_matrix --walks-per-node=1 --walk-length=10 --walk-mode=0

APP_TEST(UniformRandWalk, Citeseer, GRAPH_PATH"/citeseer.graph", 10) //SP: 0.5 ms
APP_TEST(UniformRandWalk, Mico, GRAPH_PATH"/micro.graph", 10) //SP: 1.05 ms
APP_TEST(UniformRandWalk, Reddit, GRAPH_PATH"/reddit_sampled_matrix", 10) //SP: 1.43 ms

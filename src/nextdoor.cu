#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <algorithm>
#include <numeric>
#include <string.h>
#include <assert.h>
#include <tuple>
#include <queue>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/device/device_select.cuh>
#include <hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "sample.hpp"

#ifndef __NEXTDOOR_CU__
#define __NEXTDOOR_CU__

typedef VertexID VertexID_t;

//TODO-List:
//[] Divide main() function in several small functions.
//[] Divide the code in several include files that can be included in the API.
//[] In GPU Kernels, do refactoring and move them to other places.
//[] Use vectors instead of dynamic arrays and new.
//[] Convert these vectors to a new array type that does not do initialization of data.
//[] Use MKL or cuSPARSE to do the matrix transpose or sorting
//[] A configuration that specifies all the parameters.
//[] Use Templates for cleaner code of Sampler

//Supported:
//citeseer.graph
// const int N = 3312;
// const int N_EDGES = 9074;
//micro.graph
//const int N = 100000;
//const int N_EDGES = 2160312;
//rmat.graph
// const int N = 1024;
// const int N_EDGES = 29381;
//ego-facebook
// const int N = 4039;
// const int N_EDGES = 88244;
//ego-twitter
//const int N = 81306;
//const int N_EDGES = 2420766;
//ego-gplus
//const int N = 107614;
//const int N_EDGES = 13652253;
//soc-pokec-relationships
//const int N = 1632803;
//const int N_EDGES = 30480021;
//soc-LiveJournal1
//const int N = 4847571;
//const int N_EDGES = 68556521;

//Not supportred:
//com-orkut.ungraph
// const int N = 3072441;
// const int N_EDGES = 117185083;

#include "csr.hpp"
#include "utils.hpp"
#include "sampler.cuh"
#include "rand_num_gen.cuh"
#include "libNextDoor.hpp"

using namespace utils;
using namespace GPUUtils;

#define CHECK_RESULT

//For mico, 512 works best
const size_t N_THREADS = 256;

//TODO try for larger random walks to improve results

#define WARP_HOP

const int ALL_NEIGHBORS = -1;

const bool useGridKernel = true;
const bool useSubWarpKernel = true;
const bool useThreadBlockKernel = true;

enum TransitKernelTypes {
  GridKernel = 1,
  ThreadBlockKernel = 2,
  SubWarpKernel = 3,
  IdentityKernel = 4,
  NumKernelTypes = 4
};

/**User Defined Functions**/
__host__ __device__ int steps();

__host__ __device__ 
int stepSize(int k);

__device__ inline
VertexID next(int step, const VertexID transit, const VertexID sample, 
              const float maxWeight,
              const CSR::Edge* transitEdges, const float* transitEdgeWeights,
              const EdgePos_t numEdges, const EdgePos_t neighbrID, 
              hiprandState* state);
template<int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool DECREASE_GM_LOADS>
__device__ inline
VertexID nextCached(int step, const VertexID transit, const VertexID sample, 
              const float maxWeight,
              const CSR::Edge* transitEdges, const float* transitEdgeWeights,
              const EdgePos_t numEdges, const EdgePos_t neighbrID, 
              hiprandState* state, VertexID_t* cachedEdges, float* cachedWeights,
              bool* globalLoadBV);
__host__ __device__ int steps();

__constant__ char csrPartitionBuff[sizeof(CSRPartition)];
/**********************/

__host__ __device__
EdgePos_t newNeighborsSize(int hop, EdgePos_t num_edges)
{
  return (stepSize(hop) == ALL_NEIGHBORS) ? num_edges : (EdgePos_t)stepSize(hop);
}

__host__ __device__
EdgePos_t stepSizeAtStep(int step)
{
  if (step == -1)
    return 0;

  EdgePos_t n = 1;
  for (int i = 0; i <= step; i++) {
    n = n * stepSize(i);
  }

  return n;
}


__host__ __device__ int numberOfTransits(int step) {
  return stepSizeAtStep(step);
}

#include "check_results.cu"

__global__ void samplingKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                               const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                               const size_t transitToSamplesSize, const size_t NumSamples,
                               VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                               VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                               hiprandState* randStates)
{
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  //__shared__ VertexID newNeigbhors[N_THREADS];

  if (threadId >= transitToSamplesSize)
    return;
  
  EdgePos_t transitIdx = threadId/stepSize(step);
  EdgePos_t transitNeighborIdx = threadId % stepSize(step);
  
  VertexID_t sample = transitToSamplesValues[transitIdx];
  assert(sample < NumSamples);
  VertexID_t transit = transitToSamplesKeys[transitIdx];
  VertexID_t neighbor = invalidVertex;
  graph.device_csr = (CSRPartition*)&csrPartitionBuff[0];

  if (transit != invalidVertex) {
    // if (graph.device_csr->has_vertex(transit) == false)
    //   printf("transit %d\n", transit);
    assert(graph.device_csr->has_vertex(transit));

    EdgePos_t numTransitEdges = graph.device_csr->get_n_edges_for_vertex(transit);
    
    if (numTransitEdges != 0) {
      const CSR::Edge* transitEdges = graph.device_csr->get_edges(transit);
      const float* transitEdgeWeights = graph.device_csr->get_weights(transit);
      const float maxWeight = graph.device_csr->get_max_weight(transit);

      hiprandState* randState = &randStates[threadId];
      neighbor = next(step, transit, sample, maxWeight, transitEdges, transitEdgeWeights, 
                      numTransitEdges, transitNeighborIdx, randState);
#if 0
      //search if neighbor has already been selected.
      //we can do that in register if required
      newNeigbhors[threadIdx.x] = neighbor;

      bool found = false;
      for (int i = 0; i < N_THREADS; i++) {
        if (newNeigbhors[i] == neighbor) {
          found = true;
          // break;
        }
      }

      __syncwarp();
      if (found) {
        neighbor = next(step, transit, sample, transitEdges, numTransitEdges, 
          transitNeighborIdx, randState);;
      }
#endif
    }
  }

  __syncwarp();

  EdgePos_t totalSizeOfSample = stepSizeAtStep(step - 1);

  if (step != steps() - 1) {
    //No need to store at last step
    samplesToTransitKeys[threadId] = sample;
    samplesToTransitValues[threadId] = neighbor;
  }
  
  EdgePos_t insertionPos = 0; 
  if (numberOfTransits(step) > 1) {    
    insertionPos = utils::atomicAdd(&sampleInsertionPositions[sample], 1);
  } else {
    insertionPos = step;
  }

  // if (insertionPos < finalSampleSize) {
  //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
  // }
  assert(finalSampleSize > 0);
  if (insertionPos >= finalSampleSize) {
    printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sample);
  }
  assert(insertionPos < finalSampleSize);
  finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
  // if (sample == 100) {
  //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
  // }
  //TODO: We do not need atomic instead store indices of transit in another array,
  //wich can be accessed based on sample and transitIdx.
}


__global__ void identityKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                               const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                               const size_t transitToSamplesSize, const size_t NumSamples,
                               VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                               VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                               hiprandState* randStates, const int* kernelTypeForTransit)
{
  
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  //__shared__ VertexID newNeigbhors[N_THREADS];

  if (threadId >= transitToSamplesSize)
    return;
  
  EdgePos_t transitIdx = threadId/stepSize(step);
  EdgePos_t transitNeighborIdx = threadId % stepSize(step);
  VertexID_t transit = transitToSamplesKeys[transitIdx];
  int kernelTy = kernelTypeForTransit[transit];
  
  if ((useGridKernel && kernelTy == TransitKernelTypes::GridKernel) || 
      (useSubWarpKernel && kernelTy == TransitKernelTypes::SubWarpKernel)) {
    return;
  }

  graph.device_csr = (CSRPartition*)&csrPartitionBuff[0];
  VertexID_t sample = transitToSamplesValues[transitIdx];
  assert(sample < NumSamples);
  VertexID_t neighbor = invalidVertex;

  hiprandState randState = randStates[transitIdx];

  if (transit != invalidVertex) {
    // if (graph.device_csr->has_vertex(transit) == false)
    //   printf("transit %d\n", transit);
    assert(graph.device_csr->has_vertex(transit));

    EdgePos_t numTransitEdges = graph.device_csr->get_n_edges_for_vertex(transit);
    
    if (numTransitEdges != 0) {
      const CSR::Edge* transitEdges = graph.device_csr->get_edges(transit);
      const float* transitEdgeWeights = graph.device_csr->get_weights(transit);
      const float maxWeight = graph.device_csr->get_max_weight(transit);

      neighbor = next(step, transit, sample, maxWeight, transitEdges, transitEdgeWeights, 
                      numTransitEdges, transitNeighborIdx, &randState);
#if 0
      //search if neighbor has already been selected.
      //we can do that in register if required
      newNeigbhors[threadIdx.x] = neighbor;

      bool found = false;
      for (int i = 0; i < N_THREADS; i++) {
        if (newNeigbhors[i] == neighbor) {
          found = true;
          // break;
        }
      }

      __syncwarp();
      if (found) {
        neighbor = next(step, transit, sample, transitEdges, numTransitEdges, 
          transitNeighborIdx, randState);;
      }
#endif
    }
  }

  __syncwarp();

  EdgePos_t totalSizeOfSample = stepSizeAtStep(step - 1);

  if (step != steps() - 1) {
    //No need to store at last step
    samplesToTransitKeys[threadId] = sample;
    samplesToTransitValues[threadId] = neighbor;
  }
  
  EdgePos_t insertionPos = 0; 
  if (numberOfTransits(step) > 1) {    
    insertionPos = utils::atomicAdd(&sampleInsertionPositions[sample], 1);
  } else {
    insertionPos = step;
  }

  // if (insertionPos < finalSampleSize) {
  //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
  // }
  assert(finalSampleSize > 0);
  if (insertionPos >= finalSampleSize) {
    printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sample);
  }
  assert(insertionPos < finalSampleSize);
  finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
  // if (sample == 100) {
  //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
  // }
  //TODO: We do not need atomic instead store indices of transit in another array,
  //wich can be accessed based on sample and transitIdx.
}

template<int THREADS, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool COALESCE_GL_LOADS, int TRANSITS_PER_THREAD, bool COALESCE_CURAND_LOAD>
__global__ void subWarpKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                              const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                              const size_t transitToSamplesSize, const size_t NumSamples,
                              VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                              VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                              hiprandState* randStates, const int* kernelTypeForTransit, const VertexID_t* subWarpKernelTBPositions, 
                              const EdgePos_t subWarpKernelTBPositionsNum)
{  
  __shared__ unsigned char shMemAlloc[sizeof(hiprandState)*THREADS];

  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  //__shared__ VertexID newNeigbhors[N_THREADS];
  hiprandState* curandSrcPtr;

  if (COALESCE_CURAND_LOAD) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMemAlloc[0];

    int* randStatesAsInts = (int*)randStates;
  
    for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  hiprandState localRandState = *curandSrcPtr;

  int subWarpThreadIdx = threadId % LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
  int subWarp = threadId / LoadBalancing::LoadBalancingThreshold::SubWarpLevel;

  for (int transitI = 0; transitI < TRANSITS_PER_THREAD; transitI++) {
    //TODO:*********************THIS**********************************
    EdgePos_t transitIdx = 0;
    EdgePos_t subWarpIdx = TRANSITS_PER_THREAD * subWarp + transitI;
    if (subWarpIdx >= subWarpKernelTBPositionsNum) {
      continue;
    }
    transitIdx = subWarpKernelTBPositions[subWarpIdx] + subWarpThreadIdx;
    EdgePos_t transitNeighborIdx = 0;
    VertexID_t transit = transitToSamplesKeys[transitIdx];
    VertexID_t firstThreadTransit = __shfl_sync(FULL_WARP_MASK, transit, 0, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
    __syncwarp();

    if (firstThreadTransit != transit)
      continue;

    // int kernelTy = kernelTypeForTransit[transit];
    // if (kernelTy != TransitKernelTypes::SubWarpKernel) {
    //   printf("threadId %d transitIdx %d kernelTy %d\n", threadId, transitIdx, kernelTy);
    // }
    assert(kernelTypeForTransit[transit] == TransitKernelTypes::SubWarpKernel);
    
    
    graph.device_csr = (CSRPartition*)&csrPartitionBuff[0];
    VertexID_t sample = transitToSamplesValues[transitIdx];
    assert(sample < NumSamples);
    VertexID_t neighbor = invalidVertex;

    if (transit != invalidVertex) {
      // if (graph.device_csr->has_vertex(transit) == false)
      //   printf("transit %d\n", transit);
      assert(graph.device_csr->has_vertex(transit));

      EdgePos_t numTransitEdges = graph.device_csr->get_n_edges_for_vertex(transit);
      
      if (numTransitEdges != 0) {
        const CSR::Edge* transitEdges = graph.device_csr->get_edges(transit);
        const float* transitEdgeWeights = graph.device_csr->get_weights(transit);
        const float maxWeight = graph.device_csr->get_max_weight(transit);

        neighbor = next(step, transit, sample, maxWeight, transitEdges, transitEdgeWeights, 
                        numTransitEdges, transitNeighborIdx, &localRandState);
  #if 0
        //search if neighbor has already been selected.
        //we can do that in register if required
        newNeigbhors[threadIdx.x] = neighbor;

        bool found = false;
        for (int i = 0; i < N_THREADS; i++) {
          if (newNeigbhors[i] == neighbor) {
            found = true;
            // break;
          }
        }

        __syncwarp();
        if (found) {
          neighbor = next(step, transit, sample, transitEdges, numTransitEdges, 
            transitNeighborIdx, randState);;
        }
  #endif
      }
    }

    // __syncwarp();

    //EdgePos_t totalSizeOfSample = stepSizeAtStep(step - 1);

    if (step != steps() - 1) {
      //No need to store at last step
      samplesToTransitKeys[transitIdx] = sample;
      samplesToTransitValues[transitIdx] = neighbor;
    }
    
    EdgePos_t insertionPos = 0; 
    if (false && numberOfTransits(step) > 1) {    
      insertionPos = utils::atomicAdd(&sampleInsertionPositions[sample], 1);
    } else {
      insertionPos = step;
    }

    // if (insertionPos < finalSampleSize) {
    //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
    // }
    assert(finalSampleSize > 0);
    assert(insertionPos < finalSampleSize);
    finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
    // if (sample == 100) {
    //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
    // }
    //TODO: We do not need atomic instead store indices of transit in another array,
    //wich can be accessed based on sample and transitIdx.
  }
}

template<int CACHE_SIZE, bool COALESCE_GL_LOADS, typename T>
__device__ inline VertexID_t cacheAndGet(EdgePos_t id, const T* transitEdges, T* cachedEdges, bool* globalLoadBV)
{
  VertexID_t e;

  if (id >= CACHE_SIZE)
    return transitEdges[id];
  
  if (COALESCE_GL_LOADS) {
    e = cachedEdges[id];
    if (e == -1)
      globalLoadBV[id] = true;

    __syncthreads();

    for (int i = threadIdx.x; i < CACHE_SIZE; i += blockDim.x) {
      if (globalLoadBV[i]) {
        cachedEdges[i] = transitEdges[i];
      }
    }
    
    __syncthreads();

    globalLoadBV[id] = false;
    e = cachedEdges[id];
  } else {
    e = cachedEdges[id];
    if (e == -1) {
      e = transitEdges[id];
      cachedEdges[id] = e;
    }
  }

  return e;
}

#define MAX(x,y) (((x)<(y))?(y):(x))

template<int THREADS, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool COALESCE_GL_LOADS, int TRANSITS_PER_THREAD, bool COALESCE_CURAND_LOAD>
__global__ void gridKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                           const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                           const size_t transitToSamplesSize, const size_t NumSamples,
                           VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                           VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                           hiprandState* randStates, const int* kernelTypeForTransit, const VertexID_t* gridKernelTBPositions, 
                           const EdgePos_t gridKernelTBPositionsNum)
{
  #define EDGE_CACHE_SIZE (CACHE_EDGES ? CACHE_SIZE*sizeof(CSR::Edge) : 0)
  #define WEIGHT_CACHE_SIZE (CACHE_WEIGHTS ? CACHE_SIZE*sizeof(float) : 0)
  #define CURAND_SHMEM_SIZE (sizeof(hiprandState)*THREADS)
  // #define COALESCE_GL_LOADS_SHMEM_SIZE ()

  __shared__ unsigned char shMemAlloc[MAX(EDGE_CACHE_SIZE+WEIGHT_CACHE_SIZE, CURAND_SHMEM_SIZE)];
  
  //__shared__ bool globalLoadBV[COALESCE_GL_LOADS ? CACHE_SIZE : 1];
  bool* globalLoadBV;
  __shared__ VertexID_t numEdgesInShMem;
  __shared__ bool invalidateCache;
  __shared__ VertexID_t transitForTB;
  __shared__ CSR::Edge* glTransitEdges;
  __shared__ float* glTransitEdgeWeights;
  __shared__ float maxWeight;
  __shared__ EdgePos_t mapStartPos;

  CSR::Edge* edgesInShMem = CACHE_EDGES ? (CSR::Edge*)&shMemAlloc[0] : nullptr;
  float* edgeWeightsInShMem = CACHE_WEIGHTS ? (float*)&shMemAlloc[EDGE_CACHE_SIZE] : nullptr;
  
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  
  hiprandState* curandSrcPtr;

  if (COALESCE_CURAND_LOAD) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMemAlloc[0];

    int* randStatesAsInts = (int*)randStates;
  
    for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  hiprandState localRandState = *curandSrcPtr;

  //__shared__ VertexID newNeigbhors[N_THREADS];
  //if (threadIdx.x == 0) printf("blockIdx.x %d\n", blockIdx.x);
  //shRandStates[threadIdx.x] = randStates[threadId];  
  //__syncthreads();
  
  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];

  for (int transitI = 0; transitI < TRANSITS_PER_THREAD; transitI++) {
    EdgePos_t transitIdx = 0;
    EdgePos_t transitNeighborIdx = 0;//threadId % stepSize(step); //TODO: Correct this for k-hop
    if (TRANSITS_PER_THREAD * blockIdx.x + transitI >= gridKernelTBPositionsNum) {
      continue;
    }
    if (threadIdx.x == 0) {
      mapStartPos = gridKernelTBPositions[TRANSITS_PER_THREAD * blockIdx.x + transitI];
    }
    __syncthreads();
    transitIdx = mapStartPos + threadIdx.x; //threadId/stepSize(step);
    VertexID_t transit = transitToSamplesKeys[transitIdx];

    if (threadIdx.x == 0) {
      invalidateCache = transitForTB != transit || transitI == 0;
      transitForTB = transit;
    }
    if (threadIdx.x == 0 && invalidateCache) {
      //assert(graph.device_csr->has_vertex(transit));
      //TODO: fuse below functions into one to decrease reads
      numEdgesInShMem = csr->get_n_edges_for_vertex(transit);
      glTransitEdges = (CSR::Edge*)csr->get_edges(transit);
      glTransitEdgeWeights = (float*)csr->get_weights(transit);
      maxWeight = csr->get_max_weight(transit);
    }

    __syncthreads();

    if (CACHE_EDGES && invalidateCache) {
      for (int i = threadIdx.x; i < min(CACHE_SIZE, numEdgesInShMem); i += blockDim.x) {
        edgesInShMem[i] = -1;//glTransitEdges[i];
      }
    }
  
    if (CACHE_WEIGHTS && invalidateCache) {
      for (int i = threadIdx.x; i < min(CACHE_SIZE, numEdgesInShMem); i += blockDim.x) {
        edgeWeightsInShMem[i] = -1;//glTransitEdgeWeights[i];
      }
    }

    __syncthreads();

    if (transit == transitForTB) {
      // if (threadIdx.x == 0 && kernelTypeForTransit[transit] != TransitKernelTypes::GridKernel) {
      //   printf("transit %d transitIdx %d gridDim.x %d\n", transit, transitIdx, gridDim.x);
      // }
      // assert (kernelTypeForTransit[transit] == TransitKernelTypes::GridKernel);

      VertexID_t sample = transitToSamplesValues[transitIdx];

      assert(sample < NumSamples);
      VertexID_t neighbor = invalidVertex;
      // if (graph.device_csr->has_vertex(transit) == false)
      //   printf("transit %d\n", transit);
      neighbor = nextCached<CACHE_SIZE, CACHE_EDGES, CACHE_WEIGHTS, 0>(step, transit, sample, maxWeight, 
                                                              glTransitEdges, glTransitEdgeWeights, 
                                                              numEdgesInShMem, transitNeighborIdx, &localRandState,
                                                              edgesInShMem, edgeWeightsInShMem,
                                                              &globalLoadBV[0]);
      __syncwarp();

      //EdgePos_t totalSizeOfSample = stepSizeAtStep(step - 1);

      if (step != steps() - 1) {
        //No need to store at last step
        samplesToTransitKeys[transitIdx] = sample; //TODO: Update this for khop to transitIdx + transitNeighborIdx
        samplesToTransitValues[transitIdx] = neighbor;
      }
      
      EdgePos_t insertionPos = 0; 
      if (false && numberOfTransits(step) > 1) {
        //insertionPos = utils::atomicAdd(&sampleInsertionPositions[sample], 1);
      } else {
        insertionPos = step;
      }

      // if (insertionPos < finalSampleSize) {
      //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
      // }
      assert(finalSampleSize > 0);
      if (insertionPos >= finalSampleSize) {
        printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sample);
      }
      assert(insertionPos < finalSampleSize);

      if (step %2 == 0) {
        //((uint64_t*)finalSamples)[(sample*finalSampleSize)/2 + threadIdx.x] = (uint64_t)(((uint64_t)transit) | (((uint64_t)neighbor) << 32));
      }

      finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
      // if (sample == 100) {
      //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
      // }
      //TODO: We do not need atomic instead store indices of transit in another array,
      //wich can be accessed based on sample and transitIdx.
    }
  }
}

__global__ void sampleParallelKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                               const size_t NumSamples,
                               VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                               hiprandState* randStates)
{
  //TODO: Following code assumes Random Walk

  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  //__shared__ VertexID newNeigbhors[N_THREADS];

  if (threadId >= NumSamples)
    return;
  
  VertexID_t sample = threadId;
  VertexID_t transit = (step == 0) ? sample : finalSamples[sample*finalSampleSize + step - 1];
  VertexID_t neighbor = invalidVertex;
  
  if (transit == invalidVertex) {
    return;
  }
  assert(graph.device_csr->has_vertex(transit));

  EdgePos_t numTransitEdges = graph.device_csr->get_n_edges_for_vertex(transit);
  
  if (numTransitEdges != 0) {
    const CSR::Edge* transitEdges = graph.device_csr->get_edges(transit);
    const float* transitEdgeWeights = graph.device_csr->get_weights(transit);
    const float maxWeight = graph.device_csr->get_max_weight(transit);

    hiprandState* randState = &randStates[threadId];
    neighbor = next(step, transit, sample, maxWeight, transitEdges, transitEdgeWeights, 
                    numTransitEdges, 0, randState);
#if 0
    //search if neighbor has already been selected.
    //we can do that in register if required
    newNeigbhors[threadIdx.x] = neighbor;

    bool found = false;
    for (int i = 0; i < N_THREADS; i++) {
      if (newNeigbhors[i] == neighbor) {
        found = true;
        // break;
      }
    }

    __syncwarp();
    if (found) {
      neighbor = next(step, transit, sample, transitEdges, numTransitEdges, 
        transitNeighborIdx, randState);;
    }
#endif
  }

  EdgePos_t totalSizeOfSample = stepSizeAtStep(step - 1);  
  
  EdgePos_t insertionPos = 0; 

  if (numberOfTransits(step) > 1) {    
    insertionPos = utils::atomicAdd(&sampleInsertionPositions[sample], 1);
  } else {
    insertionPos = step;
  }

  // if (insertionPos < finalSampleSize) {
  //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
  // }
  assert(finalSampleSize > 0);
  if (insertionPos >= finalSampleSize) {
    printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sample);
  }
  assert(insertionPos < finalSampleSize);
  finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
  // if (sample == 100) {
  //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
  // }
  //TODO: We do not need atomic instead store indices of transit in another array,
  //wich can be accessed based on sample and transitIdx.
}

struct functor 
{
  __device__ __host__ int operator()(int& a, int &b) 
  {
    return max(a, b);
  }
};

template<int TB_THREADS>
__global__ void partitionTransitsInKernels(int step, EdgePos_t* uniqueTransits, EdgePos_t* uniqueTransitCounts, 
                                           EdgePos_t* transitPositions,
                                           EdgePos_t uniqueTransitCountsNum, VertexID_t invalidVertex,
                                           EdgePos_t* gridKernelTransits, EdgePos_t* gridKernelTransitsNum,
                                           EdgePos_t* threadBlockKernelTransits, EdgePos_t* threadBlockKernelTransitsNum,
                                           EdgePos_t* subWarpKernelTransits, EdgePos_t* subWarpKernelTransitsNum,
                                           EdgePos_t* identityKernelTransits, EdgePos_t* identityKernelTransitsNum,
                                           int* kernelTypeForTransit, VertexID_t* transitToSamplesKeys) 
{
  //__shared__ EdgePos_t insertionPosOfThread[TB_THREADS];
  const int SHMEM_SIZE = 7*TB_THREADS;
  // __shared__ EdgePos_t trThreadBlocks[TB_THREADS];
  // __shared__ EdgePos_t trStartPos[TB_THREADS];
  typedef hipcub::BlockScan<int, TB_THREADS> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  __shared__ EdgePos_t shGridKernelTransits[SHMEM_SIZE];
  __shared__ EdgePos_t warpsLastThreadVals;
  __shared__ EdgePos_t threadToTransitPrefixSum[TB_THREADS];
  __shared__ EdgePos_t threadToTransitPos[TB_THREADS];
  __shared__ VertexID_t threadToTransit[TB_THREADS];
  __shared__ EdgePos_t totalThreadGroups;
  __shared__ EdgePos_t threadGroupsInsertionPos;
  __shared__ EdgePos_t gridKernelTransitsIter;

  int threadId = blockIdx.x * blockDim.x + threadIdx.x;

  if (threadIdx.x == 0) {
    totalThreadGroups = 0;
    gridKernelTransitsIter = 0;
  }

  for (int i = threadIdx.x; i < SHMEM_SIZE; i+= blockDim.x) {
    shGridKernelTransits[i] = 0;
  }

  __syncthreads();
  
  VertexID_t transit = uniqueTransits[threadId];
  EdgePos_t trCount = (threadId >= uniqueTransitCountsNum || transit == invalidVertex) ? -1: uniqueTransitCounts[threadId];
  EdgePos_t trPos = (threadId >= uniqueTransitCountsNum || transit == invalidVertex) ? -1: transitPositions[threadId];

  int kernelType = -1;
  EdgePos_t numThreadGroups = 0;
  if (trCount >= LoadBalancing::LoadBalancingThreshold::GridLevel) {    
    kernelType = TransitKernelTypes::GridKernel;
  } else if (trCount >= LoadBalancing::LoadBalancingThreshold::BlockLevel) {
    kernelType = TransitKernelTypes::ThreadBlockKernel;
    // numThreadGroups = 0;
    // threadToTransitPos[threadIdx.x] = 0;
  } else if (trCount >= LoadBalancing::LoadBalancingThreshold::SubWarpLevel) {
    kernelType = TransitKernelTypes::SubWarpKernel;
    
    // numThreadGroups = 0;
    // threadToTransitPos[threadIdx.x] = 0;
  } // else {
  //   kernelType = TransitKernelTypes::IdentityKernel;
  //   // numThreadGroups = 0;
  //   // threadToTransitPos[threadIdx.x] = 0;
  // }
  
  if (threadId < uniqueTransitCountsNum && transit != invalidVertex) {
    kernelTypeForTransit[transit] = kernelType;
  }

  __syncthreads();

  for (int kTy = 1; kTy < TransitKernelTypes::SubWarpKernel + 1; kTy++) {
    EdgePos_t* glKernelTransitsNum, *glKernelTransits;
    const int threadGroupSize = (kTy == TransitKernelTypes::GridKernel) ? LoadBalancing::LoadBalancingThreshold::GridLevel : 
                                (kTy == TransitKernelTypes::ThreadBlockKernel ? LoadBalancing::LoadBalancingThreshold::BlockLevel : 
                                (kTy == TransitKernelTypes::SubWarpKernel ? LoadBalancing::LoadBalancingThreshold::SubWarpLevel : -1));

    if (kTy == TransitKernelTypes::GridKernel) {
      if (kernelType == TransitKernelTypes::GridKernel) {
        numThreadGroups = DIVUP(trCount, LoadBalancing::LoadBalancingThreshold::GridLevel);
        threadToTransitPos[threadIdx.x] = trPos;
        threadToTransit[threadIdx.x] = transit;
      } else {
        numThreadGroups = 0;
        threadToTransitPos[threadIdx.x] = 0;
        threadToTransit[threadIdx.x] = -1;
      } 
      glKernelTransitsNum = gridKernelTransitsNum;
      glKernelTransits = gridKernelTransits;
    } else if (kTy == TransitKernelTypes::ThreadBlockKernel) {
      if (kernelType == TransitKernelTypes::ThreadBlockKernel) {
        numThreadGroups = DIVUP(trCount, LoadBalancing::LoadBalancingThreshold::BlockLevel);
        threadToTransitPos[threadIdx.x] = trPos;
        threadToTransit[threadIdx.x] = transit;
      } else {
        numThreadGroups = 0;
        threadToTransitPos[threadIdx.x] = 0;
        threadToTransit[threadIdx.x] = -1;
      }       
      glKernelTransitsNum = threadBlockKernelTransitsNum;
      glKernelTransits = threadBlockKernelTransits;
    } else if (kTy == TransitKernelTypes::SubWarpKernel) {
      if (kernelType == TransitKernelTypes::SubWarpKernel) {
        numThreadGroups = DIVUP(trCount, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
        threadToTransitPos[threadIdx.x] = trPos;
        threadToTransit[threadIdx.x] = transit;
        //printf("blockIdx.x %d threadIdx.x %d transit %d trCount %d numThreadgroups %d\n", threadIdx.x, transit, trCount, numThreadGroups);
      } else {
        numThreadGroups = 0;
        threadToTransitPos[threadIdx.x] = 0;
        threadToTransit[threadIdx.x] = -1;
      }       
      glKernelTransitsNum = subWarpKernelTransitsNum;
      glKernelTransits = subWarpKernelTransits;
      
    } else {
      continue;
    }

    //Get all grid kernel transits
    EdgePos_t prefixSumThreadData = 0;
    BlockScan(temp_storage).ExclusiveSum(numThreadGroups, prefixSumThreadData);
    
    __syncthreads();

    if (threadIdx.x == blockDim.x - 1) {
      totalThreadGroups = prefixSumThreadData + numThreadGroups;
      threadGroupsInsertionPos = ::atomicAdd(glKernelTransitsNum, totalThreadGroups);
    }

    threadToTransitPrefixSum[threadIdx.x] = prefixSumThreadData;
    
    __syncthreads();
    
    // if (totalThreadGroups != 0 and numThreadGroups != 0) {
    //   printf("threadIdx.x %d blockIdx.x %d tr %d trPos %d numThreadGroups %d totalThreadGroups %d prefixSumThreadData %d\n", threadIdx.x, blockIdx.x, transit, trPos, numThreadGroups, totalThreadGroups, prefixSumThreadData);
    // }
    
    for (int tgIter = 0; tgIter < totalThreadGroups; tgIter += SHMEM_SIZE) {
      for (int i = threadIdx.x; i < SHMEM_SIZE; i+= blockDim.x) {
        shGridKernelTransits[i] = 0;
      }
    
      __syncthreads();
      
      int prefixSumIndex = prefixSumThreadData - tgIter;
      if (prefixSumIndex < 0 && prefixSumIndex + numThreadGroups > 0) {
        prefixSumIndex = 0;
      }
      if (numThreadGroups > 0) {
        if (prefixSumIndex >= 0 && prefixSumIndex < SHMEM_SIZE)
          shGridKernelTransits[prefixSumIndex] = threadIdx.x;
      }
      
      __syncthreads();

      for (int tbs = threadIdx.x; tbs < DIVUP(min(SHMEM_SIZE, totalThreadGroups - tgIter), TB_THREADS)*TB_THREADS; tbs += blockDim.x) {
        int d;
        if (tbs < TB_THREADS) {
          d = (tbs < totalThreadGroups) ? shGridKernelTransits[tbs] : 0;
        } else if (threadIdx.x == 0) {
          d = (tbs < totalThreadGroups) ? max(shGridKernelTransits[tbs], shGridKernelTransits[tbs-1]): 0;
        } else {
          d = (tbs < totalThreadGroups) ? shGridKernelTransits[tbs] : 0;
        }
        
        __syncthreads();
        BlockScan(temp_storage).InclusiveScan(d, d, hipcub::Max());
        __syncthreads();

        if (tbs < totalThreadGroups)
          shGridKernelTransits[tbs] = d;
          
        __syncthreads();

        
        int previousTrPrefixSum = (tbs < totalThreadGroups && shGridKernelTransits[tbs] >= 0) ? threadToTransitPrefixSum[shGridKernelTransits[tbs]] : 0;

        if (tbs + tgIter < totalThreadGroups) {
          // if (step == 1) {
          //   printf("blockIdx.x %d shGridKernelTransits[tbs] %d tbs %d\n", blockIdx.x, shGridKernelTransits[tbs], tbs);
          // }
          EdgePos_t startPos = threadToTransitPos[shGridKernelTransits[tbs]];
          EdgePos_t pos = startPos + threadGroupSize*(tbs  + tgIter - previousTrPrefixSum);
          VertexID_t transit = threadToTransit[shGridKernelTransits[tbs]];
          glKernelTransits[threadGroupsInsertionPos + tbs + tgIter] = pos;
          assert(kernelTypeForTransit[transit] == kTy);
          // if (transitToSamplesKeys[pos] != transit) {
          //   printf("blockIdx.x %d shGridKernelTransits[tbs] %d tbs %d tgIter %d startPos %d pos %d expectedTr %d threadTr %d\n", blockIdx.x, shGridKernelTransits[tbs], tbs, tgIter, startPos, pos, transitToSamplesKeys[pos], transit);
          // }
          assert(transitToSamplesKeys[pos] == transit);
        }
      }

      __syncthreads();
    }

    // if (threadIdx.x==0){
    //   for (int i = 0; i < totalThreadGroups; i++) {
    //    // printf("blockIdx.x %d gridKernelTransits[%d] %d step %d\n", blockIdx.x, i, gridKernelTransits[threadGroupsInsertionPos + i], step);
    //   }
    // }

    __syncthreads();
  }

  // if (threadIdx.x+blockIdx.x*blockDim.x==0) {
  //   printf("subWarpKernelTransitsNum %d\n", *subWarpKernelTransitsNum);
  // }
  #if 0
  int done = 0;
  int startCopyingIteration = prefixSumThreadData/SHMEM_SIZE;
  int endCopyingIteration = (prefixSumThreadData + numThreadGroups)/SHMEM_SIZE;

  __syncthreads();

  for (int tbs = 0; tbs < gridTotalTBs; tbs += SHMEM_SIZE) {
    if (trPos >= 0 && numThreadBlocks > 0 && done < numThreadBlocks && tbs/SHMEM_SIZE >= startCopyingIteration && tbs/SHMEM_SIZE <= endCopyingIteration) {
      int todo;
      for (todo = 0; todo < min(numThreadBlocks-done, SHMEM_SIZE); todo++) {
        int idx = prefixSumThreadData + done - tbs + todo;
        if (idx >= SHMEM_SIZE) {
          break;
        }
        if (idx < 0 || idx >= SHMEM_SIZE) {
          printf("idx %d prefixSum %d done %d tbs %d todo %d\n", idx, prefixSumThreadData, done, tbs, todo);
        }
        shGridKernelTransits[idx] = trPos + LoadBalancing::LoadBalancingThreshold::GridLevel*(todo+done);
      }
      done += todo;
    }

    __syncthreads();

    for (EdgePos_t i = threadIdx.x; i < min(SHMEM_SIZE, gridTotalTBs - tbs); i+=blockDim.x) {
      gridKernelTransits[gridInsertionPos + tbs + i] = shGridKernelTransits[i];
    }
    __syncthreads();
  }
  #endif

  // if (threadIdx.x == 0) {
  //   for (EdgePos_t i = 0; i < gridTotalTBs; i+=1) {
  //     printf("%d %d, %d\n", blockIdx.x, i, gridKernelTransits[gridInsertionPos + i]);
  //   }
  // }
}

__global__ void init_curand_states(hiprandState* states, size_t num_states)
{
  int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
  if (thread_id < num_states)
    hiprand_init(thread_id, 0, 0, &states[thread_id]);
}

CSR* loadGraph(Graph& graph, char* graph_file, char* graph_type, char* graph_format)
{
  CSR* csr;

   //Load Graph
   if (strcmp(graph_type, "adj-list") == 0) {
    if (strcmp(graph_format, "text") == 0) {
      graph.load_from_adjacency_list(graph_file);
      //Convert graph to CSR format
      csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
      csr_from_graph (csr, graph);
      return csr;
    }
    else {
      printf ("graph_format '%s' not supported for graph_type '%s'\n", 
              graph_format, graph_type);
      return nullptr;
    }
  } else if (strcmp(graph_type, "edge-list") == 0) {
    if (strcmp(graph_format, "binary") == 0) {
      graph.load_from_edge_list_binary(graph_file, true);
      csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
      csr_from_graph (csr, graph);
      return csr;
    } else if (strcmp(graph_format, "text") == 0) {
      FILE* fp = fopen (graph_file, "r");
      if (fp == nullptr) {
        std::cout << "File '" << graph_file << "' not found" << std::endl;
        return nullptr;
      }
      graph.load_from_edge_list_txt(fp, true);
      fclose (fp);
      csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
      csr_from_graph (csr, graph);
      return csr;
    } else {
      printf ("graph_format '%s' not supported for graph_type '%s'\n", 
              graph_format, graph_type);
      return nullptr;
    }
  } else {
    printf("Incorrect graph file type '%s'\n", graph_type);
    return nullptr;
  }

  return nullptr;
}

GPUCSRPartition transferCSRToGPU(CSR* csr)
{
  //Assume that whole graph can be stored in GPU Memory.
  //Hence, only one Graph Partition is created.
  CSRPartition full_partition = CSRPartition (0, csr->get_n_vertices() - 1, 0, csr->get_n_edges() - 1, 
                                              csr->get_vertices(), csr->get_edges(), csr->get_weights());
  
  //Copy full graph to GPU
  GPUCSRPartition gpuCSRPartition;
  CSRPartition deviceCSRPartition = copyPartitionToGPU(full_partition, gpuCSRPartition);
  gpuCSRPartition.device_csr = (CSRPartition*)csrPartitionBuff;
  CHK_CU(hipMemcpyToSymbol(HIP_SYMBOL(csrPartitionBuff), &deviceCSRPartition, sizeof(CSRPartition)));
  return gpuCSRPartition;
}

bool allocNextDoorDataOnGPU(CSR* csr, NextDoorData& data)
{
  //Initially each sample contains only one vertex
  //Allocate one sample for each vertex
  int maxV = 0;
  for (auto vertex : csr->iterate_vertices()) {
    data.samples.push_back(vertex);
    maxV = (maxV < vertex) ? vertex : maxV;
  }

  //Size of each sample output
  size_t maxNeighborsToSample = 1;
  for (int step = 0; step < steps(); step++) {
    maxNeighborsToSample *= stepSize(step);
  }

  size_t finalSampleSize = 0;
  size_t neighborsToSampleAtStep = 1;
  for (int step = 0; step < steps(); step++) {
    neighborsToSampleAtStep *= stepSize(step);
    finalSampleSize += neighborsToSampleAtStep;
  }

  data.INVALID_VERTEX = csr->get_n_vertices();
  int maxBits = 0;
  while ((data.INVALID_VERTEX >> maxBits) != 0) {
    maxBits++;
  }
  
  data.maxBits = maxBits + 1;

  //Allocate storage for final samples on GPU
  data.hFinalSamples = std::vector<VertexID_t>(finalSampleSize*data.samples.size());

  CHK_CU(hipMalloc(&data.dFinalSamples, sizeof(VertexID_t)*data.hFinalSamples.size()));
  gpu_memset(data.dFinalSamples, data.INVALID_VERTEX, data.hFinalSamples.size());
  //Samples to Transit Map
  //TODO: hFinalSamples.size() is wrong.
  CHK_CU(hipMalloc(&data.dSamplesToTransitMapKeys, sizeof(VertexID_t)*data.samples.size()*maxNeighborsToSample));
  CHK_CU(hipMalloc(&data.dSamplesToTransitMapValues, sizeof(VertexID_t)*data.samples.size()*maxNeighborsToSample));

  //Transit to Samples Map
  //TODO: hFinalSamples.size() is wrong. It should be maximum number of transits.
  CHK_CU(hipMalloc(&data.dTransitToSampleMapKeys, sizeof(VertexID_t)*data.samples.size()*maxNeighborsToSample));
  CHK_CU(hipMalloc(&data.dTransitToSampleMapValues, sizeof(VertexID_t)*data.samples.size()*maxNeighborsToSample));

  //Same as initial values of samples for first iteration
  CHK_CU(hipMemcpy(data.dTransitToSampleMapKeys, &data.samples[0], sizeof(VertexID_t)*data.samples.size(), 
                    hipMemcpyHostToDevice));
  CHK_CU(hipMemcpy(data.dTransitToSampleMapValues, &data.samples[0], sizeof(VertexID_t)*data.samples.size(), 
                    hipMemcpyHostToDevice));

  //Insertion positions per transit vertex for each sample
  
  CHK_CU(hipMalloc(&data.dSampleInsertionPositions, sizeof(EdgePos_t)*data.samples.size()));

  CHK_CU(hipMalloc(&data.dCurandStates, maxNeighborsToSample*data.samples.size()*sizeof(hiprandState)));
  init_curand_states<<<thread_block_size(data.samples.size()*maxNeighborsToSample, 256UL), 256UL>>> (data.dCurandStates, data.samples.size()*maxNeighborsToSample);
  CHK_CU(hipDeviceSynchronize());

  return true;
}

void freeDeviceData(NextDoorData& data) 
{
  CHK_CU(hipFree(data.dSamplesToTransitMapKeys));
  CHK_CU(hipFree(data.dSamplesToTransitMapValues));
  CHK_CU(hipFree(data.dTransitToSampleMapKeys));
  CHK_CU(hipFree(data.dTransitToSampleMapValues));
  CHK_CU(hipFree(data.dSampleInsertionPositions));
  CHK_CU(hipFree(data.dCurandStates));
  CHK_CU(hipFree(data.dFinalSamples));
  CHK_CU(hipFree(data.gpuCSRPartition.device_vertex_array));
  CHK_CU(hipFree(data.gpuCSRPartition.device_edge_array));
  CHK_CU(hipFree(data.gpuCSRPartition.device_weights_array));
}

void printKernelTypes(CSR* csr, VertexID_t* dUniqueTransits, VertexID_t* dUniqueTransitsCounts, EdgePos_t* dUniqueTransitsNumRuns)
{
  EdgePos_t* hUniqueTransitsNumRuns = GPUUtils::copyDeviceMemToHostMem(dUniqueTransitsNumRuns, 1);
  VertexID_t* hUniqueTransits = GPUUtils::copyDeviceMemToHostMem(dUniqueTransits, *hUniqueTransitsNumRuns);
  VertexID_t* hUniqueTransitsCounts = GPUUtils::copyDeviceMemToHostMem(dUniqueTransitsCounts, *hUniqueTransitsNumRuns);

  size_t identityKernelTransits = 0, identityKernelSamples = 0, maxEdgesOfIdentityTransits = 0;
  size_t subWarpLevelTransits = 0, subWarpLevelSamples = 0, maxEdgesOfSubWarpTransits = 0, subWarpTransitsWithEdgesLessThan384 = 0, subWarpTransitsWithEdgesMoreThan384 = 0, numSubWarps = 0;
  size_t threadBlockLevelTransits = 0, threadBlockLevelSamples = 0, tbVerticesWithEdgesLessThan3K = 0, tbVerticesWithEdgesMoreThan3K = 0;
  size_t gridLevelTransits = 0, gridLevelSamples = 0, gridVerticesWithEdgesLessThan10K = 0, gridVerticesWithEdgesMoreThan10K = 0;
  EdgePos_t maxEdgesOfGridTransits = 0;

  for (size_t tr = 0; tr < *hUniqueTransitsNumRuns; tr++) {
    // if (tr == 0) {printf("%s:%d hUniqueTransitsCounts[0] is %d\n", __FILE__, __LINE__, hUniqueTransitsCounts[tr]);}
    if (hUniqueTransitsCounts[tr] < 8) {
      identityKernelTransits++;
      identityKernelSamples += hUniqueTransitsCounts[tr];
      maxEdgesOfIdentityTransits = max(maxEdgesOfIdentityTransits, (size_t)csr->n_edges_for_vertex(tr));
    } else if (hUniqueTransitsCounts[tr] <= LoadBalancing::LoadBalancingThreshold::BlockLevel && 
               hUniqueTransitsCounts[tr] >= 8) {
      subWarpLevelTransits++;
      subWarpLevelSamples += hUniqueTransitsCounts[tr];
      maxEdgesOfSubWarpTransits = max(maxEdgesOfSubWarpTransits, (size_t)csr->n_edges_for_vertex(tr));
      numSubWarps += DIVUP(hUniqueTransitsCounts[tr], LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
      if (csr->n_edges_for_vertex(tr) <= 384) {
        subWarpTransitsWithEdgesLessThan384 += 1;
      } else {
        subWarpTransitsWithEdgesMoreThan384 += 1;
      }
    } else if (hUniqueTransitsCounts[tr] > LoadBalancing::LoadBalancingThreshold::BlockLevel && 
               hUniqueTransitsCounts[tr] <= LoadBalancing::LoadBalancingThreshold::GridLevel) {
      threadBlockLevelTransits++;
      threadBlockLevelSamples += hUniqueTransitsCounts[tr];
      if (csr->n_edges_for_vertex(tr) <= 3*1024) {
        tbVerticesWithEdgesLessThan3K += 1;
      } else {
        tbVerticesWithEdgesMoreThan3K += 1;
      }
    } else {
      gridLevelTransits++;
      gridLevelSamples += hUniqueTransitsCounts[tr];
      if (csr->n_edges_for_vertex(tr) <= 3*1024) {
        gridVerticesWithEdgesLessThan10K += 1;
      } else {
        gridVerticesWithEdgesMoreThan10K += 1;
      }
      maxEdgesOfGridTransits = max(maxEdgesOfGridTransits, csr->n_edges_for_vertex(tr));
    }
  }

  printf("IdentityKernelTransits: %ld, IdentityKernelSamples: %ld, MaxEdgesOfIdentityTransits: %ld\n" 
         "SubWarpLevelTransits: %ld, SubWarpLevelSamples: %ld, MaxEdgesOfSubWarpTranits: %ld, VerticesWithEdges > 384: %ld, VerticesWithEdges <= 384: %ld, NumSubWarps: %ld\n"
         "ThreadBlockLevelTransits: %ld, ThreadBlockLevelSamples: %ld, VerticesWithEdges > 3K: %ld, VerticesWithEdges < 3K: %ld\n"
         "GridLevelTransits: %ld, GridLevelSamples: %ld, VerticesWithEdges > 10K: %ld, VerticesWithEdges < 10K: %ld, MaxEdgesOfTransit: %d\n", 
         identityKernelTransits, identityKernelSamples, maxEdgesOfIdentityTransits, 
         subWarpLevelTransits, subWarpLevelSamples, maxEdgesOfSubWarpTransits, 
            subWarpTransitsWithEdgesMoreThan384, subWarpTransitsWithEdgesLessThan384, numSubWarps, 
         threadBlockLevelTransits, threadBlockLevelSamples, tbVerticesWithEdgesMoreThan3K, tbVerticesWithEdgesLessThan3K,
         gridLevelTransits, gridLevelSamples, gridVerticesWithEdgesMoreThan10K, gridVerticesWithEdgesLessThan10K, maxEdgesOfGridTransits);

  delete hUniqueTransits;
  delete hUniqueTransitsCounts;
  delete hUniqueTransitsNumRuns;
}

bool doTransitParallelSampling(CSR* csr, GPUCSRPartition gpuCSRPartition, NextDoorData& nextDoorData, bool enableLoadBalancing)
{
  //Size of each sample output
  size_t maxNeighborsToSample = 1;
  for (int step = 0; step < steps(); step++) {
    maxNeighborsToSample *= stepSize(step);
  }

  size_t finalSampleSize = 0;
  size_t neighborsToSampleAtStep = 1;
  for (int step = 0; step < steps(); step++) {
    neighborsToSampleAtStep *= stepSize(step);
    finalSampleSize += neighborsToSampleAtStep;
  }
  
  neighborsToSampleAtStep = 1;
  CHK_CU(hipMemcpy(nextDoorData.dTransitToSampleMapKeys, &nextDoorData.samples[0], sizeof(VertexID_t)*nextDoorData.samples.size(), 
                  hipMemcpyHostToDevice));
  CHK_CU(hipMemcpy(nextDoorData.dTransitToSampleMapValues, &nextDoorData.samples[0], sizeof(VertexID_t)*nextDoorData.samples.size(), 
                  hipMemcpyHostToDevice));
  VertexID_t* d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  VertexID_t* dUniqueTransits = nullptr;
  VertexID_t* dUniqueTransitsCounts = nullptr;
  EdgePos_t* dUniqueTransitsNumRuns = nullptr;
  EdgePos_t* dTransitPositions = nullptr;
  EdgePos_t* uniqueTransitNumRuns = nullptr;
   
  /**Pointers for each kernel type**/
  EdgePos_t* gridKernelTransitsNum = nullptr;
  EdgePos_t* dGridKernelTransitsNum = nullptr;
  VertexID_t* dGridKernelTransits = nullptr;
  
  EdgePos_t* threadBlockKernelTransitsNum = nullptr;
  EdgePos_t* dThreadBlockKernelTransitsNum = nullptr;
  VertexID_t* dThreadBlockKernelTransits = nullptr;

  EdgePos_t* subWarpKernelTransitsNum = nullptr;
  EdgePos_t* dSubWarpKernelTransitsNum = nullptr;
  VertexID_t* dSubWarpKernelTransits = nullptr;
  /**********************************/

  /*Single Memory Location on both CPU and GPU for transferring
   *number of transits for all kernels */
  EdgePos_t* dKernelTransitNums;
  EdgePos_t* hKernelTransitNums;
  const int NUM_KERNEL_TYPES = TransitKernelTypes::NumKernelTypes;

  int* dKernelTypeForTransit = nullptr;

  CHK_CU(hipHostMalloc(&uniqueTransitNumRuns, sizeof(EdgePos_t)));
  CHK_CU(hipHostMalloc(&hKernelTransitNums, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
  
  gridKernelTransitsNum = hKernelTransitNums;
  threadBlockKernelTransitsNum = hKernelTransitNums + 1;
  subWarpKernelTransitsNum = hKernelTransitNums + 2;
  //threadBlockKernelTransitsNum = hKernelTransitNums[3];
  
  CHK_CU(hipMalloc(&dKernelTypeForTransit, sizeof(VertexID_t)*csr->get_n_vertices()));
  CHK_CU(hipMalloc(&dTransitPositions, 
                    sizeof(VertexID_t)*nextDoorData.samples.size()));
  CHK_CU(hipMalloc(&dGridKernelTransits, 
                    sizeof(VertexID_t)*nextDoorData.samples.size()*maxNeighborsToSample));
  CHK_CU(hipMalloc(&dThreadBlockKernelTransits, 
                    sizeof(VertexID_t)*nextDoorData.samples.size()*maxNeighborsToSample));
  CHK_CU(hipMalloc(&dSubWarpKernelTransits,
                    sizeof(VertexID_t)*nextDoorData.samples.size()*maxNeighborsToSample));

  CHK_CU(hipMalloc(&dKernelTransitNums, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
  CHK_CU(hipMemset(dKernelTransitNums, 0, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
  dGridKernelTransitsNum = dKernelTransitNums;
  dThreadBlockKernelTransitsNum = dKernelTransitNums + 1;
  dSubWarpKernelTransitsNum = dKernelTransitNums + 2;

  int* atomicPtrTest = nullptr;
  CHK_CU(hipMalloc(&atomicPtrTest, sizeof(int)));
  //Check if the space runs out.
  //TODO: Use DoubleBuffer version that requires O(P) space.
  //TODO: hFinalSamples.size() is wrong.
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, 
            nextDoorData.dSamplesToTransitMapValues, nextDoorData.dTransitToSampleMapKeys, 
            nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dTransitToSampleMapValues, 
            nextDoorData.samples.size()*maxNeighborsToSample);

  CHK_CU(hipMalloc(&dUniqueTransits, (csr->get_n_vertices() + 1)*sizeof(VertexID_t)));
  CHK_CU(hipMalloc(&dUniqueTransitsCounts, (csr->get_n_vertices() + 1)*sizeof(VertexID_t)));
  CHK_CU(hipMalloc(&dUniqueTransitsNumRuns, sizeof(size_t)));
  
  if (temp_storage_bytes < nextDoorData.samples.size()*maxNeighborsToSample) {
    temp_storage_bytes = nextDoorData.samples.size()*maxNeighborsToSample;
  }

  // VertexID_t* gt1, *gt2;
  // CHK_CU(hipMalloc(&gt1, nextDoorData.samples.size()*maxNeighborsToSample*sizeof(VertexID_t)));
  // CHK_CU(hipMalloc(&gt2, nextDoorData.samples.size()*maxNeighborsToSample*sizeof(VertexID_t)));

  size_t free = 0, total = 0;
  CHK_CU(hipMemGetInfo(&free, &total));
  // printf("free memory %ld temp_storage_bytes %ld nextDoorData.samples.size() %ld maxNeighborsToSample %ld\n", free, temp_storage_bytes, nextDoorData.samples.size(), maxNeighborsToSample);
  CHK_CU(hipMalloc(&d_temp_storage, temp_storage_bytes));

  std::vector<VertexID_t*> hAllSamplesToTransitMapKeys;
  std::vector<VertexID_t*> hAllTransitToSampleMapValues;

  double loadBalancingTime = 0;
  double inversionTime = 0;
  double gridKernelTime = 0;
  double subWarpKernelTime = 0;
  double identityKernelTime = 0;
  double end_to_end_t1 = convertTimeValToDouble(getTimeOfDay ());
  for (int step = 0; step < steps(); step++) {
    neighborsToSampleAtStep *= stepSize(step);
    const size_t totalThreads = nextDoorData.samples.size()*neighborsToSampleAtStep;
    
    if (step == 0 || !enableLoadBalancing) {
      //When not doing load balancing call baseline transit parallel
      samplingKernel<<<thread_block_size(totalThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
                      (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys, (const VertexID_t*)nextDoorData.dTransitToSampleMapValues,
                      totalThreads, nextDoorData.samples.size(),
                      nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dSamplesToTransitMapValues,
                      nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
                      nextDoorData.dCurandStates);
      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
    } else {
      double loadBalancingT1 = convertTimeValToDouble(getTimeOfDay ());
      void* dRunLengthEncodeTmpStorage = nullptr;
      size_t dRunLengthEncodeTmpStorageSize = 0;

      hipcub::DeviceRunLengthEncode::Encode(dRunLengthEncodeTmpStorage, dRunLengthEncodeTmpStorageSize, 
                                        nextDoorData.dTransitToSampleMapKeys,
                                        dUniqueTransits, dUniqueTransitsCounts, dUniqueTransitsNumRuns, totalThreads);

      assert(dRunLengthEncodeTmpStorageSize < temp_storage_bytes);
      dRunLengthEncodeTmpStorage = d_temp_storage;
      hipcub::DeviceRunLengthEncode::Encode(dRunLengthEncodeTmpStorage, dRunLengthEncodeTmpStorageSize, 
                                        nextDoorData.dTransitToSampleMapKeys,
                                        dUniqueTransits, dUniqueTransitsCounts, dUniqueTransitsNumRuns, totalThreads);

      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      
      CHK_CU(hipMemcpy(uniqueTransitNumRuns, dUniqueTransitsNumRuns, sizeof(*uniqueTransitNumRuns), hipMemcpyDeviceToHost));

      void* dExclusiveSumTmpStorage = nullptr;
      size_t dExclusiveSumTmpStorageSize = 0;
      
      hipcub::DeviceScan::ExclusiveSum(dExclusiveSumTmpStorage, dExclusiveSumTmpStorageSize, dUniqueTransitsCounts, dTransitPositions, *uniqueTransitNumRuns);

      assert(dExclusiveSumTmpStorageSize < temp_storage_bytes);
      dExclusiveSumTmpStorage = d_temp_storage;

      hipcub::DeviceScan::ExclusiveSum(dExclusiveSumTmpStorage, dExclusiveSumTmpStorageSize, dUniqueTransitsCounts, dTransitPositions, *uniqueTransitNumRuns);

      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      //printKernelTypes(csr, dUniqueTransits, dUniqueTransitsCounts, dUniqueTransitsNumRuns);

      CHK_CU(hipMemset(dKernelTransitNums, 0, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
      partitionTransitsInKernels<1024><<<thread_block_size((*uniqueTransitNumRuns), 1024), 1024>>>(step, dUniqueTransits, dUniqueTransitsCounts, 
          dTransitPositions, *uniqueTransitNumRuns, nextDoorData.INVALID_VERTEX, dGridKernelTransits, dGridKernelTransitsNum, 
          dThreadBlockKernelTransits, dThreadBlockKernelTransitsNum, dSubWarpKernelTransits, dSubWarpKernelTransitsNum, nullptr, nullptr, dKernelTypeForTransit,
          nextDoorData.dTransitToSampleMapKeys);

      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      CHK_CU(hipMemcpy(hKernelTransitNums, dKernelTransitNums, NUM_KERNEL_TYPES * sizeof(EdgePos_t), hipMemcpyDeviceToHost));
      
      // GPUUtils::printDeviceArray(dGridKernelTransits, *gridKernelTransitsNum, ',');
      // getchar();
      double loadBalancingT2 = convertTimeValToDouble(getTimeOfDay ());
      loadBalancingTime += (loadBalancingT2 - loadBalancingT1);
      
      double identityKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
      identityKernel<<<thread_block_size(totalThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
        (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys, (const VertexID_t*)nextDoorData.dTransitToSampleMapValues,
        totalThreads, nextDoorData.samples.size(),
        nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dSamplesToTransitMapValues,
        nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
        nextDoorData.dCurandStates, dKernelTypeForTransit);
      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      double identityKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
      identityKernelTime += (identityKernelTimeT2 - identityKernelTimeT1);
      
      const int perThreadSamplesForSubWarpKernel = 2;
      int threadBlocks = DIVUP(DIVUP(*subWarpKernelTransitsNum*LoadBalancing::LoadBalancingThreshold::SubWarpLevel, perThreadSamplesForSubWarpKernel), N_THREADS);
      //std::cout << "subWarpKernelTransitsNum " << *subWarpKernelTransitsNum << " threadBlocks " << threadBlocks << std::endl;
      double subWarpKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
      if (useSubWarpKernel) {
        subWarpKernel<N_THREADS,3*1024-3,false,true,false,perThreadSamplesForSubWarpKernel,true><<<threadBlocks, N_THREADS>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
          (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys, (const VertexID_t*)nextDoorData.dTransitToSampleMapValues,
          totalThreads, nextDoorData.samples.size(),
          nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dSamplesToTransitMapValues,
          nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
          nextDoorData.dCurandStates, dKernelTypeForTransit, dSubWarpKernelTransits, *subWarpKernelTransitsNum);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
      }
      double subWarpKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
      subWarpKernelTime += (subWarpKernelTimeT2 - subWarpKernelTimeT1);

      const int perThreadSamplesForGridKernel = 4;
      double gridKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
      threadBlocks = DIVUP(*gridKernelTransitsNum, perThreadSamplesForGridKernel);
      if (useGridKernel) {
        gridKernel<256,3*1024-3,false,true,false,perThreadSamplesForGridKernel,true><<<threadBlocks, 256>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
          (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys, (const VertexID_t*)nextDoorData.dTransitToSampleMapValues,
          totalThreads, nextDoorData.samples.size(),
          nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dSamplesToTransitMapValues,
          nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
          nextDoorData.dCurandStates, dKernelTypeForTransit, dGridKernelTransits, *gridKernelTransitsNum);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
      }
      double gridKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
      gridKernelTime += (gridKernelTimeT2 - gridKernelTimeT1);

      // atomicPointerInc<<<thread_block_size(totalThreads, N_THREADS), N_THREADS>>>(atomicPtrTest);
      // CHK_CU(hipGetLastError());
      // CHK_CU(hipDeviceSynchronize());
    }

    if (step != steps() - 1) {
      double inversionT1 = convertTimeValToDouble(getTimeOfDay ());
      //Invert sample->transit map by sorting samples based on the transit vertices
      hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, 
                                      nextDoorData.dSamplesToTransitMapValues, nextDoorData.dTransitToSampleMapKeys, 
                                      nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dTransitToSampleMapValues, 
                                      totalThreads, 0, nextDoorData.maxBits);
      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      double inversionT2 = convertTimeValToDouble(getTimeOfDay ());
      inversionTime += (inversionT2 - inversionT1);

      #if 0
      VertexID_t* hTransitToSampleMapKeys = new VertexID_t[totalThreads];
      VertexID_t* hTransitToSampleMapValues = new VertexID_t[totalThreads];
      VertexID_t* hSampleToTransitMapKeys = new VertexID_t[totalThreads];
      VertexID_t* hSampleToTransitMapValues = new VertexID_t[totalThreads];
      CHK_CU(hipMemcpy(hSampleToTransitMapKeys, nextDoorData.dSamplesToTransitMapKeys, 
        totalThreads*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      CHK_CU(hipMemcpy(hSampleToTransitMapValues, nextDoorData.dSamplesToTransitMapValues,
        totalThreads*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      CHK_CU(hipMemcpy(hTransitToSampleMapKeys, nextDoorData.dTransitToSampleMapKeys, 
                        totalThreads*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      CHK_CU(hipMemcpy(hTransitToSampleMapValues, nextDoorData.dTransitToSampleMapValues,
                        totalThreads*sizeof(VertexID_t), hipMemcpyDeviceToHost));
                  
      hAllTransitToSampleMapValues.push_back(hTransitToSampleMapValues);
      hAllSamplesToTransitMapKeys.push_back(hSampleToTransitMapKeys);
      #endif
    }
  }

  double end_to_end_t2 = convertTimeValToDouble(getTimeOfDay ());

  CHK_CU(hipMemset(nextDoorData.dSampleInsertionPositions, 0, sizeof(EdgePos_t)*nextDoorData.samples.size()));

  std::cout << "Transit Parallel: End to end time " << (end_to_end_t2 - end_to_end_t1) << " secs" << std::endl;
  std::cout << "InversionTime: " << inversionTime <<", " << "LoadBalancingTime: " << loadBalancingTime << ", " << "GridKernelTime: " << gridKernelTime << ", SubWarpKernelTime: " << subWarpKernelTime << ", IdentityKernelTime: "<< identityKernelTime << std::endl;
  CHK_CU(hipFree(d_temp_storage));
  CHK_CU(hipFree(dUniqueTransits));
  CHK_CU(hipFree(dUniqueTransitsCounts));
  CHK_CU(hipFree(dUniqueTransitsNumRuns));

  #if 0
  for (int s = 1; s < steps() - 2; s++) {
    std::unordered_set<VertexID_t> s1, s2, intersection;
    for (int i = 100000; i < 200000; i++) {
      VertexID_t v1 = hAllSamplesToTransitMapKeys[s+1][i];
      VertexID_t v2 = hAllTransitToSampleMapValues[s+2][i];
      //printf("v1 %d v2 %d\n", v1, v2);
      s1.insert(v1);
      s2.insert(v2);
    }
    
    for (auto e : s1) {
      if (s2.count(e) == 1) intersection.insert(e);
    }

    std::cout << "s: " << s << " intersection: " << intersection.size() << std::endl;
  }
  #endif
  return true;
}

bool doSampleParallelSampling(GPUCSRPartition gpuCSRPartition, NextDoorData& nextDoorData)
{
  //Size of each sample output
  size_t maxNeighborsToSample = 1;
  for (int step = 0; step < steps(); step++) {
    maxNeighborsToSample *= stepSize(step);
  }

  size_t finalSampleSize = 0;
  size_t neighborsToSampleAtStep = 1;
  for (int step = 0; step < steps(); step++) {
    neighborsToSampleAtStep *= stepSize(step);
    finalSampleSize += neighborsToSampleAtStep;
  }
  
  neighborsToSampleAtStep = 1;
  
  double end_to_end_t1 = convertTimeValToDouble(getTimeOfDay ());
  for (int step = 0; step < steps(); step++) {
    neighborsToSampleAtStep *= stepSize(step);
    const size_t totalThreads = nextDoorData.samples.size()*neighborsToSampleAtStep;
    
    //Sample neighbors of transit vertices
    sampleParallelKernel<<<thread_block_size(totalThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
                    nextDoorData.samples.size(), nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
                    nextDoorData.dCurandStates);

                    
    CHK_CU(hipGetLastError());
    CHK_CU(hipDeviceSynchronize());
  }

  double end_to_end_t2 = convertTimeValToDouble(getTimeOfDay ());
  

  CHK_CU(hipMemset(nextDoorData.dSampleInsertionPositions, 0, sizeof(EdgePos_t)*nextDoorData.samples.size()));

  std::cout << "SampleParallel: End to end time " << (end_to_end_t2 - end_to_end_t1) << " secs" << std::endl;
  return true;
}


std::vector<VertexID_t>& getFinalSamples(NextDoorData& nextDoorData)
{
  CHK_CU(hipMemcpy(&nextDoorData.hFinalSamples[0], nextDoorData.dFinalSamples, 
                    nextDoorData.hFinalSamples.size()*sizeof(nextDoorData.hFinalSamples[0]), hipMemcpyDeviceToHost));
  return nextDoorData.hFinalSamples;
}

int nextdoor(const char* graph_file, const char* graph_type, const char* graph_format, 
             const int nruns, const bool chk_results, const bool print_samples,
             const char* kernelType, const bool enableLoadBalancing)
{
  std::vector<Vertex> vertices;

  //Load Graph
  Graph graph;
  CSR* csr;
  if ((csr = loadGraph(graph, (char*)graph_file, (char*)graph_type, (char*)graph_format)) == nullptr) {
    return 1;
  }

  std::cout << "Graph has " <<graph.get_n_edges () << " edges and " << 
      graph.get_vertices ().size () << " vertices " << std::endl; 

  //graph.print(std::cout);
  GPUCSRPartition gpuCSRPartition = transferCSRToGPU(csr);
  
  NextDoorData nextDoorData;
  nextDoorData.gpuCSRPartition = gpuCSRPartition;
  allocNextDoorDataOnGPU(csr, nextDoorData);
  
  for (int i = 0; i < nruns; i++) {
    if (strcmp(kernelType, "TransitParallel") == 0)
      doTransitParallelSampling(csr, gpuCSRPartition, nextDoorData, enableLoadBalancing);
    else if (strcmp(kernelType, "SampleParallel") == 0)
      doSampleParallelSampling(gpuCSRPartition, nextDoorData);
    else
      abort();
  }
    

  std::vector<VertexID_t>& hFinalSamples = getFinalSamples(nextDoorData);

  size_t maxNeighborsToSample = 1;
  for (int step = 0; step < steps(); step++) {
    maxNeighborsToSample *= stepSize(step);
  }

  size_t finalSampleSize = 0;
  size_t neighborsToSampleAtStep = 1;
  for (int step = 0; step < steps(); step++) {
    neighborsToSampleAtStep *= stepSize(step);
    finalSampleSize += neighborsToSampleAtStep;
  }
  
  size_t totalSampledVertices = 0;
  for (auto s : hFinalSamples) {
    totalSampledVertices += (int)(s != nextDoorData.INVALID_VERTEX);
  }

  if (print_samples) {
    for (size_t s = 0; s < hFinalSamples.size(); s += finalSampleSize) {
      std::cout << "Contents of sample " << s/finalSampleSize << " [";
      for(size_t v = s; v < s + finalSampleSize; v++)
        std::cout << hFinalSamples[v] << ", ";
      std::cout << "]" << std::endl;
    }
  }

  std::cout << "totalSampledVertices " << totalSampledVertices << std::endl;
  freeDeviceData(nextDoorData);
  if (chk_results)
    return check_result(csr, nextDoorData.INVALID_VERTEX, nextDoorData.samples, finalSampleSize, hFinalSamples, 4);

  return true;
}

#endif
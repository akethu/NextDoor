#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <algorithm>
#include <numeric>
#include <string.h>
#include <assert.h>
#include <tuple>
#include <queue>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/device/device_select.cuh>
#include <hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "sample.hpp"

#ifndef __NEXTDOOR_CU__
#define __NEXTDOOR_CU__

typedef VertexID VertexID_t;

//TODO-List:
//[] Divide main() function in several small functions.
//[] Divide the code in several include files that can be included in the API.
//[] In GPU Kernels, do refactoring and move them to other places.
//[] Use vectors instead of dynamic arrays and new.
//[] Convert these vectors to a new array type that does not do initialization of data.
//[] Use MKL or cuSPARSE to do the matrix transpose or sorting
//[] A configuration that specifies all the parameters.
//[] Use Templates for cleaner code of Sampler

//Supported:
//citeseer.graph
// const int N = 3312;
// const int N_EDGES = 9074;
//micro.graph
//const int N = 100000;
//const int N_EDGES = 2160312;
//rmat.graph
// const int N = 1024;
// const int N_EDGES = 29381;
//ego-facebook
// const int N = 4039;
// const int N_EDGES = 88244;
//ego-twitter
//const int N = 81306;
//const int N_EDGES = 2420766;
//ego-gplus
//const int N = 107614;
//const int N_EDGES = 13652253;
//soc-pokec-relationships
//const int N = 1632803;
//const int N_EDGES = 30480021;
//soc-LiveJournal1
//const int N = 4847571;
//const int N_EDGES = 68556521;

//Not supportred:
//com-orkut.ungraph
// const int N = 3072441;
// const int N_EDGES = 117185083;

#include "csr.hpp"
#include "utils.hpp"
#include "sampler.cuh"
#include "rand_num_gen.cuh"
#include "libNextDoor.hpp"

using namespace utils;
using namespace GPUUtils;

#define CHECK_RESULT

//For mico, 512 works best
const size_t N_THREADS = 256;

//TODO try for larger random walks to improve results

#define WARP_HOP

const int ALL_NEIGHBORS = -1;

const bool useGridKernel = true;
const bool useSubWarpKernel = false;
const bool useThreadBlockKernel = false;
const bool combineTwoSampleStores = true;

enum TransitKernelTypes {
  GridKernel = 1,
  ThreadBlockKernel = 2,
  SubWarpKernel = 3,
  IdentityKernel = 4,
  NumKernelTypes = 4
};

/**User Defined Functions**/
enum SamplingType {
  IndividualNeighborhood,
  CollectiveNeighborhood
};

enum OutputFormat {
  SampledVertices,
  AdjacencyMatrix
};

/************Application Functions********** 
__host__ __device__ int stepSize(int k);

template<class SampleType> 
__device__ inline
VertexID next(int step, CSRPartition* csr, const VertexID* transit, 
              const VertexID sampleID, SampleType* sample,
              const float maxWeight,
              const CSR::Edge* transitEdges, const float* transitEdgeWeights,
              const EdgePos_t numEdges, const EdgePos_t neighbrID, 
              hiprandState* state);
template<class SampleType, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool DECREASE_GM_LOADS>
__device__ inline
VertexID nextCached(int step, const VertexID transit, 
              const VertexID sampleID, SampleType* sample,
              const float maxWeight,
              const CSR::Edge* transitEdges, const float* transitEdgeWeights,
              const EdgePos_t numEdges, const EdgePos_t neighbrID, 
              hiprandState* state, VertexID_t* cachedEdges, float* cachedWeights,
              bool* globalLoadBV);
__host__ __device__ int steps();
__host__ __device__ int samplingType();
__host__ __device__ bool hasExplicitTransits();
template<class SampleType>
__device__ VertexID_t stepTransits(int step, const VertexID_t sampleID, SampleType& sample, const int transitIdx, hiprandState* randState);
template<class SampleType>
__host__ SampleType initializeSample(CSR* graph, const VertexID_t sampleID);
__host__ __device__ OutputFormat outputFormat();
__host__ __device__ EdgePos_t (CSR* graph);
__host__ __device__ EdgePos_t initialSampleSize(CSR* graph);
template<class SampleType>
__host__ std::vector<VertexID_t> initialSample(int sampleIdx, CSR* graph, SampleType& sample);
*********************/

__constant__ char csrPartitionBuff[sizeof(CSRPartition)];

template<typename App>
__host__ __device__
EdgePos_t newNeighborsSize(int hop, EdgePos_t num_edges)
{
  return (App().stepSize(hop) == ALL_NEIGHBORS) ? num_edges : (EdgePos_t)App().stepSize(hop);
}

template<typename App>
__host__ __device__
EdgePos_t stepSizeAtStep(int step)
{
  if (step == -1)
    return 0;

  if (App().samplingType() == IndividualNeighborhood) {
    EdgePos_t n = App().initialSampleSize(nullptr);
    for (int i = 0; i <= step; i++) {
      n = n * App().stepSize(i);
    }

    return n;
  } else if (App().samplingType() == CollectiveNeighborhood) {
    EdgePos_t n = 0;
    for (int i = 0; i <= step; i++) {
      n += App().stepSize(i);
    }

    return n;
  }

  return -1;
}

template<typename App>
__host__ __device__ int numberOfTransits(int step) {
  if (App().samplingType() == CollectiveNeighborhood) {
    return App().stepSize(step);
  } else if (App().samplingType() == IndividualNeighborhood) {
    return stepSizeAtStep<App>(step);
  }
  assert(false);
  return -1;
}

template<class SamplingType, typename App>
__global__ void samplingKernel(const int step, GPUCSRPartition graph, const size_t threadsExecuted, const size_t currExecutionThreads,
                               const VertexID_t invalidVertex,
                               const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                               const size_t transitToSamplesSize, SamplingType* samples, const size_t NumSamples,
                               VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                               VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                               hiprandState* randStates)
{
  EdgePos_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
  //__shared__ VertexID newNeigbhors[N_THREADS];

  if (threadId >= currExecutionThreads)
    return;
  
  hiprandState* randState = &randStates[threadId];

  threadId += threadsExecuted;
  EdgePos_t transitIdx = threadId/App().stepSize(step);
  EdgePos_t transitNeighborIdx = threadId % App().stepSize(step);
  VertexID_t sampleIdx = transitToSamplesValues[transitIdx];
  assert(sampleIdx < NumSamples);
  VertexID_t transit = transitToSamplesKeys[transitIdx];
  VertexID_t neighbor = invalidVertex;
  graph.device_csr = (CSRPartition*)&csrPartitionBuff[0];

  if (transit != invalidVertex) {
    // if (graph.device_csr->has_vertex(transit) == false)
    //   printf("transit %d\n", transit);
    assert(graph.device_csr->has_vertex(transit));

    EdgePos_t numTransitEdges = graph.device_csr->get_n_edges_for_vertex(transit);
    
    if (numTransitEdges != 0) {
      const CSR::Edge* transitEdges = graph.device_csr->get_edges(transit);
      const float* transitEdgeWeights = graph.device_csr->get_weights(transit);
      const float maxWeight = graph.device_csr->get_max_weight(transit);

      neighbor = App().next(step, graph.device_csr, &transit, sampleIdx, &samples[sampleIdx], maxWeight, transitEdges, transitEdgeWeights, 
                      numTransitEdges, transitNeighborIdx, randState);
#if 0
      //search if neighbor has already been selected.
      //we can do that in register if required
      newNeigbhors[threadIdx.x] = neighbor;

      bool found = false;
      for (int i = 0; i < N_THREADS; i++) {
        if (newNeigbhors[i] == neighbor) {
          found = true;
          // break;
        }
      }

      __syncwarp();
      if (found) {
        neighbor = App().next(step, transit, sample, transitEdges, numTransitEdges, 
          transitNeighborIdx, randState);;
      }
#endif
    }
  }

  __syncwarp();

  EdgePos_t totalSizeOfSample = stepSizeAtStep<App>(step - 1);

  //TODO: templatize over hasExplicitTransits()
  if (step != App().steps() - 1) {
    //No need to store at last step
    if (App().hasExplicitTransits()) {
      VertexID_t transit = App().stepTransits(step, sampleIdx, samples[sampleIdx], transitIdx, randState);
      samplesToTransitValues[threadId] = transit;
    } else {
      samplesToTransitValues[threadId] = neighbor;
    }
    samplesToTransitKeys[threadId] = sampleIdx;
  }

  EdgePos_t insertionPos = 0;

  if (numberOfTransits<App>(step) > 1) {    
    //insertionPos = finalSampleSizeTillPreviousStep + transitNeighborIdx; //
    if (step == 0) {
      insertionPos = transitNeighborIdx;
    } else {
      EdgePos_t numTransits = numberOfTransits<App>(step);
      size_t finalSampleSizeTillPreviousStep = 0;
      size_t neighborsToSampleAtStep = 1;
      for (int _s = 0; _s < step; _s++) {
        neighborsToSampleAtStep *= App().stepSize(_s);
        finalSampleSizeTillPreviousStep += neighborsToSampleAtStep;
      }
      insertionPos = finalSampleSizeTillPreviousStep + utils::atomicAdd(&sampleInsertionPositions[sampleIdx], 1);
    }
  } else {
    insertionPos = step;
  }

  // if (insertionPo

  // if (insertionPos < finalSampleSize) {
  //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
  // }
  assert(finalSampleSize > 0);
  if (insertionPos >= finalSampleSize) {
    printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sampleIdx);
  }
  assert(insertionPos < finalSampleSize);
  if (numberOfTransits<App>(step) == 1 and combineTwoSampleStores) {
    if (step % 2 == 1) {
      finalSamples[sampleIdx*finalSampleSize + insertionPos - 1] = transit;
      finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
    } else if (step == App().steps() - 1) {
      finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
    }
  }
  else {
    finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
  }

  // if (sample == 100) {
  //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
  // }
}

template<class SampleType, typename App, int THREADS, bool COALESCE_CURAND_LOAD>
__global__ void identityKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                               const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                               const size_t transitToSamplesSize, SampleType* samples, const size_t NumSamples,
                               VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                               VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                               hiprandState* randStates, const int* kernelTypeForTransit)
{
  __shared__ unsigned char shMemCuRand[sizeof(hiprandState)*THREADS];

  int threadId = threadIdx.x + blockDim.x * blockIdx.x;

  hiprandState* curandSrcPtr;

  if (COALESCE_CURAND_LOAD) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMemCuRand[0];

    int* randStatesAsInts = (int*)randStates;
  
    for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  hiprandState localRandState = *curandSrcPtr;

  //__shared__ VertexID newNeigbhors[N_THREADS];
  EdgePos_t transitIdx;
  EdgePos_t transitNeighborIdx;
  VertexID_t transit;
  int kernelTy;

  if (threadId >= transitToSamplesSize)
    return;
  
  transitIdx = threadId/App().stepSize(step);
  transitNeighborIdx = threadId % App().stepSize(step);
  transit = transitToSamplesKeys[transitIdx];
  kernelTy = kernelTypeForTransit[transit];
  
  if ((useGridKernel && kernelTy == TransitKernelTypes::GridKernel) || 
      (useSubWarpKernel && kernelTy == TransitKernelTypes::SubWarpKernel) || 
      (useThreadBlockKernel && kernelTy == TransitKernelTypes::ThreadBlockKernel)) {
    return;
  }

  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];
  VertexID_t sampleIdx = transitToSamplesValues[transitIdx];
  assert(sampleIdx < NumSamples);
  VertexID_t neighbor = invalidVertex;

  if (transit != invalidVertex) {
    // if (graph.device_csr->has_vertex(transit) == false)
    //   printf("transit %d\n", transit);
    assert(csr->has_vertex(transit));

    EdgePos_t numTransitEdges = csr->get_n_edges_for_vertex(transit);
    
    if (numTransitEdges != 0) {
      const CSR::Edge* transitEdges = csr->get_edges(transit);
      const float* transitEdgeWeights = csr->get_weights(transit);
      const float maxWeight = csr->get_max_weight(transit);

      neighbor = App().next(step, csr, &transit, sampleIdx, &samples[sampleIdx], maxWeight, transitEdges, transitEdgeWeights, 
                      numTransitEdges, transitNeighborIdx, &localRandState);
#if 0
      //search if neighbor has already been selected.
      //we can do that in register if required
      newNeigbhors[threadIdx.x] = neighbor;

      bool found = false;
      for (int i = 0; i < N_THREADS; i++) {
        if (newNeigbhors[i] == neighbor) {
          found = true;
          // break;
        }
      }

      __syncwarp();
      if (found) {
        neighbor = next(step, transit, sample, transitEdges, numTransitEdges, 
          transitNeighborIdx, randState);;
      }
#endif
    }
  }

  __syncwarp();

//  EdgePos_t totalSizeOfSample = stepSizeAtStep<App>(step - 1);

  if (step != App().steps() - 1) {
    //No need to store at last step
    if (App().hasExplicitTransits()) {
      VertexID_t transit = App().stepTransits(step, sampleIdx, samples[sampleIdx], transitIdx, &localRandState);
      samplesToTransitValues[threadId] = transit;
    } else {
      samplesToTransitValues[threadId] = neighbor;
    }
    samplesToTransitKeys[threadId] = sampleIdx;
  }
  
  EdgePos_t insertionPos = 0; 
  if (false && numberOfTransits<App>(step) > 1) {    
    insertionPos = utils::atomicAdd(&sampleInsertionPositions[sampleIdx], 1);
  } else {
    insertionPos = step;
  }

  // if (insertionPos < finalSampleSize) {
  //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
  // }
  assert(finalSampleSize > 0);
  // if (insertionPos >= finalSampleSize) {
  //   printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sample);
  // }
  assert(insertionPos < finalSampleSize);

  if (combineTwoSampleStores) {
    if (step % 2 == 1) {
      int2 *ptr = (int2*)&finalSamples[sampleIdx*finalSampleSize + insertionPos - 1];
      int2 res;
      res.x = transit;
      res.y = neighbor;
      *ptr = res;
      //finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
    } else if (step == App().steps() - 1) {
      finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
    }
  } else {
    finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
  }
  // if (sample == 100) {
  //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
  // }
  //TODO: We do not need atomic instead store indices of transit in another array,
  //wich can be accessed based on sample and transitIdx.
}

template<class SampleType, typename App, int THREADS, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool COALESCE_GL_LOADS, int TRANSITS_PER_THREAD, bool COALESCE_CURAND_LOAD>
__global__ void subWarpKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                              const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                              const size_t transitToSamplesSize, SampleType* samples, const size_t NumSamples,
                              VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                              VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                              hiprandState* randStates, const int* kernelTypeForTransit, const VertexID_t* subWarpKernelTBPositions, 
                              const EdgePos_t subWarpKernelTBPositionsNum)
{  
  // __shared__ unsigned char shMemAlloc[sizeof(hiprandState)*THREADS];
  // __shared__ EdgePos_t shSubWarpPositions[SUBWARPS_IN_TB*TRANSITS_PER_THREAD];
  const int SUBWARPS_IN_TB = THREADS/LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
  const int EDGE_CACHE_SIZE = (CACHE_EDGES ? CACHE_SIZE * sizeof(CSR::Edge) : 0);
  const int WEIGHT_CACHE_SIZE = (CACHE_WEIGHTS ? CACHE_SIZE * sizeof(float) : 0);
  const int TOTAL_CACHE_SIZE = MAX(WEIGHT_CACHE_SIZE + EDGE_CACHE_SIZE, 1); 
  const int CACHE_SIZE_PER_SUBWARP = CACHE_SIZE/SUBWARPS_IN_TB;

  union unionShMem {
    struct {
      EdgePos_t shSubWarpPositions[SUBWARPS_IN_TB*TRANSITS_PER_THREAD];
      unsigned char edgeAndWeightCache[TOTAL_CACHE_SIZE];
    };
    unsigned char shMemAlloc[sizeof(hiprandState)*THREADS];
  };
  __shared__ unionShMem shMem;
  
  const int threadId = threadIdx.x + blockDim.x * blockIdx.x;

  const int subWarpThreadIdx = threadId % LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
  const int subWarp = threadId / LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
  const int subWarpIdxInTB = threadIdx.x/LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
  const int startSubWarpIdxInTB = (blockIdx.x*blockDim.x)/LoadBalancing::LoadBalancingThreshold::SubWarpLevel;

  EdgePos_t* edgesInShMem = (EdgePos_t*) (CACHE_EDGES ? &shMem.edgeAndWeightCache[CACHE_SIZE_PER_SUBWARP*subWarpIdxInTB] : nullptr);
  float* edgeWeightsInShMem = (float*) (CACHE_WEIGHTS ? (&shMem.edgeAndWeightCache[EDGE_CACHE_SIZE + CACHE_SIZE_PER_SUBWARP*subWarpIdxInTB]): nullptr);
  bool* globalLoadBV = nullptr;

  hiprandState* curandSrcPtr;

  if (COALESCE_CURAND_LOAD) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMem.shMemAlloc[0];

    int* randStatesAsInts = (int*)randStates;
  
    for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  hiprandState localRandState = *curandSrcPtr;
  
  for (int _subWarpIdx = threadIdx.x; _subWarpIdx < SUBWARPS_IN_TB * TRANSITS_PER_THREAD; _subWarpIdx += blockDim.x) {
    if (_subWarpIdx + startSubWarpIdxInTB * TRANSITS_PER_THREAD >= subWarpKernelTBPositionsNum) {
      continue;
    }
    shMem.shSubWarpPositions[_subWarpIdx] = subWarpKernelTBPositions[_subWarpIdx + startSubWarpIdxInTB * TRANSITS_PER_THREAD];
  }

  __syncthreads();
  bool invalidateCache;
  VertexID_t currTransit = invalidVertex;

  invalidateCache = true;
  EdgePos_t numTransitEdges;
  CSR::Edge* glTransitEdges;
  float* glTransitEdgeWeights;
  float maxWeight;

  for (int transitI = 0; transitI < TRANSITS_PER_THREAD; transitI++) {
    EdgePos_t subWarpIdx = TRANSITS_PER_THREAD * subWarp + transitI;
    if (subWarpIdx >= subWarpKernelTBPositionsNum) {
      continue;
    }

    EdgePos_t transitStartPos = shMem.shSubWarpPositions[subWarpIdxInTB * TRANSITS_PER_THREAD + transitI];
    EdgePos_t transitIdx = transitStartPos + subWarpThreadIdx;
    EdgePos_t transitNeighborIdx = 0;
    VertexID_t transit = transitIdx < NumSamples ? transitToSamplesKeys[transitIdx] : -1;
    // if ((uint64_t)(transitToSamplesKeys + transitIdx) % 32 != 0) {
    //   printf("unaligned %p %p %d %d\n", transitToSamplesKeys + transitIdx, transitToSamplesKeys, transitIdx, transitStartPos);
    // }    
    VertexID_t firstThreadTransit = __shfl_sync(FULL_WARP_MASK, transit, 0, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
    __syncwarp();

    invalidateCache = currTransit != firstThreadTransit;
    currTransit = firstThreadTransit;

    CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];
    
    int tmpReadVertexData;

    if (invalidateCache) {
      const CSR::Vertex* transitVertex = csr->get_vertex(currTransit);
      if (subWarpThreadIdx < sizeof(CSR::Vertex)/sizeof(int)) {
        tmpReadVertexData = ((const int*)transitVertex)[subWarpThreadIdx];
      }
    }
    
    __syncwarp();

    const EdgePos_t startEdgeIdx = __shfl_sync(FULL_WARP_MASK, tmpReadVertexData, 1, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
    const EdgePos_t endEdgeIdx = __shfl_sync(FULL_WARP_MASK, tmpReadVertexData, 2, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
    
    if (invalidateCache) {
      int maxWeightBuff = __shfl_sync(FULL_WARP_MASK, tmpReadVertexData, 3, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);      
      maxWeight = *((float*)&maxWeightBuff);
      numTransitEdges = (endEdgeIdx != -1) ? (endEdgeIdx - startEdgeIdx + 1) : 0; 
     
      glTransitEdges = (CSR::Edge*)((startEdgeIdx != -1) ? csr->get_edges() + startEdgeIdx : nullptr);
      glTransitEdgeWeights = (float*)((startEdgeIdx != -1) ? csr->get_weights() + startEdgeIdx : nullptr);
    }

    if (false) {
      //shMem.edgeAndWeightCache[threadIdx.x%32] = numTransitEdges + (int32_t)maxWeight + (int32_t)glTransitEdges + (int32_t)glTransitEdgeWeights;
      continue;
    }

    if (CACHE_EDGES && invalidateCache) {
      for (int e = subWarpThreadIdx; e < min((EdgePos_t)CACHE_SIZE_PER_SUBWARP, numTransitEdges); 
           e += LoadBalancing::LoadBalancingThreshold::SubWarpLevel) {
        edgesInShMem[e] = -1;
      }
    }

    if (CACHE_WEIGHTS && invalidateCache) {
      for (int e = subWarpThreadIdx; e < min((EdgePos_t)CACHE_SIZE_PER_SUBWARP, numTransitEdges); 
           e += LoadBalancing::LoadBalancingThreshold::SubWarpLevel) {
        edgeWeightsInShMem[e] = -1;
      }
    }

    __syncwarp();

    if (firstThreadTransit != transit)
      continue;

    // int kernelTy = kernelTypeForTransit[transit];
    // if (kernelTy != TransitKernelTypes::SubWarpKernel) {
    //   printf("threadId %d transitIdx %d kernelTy %d\n", threadId, transitIdx, kernelTy);
    // }
    //assert(kernelTypeForTransit[firstThreadTransit] == TransitKernelTypes::SubWarpKernel);
    VertexID_t sampleIdx = transitToSamplesValues[transitIdx];
    assert(sampleIdx < NumSamples);
    VertexID_t neighbor = invalidVertex;

    // if (graph.device_csr->has_vertex(transit) == false)
    //   printf("transit %d\n", transit);
    assert(csr->has_vertex(transit));
    
    if (numTransitEdges != 0) {
      neighbor = App().template nextCached<SampleType, CACHE_SIZE_PER_SUBWARP, CACHE_EDGES, CACHE_WEIGHTS, COALESCE_GL_LOADS>(step, transit, sampleIdx, &samples[sampleIdx], maxWeight, 
                                                                                    glTransitEdges, glTransitEdgeWeights, 
                                                                                    numTransitEdges, transitNeighborIdx, &localRandState,
                                                                                    edgesInShMem, edgeWeightsInShMem,
                                                                                    globalLoadBV);
    }

    // __syncwarp();

    //EdgePos_t totalSizeOfSample = stepSizeAtStep<App>(step - 1);

    if (step != App().steps() - 1) {
      //No need to store at last step
      samplesToTransitKeys[transitIdx] = sampleIdx;
      if (App().hasExplicitTransits()) {
        VertexID_t transit = App().stepTransits(step, sampleIdx, samples[sampleIdx], transitIdx, &localRandState);
        samplesToTransitValues[threadId] = transit;
      } else {
        samplesToTransitValues[threadId] = neighbor;
      }
    }
    
    EdgePos_t insertionPos = 0; 
    if (false && numberOfTransits<App>(step) > 1) {    
      insertionPos = utils::atomicAdd(&sampleInsertionPositions[sampleIdx], 1);
    } else {
      insertionPos = step;
    }

    // if (insertionPos < finalSampleSize) {
    //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
    // }
    assert(finalSampleSize > 0);
    assert(insertionPos < finalSampleSize);
    if (combineTwoSampleStores) {
      if (step % 2 == 1) {
        int2 *ptr = (int2*)&finalSamples[sampleIdx*finalSampleSize + insertionPos - 1];
        int2 res;
        res.x = transit;
        res.y = neighbor;
        *ptr = res;
        //finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
      } else if (step == App().steps() - 1) {
        finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
      }
    } else {
      finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
    }
    // if (sample == 100) {
    //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
    // }
    //TODO: We do not need atomic instead store indices of transit in another array,
    //wich can be accessed based on sample and transitIdx.
  }
}

template<int CACHE_SIZE, bool COALESCE_GL_LOADS, typename T>
__device__ inline VertexID_t cacheAndGet(EdgePos_t id, const T* transitEdges, T* cachedEdges, bool* globalLoadBV)
{
  VertexID_t e;
  if (id >= CACHE_SIZE) {
    return transitEdges[id];
  }
  
  if (COALESCE_GL_LOADS) {
    e = cachedEdges[id];
    if (e == -1)
      cachedEdges[id] = -2;

    int subWarpThreadIdx = threadIdx.x % LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
    //int subWarp = threadIdx.x / LoadBalancing::LoadBalancingThreshold::SubWarpLevel;
    for (int i = subWarpThreadIdx; i < CACHE_SIZE; i += LoadBalancing::LoadBalancingThreshold::SubWarpLevel) {
      if (cachedEdges[i] == -2) {
        cachedEdges[i] = transitEdges[i];
      }
    }
    
    e = cachedEdges[id];
  } else {
    e = cachedEdges[id];

    if (e == -1) {
      e = transitEdges[id];
      cachedEdges[id] = e;
    }
  }

  return e;
}

template<class SampleType, typename App, int THREADS, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool COALESCE_GL_LOADS, int TRANSITS_PER_THREAD, bool COALESCE_CURAND_LOAD>
__global__ void threadBlockKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                                  const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                                  const size_t transitToSamplesSize, SampleType* samples, const size_t NumSamples,
                                  VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                                  VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                                  hiprandState* randStates, const int* kernelTypeForTransit, 
                                  const VertexID_t* threadBlockKernelTBPositions, 
                                  const EdgePos_t threadBlockKernelTBPositionsNum)
{
  //TODO: This works with thread block size of 32 only and NEEDS to be optimized.
  #define EDGE_CACHE_SIZE (CACHE_EDGES ? CACHE_SIZE*sizeof(CSR::Edge) : 0)
  #define WEIGHT_CACHE_SIZE (CACHE_WEIGHTS ? CACHE_SIZE*sizeof(float) : 0)
  #define CURAND_SHMEM_SIZE (sizeof(hiprandState)*THREADS)
  // #define COALESCE_GL_LOADS_SHMEM_SIZE ()

  __shared__ unsigned char shMemAlloc[MAX(EDGE_CACHE_SIZE+WEIGHT_CACHE_SIZE, CURAND_SHMEM_SIZE)];
  
  //__shared__ bool globalLoadBV[COALESCE_GL_LOADS ? CACHE_SIZE : 1];
  bool* globalLoadBV;
  __shared__ VertexID_t numEdgesInShMem;
  __shared__ bool invalidateCache;
  __shared__ VertexID_t transitForTB;
  __shared__ CSR::Edge* glTransitEdges;
  __shared__ float* glTransitEdgeWeights;
  __shared__ float maxWeight;
  __shared__ EdgePos_t mapStartPos;

  CSR::Edge* edgesInShMem = CACHE_EDGES ? (CSR::Edge*)&shMemAlloc[0] : nullptr;
  float* edgeWeightsInShMem = CACHE_WEIGHTS ? (float*)&shMemAlloc[EDGE_CACHE_SIZE] : nullptr;
  
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  
  hiprandState* curandSrcPtr;

  if (COALESCE_CURAND_LOAD) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMemAlloc[0];

    int* randStatesAsInts = (int*)randStates;
  
    for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  hiprandState localRandState = *curandSrcPtr;

  //__shared__ VertexID newNeigbhors[N_THREADS];
  //if (threadIdx.x == 0) printf("blockIdx.x %d\n", blockIdx.x);
  //shRandStates[threadIdx.x] = randStates[threadId];  
  //__syncthreads();
  
  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];

  for (int transitI = 0; transitI < TRANSITS_PER_THREAD; transitI++) {
    EdgePos_t transitIdx = 0;
    EdgePos_t transitNeighborIdx = 0;//threadId % stepSize(step); //TODO: Correct this for k-hop
    if (TRANSITS_PER_THREAD * blockIdx.x + transitI >= threadBlockKernelTBPositionsNum) {
      continue;
    }
    if (threadIdx.x == 0) {
      mapStartPos = threadBlockKernelTBPositions[TRANSITS_PER_THREAD * blockIdx.x + transitI];
    }
    __syncthreads();
    transitIdx = mapStartPos + threadIdx.x; //threadId/stepSize(step);
    VertexID_t transit = transitToSamplesKeys[transitIdx];

    if (threadIdx.x == 0) {
      invalidateCache = transitForTB != transit || transitI == 0;
      transitForTB = transit;
    }
    if (threadIdx.x == 0 && invalidateCache) {
      //assert(graph.device_csr->has_vertex(transit));
      //TODO: fuse below functions into one to decrease reads
      numEdgesInShMem = csr->get_n_edges_for_vertex(transit);
      glTransitEdges = (CSR::Edge*)csr->get_edges(transit);
      glTransitEdgeWeights = (float*)csr->get_weights(transit);
      maxWeight = csr->get_max_weight(transit);
    }

    __syncthreads();

    if (CACHE_EDGES && invalidateCache) {
      for (int i = threadIdx.x; i < min(CACHE_SIZE, numEdgesInShMem); i += blockDim.x) {
        edgesInShMem[i] = -1;//glTransitEdges[i];
      }
    }
  
    if (CACHE_WEIGHTS && invalidateCache) {
      for (int i = threadIdx.x; i < min(CACHE_SIZE, numEdgesInShMem); i += blockDim.x) {
        edgeWeightsInShMem[i] = -1;//glTransitEdgeWeights[i];
      }
    }

    __syncthreads();

    if (transit == transitForTB) {
      // if (threadIdx.x == 0 && kernelTypeForTransit[transit] != TransitKernelTypes::GridKernel) {
      //   printf("transit %d transitIdx %d gridDim.x %d\n", transit, transitIdx, gridDim.x);
      // }
      // assert (kernelTypeForTransit[transit] == TransitKernelTypes::GridKernel);

      VertexID_t sampleIdx = transitToSamplesValues[transitIdx];

      assert(sampleIdx < NumSamples);
      VertexID_t neighbor = invalidVertex;
      // if (graph.device_csr->has_vertex(transit) == false)
      //   printf("transit %d\n", transit);
      neighbor = App().template nextCached<SampleType, CACHE_SIZE, CACHE_EDGES, CACHE_WEIGHTS, 0>(step, transit, sampleIdx, &samples[sampleIdx], maxWeight, 
                                                              glTransitEdges, glTransitEdgeWeights, 
                                                              numEdgesInShMem, transitNeighborIdx, &localRandState,
                                                              edgesInShMem, edgeWeightsInShMem,
                                                              &globalLoadBV[0]);
      __syncwarp();

      //EdgePos_t totalSizeOfSample = stepSizeAtStep<App>(step - 1);

      if (step != App().steps() - 1) {
        //No need to store at last step
        samplesToTransitKeys[transitIdx] = sampleIdx; //TODO: Update this for khop to transitIdx + transitNeighborIdx
        if (App().hasExplicitTransits()) {
          VertexID_t transit = App().stepTransits(step, sampleIdx, samples[sampleIdx], transitIdx, &localRandState);
          samplesToTransitValues[transitIdx] = transit;
        } else {
          samplesToTransitValues[transitIdx] = neighbor;
        }
      }
      
      EdgePos_t insertionPos = 0; 
      if (false && numberOfTransits<App>(step) > 1) {
        //insertionPos = utils::atomicAdd(&sampleInsertionPositions[sample], 1);
      } else {
        insertionPos = step;
      }

      // if (insertionPos < finalSampleSize) {
      //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
      // }
      assert(finalSampleSize > 0);
      if (insertionPos >= finalSampleSize) {
        printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sampleIdx);
      }
      assert(insertionPos < finalSampleSize);

      if (combineTwoSampleStores) {
        if (step % 2 == 1) {
          int2 *ptr = (int2*)&finalSamples[sampleIdx*finalSampleSize + insertionPos - 1];
          int2 res;
          res.x = transit;
          res.y = neighbor;
          *ptr = res;
          //finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
        } else if (step == App().steps() - 1) {
          finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
        }
      } else {
        finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
      }
      // if (sample == 100) {
      //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
      // }
      //TODO: We do not need atomic instead store indices of transit in another array,
      //wich can be accessed based on sample and transitIdx.
    }
  }
}

template<class SampleType, typename App, int THREADS, int CACHE_SIZE, bool CACHE_EDGES, bool CACHE_WEIGHTS, bool COALESCE_GL_LOADS, int TRANSITS_PER_THREAD, bool COALESCE_CURAND_LOAD>
__global__ void gridKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                           const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                           const size_t transitToSamplesSize, SampleType* samples, const size_t NumSamples,
                           VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                           VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                           hiprandState* randStates, const int* kernelTypeForTransit, const VertexID_t* gridKernelTBPositions, 
                           const EdgePos_t gridKernelTBPositionsNum)
{
  #define EDGE_CACHE_SIZE (CACHE_EDGES ? CACHE_SIZE*sizeof(CSR::Edge) : 0)
  #define WEIGHT_CACHE_SIZE (CACHE_WEIGHTS ? CACHE_SIZE*sizeof(float) : 0)
  #define CURAND_SHMEM_SIZE (sizeof(hiprandState)*THREADS)
  // #define COALESCE_GL_LOADS_SHMEM_SIZE ()

  __shared__ unsigned char shMemAlloc[MAX(EDGE_CACHE_SIZE+WEIGHT_CACHE_SIZE, CURAND_SHMEM_SIZE)];
  
  //__shared__ bool globalLoadBV[COALESCE_GL_LOADS ? CACHE_SIZE : 1];
  bool* globalLoadBV;
  __shared__ VertexID_t numEdgesInShMem;
  __shared__ bool invalidateCache;
  __shared__ VertexID_t transitForTB;
  __shared__ CSR::Edge* glTransitEdges;
  __shared__ float* glTransitEdgeWeights;
  __shared__ float maxWeight;
  __shared__ EdgePos_t mapStartPos;

  CSR::Edge* edgesInShMem = CACHE_EDGES ? (CSR::Edge*)&shMemAlloc[0] : nullptr;
  float* edgeWeightsInShMem = CACHE_WEIGHTS ? (float*)&shMemAlloc[EDGE_CACHE_SIZE] : nullptr;
  
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  
  hiprandState* curandSrcPtr;

  if (COALESCE_CURAND_LOAD) {
    const int intsInRandState = sizeof(hiprandState)/sizeof(int);
    int* shStateBuff = (int*)&shMemAlloc[0];

    int* randStatesAsInts = (int*)randStates;
  
    for (int i = threadIdx.x; i < intsInRandState*blockDim.x; i += blockDim.x) {
      shStateBuff[i] = randStatesAsInts[i + blockDim.x*blockIdx.x];
    }

    __syncthreads();
    curandSrcPtr = (hiprandState*)(&shStateBuff[threadIdx.x*intsInRandState]);
  } else {
    curandSrcPtr = &randStates[threadId];
  }

  hiprandState localRandState = *curandSrcPtr;

  //__shared__ VertexID newNeigbhors[N_THREADS];
  //if (threadIdx.x == 0) printf("blockIdx.x %d\n", blockIdx.x);
  //shRandStates[threadIdx.x] = randStates[threadId];  
  //__syncthreads();
  
  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];

  for (int transitI = 0; transitI < TRANSITS_PER_THREAD; transitI++) {
    EdgePos_t transitIdx = 0;
    EdgePos_t transitNeighborIdx = 0;//threadId % stepSize(step); //TODO: Correct this for k-hop
    if (TRANSITS_PER_THREAD * blockIdx.x + transitI >= gridKernelTBPositionsNum) {
      continue;
    }
    if (threadIdx.x == 0) {
      mapStartPos = gridKernelTBPositions[TRANSITS_PER_THREAD * blockIdx.x + transitI];
    }
    __syncthreads();
    transitIdx = mapStartPos + threadIdx.x; //threadId/stepSize(step);
    VertexID_t transit = transitToSamplesKeys[transitIdx];

    if (threadIdx.x == 0) {
      invalidateCache = transitForTB != transit || transitI == 0;
      transitForTB = transit;
    }
    if (threadIdx.x == 0 && invalidateCache) {
      //assert(graph.device_csr->has_vertex(transit));
      //TODO: fuse below functions into one to decrease reads
      numEdgesInShMem = csr->get_n_edges_for_vertex(transit);
      glTransitEdges = (CSR::Edge*)csr->get_edges(transit);
      glTransitEdgeWeights = (float*)csr->get_weights(transit);
      maxWeight = csr->get_max_weight(transit);
    }

    __syncthreads();

    if (CACHE_EDGES && invalidateCache) {
      for (int i = threadIdx.x; i < min(CACHE_SIZE, numEdgesInShMem); i += blockDim.x) {
        edgesInShMem[i] = -1;//glTransitEdges[i];
      }
    }
  
    if (CACHE_WEIGHTS && invalidateCache) {
      for (int i = threadIdx.x; i < min(CACHE_SIZE, numEdgesInShMem); i += blockDim.x) {
        edgeWeightsInShMem[i] = -1;//glTransitEdgeWeights[i];
      }
    }

    __syncthreads();

    if (transit == transitForTB) {
      // if (threadIdx.x == 0 && kernelTypeForTransit[transit] != TransitKernelTypes::GridKernel) {
      //   printf("transit %d transitIdx %d gridDim.x %d\n", transit, transitIdx, gridDim.x);
      // }
      // assert (kernelTypeForTransit[transit] == TransitKernelTypes::GridKernel);

      VertexID_t sampleIdx = transitToSamplesValues[transitIdx];

      assert(sampleIdx < NumSamples);
      VertexID_t neighbor = invalidVertex;
      // if (graph.device_csr->has_vertex(transit) == false)
      //   printf("transit %d\n", transit);
      neighbor = App().template nextCached<SampleType, CACHE_SIZE, CACHE_EDGES, CACHE_WEIGHTS, 0>(step, transit, sampleIdx, &samples[sampleIdx], maxWeight, 
                                                              glTransitEdges, glTransitEdgeWeights, 
                                                              numEdgesInShMem, transitNeighborIdx, &localRandState,
                                                              edgesInShMem, edgeWeightsInShMem,
                                                              &globalLoadBV[0]);
      __syncwarp();

      //EdgePos_t totalSizeOfSample = stepSizeAtStep<App>(step - 1);

      if (step != App().steps() - 1) {
        //No need to store at last step
        samplesToTransitKeys[transitIdx] = sampleIdx; //TODO: Update this for khop to transitIdx + transitNeighborIdx
        if (App().hasExplicitTransits()) {
          VertexID_t transit = App().stepTransits(step, sampleIdx, samples[sampleIdx], transitIdx, &localRandState);
          samplesToTransitValues[transitIdx] = transit;
        } else {
          samplesToTransitValues[transitIdx] = neighbor;
        }
      }
      
      EdgePos_t insertionPos = 0; 
      if (false && numberOfTransits<App>(step) > 1) {
        //insertionPos = utils::atomicAdd(&sampleInsertionPositions[sample], 1);
      } else {
        insertionPos = step;
      }

      // if (insertionPos < finalSampleSize) {
      //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
      // }
      assert(finalSampleSize > 0);
      if (insertionPos >= finalSampleSize) {
        printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sampleIdx);
      }
      assert(insertionPos < finalSampleSize);

      if (combineTwoSampleStores) {
        if (step % 2 == 1) {
          int2 *ptr = (int2*)&finalSamples[sampleIdx*finalSampleSize + insertionPos - 1];
          int2 res;
          res.x = transit;
          res.y = neighbor;
          *ptr = res;
          //finalSamples[sample*finalSampleSize + insertionPos] = neighbor;
        } else if (step == App().steps() - 1) {
          finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
        }
      } else {
        finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
      }
      // if (sample == 100) {
      //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
      // }
      //TODO: We do not need atomic instead store indices of transit in another array,
      //wich can be accessed based on sample and transitIdx.
    }
  }
}

template<typename App>
__global__ void collectiveNeighbrsSize(const int step, GPUCSRPartition graph, 
                                       const VertexID_t invalidVertex,
                                       VertexID_t* initialSamples, 
                                       VertexID_t* finalSamples, 
                                       const size_t finalSampleSize, 
                                       EdgePos_t* sampleNeighborhoodPos,
                                       EdgePos_t* sumNeighborhoodSizes)
{
  //Assign one thread block to a sample
  __shared__ EdgePos_t neighborhoodSize;

  if (threadIdx.x == 0) {
    neighborhoodSize = 0;
  }

  __syncthreads();

  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];  
  VertexID_t sampleIdx = blockIdx.x;
  EdgePos_t numTransits = App().initialSampleSize(nullptr);
  //EdgePos_t numTransitsInPrevStep = numberOfTransits(step - 1);

  //TODO: Assuming step is 0
  for (int transitIdx = threadIdx.x; transitIdx < numTransits; transitIdx += blockDim.x) {
    VertexID_t transit = initialSamples[sampleIdx*App().initialSampleSize(nullptr) + transitIdx];
    if (transit != invalidVertex) {
      ::atomicAdd(&neighborhoodSize, csr->get_n_edges_for_vertex(transit)); 
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    sampleNeighborhoodPos[sampleIdx] = ::atomicAdd(sumNeighborhoodSizes, neighborhoodSize);
  }
}

template<typename App>
__global__ void collectiveNeighborhood(const int step, GPUCSRPartition graph, 
                                       const VertexID_t invalidVertex,
                                       VertexID_t* initialSamples, 
                                       VertexID_t* finalSamples, 
                                       const size_t finalSampleSize, 
                                       EdgePos_t* sampleNeighborhoodCSRRows,
                                       VertexID_t* sampleNeighborhoodCSRCols,
                                       EdgePos_t* sampleNeighborhoodPos,
                                       EdgePos_t* sumNeighborhoodSizes)
{
  //Assign one thread block to a sample
  EdgePos_t insertionPos = 0;
  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];  
  VertexID_t sampleIdx = blockIdx.x;
  EdgePos_t numTransits = App().initialSampleSize(nullptr);
  //EdgePos_t numTransitsInPrevStep = numberOfTransits(step - 1);

  //TODO: Assuming step is 0
  //Copy edges from graph, vertex by vertex
  for (int transitIdx = 0; transitIdx < numTransits; transitIdx++) {
    VertexID_t transit = initialSamples[sampleIdx*App().initialSampleSize(nullptr) + transitIdx];
    EdgePos_t nEdges = csr->get_n_edges_for_vertex(transit);
    const CSR::Edge* edges = csr->get_edges(transit);
    
    sampleNeighborhoodCSRRows[sampleIdx*App().initialSampleSize(nullptr) + transitIdx] = insertionPos;

    for (int e = threadIdx.x; e < nEdges; e += blockDim.x) {
      EdgePos_t pos = sampleNeighborhoodPos[sampleIdx] + insertionPos + e;
      sampleNeighborhoodCSRCols[pos] = edges[e];
    }

    insertionPos += nEdges;
    __syncthreads();
  }
}

template<class SampleType, typename App, bool StoreAsMap>
__global__ void explicitTransitsKernel(const int step, GPUCSRPartition graph, 
                                     const VertexID_t invalidVertex,
                                     const size_t threadsExecuted, 
                                     const size_t currExecutionThreads,
                                     const size_t totalThreads,
                                     SampleType* samples,
                                     const size_t NumSamples,
                                     VertexID_t* samplesToTransitKeys,
                                     VertexID_t* samplesToTransitValues,
                                     hiprandState* randStates)
{
  //Number of threads executed are: Num of Samples * Number of Transits
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  //__shared__ VertexID newNeigbhors[N_THREADS];
  CSRPartition* csr = (CSRPartition*)&csrPartitionBuff[0];
  if (threadId >= currExecutionThreads)
    return;
  
  hiprandState* randState = &randStates[threadId];
  threadId += threadsExecuted;
  EdgePos_t numTransits = numberOfTransits<App>(step - 1);
  EdgePos_t sampleIdx = threadId/numTransits;
  EdgePos_t transitIdx = threadId % numTransits;
  if (App().samplingType() == CollectiveNeighborhood) {
    assert(!App().hasExplicitTransits());
  } else {
    VertexID_t transit = App().stepTransits(step, sampleIdx, samples[sampleIdx], transitIdx, randState);
    samplesToTransitValues[threadId] = transit;

    if (StoreAsMap) {
      samplesToTransitKeys[threadId] = sampleIdx;
    }
  }
}

template<class SampleType, typename App>
__global__ void sampleParallelKernel(const int step, GPUCSRPartition graph, 
                                     const VertexID_t invalidVertex,
                                     const size_t threadsExecuted, 
                                     const size_t currExecutionThreads,
                                     const size_t totalThreads,
                                     VertexID_t* initialSamples,
                                     SampleType* samples,
                                     const size_t NumSamples,
                                     VertexID_t* finalSamplesCSRCol, 
                                     EdgePos_t* finalSamplesCSRRow,
                                     float* finalSamplesCSRVal,
                                     VertexID_t* finalSamples,
                                     const size_t finalSampleSize, 
                                     VertexID_t* explicitTransits,
                                     EdgePos_t* sampleInsertionPositions,
                                     hiprandState* randStates)
{
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  //__shared__ VertexID newNeigbhors[N_THREADS];
  graph.device_csr = (CSRPartition*)&csrPartitionBuff[0];
  if (threadId >= currExecutionThreads)
    return;

  hiprandState* randState = &randStates[threadId];
  threadId += threadsExecuted;
  EdgePos_t numTransits = numberOfTransits<App>(step);
  EdgePos_t numTransitsInPrevStep = numberOfTransits<App>(step - 1);
  VertexID_t sampleIdx = threadId / numTransits;
  VertexID_t* transits = nullptr;
  EdgePos_t numTransitsInNeghbrhood = 0;
  //TODO: Template this kernel based on the sampling type
  if (App().samplingType() == CollectiveNeighborhood) {
    assert(!App().hasExplicitTransits());
    numTransitsInNeghbrhood = numberOfTransits<App>(step);
    if (step == 0) {
      transits = &initialSamples[sampleIdx*App().initialSampleSize(nullptr)];
    } else {
      size_t verticesAddTillPreviousStep = stepSizeAtStep<App>(step - 2);
      //printf("verticesAddTillPreviousStep %ld\n", verticesAddTillPreviousStep); 

      transits = &finalSamples[sampleIdx*finalSampleSize + verticesAddTillPreviousStep];
    }
  } else {
    if (step == 0) {
      EdgePos_t transitIdx = threadId % numTransits;
      transits = &initialSamples[sampleIdx*App().initialSampleSize(nullptr) + transitIdx];
    } else if (App().hasExplicitTransits()) {
      transits = &explicitTransits[sampleIdx*numTransitsInPrevStep + (threadId % numTransits) % numTransitsInPrevStep];
    } else {
      transits = &finalSamples[sampleIdx*finalSampleSize + (step - 1) * numTransits + (threadId % numTransits) % numTransitsInPrevStep];
    }

    numTransitsInNeghbrhood = 1;
  }

  VertexID_t neighbor = invalidVertex;
  VertexID_t neighbrID = threadId % App().stepSize(step) ;//(threadId % numTransits) % numTransitsInPrevStep;
  VertexID_t transitID = (threadId % numTransits) / App().stepSize(step);

  if (*transits == invalidVertex) {
    return;
  }
  
  EdgePos_t numTransitEdges = 0;

  for (int i = 0; i < numTransitsInNeghbrhood; i++) {
    assert(graph.device_csr->has_vertex(transits[i]));
    numTransitEdges += graph.device_csr->get_n_edges_for_vertex(transits[i]);
  }
  
  if (numTransitEdges != 0) {
    const CSR::Edge* transitEdges = (App().samplingType() == CollectiveNeighborhood) ? nullptr : graph.device_csr->get_edges(*transits);
    const float* transitEdgeWeights = (App().samplingType() == CollectiveNeighborhood) ? nullptr : graph.device_csr->get_weights(*transits);
    const float maxWeight = (App().samplingType() == CollectiveNeighborhood) ? 0.0 : graph.device_csr->get_max_weight(*transits);

    neighbor = App().next(step, graph.device_csr, transits, sampleIdx, &samples[sampleIdx], maxWeight, transitEdges, transitEdgeWeights, 
                    numTransitEdges, neighbrID, randState);
#if 0
    //search if neighbor has already been selected.
    //we can do that in register if required
    newNeigbhors[threadIdx.x] = neighbor;

    bool found = false;
    for (int i = 0; i < N_THREADS; i++) {
      if (newNeigbhors[i] == neighbor) {
        found = true;
        // break;
      }
    }

    __syncwarp();
    if (found) {
      neighbor = next(step, transit, sample, transitEdges, numTransitEdges, 
        transitNeighborIdx, randState);;
    }
#endif
  }
  
  EdgePos_t insertionPos = 0; 

  size_t finalSampleSizeTillPreviousStep = 0;
  size_t neighborsToSampleAtStep = 1;
  for (int _s = 0; _s < step; _s++) {
    neighborsToSampleAtStep *= App().stepSize(_s);
    finalSampleSizeTillPreviousStep += neighborsToSampleAtStep;
  }

  

  // if (insertionPos < finalSampleSize) {
  //   printf("insertionPos %d finalSampleSize %d\n", insertionPos, finalSampleSize);
  // }
  // assert(finalSampleSize > 0);
  // if (insertionPos >= finalSampleSize) {
  //   printf("insertionPos %d finalSampleSize %ld sample %d\n", insertionPos, finalSampleSize, sample);
  // }
  // assert(insertionPos < finalSampleSize);
  
  if (App().outputFormat() == AdjacencyMatrix && App().samplingType() == CollectiveNeighborhood) {
    finalSamples[sampleIdx*finalSampleSize + stepSizeAtStep<App>(step - 1) + neighbrID] = neighbor;
  } else if (App().outputFormat() == SampledVertices && App().samplingType() == IndividualNeighborhood) {
    if (numberOfTransits<App>(step) > 1) {    
      insertionPos = finalSampleSizeTillPreviousStep + (threadId % numTransits);
    } else {
      insertionPos = step;
    }
    finalSamples[sampleIdx*finalSampleSize + insertionPos] = neighbor;
  }
  // if (sample == 100) {
  //   printf("neighbor for 100 %d insertionPos %ld transit %d\n", neighbor, (long)insertionPos, transit);
  // }
  //TODO: We do not need atomic instead store indices of transit in another array,
  //wich can be accessed based on sample and transitIdx.
}

template<int TB_THREADS>
__global__ void partitionTransitsInKernels(int step, EdgePos_t* uniqueTransits, EdgePos_t* uniqueTransitCounts, 
                                           EdgePos_t* transitPositions,
                                           EdgePos_t uniqueTransitCountsNum, VertexID_t invalidVertex,
                                           EdgePos_t* gridKernelTransits, EdgePos_t* gridKernelTransitsNum,
                                           EdgePos_t* threadBlockKernelTransits, EdgePos_t* threadBlockKernelTransitsNum,
                                           EdgePos_t* subWarpKernelTransits, EdgePos_t* subWarpKernelTransitsNum,
                                           EdgePos_t* identityKernelTransits, EdgePos_t* identityKernelTransitsNum,
                                           int* kernelTypeForTransit, VertexID_t* transitToSamplesKeys) 
{
  //__shared__ EdgePos_t insertionPosOfThread[TB_THREADS];
  const int SHMEM_SIZE = 7*TB_THREADS;
  // __shared__ EdgePos_t trThreadBlocks[TB_THREADS];
  // __shared__ EdgePos_t trStartPos[TB_THREADS];
  typedef hipcub::BlockScan<int, TB_THREADS> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  __shared__ EdgePos_t shGridKernelTransits[SHMEM_SIZE];
  //__shared__ EdgePos_t warpsLastThreadVals;
  __shared__ EdgePos_t threadToTransitPrefixSum[TB_THREADS];
  __shared__ EdgePos_t threadToTransitPos[TB_THREADS];
  __shared__ VertexID_t threadToTransit[TB_THREADS];
  __shared__ EdgePos_t totalThreadGroups;
  __shared__ EdgePos_t threadGroupsInsertionPos;
//  __shared__ EdgePos_t gridKernelTransitsIter;

  int threadId = blockIdx.x * blockDim.x + threadIdx.x;

  if (threadIdx.x == 0) {
    totalThreadGroups = 0;
  //  gridKernelTransitsIter = 0;
  }

  for (int i = threadIdx.x; i < SHMEM_SIZE; i+= blockDim.x) {
    shGridKernelTransits[i] = 0;
  }

  __syncthreads();
  
  VertexID_t transit = uniqueTransits[threadId];
  EdgePos_t trCount = (threadId >= uniqueTransitCountsNum || transit == invalidVertex) ? -1: uniqueTransitCounts[threadId];
  EdgePos_t trPos = (threadId >= uniqueTransitCountsNum || transit == invalidVertex) ? -1: transitPositions[threadId];

  int kernelType = -1;
  EdgePos_t numThreadGroups = 0;
  if (trCount >= LoadBalancing::LoadBalancingThreshold::GridLevel) {    
    kernelType = TransitKernelTypes::GridKernel;
  } else if (trCount > LoadBalancing::LoadBalancingThreshold::BlockLevel) {
    kernelType = TransitKernelTypes::ThreadBlockKernel;
    // numThreadGroups = 0;
    // threadToTransitPos[threadIdx.x] = 0;
  } else if (trCount >= LoadBalancing::LoadBalancingThreshold::SubWarpLevel) {
    kernelType = TransitKernelTypes::SubWarpKernel;
    
    // numThreadGroups = 0;
    // threadToTransitPos[threadIdx.x] = 0;
  } // else {
  //   kernelType = TransitKernelTypes::IdentityKernel;
  //   // numThreadGroups = 0;
  //   // threadToTransitPos[threadIdx.x] = 0;
  // }
  
  if (threadId < uniqueTransitCountsNum && transit != invalidVertex) {
    kernelTypeForTransit[transit] = kernelType;
  }

  __syncthreads();

  for (int kTy = 1; kTy < TransitKernelTypes::SubWarpKernel + 1; kTy++) {
    EdgePos_t* glKernelTransitsNum, *glKernelTransits;
    const int threadGroupSize = (kTy == TransitKernelTypes::GridKernel) ? LoadBalancing::LoadBalancingThreshold::GridLevel : 
                                (kTy == TransitKernelTypes::ThreadBlockKernel ? LoadBalancing::LoadBalancingThreshold::BlockLevel : 
                                (kTy == TransitKernelTypes::SubWarpKernel ? LoadBalancing::LoadBalancingThreshold::SubWarpLevel : -1));

    if (kTy == TransitKernelTypes::GridKernel && useGridKernel) {
      if (kernelType == TransitKernelTypes::GridKernel) {
        numThreadGroups = DIVUP(trCount, LoadBalancing::LoadBalancingThreshold::GridLevel);
        threadToTransitPos[threadIdx.x] = trPos;
        threadToTransit[threadIdx.x] = transit;
      } else {
        numThreadGroups = 0;
        threadToTransitPos[threadIdx.x] = 0;
        threadToTransit[threadIdx.x] = -1;
      } 
      glKernelTransitsNum = gridKernelTransitsNum;
      glKernelTransits = gridKernelTransits;
    } else if (kTy == TransitKernelTypes::ThreadBlockKernel && useThreadBlockKernel) {
      if (kernelType == TransitKernelTypes::ThreadBlockKernel) {
        numThreadGroups = DIVUP(trCount, LoadBalancing::LoadBalancingThreshold::BlockLevel);
        threadToTransitPos[threadIdx.x] = trPos;
        threadToTransit[threadIdx.x] = transit;
      } else {
        numThreadGroups = 0;
        threadToTransitPos[threadIdx.x] = 0;
        threadToTransit[threadIdx.x] = -1;
      }       
      glKernelTransitsNum = threadBlockKernelTransitsNum;
      glKernelTransits = threadBlockKernelTransits;
    } else if (kTy == TransitKernelTypes::SubWarpKernel && useSubWarpKernel) {
      if (kernelType == TransitKernelTypes::SubWarpKernel) {
        numThreadGroups = DIVUP(trCount, LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
        threadToTransitPos[threadIdx.x] = trPos;
        threadToTransit[threadIdx.x] = transit;
        //printf("blockIdx.x %d threadIdx.x %d transit %d trCount %d numThreadgroups %d\n", threadIdx.x, transit, trCount, numThreadGroups);
      } else {
        numThreadGroups = 0;
        threadToTransitPos[threadIdx.x] = 0;
        threadToTransit[threadIdx.x] = -1;
      }       
      glKernelTransitsNum = subWarpKernelTransitsNum;
      glKernelTransits = subWarpKernelTransits;
    } else {
      continue;
    }

    //Get all grid kernel transits
    EdgePos_t prefixSumThreadData = 0;
    BlockScan(temp_storage).ExclusiveSum(numThreadGroups, prefixSumThreadData);
    
    __syncthreads();

    if (threadIdx.x == blockDim.x - 1) {
      totalThreadGroups = prefixSumThreadData + numThreadGroups;
      threadGroupsInsertionPos = ::atomicAdd(glKernelTransitsNum, totalThreadGroups);
    }

    threadToTransitPrefixSum[threadIdx.x] = prefixSumThreadData;
    
    __syncthreads();
    
    // if (totalThreadGroups != 0 and numThreadGroups != 0) {
    //   printf("threadIdx.x %d blockIdx.x %d tr %d trPos %d numThreadGroups %d totalThreadGroups %d prefixSumThreadData %d\n", threadIdx.x, blockIdx.x, transit, trPos, numThreadGroups, totalThreadGroups, prefixSumThreadData);
    // }
    
    for (int tgIter = 0; tgIter < totalThreadGroups; tgIter += SHMEM_SIZE) {
      for (int i = threadIdx.x; i < SHMEM_SIZE; i+= blockDim.x) {
        shGridKernelTransits[i] = 0;
      }
    
      __syncthreads();
      
      int prefixSumIndex = prefixSumThreadData - tgIter;
      if (prefixSumIndex < 0 && prefixSumIndex + numThreadGroups > 0) {
        prefixSumIndex = 0;
      }
      if (numThreadGroups > 0) {
        if (prefixSumIndex >= 0 && prefixSumIndex < SHMEM_SIZE)
          shGridKernelTransits[prefixSumIndex] = threadIdx.x;
      }
      
      __syncthreads();

      for (int tbs = threadIdx.x; tbs < DIVUP(min(SHMEM_SIZE, totalThreadGroups - tgIter), TB_THREADS)*TB_THREADS; tbs += blockDim.x) {
        int d;
        if (tbs < TB_THREADS) {
          d = (tbs < totalThreadGroups) ? shGridKernelTransits[tbs] : 0;
        } else if (threadIdx.x == 0) {
          d = (tbs < totalThreadGroups) ? max(shGridKernelTransits[tbs], shGridKernelTransits[tbs-1]): 0;
        } else {
          d = (tbs < totalThreadGroups) ? shGridKernelTransits[tbs] : 0;
        }
        
        __syncthreads();
        BlockScan(temp_storage).InclusiveScan(d, d, hipcub::Max());
        __syncthreads();

        if (tbs < totalThreadGroups)
          shGridKernelTransits[tbs] = d;
          
        __syncthreads();

        
        int previousTrPrefixSum = (tbs < totalThreadGroups && shGridKernelTransits[tbs] >= 0) ? threadToTransitPrefixSum[shGridKernelTransits[tbs]] : 0;

        if (tbs + tgIter < totalThreadGroups) {
          // if (step == 1) {
          //   printf("blockIdx.x %d shGridKernelTransits[tbs] %d tbs %d\n", blockIdx.x, shGridKernelTransits[tbs], tbs);
          // }
          EdgePos_t startPos = threadToTransitPos[shGridKernelTransits[tbs]];
          EdgePos_t pos = startPos + threadGroupSize*(tbs  + tgIter - previousTrPrefixSum);
          VertexID_t transit = threadToTransit[shGridKernelTransits[tbs]];
          glKernelTransits[threadGroupsInsertionPos + tbs + tgIter] = pos;
          assert(kernelTypeForTransit[transit] == kTy);
          // if (transitToSamplesKeys[pos] != transit) {
          //   printf("blockIdx.x %d shGridKernelTransits[tbs] %d tbs %d tgIter %d startPos %d pos %d expectedTr %d threadTr %d\n", blockIdx.x, shGridKernelTransits[tbs], tbs, tgIter, startPos, pos, transitToSamplesKeys[pos], transit);
          // }
          assert(transitToSamplesKeys[pos] == transit);
        }
      }

      __syncthreads();
    }

    // if (threadIdx.x==0){
    //   for (int i = 0; i < totalThreadGroups; i++) {
    //    // printf("blockIdx.x %d gridKernelTransits[%d] %d step %d\n", blockIdx.x, i, gridKernelTransits[threadGroupsInsertionPos + i], step);
    //   }
    // }

    __syncthreads();
  }

  // if (threadIdx.x+blockIdx.x*blockDim.x==0) {
  //   printf("subWarpKernelTransitsNum %d\n", *subWarpKernelTransitsNum);
  // }
  #if 0
  int done = 0;
  int startCopyingIteration = prefixSumThreadData/SHMEM_SIZE;
  int endCopyingIteration = (prefixSumThreadData + numThreadGroups)/SHMEM_SIZE;

  __syncthreads();

  for (int tbs = 0; tbs < gridTotalTBs; tbs += SHMEM_SIZE) {
    if (trPos >= 0 && numThreadBlocks > 0 && done < numThreadBlocks && tbs/SHMEM_SIZE >= startCopyingIteration && tbs/SHMEM_SIZE <= endCopyingIteration) {
      int todo;
      for (todo = 0; todo < min(numThreadBlocks-done, SHMEM_SIZE); todo++) {
        int idx = prefixSumThreadData + done - tbs + todo;
        if (idx >= SHMEM_SIZE) {
          break;
        }
        if (idx < 0 || idx >= SHMEM_SIZE) {
          printf("idx %d prefixSum %d done %d tbs %d todo %d\n", idx, prefixSumThreadData, done, tbs, todo);
        }
        shGridKernelTransits[idx] = trPos + LoadBalancing::LoadBalancingThreshold::GridLevel*(todo+done);
      }
      done += todo;
    }

    __syncthreads();

    for (EdgePos_t i = threadIdx.x; i < min(SHMEM_SIZE, gridTotalTBs - tbs); i+=blockDim.x) {
      gridKernelTransits[gridInsertionPos + tbs + i] = shGridKernelTransits[i];
    }
    __syncthreads();
  }
  #endif

  // if (threadIdx.x == 0) {
  //   for (EdgePos_t i = 0; i < gridTotalTBs; i+=1) {
  //     printf("%d %d, %d\n", blockIdx.x, i, gridKernelTransits[gridInsertionPos + i]);
  //   }
  // }
}

__global__ void init_curand_states(hiprandState* states, size_t num_states)
{
  int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
  if (thread_id < num_states)
    hiprand_init(thread_id, 0, 0, &states[thread_id]);
}

CSR* loadGraph(Graph& graph, char* graph_file, char* graph_type, char* graph_format)
{
  CSR* csr;

   //Load Graph
   if (strcmp(graph_type, "adj-list") == 0) {
    if (strcmp(graph_format, "text") == 0) {
      graph.load_from_adjacency_list(graph_file);
      //Convert graph to CSR format
      csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
      csr_from_graph (csr, graph);
      return csr;
    }
    else {
      printf ("graph_format '%s' not supported for graph_type '%s'\n", 
              graph_format, graph_type);
      return nullptr;
    }
  } else if (strcmp(graph_type, "edge-list") == 0) {
    if (strcmp(graph_format, "binary") == 0) {
      graph.load_from_edge_list_binary(graph_file, true);
      csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
      csr_from_graph (csr, graph);
      return csr;
    } else if (strcmp(graph_format, "text") == 0) {
      FILE* fp = fopen (graph_file, "r");
      if (fp == nullptr) {
        std::cout << "File '" << graph_file << "' not found" << std::endl;
        return nullptr;
      }
      graph.load_from_edge_list_txt(fp, true);
      fclose (fp);
      csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
      csr_from_graph (csr, graph);
      return csr;
    } else {
      printf ("graph_format '%s' not supported for graph_type '%s'\n", 
              graph_format, graph_type);
      return nullptr;
    }
  } else {
    printf("Incorrect graph file type '%s'\n", graph_type);
    return nullptr;
  }

  return nullptr;
}

GPUCSRPartition transferCSRToGPU(CSR* csr)
{
  //Assume that whole graph can be stored in GPU Memory.
  //Hence, only one Graph Partition is created.
  CSRPartition full_partition = CSRPartition (0, csr->get_n_vertices() - 1, 0, csr->get_n_edges() - 1, 
                                              csr->get_vertices(), csr->get_edges(), csr->get_weights());
  
  //Copy full graph to GPU
  GPUCSRPartition gpuCSRPartition;
  CSRPartition deviceCSRPartition = copyPartitionToGPU(full_partition, gpuCSRPartition);
  gpuCSRPartition.device_csr = (CSRPartition*)csrPartitionBuff;
  CHK_CU(hipMemcpyToSymbol(HIP_SYMBOL(csrPartitionBuff), &deviceCSRPartition, sizeof(CSRPartition)));
  return gpuCSRPartition;
}

template<typename App>
int getFinalSampleSize()
{
  size_t finalSampleSize = 0;
  size_t neighborsToSampleAtStep = 1;
  for (int step = 0; step < App().steps(); step++) {
    if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
      neighborsToSampleAtStep = App().stepSize(step);
    } else {
      neighborsToSampleAtStep *= App().stepSize(step);
    }

    finalSampleSize += neighborsToSampleAtStep;
  }

  return finalSampleSize;
}

template<typename SampleType, typename App>
bool allocNextDoorDataOnGPU(CSR* csr, NextDoorData<SampleType>& data)
{
  //Initially each sample contains only one vertex
  //Allocate one sample for each vertex
  int maxV = 0;
  for (int sampleIdx = 0; sampleIdx < App().numSamples(csr); sampleIdx++) {
    SampleType sample = App().template initializeSample<SampleType>(csr, sampleIdx);
    data.samples.push_back(sample);
    auto initialVertices = App().initialSample(sampleIdx, csr, data.samples[data.samples.size() - 1]);
    if ((EdgePos_t)initialVertices.size() != App().initialSampleSize(csr)) {
      //We require that number of vertices in sample initially are equal to the initialSampleSize
      printf ("initialSampleSize '%d' != initialSample(%d).size() '%ld'\n", 
              App().initialSampleSize(csr), sampleIdx, initialVertices.size());
      abort();
    }

    data.initialContents.insert(data.initialContents.end(), initialVertices.begin(), initialVertices.end());
    for (auto v : initialVertices)
      data.initialTransitToSampleValues.push_back(sampleIdx);
  }

  for (auto vertex : csr->iterate_vertices()) {
    maxV = (maxV < vertex) ? vertex : maxV;
  }
  //Size of each sample output
  size_t maxNeighborsToSample = App().initialSampleSize(csr); //TODO: Set initial vertices
  for (int step = 0; step < App().steps() - 1; step++) {
    if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
      maxNeighborsToSample = max((long)App().stepSize(step), maxNeighborsToSample);
    } else {
      maxNeighborsToSample *= App().stepSize(step);
    }
  }

  int finalSampleSize = getFinalSampleSize<App>();
  std::cout << "Final Size of each sample: " << finalSampleSize << std::endl;
  std::cout << "Maximum Neighbors Sampled at each step: " << maxNeighborsToSample << std::endl;
  std::cout << "Number of Samples: " << App().numSamples(csr) << std::endl;
  data.INVALID_VERTEX = csr->get_n_vertices();
  int maxBits = 0;
  while ((data.INVALID_VERTEX >> maxBits) != 0) {
    maxBits++;
  }
  
  data.maxBits = maxBits;
  
  // size_t free = 0, total = 0;
  // CHK_CU(hipMemGetInfo(&free, &total));
  // printf("free memory %ld nextDoorData.samples.size() %ld maxNeighborsToSample %ld\n", free, data.samples.size(), maxNeighborsToSample);
  const size_t numSamples = data.samples.size();

  //Allocate storage and copy initial samples on GPU
  CHK_CU(hipMalloc(&data.dInitialSamples, sizeof(VertexID_t)*data.initialContents.size()));
  CHK_CU(hipMemcpy(data.dInitialSamples, &data.initialContents[0], 
                    sizeof(VertexID_t)*data.initialContents.size(), hipMemcpyHostToDevice));

  //Allocate storage for samples on GPU
  if (sizeof(SampleType) > 0) {
    CHK_CU(hipMalloc(&data.dOutputSamples, sizeof(SampleType)*numSamples));
    CHK_CU(hipMemcpy(data.dOutputSamples, &data.samples[0], sizeof(SampleType)*numSamples, 
                      hipMemcpyHostToDevice));
  }

  //Allocate storage for final samples on GPU
  data.hFinalSamples = std::vector<VertexID_t>(finalSampleSize*numSamples);

  //TODO: Do not need this when output is adjacency matrix
  CHK_CU(hipMalloc(&data.dFinalSamples, sizeof(VertexID_t)*data.hFinalSamples.size()));
  gpu_memset(data.dFinalSamples, data.INVALID_VERTEX, data.hFinalSamples.size());
  //Samples to Transit Map

  CHK_CU(hipMalloc(&data.dSamplesToTransitMapKeys, sizeof(VertexID_t)*numSamples*maxNeighborsToSample));

  CHK_CU(hipMalloc(&data.dSamplesToTransitMapValues, sizeof(VertexID_t)*numSamples*maxNeighborsToSample));

  //Transit to Samples Map
  CHK_CU(hipMalloc(&data.dTransitToSampleMapKeys, sizeof(VertexID_t)*numSamples*maxNeighborsToSample));
  CHK_CU(hipMalloc(&data.dTransitToSampleMapValues, sizeof(VertexID_t)*numSamples*maxNeighborsToSample));

  //Same as initial values of samples for first iteration
  CHK_CU(hipMemcpy(data.dTransitToSampleMapKeys, &data.samples[0], sizeof(VertexID_t)*numSamples, 
                    hipMemcpyHostToDevice));
  CHK_CU(hipMemcpy(data.dTransitToSampleMapValues, &data.samples[0], sizeof(VertexID_t)*numSamples, 
                    hipMemcpyHostToDevice));
  //Insertion positions per transit vertex for each sample
  CHK_CU(hipMalloc(&data.dSampleInsertionPositions, sizeof(EdgePos_t)*numSamples));
  size_t curandDataSize = maxNeighborsToSample*numSamples*sizeof(hiprandState);
  const size_t curandSizeLimit = 5L*1024L*1024L*8*sizeof(hiprandState);
  if (curandDataSize < curandSizeLimit) {
    data.maxThreadsPerKernel = maxNeighborsToSample*numSamples;
  } else {
    data.maxThreadsPerKernel = curandSizeLimit/sizeof(hiprandState);
    curandDataSize = curandSizeLimit;
  }
  printf("Maximum Threads Per Kernel: %ld\n", data.maxThreadsPerKernel);
  CHK_CU(hipMalloc(&data.dCurandStates, curandDataSize));
  init_curand_states<<<thread_block_size(data.maxThreadsPerKernel, 256UL), 256UL>>> (data.dCurandStates, data.maxThreadsPerKernel);
  CHK_CU(hipDeviceSynchronize());

  if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
    CHK_CU(hipMalloc(&data.dNeighborhoodSizes, sizeof(EdgePos_t)*numSamples));
  }

  return true;
}

template<class SampleType>
void freeDeviceData(NextDoorData<SampleType>& data) 
{
  CHK_CU(hipFree(data.dSamplesToTransitMapKeys));
  CHK_CU(hipFree(data.dSamplesToTransitMapValues));
  CHK_CU(hipFree(data.dTransitToSampleMapKeys));
  CHK_CU(hipFree(data.dTransitToSampleMapValues));
  CHK_CU(hipFree(data.dSampleInsertionPositions));
  CHK_CU(hipFree(data.dCurandStates));
  CHK_CU(hipFree(data.dFinalSamples));
  CHK_CU(hipFree(data.gpuCSRPartition.device_vertex_array));
  CHK_CU(hipFree(data.gpuCSRPartition.device_edge_array));
  CHK_CU(hipFree(data.gpuCSRPartition.device_weights_array));
}

void printKernelTypes(CSR* csr, VertexID_t* dUniqueTransits, VertexID_t* dUniqueTransitsCounts, EdgePos_t* dUniqueTransitsNumRuns)
{
  EdgePos_t* hUniqueTransitsNumRuns = GPUUtils::copyDeviceMemToHostMem(dUniqueTransitsNumRuns, 1);
  VertexID_t* hUniqueTransits = GPUUtils::copyDeviceMemToHostMem(dUniqueTransits, *hUniqueTransitsNumRuns);
  VertexID_t* hUniqueTransitsCounts = GPUUtils::copyDeviceMemToHostMem(dUniqueTransitsCounts, *hUniqueTransitsNumRuns);

  size_t identityKernelTransits = 0, identityKernelSamples = 0, maxEdgesOfIdentityTransits = 0;
  size_t subWarpLevelTransits = 0, subWarpLevelSamples = 0, maxEdgesOfSubWarpTransits = 0, subWarpTransitsWithEdgesLessThan384 = 0, subWarpTransitsWithEdgesMoreThan384 = 0, numSubWarps = 0;
  size_t threadBlockLevelTransits = 0, threadBlockLevelSamples = 0, tbVerticesWithEdgesLessThan3K = 0, tbVerticesWithEdgesMoreThan3K = 0;
  size_t gridLevelTransits = 0, gridLevelSamples = 0, gridVerticesWithEdgesLessThan10K = 0, gridVerticesWithEdgesMoreThan10K = 0;
  EdgePos_t maxEdgesOfGridTransits = 0;

  for (EdgePos_t tr = 0; tr < *hUniqueTransitsNumRuns; tr++) {
    // if (tr == 0) {printf("%s:%d hUniqueTransitsCounts[0] is %d\n", __FILE__, __LINE__, hUniqueTransitsCounts[tr]);}
    if (hUniqueTransitsCounts[tr] < 8) {
      identityKernelTransits++;
      identityKernelSamples += hUniqueTransitsCounts[tr];
      maxEdgesOfIdentityTransits = max(maxEdgesOfIdentityTransits, (size_t)csr->n_edges_for_vertex(tr));
    } else if (hUniqueTransitsCounts[tr] <= LoadBalancing::LoadBalancingThreshold::BlockLevel && 
               hUniqueTransitsCounts[tr] >= 8) {
      subWarpLevelTransits++;
      subWarpLevelSamples += hUniqueTransitsCounts[tr];
      maxEdgesOfSubWarpTransits = max(maxEdgesOfSubWarpTransits, (size_t)csr->n_edges_for_vertex(tr));
      numSubWarps += DIVUP(hUniqueTransitsCounts[tr], LoadBalancing::LoadBalancingThreshold::SubWarpLevel);
      if (csr->n_edges_for_vertex(tr) <= 96) {
        subWarpTransitsWithEdgesLessThan384 += 1;
      } else {
        subWarpTransitsWithEdgesMoreThan384 += 1;
      }
    } else if (hUniqueTransitsCounts[tr] > LoadBalancing::LoadBalancingThreshold::BlockLevel && 
               hUniqueTransitsCounts[tr] <= LoadBalancing::LoadBalancingThreshold::GridLevel) {
      threadBlockLevelTransits++;
      threadBlockLevelSamples += hUniqueTransitsCounts[tr];
      if (csr->n_edges_for_vertex(tr) <= 3*1024) {
        tbVerticesWithEdgesLessThan3K += 1;
      } else {
        tbVerticesWithEdgesMoreThan3K += 1;
      }
    } else {
      gridLevelTransits++;
      gridLevelSamples += hUniqueTransitsCounts[tr];
      if (csr->n_edges_for_vertex(tr) <= 3*1024) {
        gridVerticesWithEdgesLessThan10K += 1;
      } else {
        gridVerticesWithEdgesMoreThan10K += 1;
      }
      maxEdgesOfGridTransits = max(maxEdgesOfGridTransits, csr->n_edges_for_vertex(tr));
    }
  }

  printf("IdentityKernelTransits: %ld, IdentityKernelSamples: %ld, MaxEdgesOfIdentityTransits: %ld\n" 
         "SubWarpLevelTransits: %ld, SubWarpLevelSamples: %ld, MaxEdgesOfSubWarpTranits: %ld, VerticesWithEdges > 384: %ld, VerticesWithEdges <= 384: %ld, NumSubWarps: %ld\n"
         "ThreadBlockLevelTransits: %ld, ThreadBlockLevelSamples: %ld, VerticesWithEdges > 3K: %ld, VerticesWithEdges < 3K: %ld\n"
         "GridLevelTransits: %ld, GridLevelSamples: %ld, VerticesWithEdges > 10K: %ld, VerticesWithEdges < 10K: %ld, MaxEdgesOfTransit: %d\n", 
         identityKernelTransits, identityKernelSamples, maxEdgesOfIdentityTransits, 
         subWarpLevelTransits, subWarpLevelSamples, maxEdgesOfSubWarpTransits, 
            subWarpTransitsWithEdgesMoreThan384, subWarpTransitsWithEdgesLessThan384, numSubWarps, 
         threadBlockLevelTransits, threadBlockLevelSamples, tbVerticesWithEdgesMoreThan3K, tbVerticesWithEdgesLessThan3K,
         gridLevelTransits, gridLevelSamples, gridVerticesWithEdgesMoreThan10K, gridVerticesWithEdgesLessThan10K, maxEdgesOfGridTransits);

  delete hUniqueTransits;
  delete hUniqueTransitsCounts;
  delete hUniqueTransitsNumRuns;
}

template<class SampleType, typename App>
bool doTransitParallelSampling(CSR* csr, GPUCSRPartition gpuCSRPartition, NextDoorData<SampleType>& nextDoorData, bool enableLoadBalancing)
{
  //Size of each sample output
  size_t maxNeighborsToSample = (App().samplingType() == CollectiveNeighborhood) ? 1 : App().initialSampleSize(csr);
  for (int step = 0; step < App().steps() - 1; step++) {
    if (App().samplingType() == CollectiveNeighborhood) {
      maxNeighborsToSample = max(maxNeighborsToSample, (size_t)App().stepSize(step));
    } else {
      maxNeighborsToSample *= App().stepSize(step);
    }
  }
  
  size_t finalSampleSize = getFinalSampleSize<App>();
  CHK_CU(hipMemcpy(nextDoorData.dTransitToSampleMapKeys, &nextDoorData.initialContents[0], 
                    sizeof(VertexID_t)*nextDoorData.initialContents.size(), 
                    hipMemcpyHostToDevice));
  CHK_CU(hipMemcpy(nextDoorData.dTransitToSampleMapValues, &nextDoorData.initialTransitToSampleValues[0], 
                    sizeof(VertexID_t)*nextDoorData.initialTransitToSampleValues.size(), 
                    hipMemcpyHostToDevice));
  VertexID_t* d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;

  VertexID_t* dUniqueTransits = nullptr;
  VertexID_t* dUniqueTransitsCounts = nullptr;
  EdgePos_t* dUniqueTransitsNumRuns = nullptr;
  EdgePos_t* dTransitPositions = nullptr;
  EdgePos_t* uniqueTransitNumRuns = nullptr;
   
  /**Pointers for each kernel type**/
  EdgePos_t* gridKernelTransitsNum = nullptr;
  EdgePos_t* dGridKernelTransitsNum = nullptr;
  VertexID_t* dGridKernelTransits = nullptr;
  
  EdgePos_t* threadBlockKernelTransitsNum = nullptr;
  EdgePos_t* dThreadBlockKernelTransitsNum = nullptr;
  VertexID_t* dThreadBlockKernelTransits = nullptr;

  EdgePos_t* subWarpKernelTransitsNum = nullptr;
  EdgePos_t* dSubWarpKernelTransitsNum = nullptr;
  VertexID_t* dSubWarpKernelTransits = nullptr;
  /**********************************/

  /*Single Memory Location on both CPU and GPU for transferring
   *number of transits for all kernels */
  EdgePos_t* dKernelTransitNums;
  EdgePos_t* hKernelTransitNums;
  const int NUM_KERNEL_TYPES = TransitKernelTypes::NumKernelTypes;

  int* dKernelTypeForTransit = nullptr;

  CHK_CU(hipHostMalloc(&uniqueTransitNumRuns, sizeof(EdgePos_t)));
  CHK_CU(hipHostMalloc(&hKernelTransitNums, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
  
  gridKernelTransitsNum = hKernelTransitNums;
  threadBlockKernelTransitsNum = hKernelTransitNums + 1;
  subWarpKernelTransitsNum = hKernelTransitNums + 2;
  //threadBlockKernelTransitsNum = hKernelTransitNums[3];
  
  CHK_CU(hipMalloc(&dKernelTypeForTransit, sizeof(VertexID_t)*csr->get_n_vertices()));
  CHK_CU(hipMalloc(&dTransitPositions, 
                    sizeof(VertexID_t)*nextDoorData.samples.size()));
  CHK_CU(hipMalloc(&dGridKernelTransits, 
                    sizeof(VertexID_t)*nextDoorData.samples.size()*maxNeighborsToSample));
  CHK_CU(hipMalloc(&dThreadBlockKernelTransits, 
                    sizeof(VertexID_t)*nextDoorData.samples.size()*maxNeighborsToSample));
  CHK_CU(hipMalloc(&dSubWarpKernelTransits,
                    sizeof(VertexID_t)*nextDoorData.samples.size()*maxNeighborsToSample));

  CHK_CU(hipMalloc(&dKernelTransitNums, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
  CHK_CU(hipMemset(dKernelTransitNums, 0, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
  dGridKernelTransitsNum = dKernelTransitNums;
  dThreadBlockKernelTransitsNum = dKernelTransitNums + 1;
  dSubWarpKernelTransitsNum = dKernelTransitNums + 2;

  int* atomicPtrTest = nullptr;
  CHK_CU(hipMalloc(&atomicPtrTest, sizeof(int)));
  //Check if the space runs out.
  //TODO: Use DoubleBuffer version that requires O(P) space.
  //TODO: hFinalSamples.size() is wrong.
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, 
            nextDoorData.dSamplesToTransitMapValues, nextDoorData.dTransitToSampleMapKeys, 
            nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dTransitToSampleMapValues, 
            nextDoorData.samples.size()*maxNeighborsToSample);

  CHK_CU(hipMalloc(&dUniqueTransits, (csr->get_n_vertices() + 1)*sizeof(VertexID_t)));
  CHK_CU(hipMalloc(&dUniqueTransitsCounts, (csr->get_n_vertices() + 1)*sizeof(VertexID_t)));
  CHK_CU(hipMalloc(&dUniqueTransitsNumRuns, sizeof(size_t)));
  
  if (temp_storage_bytes < nextDoorData.samples.size()*maxNeighborsToSample) {
    temp_storage_bytes = nextDoorData.samples.size()*maxNeighborsToSample;
  }

  // VertexID_t* gt1, *gt2;
  // CHK_CU(hipMalloc(&gt1, nextDoorData.samples.size()*maxNeighborsToSample*sizeof(VertexID_t)));
  // CHK_CU(hipMalloc(&gt2, nextDoorData.samples.size()*maxNeighborsToSample*sizeof(VertexID_t)));

  size_t free = 0, total = 0;
  CHK_CU(hipMemGetInfo(&free, &total));
  // printf("free memory %ld temp_storage_bytes %ld nextDoorData.samples.size() %ld maxNeighborsToSample %ld\n", free, temp_storage_bytes, nextDoorData.samples.size(), maxNeighborsToSample);
  CHK_CU(hipMalloc(&d_temp_storage, temp_storage_bytes));

  std::vector<VertexID_t*> hAllSamplesToTransitMapKeys;
  std::vector<VertexID_t*> hAllTransitToSampleMapValues;

  double loadBalancingTime = 0;
  double inversionTime = 0;
  double gridKernelTime = 0;
  double subWarpKernelTime = 0;
  double identityKernelTime = 0;
  double threadBlockKernelTime = 0;
  size_t neighborsToSampleAtStep = (App().samplingType() == CollectiveNeighborhood) ? 1 : App().initialSampleSize(csr);

  double end_to_end_t1 = convertTimeValToDouble(getTimeOfDay ());
  for (int step = 0; step < App().steps(); step++) {
    const size_t numTransits = (App().samplingType() == CollectiveNeighborhood) ? 1 : neighborsToSampleAtStep;
    neighborsToSampleAtStep = (App().samplingType() == CollectiveNeighborhood) ? App().stepSize(step) : neighborsToSampleAtStep * App().stepSize(step);    
    const size_t totalThreads = App().numSamples(csr)*neighborsToSampleAtStep;

    if (step == 0 || !enableLoadBalancing) {
      //When not doing load balancing call baseline transit parallel
      for (int threadsExecuted = 0; threadsExecuted < totalThreads; threadsExecuted += nextDoorData.maxThreadsPerKernel) {
        size_t currExecutionThreads = min((size_t)nextDoorData.maxThreadsPerKernel, totalThreads - threadsExecuted);
        samplingKernel<SampleType, App><<<thread_block_size(currExecutionThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartition, 
                        threadsExecuted, currExecutionThreads, nextDoorData.INVALID_VERTEX,
                        (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys, (const VertexID_t*)nextDoorData.dTransitToSampleMapValues,
                        totalThreads, nextDoorData.dOutputSamples, nextDoorData.samples.size(),
                        nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dSamplesToTransitMapValues,
                        nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
                        nextDoorData.dCurandStates);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
      }
    } else {
      double loadBalancingT1 = convertTimeValToDouble(getTimeOfDay ());
      void* dRunLengthEncodeTmpStorage = nullptr;
      size_t dRunLengthEncodeTmpStorageSize = 0;

      hipcub::DeviceRunLengthEncode::Encode(dRunLengthEncodeTmpStorage, dRunLengthEncodeTmpStorageSize, 
                                        nextDoorData.dTransitToSampleMapKeys,
                                        dUniqueTransits, dUniqueTransitsCounts, dUniqueTransitsNumRuns, totalThreads);

      assert(dRunLengthEncodeTmpStorageSize < temp_storage_bytes);
      dRunLengthEncodeTmpStorage = d_temp_storage;
      hipcub::DeviceRunLengthEncode::Encode(dRunLengthEncodeTmpStorage, dRunLengthEncodeTmpStorageSize, 
                                        nextDoorData.dTransitToSampleMapKeys,
                                        dUniqueTransits, dUniqueTransitsCounts, dUniqueTransitsNumRuns, totalThreads);

      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      
      CHK_CU(hipMemcpy(uniqueTransitNumRuns, dUniqueTransitsNumRuns, sizeof(*uniqueTransitNumRuns), hipMemcpyDeviceToHost));

      void* dExclusiveSumTmpStorage = nullptr;
      size_t dExclusiveSumTmpStorageSize = 0;
      
      hipcub::DeviceScan::ExclusiveSum(dExclusiveSumTmpStorage, dExclusiveSumTmpStorageSize, dUniqueTransitsCounts, dTransitPositions, *uniqueTransitNumRuns);

      assert(dExclusiveSumTmpStorageSize < temp_storage_bytes);
      dExclusiveSumTmpStorage = d_temp_storage;

      hipcub::DeviceScan::ExclusiveSum(dExclusiveSumTmpStorage, dExclusiveSumTmpStorageSize, dUniqueTransitsCounts, dTransitPositions, *uniqueTransitNumRuns);

      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      // printKernelTypes(csr, dUniqueTransits, dUniqueTransitsCounts, dUniqueTransitsNumRuns);

      CHK_CU(hipMemset(dKernelTransitNums, 0, NUM_KERNEL_TYPES * sizeof(EdgePos_t)));
      partitionTransitsInKernels<1024><<<thread_block_size((*uniqueTransitNumRuns), 1024), 1024>>>(step, dUniqueTransits, dUniqueTransitsCounts, 
          dTransitPositions, *uniqueTransitNumRuns, nextDoorData.INVALID_VERTEX, dGridKernelTransits, dGridKernelTransitsNum, 
          dThreadBlockKernelTransits, dThreadBlockKernelTransitsNum, dSubWarpKernelTransits, dSubWarpKernelTransitsNum, nullptr, nullptr, dKernelTypeForTransit,
          nextDoorData.dTransitToSampleMapKeys);

      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      CHK_CU(hipMemcpy(hKernelTransitNums, dKernelTransitNums, NUM_KERNEL_TYPES * sizeof(EdgePos_t), hipMemcpyDeviceToHost));

      // GPUUtils::printDeviceArray(dGridKernelTransits, *gridKernelTransitsNum, ',');
      // getchar();
      double loadBalancingT2 = convertTimeValToDouble(getTimeOfDay ());
      loadBalancingTime += (loadBalancingT2 - loadBalancingT1);
      
      double identityKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
      identityKernel<SampleType, App, N_THREADS, true><<<DIVUP(totalThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
        (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys, (const VertexID_t*)nextDoorData.dTransitToSampleMapValues,
        totalThreads, nextDoorData.dOutputSamples, nextDoorData.samples.size(),
        nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dSamplesToTransitMapValues,
        nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
        nextDoorData.dCurandStates, dKernelTypeForTransit);
      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      double identityKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
      identityKernelTime += (identityKernelTimeT2 - identityKernelTimeT1);
      
      const int perThreadSamplesForSubWarpKernel = 1;
      int threadBlocks = DIVUP(DIVUP(*subWarpKernelTransitsNum*LoadBalancing::LoadBalancingThreshold::SubWarpLevel, perThreadSamplesForSubWarpKernel), 256);
      //std::cout << "subWarpKernelTransitsNum " << *subWarpKernelTransitsNum << " threadBlocks " << threadBlocks << std::endl;
      double subWarpKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
      if (useSubWarpKernel) {
        subWarpKernel<SampleType, App, 256,3*1024,false,false,false,perThreadSamplesForSubWarpKernel,true><<<threadBlocks, 256>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
          (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys, (const VertexID_t*)nextDoorData.dTransitToSampleMapValues,
          totalThreads, nextDoorData.dOutputSamples, nextDoorData.samples.size(),
          nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dSamplesToTransitMapValues,
          nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
          nextDoorData.dCurandStates, dKernelTypeForTransit, dSubWarpKernelTransits, *subWarpKernelTransitsNum);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
      }
      double subWarpKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
      subWarpKernelTime += (subWarpKernelTimeT2 - subWarpKernelTimeT1);

      double threadBlockKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
      const int perThreadSamplesForThreadBlockKernel = 1;
      threadBlocks = DIVUP(*threadBlockKernelTransitsNum, perThreadSamplesForThreadBlockKernel);
      if (useThreadBlockKernel) {
        threadBlockKernel<SampleType, App, 256,3*1024-3,true,false,false,perThreadSamplesForThreadBlockKernel,true><<<threadBlocks, 32>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
          (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys, (const VertexID_t*)nextDoorData.dTransitToSampleMapValues,
          totalThreads, nextDoorData.dOutputSamples, nextDoorData.samples.size(),
          nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dSamplesToTransitMapValues,
          nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
          nextDoorData.dCurandStates, dKernelTypeForTransit, dThreadBlockKernelTransits, *threadBlockKernelTransitsNum);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
      }
      double threadBlockKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
      threadBlockKernelTime += (threadBlockKernelTimeT2 - threadBlockKernelTimeT1);

      const int perThreadSamplesForGridKernel = 4;
      double gridKernelTimeT1 = convertTimeValToDouble(getTimeOfDay ());
      threadBlocks = DIVUP(*gridKernelTransitsNum, perThreadSamplesForGridKernel);
      if (useGridKernel) {
        gridKernel<SampleType,App,256,3*1024-3,false,false,false,perThreadSamplesForGridKernel,true><<<threadBlocks, 256>>>(step, gpuCSRPartition, nextDoorData.INVALID_VERTEX,
          (const VertexID_t*)nextDoorData.dTransitToSampleMapKeys, (const VertexID_t*)nextDoorData.dTransitToSampleMapValues,
          totalThreads,  nextDoorData.dOutputSamples, nextDoorData.samples.size(),
          nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dSamplesToTransitMapValues,
          nextDoorData.dFinalSamples, finalSampleSize, nextDoorData.dSampleInsertionPositions,
          nextDoorData.dCurandStates, dKernelTypeForTransit, dGridKernelTransits, *gridKernelTransitsNum);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
      }
      double gridKernelTimeT2 = convertTimeValToDouble(getTimeOfDay ());
      gridKernelTime += (gridKernelTimeT2 - gridKernelTimeT1);
    }

    if (step != App().steps() - 1) {
      double inversionT1 = convertTimeValToDouble(getTimeOfDay ());
      //Invert sample->transit map by sorting samples based on the transit vertices
      hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, 
                                      nextDoorData.dSamplesToTransitMapValues, nextDoorData.dTransitToSampleMapKeys, 
                                      nextDoorData.dSamplesToTransitMapKeys, nextDoorData.dTransitToSampleMapValues, 
                                      totalThreads, 0, nextDoorData.maxBits);
      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
      double inversionT2 = convertTimeValToDouble(getTimeOfDay ());
      inversionTime += (inversionT2 - inversionT1);

      #if 0
      VertexID_t* hTransitToSampleMapKeys = new VertexID_t[totalThreads];
      VertexID_t* hTransitToSampleMapValues = new VertexID_t[totalThreads];
      VertexID_t* hSampleToTransitMapKeys = new VertexID_t[totalThreads];
      VertexID_t* hSampleToTransitMapValues = new VertexID_t[totalThreads];

      
      CHK_CU(hipMemcpy(hSampleToTransitMapKeys, nextDoorData.dSamplesToTransitMapKeys, 
        totalThreads*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      CHK_CU(hipMemcpy(hSampleToTransitMapValues, nextDoorData.dSamplesToTransitMapValues,
        totalThreads*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      CHK_CU(hipMemcpy(hTransitToSampleMapKeys, nextDoorData.dTransitToSampleMapKeys, 
                        totalThreads*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      CHK_CU(hipMemcpy(hTransitToSampleMapValues, nextDoorData.dTransitToSampleMapValues,
                        totalThreads*sizeof(VertexID_t), hipMemcpyDeviceToHost));
      hAllTransitToSampleMapValues.push_back(hTransitToSampleMapValues);
      hAllSamplesToTransitMapKeys.push_back(hSampleToTransitMapKeys);
      #endif
    }
  }

  double end_to_end_t2 = convertTimeValToDouble(getTimeOfDay ());

  CHK_CU(hipMemset(nextDoorData.dSampleInsertionPositions, 0, sizeof(EdgePos_t)*nextDoorData.samples.size()));

  std::cout << "Transit Parallel: End to end time " << (end_to_end_t2 - end_to_end_t1) << " secs" << std::endl;
  std::cout << "InversionTime: " << inversionTime <<", " << "LoadBalancingTime: " << loadBalancingTime << ", " << "GridKernelTime: " << gridKernelTime << ", ThreadBlockKernelTime: " << threadBlockKernelTime << ", SubWarpKernelTime: " << subWarpKernelTime << ", IdentityKernelTime: "<< identityKernelTime << std::endl;
  CHK_CU(hipFree(d_temp_storage));
  CHK_CU(hipFree(dUniqueTransits));
  CHK_CU(hipFree(dUniqueTransitsCounts));
  CHK_CU(hipFree(dUniqueTransitsNumRuns));

  #if 0
  for (int s = 1; s < App().steps() - 2; s++) {
    std::unordered_set<VertexID_t> s1, s2, intersection;
    for (int i = 100000; i < 200000; i++) {
      VertexID_t v1 = hAllSamplesToTransitMapKeys[s+1][i];
      VertexID_t v2 = hAllTransitToSampleMapValues[s+2][i];
      //printf("v1 %d v2 %d\n", v1, v2);
      s1.insert(v1);
      s2.insert(v2);
    }
    
    for (auto e : s1) {
      if (s2.count(e) == 1) intersection.insert(e);
    }

    std::cout << "s: " << s << " intersection: " << intersection.size() << std::endl;
  }
  #endif
  return true;
}

template<class SampleType, typename App>
bool doSampleParallelSampling(CSR* csr, GPUCSRPartition gpuCSRPartition, NextDoorData<SampleType>& nextDoorData)
{
  //Size of each sample output
  int finalSampleSize = getFinalSampleSize<App>();
  int neighborsToSampleAtStep = App().initialSampleSize(csr);

  EdgePos_t* hSumNeighborhoodSizes;
  EdgePos_t* dSumNeighborhoodSizes;
  EdgePos_t* dSampleNeighborhoodPos;
  VertexID_t* dCollectiveNeighborhoodCSRCols;
  EdgePos_t* dCollectiveNeighborhoodCSRRows;

  if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
    CHK_CU(hipHostMalloc(&hSumNeighborhoodSizes, sizeof(EdgePos_t)));
    CHK_CU(hipMalloc(&dSumNeighborhoodSizes, sizeof(EdgePos_t)));
    CHK_CU(hipMalloc(&dSampleNeighborhoodPos, sizeof(EdgePos_t)*nextDoorData.samples.size()));
  }

  double end_to_end_t1 = convertTimeValToDouble(getTimeOfDay ());
  for (int step = 0; step < App().steps(); step++) {
    //Number of threads created are equal to number of new neighbors to be sampled at a step.
    //In collective neighborhood we sample stepSize(step) vertices at each step
    //Otherwise need to sample product.
    const size_t numTransits = (App().samplingType() == CollectiveNeighborhood) ? 1 : neighborsToSampleAtStep;
    neighborsToSampleAtStep = (App().samplingType() == CollectiveNeighborhood) ? App().stepSize(step) : neighborsToSampleAtStep * App().stepSize(step);
    const size_t totalThreads = App().numSamples(csr)*neighborsToSampleAtStep;
    // std::cout << "totalThreads " << totalThreads << std::endl;
    for (int threadsExecuted = 0; threadsExecuted < totalThreads; threadsExecuted += nextDoorData.maxThreadsPerKernel) {
      size_t currExecutionThreads = min(nextDoorData.maxThreadsPerKernel, totalThreads - threadsExecuted);
      if (App().samplingType() == SamplingType::CollectiveNeighborhood) {
        //TODO: No need to do this right now.
        //Create collective neighborhood for all transits related to a sample
        collectiveNeighbrsSize<App><<<nextDoorData.samples.size(), N_THREADS>>>(step, gpuCSRPartition, 
                                                                           nextDoorData.INVALID_VERTEX,
                                                                           nextDoorData.dInitialSamples, 
                                                                           nextDoorData.dFinalSamples, 
                                                                           nextDoorData.samples.size(),
                                                                           dSampleNeighborhoodPos,
                                                                           dSumNeighborhoodSizes);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
        CHK_CU(hipMemcpy(hSumNeighborhoodSizes, dSumNeighborhoodSizes, sizeof(EdgePos_t), hipMemcpyDeviceToHost));
        CHK_CU(hipMalloc(&dCollectiveNeighborhoodCSRCols, sizeof(VertexID_t)*(*hSumNeighborhoodSizes)));
        CHK_CU(hipMalloc(&dCollectiveNeighborhoodCSRRows, sizeof(EdgePos_t)*App().initialSampleSize(csr)*nextDoorData.samples.size()));

        collectiveNeighborhood<App><<<nextDoorData.samples.size(), N_THREADS>>>(step, gpuCSRPartition, 
                                                                           nextDoorData.INVALID_VERTEX,
                                                                           nextDoorData.dInitialSamples,
                                                                           nextDoorData.dFinalSamples, 
                                                                           nextDoorData.samples.size(),
                                                                           dCollectiveNeighborhoodCSRRows,
                                                                           dCollectiveNeighborhoodCSRCols,
                                                                           dSampleNeighborhoodPos,
                                                                           dSumNeighborhoodSizes);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
        
    #if 0
        //Check if the CSR is correct
        EdgePos_t* csrRows = new EdgePos_t[App().initialSampleSize(csr)*nextDoorData.samples.size()];
        EdgePos_t* csrCols = new VertexID_t[(*hSumNeighborhoodSizes)];
        EdgePos_t* samplePos = new EdgePos_t[nextDoorData.samples.size()];
        
        CHK_CU(hipMemcpy(csrCols, dCollectiveNeighborhoodCSRCols, sizeof(VertexID_t)*(*hSumNeighborhoodSizes), 
                          hipMemcpyDeviceToHost));
        CHK_CU(hipMemcpy(csrRows, dCollectiveNeighborhoodCSRRows, sizeof(EdgePos_t)*App().initialSampleSize(csr)*nextDoorData.samples.size(), 
                          hipMemcpyDeviceToHost));
        CHK_CU(hipMemcpy(samplePos, dSampleNeighborhoodPos, sizeof(EdgePos_t)*nextDoorData.samples.size(), 
                          hipMemcpyDeviceToHost));
        const int SZ = App().initialSampleSize(csr)*nextDoorData.samples.size();
        for (int sample = 0; sample < nextDoorData.samples.size(); sample++) {
          for (int v = 0; v < App().initialSampleSize(csr); v++) {
            EdgePos_t edgeStart = csrRows[sample * App().initialSampleSize(csr) + v];
            EdgePos_t edgeEnd = -1;
            EdgePos_t idxInRows = sample * App().initialSampleSize(csr) + v;
            
            //TODO: Add one more field to a vertex to each sample that is the length of all edges.
            if (v + 1 == App().initialSampleSize(csr)) {
              continue;
            }
            if (v + 1 < App().initialSampleSize(csr)) {
              edgeEnd = csrRows[idxInRows + 1];
            } else if (sample + 1 < nextDoorData.samples.size()) {
              edgeEnd = samplePos[sample + 1];
            } else {
              edgeEnd = (*hSumNeighborhoodSizes);
            }
             
            VertexID transit = nextDoorData.initialContents[sample * App().initialSampleSize(csr) + v];
            if (edgeEnd - edgeStart != csr->n_edges_for_vertex(transit)) {
              printf("transit %d edgeEnd %d edgeStart %d csr->n_edges_for_vertex(transit) %d\n", transit, edgeEnd, edgeStart, csr->n_edges_for_vertex(transit));
            }
            assert(edgeEnd - edgeStart == csr->n_edges_for_vertex(transit));
          }
        }
    #endif
        /*Sorting takes a ton of time (2-3x more). So, it probably be benificial to 
         * create a CSR matrix of the neighborhood of transit vertices.*/
        //Sort these edges of neighborhood
        /****************************
        void* dTempStorage = nullptr;
        size_t dTempStorageBytes = 0;
        hipcub::DeviceSegmentedRadixSort::SortKeys(dTempStorage, dTempStorageBytes, (const VertexID_t*)dCollectiveNeighborhood, 
                                                dCollectiveNeighborhood + sizeof(VertexID_t)*(*hSumNeighborhoodSizes), 
                                                *hSumNeighborhoodSizes, (int)nextDoorData.samples.size(),
                                                dSampleNeighborhoodPos, dSampleNeighborhoodPos + 1, 0, nextDoorData.maxBits);
        
        CHK_CU(hipMalloc(&dTempStorage, dTempStorageBytes));
        hipcub::DeviceSegmentedRadixSort::SortKeys(dTempStorage, dTempStorageBytes, (const VertexID_t*)dCollectiveNeighborhood, 
                                                dCollectiveNeighborhood + sizeof(VertexID_t)*(*hSumNeighborhoodSizes), 
                                                *hSumNeighborhoodSizes, (int)nextDoorData.samples.size(),
                                                dSampleNeighborhoodPos, dSampleNeighborhoodPos + 1, 0, nextDoorData.maxBits);
        CHK_CU(hipGetLastError());
        CHK_CU(hipDeviceSynchronize());
        ****************************/
      }
      if (App().hasExplicitTransits() and step > 0) {
        const size_t totalThreads = App().numSamples(csr)*neighborsToSampleAtStep;
        for (int _thExecs = 0; _thExecs < totalThreads; _thExecs += nextDoorData.maxThreadsPerKernel) {
          const size_t currExecThreads = min(nextDoorData.maxThreadsPerKernel, totalThreads - _thExecs);

          explicitTransitsKernel<SampleType, App, false><<<DIVUP(currExecThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartition, 
                                                                                                      nextDoorData.INVALID_VERTEX,
                                                                                                      _thExecs, currExecThreads,
                                                                                                      totalThreads,
                                                                                                      nextDoorData.dOutputSamples,
                                                                                                      nextDoorData.samples.size(),
                                                                                                      nullptr,
                                                                                                      nextDoorData.dSamplesToTransitMapValues,
                                                                                                      nextDoorData.dCurandStates);
          
          CHK_CU(hipGetLastError());
          CHK_CU(hipDeviceSynchronize());
        }
      }
//       template<class SampleType>
// __global__ void sampleParallelKernel(const int step, GPUCSRPartition graph, 
//                                      const VertexID_t invalidVertex,
//                                      const size_t threadsExecuted, 
//                                      const size_t currExecutionThreads,
//                                      const size_t totalThreads,
//                                      VertexID_t* initialSamples,
//                                      SampleType* samples,
//                                      const size_t NumSamples,
//                                      VertexID_t* finalSamplesCSRCol, 
//                                      EdgePos_t* finalSamplesCSRRow,
//                                      float* finalSamplesCSRVal,
//                                      VertexID_t* finalSamples,
//                                      const size_t finalSampleSize, 
//                                      VertexID_t* explicitTransits,
//                                      EdgePos_t* sampleInsertionPositions,
//                                      hiprandState* randStates)
      //Perform SampleParallel Sampling
      sampleParallelKernel<SampleType, App><<<thread_block_size(currExecutionThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartition, 
                    nextDoorData.INVALID_VERTEX,
                    threadsExecuted, currExecutionThreads, totalThreads, 
                    nextDoorData.dInitialSamples, nextDoorData.dOutputSamples, nextDoorData.samples.size(),
                    nextDoorData.dFinalSamplesCSRCol, nextDoorData.dFinalSamplesCSRRow, nextDoorData.dFinalSamplesCSRVal, 
                    nextDoorData.dFinalSamples, finalSampleSize, 
                    nextDoorData.dSamplesToTransitMapValues,
                    nextDoorData.dSampleInsertionPositions, nextDoorData.dCurandStates);
      CHK_CU(hipGetLastError());
      CHK_CU(hipDeviceSynchronize());
    }
  }

  double end_to_end_t2 = convertTimeValToDouble(getTimeOfDay ());
  

  CHK_CU(hipMemset(nextDoorData.dSampleInsertionPositions, 0, sizeof(EdgePos_t)*nextDoorData.samples.size()));

  std::cout << "SampleParallel: End to end time " << (end_to_end_t2 - end_to_end_t1) << " secs" << std::endl;
  return true;
}

template<class SampleType>
std::vector<VertexID_t>& getFinalSamples(NextDoorData<SampleType>& nextDoorData)
{
  CHK_CU(hipMemcpy(&nextDoorData.hFinalSamples[0], nextDoorData.dFinalSamples, 
                    nextDoorData.hFinalSamples.size()*sizeof(nextDoorData.hFinalSamples[0]), hipMemcpyDeviceToHost));
  CHK_CU(hipMemcpy(&nextDoorData.samples[0], nextDoorData.dOutputSamples, 
                    nextDoorData.samples.size()*sizeof(SampleType), hipMemcpyDeviceToHost));
  return nextDoorData.hFinalSamples;
}

template<class SampleType, typename App>
bool nextdoor(const char* graph_file, const char* graph_type, const char* graph_format, 
             const int nruns, const bool chk_results, const bool print_samples,
             const char* kernelType, const bool enableLoadBalancing,
             bool (*checkResultsFunc)(NextDoorData<SampleType>&))
{
  std::vector<Vertex> vertices;

  //Load Graph
  Graph graph;
  CSR* csr;
  if ((csr = loadGraph(graph, (char*)graph_file, (char*)graph_type, (char*)graph_format)) == nullptr) {
    return false;
  }

  std::cout << "Graph has " <<graph.get_n_edges () << " edges and " << 
      graph.get_vertices ().size () << " vertices " << std::endl; 

  //graph.print(std::cout);
  GPUCSRPartition gpuCSRPartition = transferCSRToGPU(csr);
  
  NextDoorData<SampleType> nextDoorData;
  nextDoorData.csr = csr;
  nextDoorData.gpuCSRPartition = gpuCSRPartition;
  allocNextDoorDataOnGPU<SampleType, App>(csr, nextDoorData);
  
  for (int i = 0; i < nruns; i++) {
    if (strcmp(kernelType, "TransitParallel") == 0)
      doTransitParallelSampling<SampleType, App>(csr, gpuCSRPartition, nextDoorData, enableLoadBalancing);
    else if (strcmp(kernelType, "SampleParallel") == 0)
      doSampleParallelSampling<SampleType, App>(csr, gpuCSRPartition, nextDoorData);
    else
      abort();
  }
    

  getFinalSamples(nextDoorData);

  size_t maxNeighborsToSample = 1;
  for (int step = 0; step < App().steps(); step++) {
    maxNeighborsToSample *= App().stepSize(step);
  }

  size_t finalSampleSize = getFinalSampleSize<App>();
  
  size_t totalSampledVertices = 0;

  for (auto s : nextDoorData.hFinalSamples) {
    totalSampledVertices += (int)(s != nextDoorData.INVALID_VERTEX);
  }

  if (print_samples) {
    for (size_t s = 0; s < nextDoorData.hFinalSamples.size(); s += finalSampleSize) {
      std::cout << "Contents of sample " << s/finalSampleSize << " [";
      for(size_t v = s; v < s + finalSampleSize; v++)
        std::cout << nextDoorData.hFinalSamples[v] << ", ";
      std::cout << "]" << std::endl;
    }
  }

  std::cout << "totalSampledVertices " << totalSampledVertices << std::endl;
  freeDeviceData(nextDoorData);
  if (chk_results) {
      return checkResultsFunc(nextDoorData);
  }
  
  return true;
}

#endif
#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <stdio.h>
#include <vector>
#include <bitset>
#include <unordered_set>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <algorithm>
#include <numeric>
#include <string.h>
#include <assert.h>
#include <tuple>
#include <queue>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/device/device_select.cuh>
#include <hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include <anyoption.h>
#include "sample.hpp"

typedef VertexID VertexID_t;

//TODO-List:
//[] Divide main() function in several small functions.
//[] Divide the code in several include files that can be included in the API.
//[] In GPU Kernels, do refactoring and move them to other places.
//[] Use vectors instead of dynamic arrays and new.
//[] Convert these vectors to a new array type that does not do initialization of data.
//[] Use MKL or cuSPARSE to do the matrix transpose or sorting
//[] A configuration that specifies all the parameters.
//[] Use Templates for cleaner code of Sampler

//Supported:
//citeseer.graph
// const int N = 3312;
// const int N_EDGES = 9074;
//micro.graph
//const int N = 100000;
//const int N_EDGES = 2160312;
//rmat.graph
// const int N = 1024;
// const int N_EDGES = 29381;
//ego-facebook
// const int N = 4039;
// const int N_EDGES = 88244;
//ego-twitter
//const int N = 81306;
//const int N_EDGES = 2420766;
//ego-gplus
//const int N = 107614;
//const int N_EDGES = 13652253;
//soc-pokec-relationships
//const int N = 1632803;
//const int N_EDGES = 30480021;
//soc-LiveJournal1
//const int N = 4847571;
//const int N_EDGES = 68556521;

//Not supportred:
//com-orkut.ungraph
// const int N = 3072441;
// const int N_EDGES = 117185083;

#include "csr.hpp"
#include "utils.hpp"
#include "pinned_memory_alloc.hpp"
#include "sampler.cuh"
#include "rand_num_gen.cuh"

using namespace utils;
using namespace GPUUtils;

#define CHECK_RESULT

//For mico, 512 works best
const int N_THREADS = 256;

//TODO try for larger random walks to improve results

#define WARP_HOP

const int ALL_NEIGHBORS = -1;

/**User Defined Functions**/

//GraphSage 2-hop sampling
const bool has_random = true;
__host__ __device__ int steps() {return 2;}

__host__ __device__ 
int stepSize(int k) {
  return ((k == 0) ? 25 : 10);
}

__device__ inline
VertexID next(int step, const VertexID transit, const VertexID sample, 
              const CSR::Edge* transitEdges, const EdgePos_t numEdges,
              const EdgePos_t neighbrID, 
              hiprandState* state)
{
  EdgePos_t id = RandNumGen::rand_int(state, numEdges);
  return transitEdges[0];
}

/**********************/

#include "check_results.cu"

__host__ __device__
EdgePos_t newNeighborsSize(int hop, EdgePos_t num_edges)
{
  return (stepSize(hop) == ALL_NEIGHBORS) ? num_edges : (EdgePos_t)stepSize(hop);
}

__host__ __device__
EdgePos_t stepSizeAtStep(int step)
{
  if (step == -1)
    return 0;

  EdgePos_t n = 1;
  for (int i = 0; i <= step; i++) {
    n = n * stepSize(i);
  }

  return n;
}

__global__ void samplingKernel(const int step, GPUCSRPartition graph, const VertexID_t invalidVertex,
                               const VertexID_t* transitToSamplesKeys, const VertexID_t* transitToSamplesValues,
                               const size_t transitToSamplesSize, const size_t NumSamples,
                               VertexID_t* samplesToTransitKeys, VertexID_t* samplesToTransitValues,
                               VertexID_t* finalSamples, const size_t finalSampleSize, EdgePos_t* sampleInsertionPositions,
                               hiprandState* randStates)
{
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;

  if (threadId >= transitToSamplesSize)
    return;
  
  EdgePos_t transitIdx = threadId/stepSize(step);
  EdgePos_t transitNeighborIdx = threadId % stepSize(step);
  
  VertexID_t sample = transitToSamplesValues[transitIdx];
  assert(sample < NumSamples);
  VertexID_t transit = transitToSamplesKeys[transitIdx];
  VertexID_t neighbor = invalidVertex;

  if (transit != invalidVertex) {
    assert(graph.device_csr->has_vertex(transit));

    EdgePos_t numTransitEdges = graph.device_csr->get_n_edges_for_vertex(transit);
    const CSR::Edge* transitEdges = graph.device_csr->get_edges(transit);

    if (numTransitEdges != 0) {
      hiprandState* randState = &randStates[threadId];
      neighbor = next(step, transit, sample, transitEdges, numTransitEdges, 
                      transitNeighborIdx, randState);
    }
  }

  __syncwarp();

  EdgePos_t totalSizeOfSample = stepSizeAtStep(step - 1);

  samplesToTransitKeys[transitIdx*stepSize(step) + transitNeighborIdx] = sample;
  samplesToTransitValues[transitIdx*stepSize(step) + transitNeighborIdx] = neighbor;
  
  EdgePos_t insertionPos = 0;
  if (stepSize(step) > 1)
    insertionPos = utils::atomicAdd(&sampleInsertionPositions[sample], 1);

  assert(insertionPos < finalSampleSize);
  finalSamples[sample*finalSampleSize + insertionPos] = neighbor;

  //TODO: We do not need atomic instead store indices of transit in another array,
  //wich can be accessed based on sample and transitIdx.
}

__global__ void init_curand_states(hiprandState* states, size_t num_states)
{
  int thread_id = blockIdx.x*blockDim.x + threadIdx.x;
  if (thread_id < num_states)
    hiprand_init(1234, 0, 0, &states[thread_id]);
}

bool loadGraph(Graph& graph, AnyOption* opt) 
{
  char* graph_file = opt->getValue('g');
  char* graph_type = opt->getValue('t');
  char* graph_format = opt->getValue('f');

  if (graph_file == nullptr || graph_type == nullptr || 
      graph_format == nullptr) {
    opt->printUsage();
    delete opt;
    return 0;
  }

  //Load Graph
  if (strcmp(graph_type, "adj-list") == 0) {
    if (strcmp(graph_format, "text") == 0) {
      graph.load_from_adjacency_list(graph_file);
      return true;
    }
    else {
      printf ("graph_format '%s' not supported for graph_type '%s'\n", 
              graph_format, graph_type);
      return false;
    }
  } else if (strcmp(graph_type, "edge-list") == 0) {
    if (strcmp(graph_format, "binary") == 0) {
      graph.load_from_edge_list_binary(graph_file, true);
      return true;
    } else if (strcmp(graph_format, "text") == 0) {
      FILE* fp = fopen (graph_file, "r");
      if (fp == nullptr) {
        std::cout << "File '" << graph_file << "' not found" << std::endl;
        return false;
      }
      graph.load_from_edge_list_txt(fp, true);
      fclose (fp);
      return true;
    } else {
      printf ("graph_format '%s' not supported for graph_type '%s'\n", 
              graph_format, graph_type);
      return false;
    }
  } else {
    printf("Incorrect graph file type '%s'\n", graph_type);
    return false;
  }

  return false;
}

int main(int argc, char* argv[])
{
  std::vector<Vertex> vertices;

  AnyOption *opt = new AnyOption();
  opt->addUsage("usage: ");
  opt->addUsage("");
  opt->addUsage("-h --help        Prints this help");
  opt->addUsage("-g --graph-file  File containing graph");
  opt->addUsage("-t --graph-type <type> Format of graph file: 'adj-list' or 'edge-list'");
  opt->addUsage("-f --format <format> Format of graph file: 'binary' or 'text'");
  opt->addUsage("-chk --check-results Check results using an algorithm");
  opt->addUsage("-p --print-samples Print Samples");

  opt->setFlag("help", 'h');
  opt->setOption("graph-file",  'g');
  opt->setOption("graph-type", 't');
  opt->setOption("graph-format", 'f');
  opt->setFlag("print-samples", 'p');
  opt->setFlag("check-results", 'chk');

  opt->processCommandArgs(argc, argv);

  if (!opt->hasOptions()) {
    opt->printUsage();
    delete opt;
    return 0;
  }  

  //Load Graph
  Graph graph;
  if (loadGraph(graph, opt) == false) {
    return 1;
  }

  std::cout << "Graph has " <<graph.get_n_edges () << " edges and " << 
      graph.get_vertices ().size () << " vertices " << std::endl; 

  //Convert graph to CSR format
  CSR* csr = new CSR(graph.get_vertices().size(), graph.get_n_edges());
  csr_from_graph (csr, graph);

  double total_stream_time = 0;

  double_t kernelTotalTime = 0.0;
  std::vector<CSRPartition> csr_partitions;

  //Assume that whole graph can be stored in GPU Memory.
  //Hence, only one Graph Partition is created.
  CSRPartition full_partition = CSRPartition (0, csr->get_n_vertices () - 1, 0, csr->get_n_edges () - 1, 
                                              csr->get_vertices (), csr->get_edges ());
  csr_partitions.push_back (full_partition);

  assert(csr_partitions.size() == 1);
  
  //Copy full graph to GPU
  GPUCSRPartition gpuCSRPartition;
  copy_partition_to_gpu(csr_partitions[0], gpuCSRPartition);

  //Initially each sample contains only one vertex
  std::vector<VertexID_t> samples;

  //Allocate one sample for each vertex
  for (auto vertex : csr->iterate_vertices()) {
    samples.push_back(vertex);
  }
  
  //Size of each sample output
  size_t maxNeighborsToSample = 1;
  for (int step = 0; step < steps(); step++) {
    maxNeighborsToSample *= stepSize(step);
  }

  size_t finalSampleSize = 0;
  size_t neighborsToSampleAtStep = 1;
  for (int step = 0; step < steps(); step++) {
    neighborsToSampleAtStep *= stepSize(step);
    finalSampleSize += neighborsToSampleAtStep;
  }

  //Allocate storage for final samples on GPU
  std::vector<VertexID_t> hFinalSamples(finalSampleSize*samples.size());

  VertexID_t* dFinalSamples;
  CHK_CU(hipMalloc(&dFinalSamples, sizeof(VertexID_t)*hFinalSamples.size()));

  //Samples to Transit Map
  VertexID_t* dSamplesToTransitMapKeys;
  VertexID_t* dSamplesToTransitMapValues;
  CHK_CU(hipMalloc(&dSamplesToTransitMapKeys, sizeof(VertexID_t)*hFinalSamples.size()));
  CHK_CU(hipMalloc(&dSamplesToTransitMapValues, sizeof(VertexID_t)*hFinalSamples.size()));

  //Transit to Samples Map
  VertexID_t* dTransitToSampleMapKeys;
  VertexID_t* dTransitToSampleMapValues;
  CHK_CU(hipMalloc(&dTransitToSampleMapKeys, sizeof(VertexID_t)*hFinalSamples.size()));
  CHK_CU(hipMalloc(&dTransitToSampleMapValues, sizeof(VertexID_t)*hFinalSamples.size()));

  //Same as initial values of samples for first iteration
  CHK_CU(hipMemcpy(dTransitToSampleMapKeys, &samples[0], sizeof(VertexID_t)*samples.size(), 
                    hipMemcpyHostToDevice));
  CHK_CU(hipMemcpy(dTransitToSampleMapValues, &samples[0], sizeof(VertexID_t)*samples.size(), 
                    hipMemcpyHostToDevice));

  //Insertion positions per transit vertex for each sample
  EdgePos_t* dSampleInsertionPositions;
  CHK_CU(hipMalloc(&dSampleInsertionPositions, sizeof(EdgePos_t)*samples.size()));

  const VertexID_t INVALID_VERTEX = graph.get_vertices().size();
  
  hiprandState* dCurandStates;
  
  CHK_CU(hipMalloc(&dCurandStates, maxNeighborsToSample*samples.size()*sizeof(hiprandState)));
  init_curand_states<<<next_multiple(samples.size()*maxNeighborsToSample, 256), 256>>> (dCurandStates, samples.size()*maxNeighborsToSample);
  CHK_CU(hipDeviceSynchronize());

  neighborsToSampleAtStep = 1;
  
  double end_to_end_t1 = convertTimeValToDouble(getTimeOfDay ());
  for (int step = 0; step < steps(); step++) {
    neighborsToSampleAtStep *= stepSize(step);
    const size_t totalThreads = samples.size()*neighborsToSampleAtStep;
    
    //Sample neighbors of transit vertices
    samplingKernel<<<next_multiple(totalThreads, N_THREADS), N_THREADS>>>(step, gpuCSRPartition, INVALID_VERTEX,
                   (const VertexID_t*)dTransitToSampleMapKeys, (const VertexID_t*)dTransitToSampleMapValues,
                   totalThreads, samples.size(),
                   dSamplesToTransitMapKeys, dSamplesToTransitMapValues,
                   dFinalSamples, finalSampleSize, dSampleInsertionPositions,
                   dCurandStates);
    CHK_CU(hipGetLastError());
    CHK_CU(hipDeviceSynchronize());

    if (step != steps() - 1) {
      //Invert sample->transit map by sorting samples based on the transit vertices
      VertexID_t* d_temp_storage = nullptr;
      size_t temp_storage_bytes = 0;
      
      //Check if the space runs out.
      //TODO: Use DoubleBuffer version that requires O(P) space.
      hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, 
                                      dSamplesToTransitMapValues, dTransitToSampleMapKeys, dSamplesToTransitMapKeys, dTransitToSampleMapValues, totalThreads);
      
      CHK_CU (hipMalloc(&d_temp_storage, temp_storage_bytes));

      hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, 
                                      dSamplesToTransitMapValues, dTransitToSampleMapKeys, dSamplesToTransitMapKeys, dTransitToSampleMapValues, totalThreads);
    }
  }

  double end_to_end_t2 = convertTimeValToDouble(getTimeOfDay ());
  

  CHK_CU(hipMemcpy(&hFinalSamples[0], dFinalSamples, hFinalSamples.size()*sizeof(hFinalSamples[0]), hipMemcpyDeviceToHost));

  size_t totalSampledVertices = 0;
  for (auto s : hFinalSamples) {
    totalSampledVertices += (int)(s != INVALID_VERTEX);
  }

  std::cout << "totalSampledVertices " << totalSampledVertices << std::endl;
  if (opt->getFlag('check-results'))
    assert(check_result(csr, INVALID_VERTEX, samples, finalSampleSize, hFinalSamples));

  if (opt->getFlag('print-samples')) {
    for (size_t s = 0; s < hFinalSamples.size(); s += finalSampleSize) {
      std::cout << "Contents of sample " << s/finalSampleSize << " [";
      for(size_t v = s; v < s + finalSampleSize; v++)
        std::cout << hFinalSamples[v] << ", ";
      std::cout << "]" << std::endl;
    }
  }
  
  // std::cout << "GPU Time: " << gpu_time << " secs" << std::endl;
  std::cout << "End to end time " << (end_to_end_t2 - end_to_end_t1) << " secs" << std::endl;
  // std::cout << "Total " << N_HOPS << "-hop neighbors " << total_neighbors << std::endl;

  // std::cout << "Results are correct? " <<check_result(csr, additions_sizes, neighbors) << std::endl;
  // std::cout << "Time spent in GPU kernel execution " << kernelTotalTime << std::endl;
  // std::cout << "Time spent in Streams " << total_stream_time << std::endl;
}
